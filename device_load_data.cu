#include <cutil_inline.h>

extern "C" void load_data(float *device_pointer, float *host_pointer, size_t size, int nsamp, int maxshift);

//{{{ load_data_from_host_to_device

void load_data(float *device_pointer, float *host_pointer, size_t size, int nsamp, int maxshift) {

	//{{{ Copy data and set up the GPU constants/variables.

	//hipEvent_t start, stop;
	//float time;
	//hipEventCreate(&start);
	//hipEventCreate(&stop);

	//printf("\n\tmemStart"),fflush(stdout);
	//hipEventRecord(start,0);
	
	cutilSafeCall( hipMemcpy(device_pointer, host_pointer, size, hipMemcpyHostToDevice) );
	cutilSafeCall( hipMemcpyToSymbol(HIP_SYMBOL("i_nsamp"), &nsamp, sizeof(int)) );
	cutilSafeCall( hipMemcpyToSymbol(HIP_SYMBOL("i_maxshift"), &maxshift, sizeof(int)) );

	//hipEventRecord(stop, 0);
	//hipEventSynchronize(stop);
	//hipEventElapsedTime(&time, start, stop);
	//printf("\n\tmemStop"),fflush(stdout);
	//printf("\n\tCopied data to GPU:\t\t\t\t%lf ms", time);    
	//printf("\n\n\tEffective bandwidth in GB per second (input):\t%f\n", (((float)size)/1000000000)/(time/1000));

	//}}}
	
}

//}}}
