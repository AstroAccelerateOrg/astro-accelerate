#include <cutil_inline.h>

extern "C" void free_device_memory(float *device_pointer);

//{{{ init_gpu

void free_device_memory(float *device_pointer) {

	//{{{ Free the memory

//	hipEvent_t start, stop;
//	float time;
//	hipEventCreate(&start);
//	hipEventCreate(&stop);

//	printf("\n\n\tfreeStart"),fflush(stdout);	
//	hipEventRecord(start,0);
	
	cutilSafeCall( hipFree(device_pointer));

//	hipEventRecord(stop, 0);
//	hipEventSynchronize(stop);
//	hipEventElapsedTime(&time, start, stop);
//	printf("\n\tfreeStart"),fflush(stdout);	
//	printf("\n\tGPU memory free:\t\t\t\t%lf ms", time);

//	hipEventDestroy(start); 
//	hipEventDestroy(stop);

	//}}}
	
}

//}}}

