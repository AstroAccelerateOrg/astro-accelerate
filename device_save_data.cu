#include <cutil_inline.h>

extern "C" void save_data(float *device_pointer, float *host_pointer, size_t size);

//{{{ save_data_from_device_to_host

void save_data(float *device_pointer, float *host_pointer, size_t size) {

	//{{{ Copy data and set up the GPU constants/variables.

//	hipEvent_t start, stop;
//	float time;
//	hipEventCreate(&start);
//	hipEventCreate(&stop);

//	printf("\n\tmemStart"),fflush(stdout);
//	hipEventRecord(start,0);

	cutilSafeCall(hipMemcpy(host_pointer, device_pointer, size,hipMemcpyDeviceToHost));

//	hipEventRecord(stop, 0);
//	hipEventSynchronize(stop);
//	hipEventElapsedTime(&time, start, stop);
//	printf("\n\tmemStop"),fflush(stdout);
//	printf("\n\tCopied data to GPU:\t\t\t\t%lf ms", time);    
//	printf("\n\n\tEffective bandwidth in GB per second (input):\t%f\n", (((float)inputsize)/1000000000)/(time/1000));

	//}}}
	
}

//}}}
