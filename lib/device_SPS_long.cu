#include "hip/hip_runtime.h"
//Added by Karel Adamek
//#define SPS_LONG_DEBUG
#define SPS_LONG_LOG

#include <vector>

#include "headers/params.h"
#include "headers/device_BC_plan.h"
#include "headers/device_MSD_BLN_pw.h"
//#include "headers/device_MSD_BLN_pw_dp.h"
#include "headers/device_MSD_limited.h"
#include "device_SPS_long_kernel.cu"


#ifdef SPS_LONG_LOG
class MSD_values {
public:
	float mean;
	float sd;
	float modifier;
	int nTaps;
	int start_taps;
	int DIT_value;
};

void Export_LA_values(std::vector<MSD_values> log){
	char str[200];
	FILE *file_out;
	sprintf(str,"MSD_LA_values_ALL.dat");
	if (( file_out = fopen(str, "a") ) == NULL)	{
		fprintf(stderr, "Error opening output file!\n");
		exit(0);
	}


	fprintf(file_out, "%d %f\n", 1, log[0].sd);
	fprintf(file_out, "%d %f\n", log[0].start_taps + log[0].DIT_value*log[0].nTaps, log[0].sd + (log[0].nTaps-1)*log[0].modifier);	
	for(size_t f=1; f<log.size(); f++){
		fprintf(file_out, "%d %f\n", log[f].start_taps, log[f].sd);
		fprintf(file_out, "%d %f\n", log[f].start_taps + log[f].DIT_value*log[f].nTaps, log[f].sd + (log[f].nTaps)*log[f].modifier);
	}
	fprintf(file_out, "\n\n");
	fclose(file_out);
}

void Export_BLN_LA_values(std::vector<MSD_values> log){
	char str[200];
	FILE *file_out;
	sprintf(str,"MSD_BLN_LA_values_ALL.dat");
	if (( file_out = fopen(str, "a") ) == NULL)	{
		fprintf(stderr, "Error opening output file!\n");
		exit(0);
	}


	fprintf(file_out, "%d %f\n", 1, log[0].sd);
	fprintf(file_out, "%d %f\n", log[0].start_taps + log[0].DIT_value*log[0].nTaps, log[0].sd + (log[0].nTaps-1)*log[0].modifier);	
	for(size_t f=1; f<log.size(); f++){
		fprintf(file_out, "%d %f\n", log[f].start_taps, log[f].sd);
		fprintf(file_out, "%d %f\n", log[f].start_taps + log[f].DIT_value*log[f].nTaps, log[f].sd + (log[f].nTaps)*log[f].modifier);
	}
	fprintf(file_out, "\n\n");
	fclose(file_out);
}
#endif


size_t Get_memory_requirement_of_SPS(){
	return((size_t) (5.5*sizeof(float) + 2*sizeof(ushort)));
}

void Assign_parameters(int f, std::vector<PulseDetection_plan> *PD_plan, int *decimated_timesamples, int *dtm, int *iteration, int *nBoxcars, int *nBlocks, int *output_shift, int *shift, int *startTaps, int *unprocessed_samples, int *total_ut){
	*decimated_timesamples = PD_plan->operator[](f).decimated_timesamples;
	*dtm                   = PD_plan->operator[](f).dtm;
	*iteration             = PD_plan->operator[](f).iteration;
	*nBoxcars              = PD_plan->operator[](f).nBoxcars;
	*nBlocks               = PD_plan->operator[](f).nBlocks;
	*output_shift          = PD_plan->operator[](f).output_shift;
	*shift                 = PD_plan->operator[](f).shift;           
	*startTaps             = PD_plan->operator[](f).startTaps; 
	*unprocessed_samples   = PD_plan->operator[](f).unprocessed_samples;
	*total_ut              = PD_plan->operator[](f).total_ut;	
}

void PD_SEARCH_LONG_init() {
	//---------> Specific nVidia stuff
	hipDeviceSetCacheConfig (hipFuncCachePreferShared);
	hipDeviceSetSharedMemConfig (hipSharedMemBankSizeEightByte);
}


int PD_SEARCH_LONG_BLN(float *d_input, float *d_boxcar_values, float *d_decimated, float *d_output_SNR, ushort *d_output_taps, float *d_MSD, std::vector<PulseDetection_plan> *PD_plan, int max_iteration, int nDMs, int nTimesamples) {
	//---------> Task specific
	
	//---------> CUDA block and CUDA grid parameters
	dim3 gridSize(1, 1, 1);
	dim3 blockSize(PD_NTHREADS, 1, 1);
	
	//---------> Pulse detection FIR
	PD_SEARCH_LONG_init();
	
	int f;
	int decimated_timesamples, dtm, iteration, nBoxcars, nBlocks, output_shift, shift, startTaps, unprocessed_samples, total_ut;
	
	// ----------> First iteration
	Assign_parameters(0, PD_plan, &decimated_timesamples, &dtm, &iteration, &nBoxcars, &nBlocks, &output_shift, &shift, &startTaps, &unprocessed_samples, &total_ut);
	gridSize.x=nBlocks; gridSize.y=nDMs; gridSize.z=1;
	blockSize.x=PD_NTHREADS; blockSize.y=1; blockSize.z=1;
	#ifdef SPS_LONG_DEBUG
	printf("decimated_timesamples:%d; dtm:%d; iteration:%d; nBoxcars:%d; nBlocks:%d; output_shift:%d; shift:%d; startTaps:%d; unprocessed_samples:%d; total_ut:%d;\n",decimated_timesamples, dtm, iteration ,nBoxcars ,nBlocks ,output_shift ,shift ,startTaps ,unprocessed_samples ,total_ut);
	#endif
	if(nBlocks>0) PD_GPU_1st_BLN<<<gridSize,blockSize>>>( d_input, d_boxcar_values, d_decimated, d_output_SNR, d_output_taps, d_MSD, decimated_timesamples, nBoxcars, dtm);
	
	
	for(f=1; f<max_iteration; f++){
		Assign_parameters(f, PD_plan, &decimated_timesamples, &dtm, &iteration, &nBoxcars, &nBlocks, &output_shift, &shift, &startTaps, &unprocessed_samples, &total_ut);
		gridSize.x=nBlocks; gridSize.y=nDMs; gridSize.z=1;
		blockSize.x=PD_NTHREADS; blockSize.y=1; blockSize.z=1;
		#ifdef SPS_LONG_DEBUG
		printf("decimated_timesamples:%d; dtm:%d; iteration:%d; nBoxcars:%d; nBlocks:%d; output_shift:%d; shift:%d; startTaps:%d; unprocessed_samples:%d; total_ut:%d;\n",decimated_timesamples, dtm, iteration, nBoxcars ,nBlocks ,output_shift ,shift ,startTaps ,unprocessed_samples ,total_ut);
		#endif
		if( (f%2) == 0 ) {
			if(nBlocks>0) PD_GPU_Nth_BLN<<<gridSize,blockSize>>>(&d_input[shift], &d_boxcar_values[nDMs*(nTimesamples>>1)], d_boxcar_values, d_decimated, &d_output_SNR[nDMs*output_shift], &d_output_taps[nDMs*output_shift], d_MSD, decimated_timesamples, nBoxcars, startTaps, (1<<iteration), dtm);
		}
		else {
			if(nBlocks>0) PD_GPU_Nth_BLN<<<gridSize,blockSize>>>(&d_decimated[shift], d_boxcar_values, &d_boxcar_values[nDMs*(nTimesamples>>1)], d_input, &d_output_SNR[nDMs*output_shift], &d_output_taps[nDMs*output_shift], d_MSD, decimated_timesamples, nBoxcars, startTaps, (1<<iteration), dtm);
		}
	}

	return(0);
}


// TODO: this also needs modified kernel since number of tams must be kept local (taps) and not global (star_taps+taps)
int PD_SEARCH_LONG_BLN_EACH(float *d_input, float *d_boxcar_values, float *d_decimated, float *d_output_SNR, ushort *d_output_taps, std::vector<PulseDetection_plan> *PD_plan, int max_iteration, int nDMs, int nTimesamples, float sigma_constant) {
	//---------> Task specific
	float *d_MSD_BV, *d_MSD_DIT;
	if ( hipSuccess != hipMalloc((void **) &d_MSD_BV, sizeof(float)*3))  {printf("Allocation error!\n"); exit(1001);}
	if ( hipSuccess != hipMalloc((void **) &d_MSD_DIT, sizeof(float)*3)) {printf("Allocation error!\n"); exit(1001);}
	
	#ifdef SPS_LONG_DEBUG
	float h_MSD_BV[3], h_MSD_DIT[3];
	#endif
	
	//---------> CUDA block and CUDA grid parameters
	dim3 gridSize(1, 1, 1);
	dim3 blockSize(PD_NTHREADS, 1, 1);
	
	//---------> Pulse detection FIR
	PD_SEARCH_LONG_init();
	
	int f;
	int decimated_timesamples, dtm, iteration, nBoxcars, nBlocks, output_shift, shift, startTaps, unprocessed_samples, total_ut;
	
	// ----------> First iteration
	Assign_parameters(0, PD_plan, &decimated_timesamples, &dtm, &iteration, &nBoxcars, &nBlocks, &output_shift, &shift, &startTaps, &unprocessed_samples, &total_ut);
	gridSize.x=nBlocks; gridSize.y=nDMs; gridSize.z=1;
	blockSize.x=PD_NTHREADS; blockSize.y=1; blockSize.z=1;
	//Note: Musim udelat dve SNR jeden pro BV_in and dalsi pro decimated values. Celkove rms je pak rms(BV) + sqrt(ntaps)*rms(decimated)
	MSD_BLN_pw(d_input, d_MSD_BV, nDMs, decimated_timesamples, 0, sigma_constant);
	#ifdef SPS_LONG_DEBUG

	hipMemcpy(h_MSD_BV, d_MSD_BV, 3*sizeof(float), hipMemcpyDeviceToHost);
	//printf("     MSD BLN point-wise: Mean: %f, Stddev: %f, modifier: %f\n", h_MSD_BV[0], h_MSD_BV[1], h_MSD_BV[2]);
	//printf("decimated_timesamples:%d; dtm:%d; iteration:%d; nBoxcars:%d; nBlocks:%d; output_shift:%d; shift:%d; startTaps:%d; unprocessed_samples:%d; total_ut:%d;\n",decimated_timesamples, dtm, iteration ,nBoxcars ,nBlocks ,output_shift ,shift ,startTaps ,unprocessed_samples ,total_ut);
	#endif
	
	if(nBlocks>0) PD_GPU_1st_BLN<<<gridSize,blockSize>>>( d_input, d_boxcar_values, d_decimated, d_output_SNR, d_output_taps, d_MSD_BV, decimated_timesamples, nBoxcars, dtm);
	
	
	for(f=1; f<max_iteration; f++){
		Assign_parameters(f, PD_plan, &decimated_timesamples, &dtm, &iteration, &nBoxcars, &nBlocks, &output_shift, &shift, &startTaps, &unprocessed_samples, &total_ut);
		gridSize.x=nBlocks; gridSize.y=nDMs; gridSize.z=1;
		blockSize.x=PD_NTHREADS; blockSize.y=1; blockSize.z=1;
		#ifdef SPS_LONG_DEBUG
		//printf("decimated_timesamples:%d; dtm:%d; iteration:%d; nBoxcars:%d; nBlocks:%d; output_shift:%d; shift:%d; startTaps:%d; unprocessed_samples:%d; total_ut:%d;\n",decimated_timesamples, dtm, iteration, nBoxcars ,nBlocks ,output_shift ,shift ,startTaps ,unprocessed_samples ,total_ut);
		#endif
		if( (f%2) == 0 ) {
			MSD_BLN_pw(d_input, d_MSD_DIT, nDMs, decimated_timesamples, 0, sigma_constant);
			MSD_BLN_pw(&d_boxcar_values[nDMs*(nTimesamples>>1)], d_MSD_BV, nDMs, decimated_timesamples, PD_plan->operator[](f-1).unprocessed_samples, sigma_constant);
			if(nBlocks>0) PD_GPU_Nth_BLN_EACH<<<gridSize,blockSize>>>(&d_input[shift], &d_boxcar_values[nDMs*(nTimesamples>>1)], d_boxcar_values, d_decimated, &d_output_SNR[nDMs*output_shift], &d_output_taps[nDMs*output_shift], d_MSD_BV, d_MSD_DIT, decimated_timesamples, nBoxcars, startTaps, (1<<iteration), dtm);
		}
		else {
			MSD_BLN_pw(d_input, d_MSD_DIT, nDMs, decimated_timesamples, 0, sigma_constant);
			MSD_BLN_pw(d_boxcar_values, d_MSD_BV, nDMs, decimated_timesamples, PD_plan->operator[](f-1).unprocessed_samples, sigma_constant);
			if(nBlocks>0) PD_GPU_Nth_BLN_EACH<<<gridSize,blockSize>>>(&d_decimated[shift], d_boxcar_values, &d_boxcar_values[nDMs*(nTimesamples>>1)], d_input, &d_output_SNR[nDMs*output_shift], &d_output_taps[nDMs*output_shift], d_MSD_BV, d_MSD_DIT, decimated_timesamples, nBoxcars, startTaps, (1<<iteration), dtm);
		}
		
		#ifdef SPS_LONG_DEBUG
		hipMemcpy(h_MSD_BV, d_MSD_BV, 3*sizeof(float), hipMemcpyDeviceToHost);
		hipMemcpy(h_MSD_DIT, d_MSD_DIT, 3*sizeof(float), hipMemcpyDeviceToHost);
		printf("     MSD BV point-wise: Mean: %f, Stddev: %f, modifier: %f\n", h_MSD_BV[0], h_MSD_BV[1], h_MSD_BV[2]);
		printf("     MSD DIT point-wise: Mean: %f, Stddev: %f, modifier: %f\n", h_MSD_DIT[0], h_MSD_DIT[1], h_MSD_DIT[2]);
		#endif
	}

	
	hipFree(d_MSD_BV);
	hipFree(d_MSD_DIT);
	return(0);
}





int PD_SEARCH_LONG_LINAPPROX(float *d_input, float *d_boxcar_values, float *d_decimated, float *d_output_SNR, ushort *d_output_taps, float *d_MSD, std::vector<PulseDetection_plan> *PD_plan, int max_iteration, int nDMs, int nTimesamples) {
	//---------> Task specific
	
	//---------> CUDA block and CUDA grid parameters
	dim3 gridSize(1, 1, 1);
	dim3 blockSize(PD_NTHREADS, 1, 1);
	
	//---------> Pulse detection FIR
	PD_SEARCH_LONG_init();
	
	int f;
	int decimated_timesamples, dtm, iteration, nBoxcars, nBlocks, output_shift, shift, startTaps, unprocessed_samples, total_ut;
	
	// ----------> First iteration
	Assign_parameters(0, PD_plan, &decimated_timesamples, &dtm, &iteration, &nBoxcars, &nBlocks, &output_shift, &shift, &startTaps, &unprocessed_samples, &total_ut);
	gridSize.x=nBlocks; gridSize.y=nDMs; gridSize.z=1;
	blockSize.x=PD_NTHREADS; blockSize.y=1; blockSize.z=1;
	#ifdef SPS_LONG_DEBUG
	printf("decimated_timesamples:%d; dtm:%d; iteration:%d; nBoxcars:%d; nBlocks:%d; output_shift:%d; shift:%d; startTaps:%d; unprocessed_samples:%d; total_ut:%d;\n",decimated_timesamples, dtm, iteration ,nBoxcars ,nBlocks ,output_shift ,shift ,startTaps ,unprocessed_samples ,total_ut);
	#endif
	if(nBlocks>0) PD_GPU_1st_LA<<<gridSize,blockSize>>>( d_input, d_boxcar_values, d_decimated, d_output_SNR, d_output_taps, d_MSD, decimated_timesamples, nBoxcars, dtm);
	
	
	for(f=1; f<max_iteration; f++){
		Assign_parameters(f, PD_plan, &decimated_timesamples, &dtm, &iteration, &nBoxcars, &nBlocks, &output_shift, &shift, &startTaps, &unprocessed_samples, &total_ut);
		gridSize.x=nBlocks; gridSize.y=nDMs; gridSize.z=1;
		blockSize.x=PD_NTHREADS; blockSize.y=1; blockSize.z=1;
		#ifdef SPS_LONG_DEBUG
		printf("decimated_timesamples:%d; dtm:%d; iteration:%d; nBoxcars:%d; nBlocks:%d; output_shift:%d; shift:%d; startTaps:%d; unprocessed_samples:%d; total_ut:%d;\n",decimated_timesamples, dtm, iteration, nBoxcars ,nBlocks ,output_shift ,shift ,startTaps ,unprocessed_samples ,total_ut);
		#endif
		if( (f%2) == 0 ) {
			if(nBlocks>0) PD_GPU_Nth_LA<<<gridSize,blockSize>>>(&d_input[shift], &d_boxcar_values[nDMs*(nTimesamples>>1)], d_boxcar_values, d_decimated, &d_output_SNR[nDMs*output_shift], &d_output_taps[nDMs*output_shift], d_MSD, decimated_timesamples, nBoxcars, startTaps, (1<<iteration), dtm);
		}
		else {
			if(nBlocks>0) PD_GPU_Nth_LA<<<gridSize,blockSize>>>(&d_decimated[shift], d_boxcar_values, &d_boxcar_values[nDMs*(nTimesamples>>1)], d_input, &d_output_SNR[nDMs*output_shift], &d_output_taps[nDMs*output_shift], d_MSD, decimated_timesamples, nBoxcars, startTaps, (1<<iteration), dtm);
		}
	}

	return(0);
}


// TODO: this also needs modified kernel since number of tams must be kept local (taps) and not global (star_taps+taps)
int PD_SEARCH_LONG_LINAPPROX_EACH(float *d_input, float *d_boxcar_values, float *d_decimated, float *d_output_SNR, ushort *d_output_taps, std::vector<PulseDetection_plan> *PD_plan, int max_iteration, int nDMs, int nTimesamples) {
	//---------> Task specific
	float *d_MSD, *d_MSD_Nth;
	if ( hipSuccess != hipMalloc((void **) &d_MSD, sizeof(float)*4))  {printf("Allocation error!\n"); exit(1001);}
	if ( hipSuccess != hipMalloc((void **) &d_MSD_Nth, sizeof(float)*4))  {printf("Allocation error!\n"); exit(1001);}
	#ifdef SPS_LONG_DEBUG
	float h_MSD[4];
	#endif
	#ifdef SPS_LONG_LOG
	float h_MSD_LOG[4];
	int log_DIT_value;
	std::vector<MSD_values> log;
	MSD_values log_temp;
	#endif
	
	//---------> CUDA block and CUDA grid parameters
	dim3 gridSize(1, 1, 1);
	dim3 blockSize(PD_NTHREADS, 1, 1);
	
	//---------> Pulse detection FIR
	PD_SEARCH_LONG_init();
	
	int f;
	int decimated_timesamples, dtm, iteration, nBoxcars, nBlocks, output_shift, shift, startTaps, unprocessed_samples, total_ut;
	
	// ----------> First iteration
	Assign_parameters(0, PD_plan, &decimated_timesamples, &dtm, &iteration, &nBoxcars, &nBlocks, &output_shift, &shift, &startTaps, &unprocessed_samples, &total_ut);
	gridSize.x=nBlocks; gridSize.y=nDMs; gridSize.z=1;
	blockSize.x=PD_NTHREADS; blockSize.y=1; blockSize.z=1;
	MSD_linear_approximation(d_input, d_MSD, nBoxcars, nDMs, decimated_timesamples, 0);
	#ifdef SPS_LONG_DEBUG

	hipMemcpy(h_MSD, d_MSD, 3*sizeof(float), hipMemcpyDeviceToHost);
	//printf("     MSD linear approximation: Mean: %f, Stddev: %f, modifier: %f\n", h_MSD[0], h_MSD[1], h_MSD[2]);
	//printf("decimated_timesamples:%d; dtm:%d; iteration:%d; nBoxcars:%d; nBlocks:%d; output_shift:%d; shift:%d; startTaps:%d; unprocessed_samples:%d; total_ut:%d;\n",decimated_timesamples, dtm, iteration ,nBoxcars ,nBlocks ,output_shift ,shift ,startTaps ,unprocessed_samples ,total_ut);
	#endif
	#ifdef SPS_LONG_LOG
	hipMemcpy(h_MSD_LOG, d_MSD, 3*sizeof(float), hipMemcpyDeviceToHost);
	log_DIT_value = 1;
	log_temp.mean       = h_MSD_LOG[0];
	log_temp.sd         = h_MSD_LOG[1];
	log_temp.modifier   = h_MSD_LOG[2];
	log_temp.nTaps      = nBoxcars;
	log_temp.start_taps = startTaps;
	log_temp.DIT_value  = log_DIT_value;
	log.push_back(log_temp);
	#endif
	if(nBlocks>0) PD_GPU_1st_LA<<<gridSize,blockSize>>>( d_input, d_boxcar_values, d_decimated, d_output_SNR, d_output_taps, d_MSD, decimated_timesamples, nBoxcars, dtm);
	
	
	for(f=1; f<max_iteration; f++){
		Assign_parameters(f, PD_plan, &decimated_timesamples, &dtm, &iteration, &nBoxcars, &nBlocks, &output_shift, &shift, &startTaps, &unprocessed_samples, &total_ut);
		gridSize.x=nBlocks; gridSize.y=nDMs; gridSize.z=1;
		blockSize.x=PD_NTHREADS; blockSize.y=1; blockSize.z=1;
		#ifdef SPS_LONG_DEBUG
		//printf("decimated_timesamples:%d; dtm:%d; iteration:%d; nBoxcars:%d; nBlocks:%d; output_shift:%d; shift:%d; startTaps:%d; unprocessed_samples:%d; total_ut:%d;\n",decimated_timesamples, dtm, iteration, nBoxcars ,nBlocks ,output_shift ,shift ,startTaps ,unprocessed_samples ,total_ut);
		#endif
		if( (f%2) == 0 ) {
			MSD_LA_Nth(&d_input[shift], &d_boxcar_values[nDMs*(nTimesamples>>1)], d_MSD_Nth, d_MSD, nBoxcars, nDMs, decimated_timesamples, 3*unprocessed_samples, (1<<iteration));
			if(nBlocks>0) PD_GPU_Nth_LA_EACH<<<gridSize,blockSize>>>(&d_input[shift], &d_boxcar_values[nDMs*(nTimesamples>>1)], d_boxcar_values, d_decimated, &d_output_SNR[nDMs*output_shift], &d_output_taps[nDMs*output_shift], d_MSD_Nth, decimated_timesamples, nBoxcars, startTaps, (1<<iteration), dtm);
		}
		else {
			MSD_LA_Nth(&d_decimated[shift], d_boxcar_values, d_MSD_Nth, d_MSD, nBoxcars, nDMs, decimated_timesamples, 3*unprocessed_samples, (1<<iteration));
			if(nBlocks>0) PD_GPU_Nth_LA_EACH<<<gridSize,blockSize>>>(&d_decimated[shift], d_boxcar_values, &d_boxcar_values[nDMs*(nTimesamples>>1)], d_input, &d_output_SNR[nDMs*output_shift], &d_output_taps[nDMs*output_shift], d_MSD_Nth, decimated_timesamples, nBoxcars, startTaps, (1<<iteration), dtm);
		}
		
		#ifdef SPS_LONG_DEBUG
		hipMemcpy(h_MSD, d_MSD_Nth, 4*sizeof(float), hipMemcpyDeviceToHost);
		printf("     MSD linear approximation: BV Mean: %f, Stddev: %f, modifier: %f; DIT Mean:%f;\n", h_MSD[0], h_MSD[1], h_MSD[2], h_MSD[3]);
		#endif
		#ifdef SPS_LONG_LOG
		hipMemcpy(h_MSD_LOG, d_MSD_Nth, 4*sizeof(float), hipMemcpyDeviceToHost);
		log_DIT_value = log_DIT_value*2;
		log_temp.mean       = h_MSD_LOG[0];
		log_temp.sd         = h_MSD_LOG[1];
		log_temp.modifier   = h_MSD_LOG[2];
		log_temp.nTaps      = nBoxcars;
		log_temp.start_taps = startTaps;
		log_temp.DIT_value  = log_DIT_value;
		log.push_back(log_temp);
		#endif
	}
	#ifdef SPS_LONG_LOG
	Export_LA_values(log);
	#endif

	hipFree(d_MSD);
	return(0);
}

int PD_SEARCH_LONG_BLN_LINAPPROX_EACH(float *d_input, float *d_boxcar_values, float *d_decimated, float *d_output_SNR, ushort *d_output_taps, std::vector<PulseDetection_plan> *PD_plan, int max_iteration, int nDMs, int nTimesamples, float sigma_constant) {
	//---------> Task specific
	float *d_MSD, *d_MSD_Nth;
	if ( hipSuccess != hipMalloc((void **) &d_MSD, sizeof(float)*4))  {printf("Allocation error!\n"); exit(1001);}
	if ( hipSuccess != hipMalloc((void **) &d_MSD_Nth, sizeof(float)*4))  {printf("Allocation error!\n"); exit(1001);}
	#ifdef SPS_LONG_DEBUG
	float h_MSD[4];
	#endif
	#ifdef SPS_LONG_LOG
	float h_MSD_LOG[4];
	int log_DIT_value;
	std::vector<MSD_values> log;
	MSD_values log_temp;
	#endif
	
	//---------> CUDA block and CUDA grid parameters
	dim3 gridSize(1, 1, 1);
	dim3 blockSize(PD_NTHREADS, 1, 1);
	
	//---------> Pulse detection FIR
	PD_SEARCH_LONG_init();
	
	int f;
	int decimated_timesamples, dtm, iteration, nBoxcars, nBlocks, output_shift, shift, startTaps, unprocessed_samples, total_ut;
	
	// ----------> First iteration
	Assign_parameters(0, PD_plan, &decimated_timesamples, &dtm, &iteration, &nBoxcars, &nBlocks, &output_shift, &shift, &startTaps, &unprocessed_samples, &total_ut);
	gridSize.x=nBlocks; gridSize.y=nDMs; gridSize.z=1;
	blockSize.x=PD_NTHREADS; blockSize.y=1; blockSize.z=1;
	MSD_BLN_LA_pw_normal(d_input, d_MSD, nBoxcars, nDMs, decimated_timesamples, 0, sigma_constant);
	#ifdef SPS_LONG_DEBUG
	hipMemcpy(h_MSD, d_MSD, 3*sizeof(float), hipMemcpyDeviceToHost);
	printf("     MSD linear approximation: Mean: %f, Stddev: %f, modifier: %f\n", h_MSD[0], h_MSD[1], h_MSD[2]);
	printf("decimated_timesamples:%d; dtm:%d; iteration:%d; nBoxcars:%d; nBlocks:%d; output_shift:%d; shift:%d; startTaps:%d; unprocessed_samples:%d; total_ut:%d;\n",decimated_timesamples, dtm, iteration ,nBoxcars ,nBlocks ,output_shift ,shift ,startTaps ,unprocessed_samples ,total_ut);
	#endif
	#ifdef SPS_LONG_LOG
	hipMemcpy(h_MSD_LOG, d_MSD, 3*sizeof(float), hipMemcpyDeviceToHost);
	log_DIT_value = 1;
	log_temp.mean       = h_MSD_LOG[0];
	log_temp.sd         = h_MSD_LOG[1];
	log_temp.modifier   = h_MSD_LOG[2];
	log_temp.nTaps      = nBoxcars;
	log_temp.start_taps = startTaps;
	log_temp.DIT_value  = log_DIT_value;
	log.push_back(log_temp);
	#endif
	if(nBlocks>0) PD_GPU_1st_LA<<<gridSize,blockSize>>>( d_input, d_boxcar_values, d_decimated, d_output_SNR, d_output_taps, d_MSD, decimated_timesamples, nBoxcars, dtm);
	
	
	for(f=1; f<max_iteration; f++){
		Assign_parameters(f, PD_plan, &decimated_timesamples, &dtm, &iteration, &nBoxcars, &nBlocks, &output_shift, &shift, &startTaps, &unprocessed_samples, &total_ut);
		gridSize.x=nBlocks; gridSize.y=nDMs; gridSize.z=1;
		blockSize.x=PD_NTHREADS; blockSize.y=1; blockSize.z=1;
		#ifdef SPS_LONG_DEBUG
		printf("decimated_timesamples:%d; dtm:%d; iteration:%d; nBoxcars:%d; nBlocks:%d; output_shift:%d; shift:%d; startTaps:%d; unprocessed_samples:%d; total_ut:%d;\n",decimated_timesamples, dtm, iteration, nBoxcars ,nBlocks ,output_shift ,shift ,startTaps ,unprocessed_samples ,total_ut);
		#endif
		if( (f%2) == 0 ) {
			MSD_BLN_LA_Nth_pw_normal(&d_input[shift], &d_boxcar_values[nDMs*(nTimesamples>>1)], d_MSD_Nth, d_MSD, nBoxcars, nDMs, decimated_timesamples, 3*unprocessed_samples, (1<<iteration), sigma_constant);
			if(nBlocks>0) PD_GPU_Nth_LA_EACH<<<gridSize,blockSize>>>(&d_input[shift], &d_boxcar_values[nDMs*(nTimesamples>>1)], d_boxcar_values, d_decimated, &d_output_SNR[nDMs*output_shift], &d_output_taps[nDMs*output_shift], d_MSD_Nth, decimated_timesamples, nBoxcars, startTaps, (1<<iteration), dtm);
		}
		else {
			MSD_BLN_LA_Nth_pw_normal(&d_decimated[shift], d_boxcar_values, d_MSD_Nth, d_MSD, nBoxcars, nDMs, decimated_timesamples, 3*unprocessed_samples, (1<<iteration), sigma_constant);
			if(nBlocks>0) PD_GPU_Nth_LA_EACH<<<gridSize,blockSize>>>(&d_decimated[shift], d_boxcar_values, &d_boxcar_values[nDMs*(nTimesamples>>1)], d_input, &d_output_SNR[nDMs*output_shift], &d_output_taps[nDMs*output_shift], d_MSD_Nth, decimated_timesamples, nBoxcars, startTaps, (1<<iteration), dtm);
		}
		
		#ifdef SPS_LONG_DEBUG
		hipMemcpy(h_MSD, d_MSD_Nth, 4*sizeof(float), hipMemcpyDeviceToHost);
		printf("     MSD linear approximation: BV Mean: %f, Stddev: %f, modifier: %f; DIT Mean:%f;\n", h_MSD[0], h_MSD[1], h_MSD[2], h_MSD[3]);
		#endif
		#ifdef SPS_LONG_LOG
		hipMemcpy(h_MSD_LOG, d_MSD_Nth, 4*sizeof(float), hipMemcpyDeviceToHost);
		log_DIT_value = log_DIT_value*2;
		log_temp.mean       = h_MSD_LOG[0];
		log_temp.sd         = h_MSD_LOG[1];
		log_temp.modifier   = h_MSD_LOG[2];
		log_temp.nTaps      = nBoxcars;
		log_temp.start_taps = startTaps;
		log_temp.DIT_value  = log_DIT_value;
		log.push_back(log_temp);
		#endif
	}
	#ifdef SPS_LONG_LOG
	Export_BLN_LA_values(log);
	#endif

	hipFree(d_MSD);
	return(0);
}
