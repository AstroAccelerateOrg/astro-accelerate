#include "hip/hip_runtime.h"
//#define GPU_ANALYSIS_DEBUG
//#define MSD_BOXCAR_TEST
//#define GPU_PARTIAL_TIMER
#define GPU_TIMER
#define VERBOSE 1

#include <vector>
#include <stdio.h>
#include <stdlib.h>
#include "headers/params.h"

#include "headers/device_MSD_Parameters.h"
#include "headers/device_MSD_plane_profile.h"

#include "headers/device_SPS_DataDescription.h"
#include "headers/device_SPS_Parameters.h"
#include "headers/device_SPS_BC_plan.h"
#include "headers/device_SPS_long.h"
#include "headers/device_peak_find.h"
#include "headers/device_threshold.h"

#include "timer.h"

//TODO:
// Make BC_plan for arbitrary long pulses, by reusing last element in the plane


void Create_list_of_boxcar_widths(std::vector<int> *boxcar_widths, std::vector<int> *BC_widths, int max_boxcar_width){
	int DIT_value, DIT_factor, width;
	DIT_value = 1;
	DIT_factor = 2;
	width = 0;
	for(int f=0; f<(int) BC_widths->size(); f++){
		for(int b=0; b<BC_widths->operator[](f); b++){
			width = width + DIT_value;
			if(width<=max_boxcar_width){
				boxcar_widths->push_back(width);
			}
		}
		DIT_value = DIT_value*DIT_factor;
	}
}


// Extend this to arbitrary size plans
void Create_PD_plan(std::vector<PulseDetection_plan> *PD_plan, std::vector<int> *BC_widths, int nTimesamples){
	int Elements_per_block, itemp, nRest;
	PulseDetection_plan PDmp;
	
	if(BC_widths->size()>0){
		PDmp.shift        = 0;
		PDmp.output_shift = 0;
		PDmp.startTaps    = 0;
		PDmp.iteration    = 0;
		
		PDmp.decimated_timesamples = nTimesamples;
		PDmp.dtm = (nTimesamples>>(PDmp.iteration+1));
		PDmp.dtm = PDmp.dtm - (PDmp.dtm&1);
		
		PDmp.nBoxcars = BC_widths->operator[](0);
		Elements_per_block = PD_NTHREADS*2 - PDmp.nBoxcars;
		itemp = PDmp.decimated_timesamples;
		PDmp.nBlocks = itemp/Elements_per_block;
		nRest = itemp - PDmp.nBlocks*Elements_per_block;
		if(nRest>0) PDmp.nBlocks++;
		PDmp.unprocessed_samples = PDmp.nBoxcars + 6;
		if(PDmp.decimated_timesamples<PDmp.unprocessed_samples) PDmp.nBlocks=0;
		PDmp.total_ut = PDmp.unprocessed_samples;
		
		
		PD_plan->push_back(PDmp);
		
		for(int f=1; f< (int) BC_widths->size(); f++){
			// These are based on previous values of PDmp
			PDmp.shift        = PDmp.nBoxcars/2;
			PDmp.output_shift = PDmp.output_shift + PDmp.decimated_timesamples;
			PDmp.startTaps    = PDmp.startTaps + PDmp.nBoxcars*(1<<PDmp.iteration);
			PDmp.iteration    = PDmp.iteration + 1;
			
			// Definition of new PDmp values
			PDmp.decimated_timesamples = PDmp.dtm;
			PDmp.dtm = (nTimesamples>>(PDmp.iteration+1));
			PDmp.dtm = PDmp.dtm - (PDmp.dtm&1);
			
			PDmp.nBoxcars = BC_widths->operator[](f);
			Elements_per_block=PD_NTHREADS*2 - PDmp.nBoxcars;
			itemp = PDmp.decimated_timesamples;
			PDmp.nBlocks = itemp/Elements_per_block;
			nRest = itemp - PDmp.nBlocks*Elements_per_block;
			if(nRest>0) PDmp.nBlocks++;
			PDmp.unprocessed_samples = PDmp.unprocessed_samples/2 + PDmp.nBoxcars + 6; //
			if(PDmp.decimated_timesamples<PDmp.unprocessed_samples) PDmp.nBlocks=0;
			PDmp.total_ut = PDmp.unprocessed_samples*(1<<PDmp.iteration);
			
			PD_plan->push_back(PDmp);
		}
	}
}


int Get_max_iteration(int max_boxcar_width, std::vector<int> *BC_widths, int *max_width_performed){
	int startTaps, iteration;
	
	startTaps = 0;
	iteration = 0;
	for(int f=0; f<(int) BC_widths->size(); f++){
		startTaps = startTaps + BC_widths->operator[](f)*(1<<f);
		if(startTaps>=max_boxcar_width) {
			iteration = f+1;
			break;
		}
	}
	
	if(max_boxcar_width>startTaps) {
		iteration=(int) BC_widths->size();
	}
	
	*max_width_performed=startTaps;
	return(iteration);
}


void analysis_GPU( float *h_peak_list, size_t *peak_pos, size_t max_peak_size, SPS_DataDescription SPS_data, float *d_SPS_input, SPS_Parameters *SPS_params, MSD_Parameters *MSD_params){
	// Definition of some local variables
	float  local_tsamp  = SPS_data.sampling_time*SPS_data.inBin; // corrected sampling time
	size_t nTimesamples = SPS_data.nTimesamples;
	size_t nDMs         = SPS_data.nDMs;
	if(SPS_params->verbose) {
		printf("\n----------> GPU analysis part\n");
		printf("  Dimensions: nTimesamples:%zu; nDMs:%zu; inBin:%d; sampling time: %f; corrected s. time: %f;\n", nTimesamples, nDMs, SPS_data.inBin, SPS_data.sampling_time, local_tsamp);
	}
	
	//--------> Definition of SPDT boxcar plan
	int max_desired_boxcar_width = (int) (SPS_params->max_boxcar_width_in_sec/local_tsamp);
	int max_width_performed = 0;
	int t_BC_widths[10]={PD_MAXTAPS,16,16,16,8,8,8,8,8,8};
	std::vector<int> BC_widths(t_BC_widths,t_BC_widths+sizeof(t_BC_widths)/sizeof(int));
	std::vector<PulseDetection_plan> PD_plan;
	Create_PD_plan(&PD_plan, &BC_widths, nTimesamples); //PD_plan is independent on maximum boxcar width. which is wrong?
	int max_iteration = Get_max_iteration(max_desired_boxcar_width, &BC_widths, &max_width_performed);
	if(SPS_params->verbose) 
		printf("  Selected iteration:%d; maximum boxcar width requested:%d; maximum boxcar width performed:%d;\n", max_iteration, max_desired_boxcar_width, max_width_performed);
	std::vector<int> h_boxcar_widths;
	Create_list_of_boxcar_widths(&h_boxcar_widths, &BC_widths, max_width_performed);
	
	// It should be like this:
	//   SPS_params should contain BC_widths
	//   SPS_params should also contain function get_maximum_iteration which would give number of iterations required to achieve user defined value in form of max_desired_boxcar_width
	//   Based on maximum_iteration SPS should build PD_plan
	//   Proper error check must be placed so SPS would not die if user chooses wrong maximum search width
	
	//--------> Benchmarking
	double total_time=0, MSD_time=0, SPDT_time=0, PF_time=0;
	

	//---------------------------------------------------------------------------
	//----------> GPU part
	GpuTimer total_timer, timer;
	total_timer.Start();
	
	
	size_t free_mem,total_mem;
	hipMemGetInfo(&free_mem,&total_mem);
	if(SPS_params->verbose) printf("  Memory required by boxcar filters:%0.3f MB\n",(4.5*nTimesamples*nDMs*sizeof(float) + 2*nTimesamples*nDMs*sizeof(ushort))/(1024.0*1024) );
	if(SPS_params->verbose) printf("  Memory available:%0.3f MB \n", ((float) free_mem)/(1024.0*1024.0) );
	
	
	//-------------------------------------------------------------------------
	//---------> Comparison between interpolated values and computed values
	#ifdef MSD_BOXCAR_TEST
		MSD_plane_profile_boxcars(d_SPS_input, nTimesamples, nDMs, &h_boxcar_widths, MSD_params->OR_sigma_multiplier, SPS_data.d,_low, SPS_data.dm_high, SPS_data.time_start);
	#endif
	//---------> Comparison between interpolated values and computed values
	//-------------------------------------------------------------------------
	
	
	
	//-------------------------------------------------------------------------
	//------------ Using MSD_plane_profile
	size_t MSD_profile_size_in_bytes, MSD_DIT_profile_size_in_bytes, workarea_size_in_bytes;
	hipMemGetInfo(&free_mem,&total_mem);
	Get_MSD_plane_profile_memory_requirements(&MSD_profile_size_in_bytes, &MSD_DIT_profile_size_in_bytes, &workarea_size_in_bytes, nTimesamples, nDMs, &h_boxcar_widths);
	double dit_time, MSD_only_time;
	float *d_MSD_interpolated;
	float *d_MSD_DIT = NULL;
	float *temporary_workarea;
	hipMalloc((void **) &d_MSD_interpolated, MSD_profile_size_in_bytes);
	hipMalloc((void **) &temporary_workarea, workarea_size_in_bytes);
	
	MSD_plane_profile(d_MSD_interpolated, d_SPS_input, d_MSD_DIT, temporary_workarea, false, nTimesamples, nDMs, &h_boxcar_widths, SPS_data.time_start, SPS_data.dm_low, SPS_data.dm_high, MSD_params->OR_sigma_multiplier, MSD_params->enable_outlier_rejection, false, &MSD_time, &dit_time, &MSD_only_time);
	
	#ifdef GPU_PARTIAL_TIMER
		printf("    MSD time: Total: %f ms; DIT: %f ms; MSD: %f ms;\n", MSD_time, dit_time, MSD_only_time);
	#endif
	
	hipFree(temporary_workarea);
	//------------ Using MSD_plane_profile
	//-------------------------------------------------------------------------	
	
	
	//-------------------------------------------------------------------------
	//------------ Splitting input data into chunks
	std::vector<int> DM_list;
	unsigned long int max_timesamples=(free_mem*0.95)/(5.5*sizeof(float) + 2*sizeof(ushort));
	int DMs_per_cycle = max_timesamples/nTimesamples;
	int nRepeats, nRest, DM_shift, itemp, local_max_list_size;//BC_shift,
	
	itemp = (int) (DMs_per_cycle/THR_WARPS_PER_BLOCK);
	DMs_per_cycle = itemp*THR_WARPS_PER_BLOCK;
	
	nRepeats = nDMs/DMs_per_cycle;
	nRest = nDMs - nRepeats*DMs_per_cycle;
	local_max_list_size = (DMs_per_cycle*nTimesamples)/4;
	
	for(int f=0; f<nRepeats; f++) DM_list.push_back(DMs_per_cycle);
	if(nRest>0) DM_list.push_back(nRest);
	
	if( (int) DM_list.size() > 1 ) 
		printf("  SPS will run %d batches each containing %d DM trials. Remainder %d DM trials\n", (int) DM_list.size(), DMs_per_cycle, nRest);
	else 
		printf("  SPS will run %d batch containing %d DM trials.\n", (int) DM_list.size(), nRest);
	//------------ Splitting input data into chunks
	//-------------------------------------------------------------------------	
	
	
	
	if(DM_list.size()>0){
		DMs_per_cycle = DM_list[0];
		
		float *d_peak_list;
		if ( hipSuccess != hipMalloc((void**) &d_peak_list, sizeof(float)*DMs_per_cycle*nTimesamples)) printf("Allocation error! peaks\n");
		
		float *d_decimated;
		if ( hipSuccess != hipMalloc((void **) &d_decimated,  sizeof(float)*(((DMs_per_cycle*nTimesamples)/2)+PD_MAXTAPS) )) printf("Allocation error! dedispered\n");
		
		float *d_boxcar_values;
		if ( hipSuccess != hipMalloc((void **) &d_boxcar_values,  sizeof(float)*DMs_per_cycle*nTimesamples)) printf("Allocation error! boxcars\n");
		
		float *d_output_SNR;
		if ( hipSuccess != hipMalloc((void **) &d_output_SNR, sizeof(float)*2*DMs_per_cycle*nTimesamples)) printf("Allocation error! SNR\n");
		
		ushort *d_output_taps;
		if ( hipSuccess != hipMalloc((void **) &d_output_taps, sizeof(ushort)*2*DMs_per_cycle*nTimesamples)) printf("Allocation error! taps\n");
		
		int *gmem_peak_pos;
		hipMalloc((void**) &gmem_peak_pos, 1*sizeof(int));
		hipMemset((void*) gmem_peak_pos, 0, sizeof(int));
		
		DM_shift = 0;
		for(int f=0; f<DM_list.size(); f++) {
			//-------------- SPDT
			timer.Start();
			SPDT_search_long_MSD_plane(&d_SPS_input[DM_shift*nTimesamples], d_boxcar_values, d_decimated, d_output_SNR, d_output_taps, d_MSD_interpolated, &PD_plan, max_iteration, nTimesamples, DM_list[f]);
			timer.Stop();
			SPDT_time += timer.Elapsed();
			#ifdef GPU_PARTIAL_TIMER
			printf("    SPDT took:%f ms\n", timer.Elapsed());
			#endif
			//-------------- SPDT
			
			checkCudaErrors(hipGetLastError());
			
			#ifdef GPU_ANALYSIS_DEBUG
			printf("    BC_shift:%d; DMs_per_cycle:%d; f*DMs_per_cycle:%d; max_iteration:%d;\n", DM_shift*nTimesamples, DM_list[f], DM_shift, max_iteration);
			#endif
			
			if(SPS_params->candidate_algorithm==1){
				//-------------- Thresholding
				timer.Start();
				THRESHOLD(d_output_SNR, d_output_taps, d_peak_list, gmem_peak_pos, SPS_params->sigma_cutoff, DM_list[f], nTimesamples, DM_shift, &PD_plan, max_iteration, local_max_list_size, SPS_data.dm_step, SPS_data.dm_low, local_tsamp, SPS_data.inBin, SPS_data.time_start);
				timer.Stop();
				PF_time += timer.Elapsed();
				#ifdef GPU_PARTIAL_TIMER
				printf("    Thresholding took:%f ms\n", timer.Elapsed());
				#endif
				//-------------- Thresholding
			}
			else {
				//-------------- Peak finding
				timer.Start();
				PEAK_FIND(d_output_SNR, d_output_taps, d_peak_list, DM_list[f], nTimesamples, SPS_params->sigma_cutoff, local_max_list_size, gmem_peak_pos, DM_shift, &PD_plan, max_iteration, SPS_data.dm_step, SPS_data.dm_low, local_tsamp, SPS_data.inBin, SPS_data.time_start);
				timer.Stop();
				PF_time = timer.Elapsed();
				#ifdef GPU_PARTIAL_TIMER
				printf("    Peak finding took:%f ms\n", timer.Elapsed());
				#endif
				//-------------- Peak finding
			}
			
			checkCudaErrors(hipGetLastError());
			
			int temp_peak_pos = 0;
			checkCudaErrors(hipMemcpy(&temp_peak_pos, gmem_peak_pos, sizeof(int), hipMemcpyDeviceToHost));
			#ifdef GPU_ANALYSIS_DEBUG
			printf("    Candidates found:%d; Total #candidates for this chunk:%zu; Maximum #candidates:%zu; Local max. #candidates:%d;\n", temp_peak_pos, (*peak_pos), max_peak_size, local_max_list_size);
			#endif
			if( temp_peak_pos>=local_max_list_size ) {
				printf("    WARNING: Maximum list size reached! Not all candidates will be saved. You can increase sigma cutoff.\n");
				temp_peak_pos = local_max_list_size;
			}
			if( ((*peak_pos) + temp_peak_pos)<max_peak_size){
				checkCudaErrors(hipMemcpy(&h_peak_list[(*peak_pos)*4], d_peak_list, temp_peak_pos*4*sizeof(float), hipMemcpyDeviceToHost));
				*peak_pos = (*peak_pos) + temp_peak_pos;
			}
			else printf("    ERROR: Not enough memory to store all candidates on the host!\n");

			DM_shift = DM_shift + DM_list[f];
			hipMemset((void*) gmem_peak_pos, 0, sizeof(int));
		}
		
		hipFree(d_peak_list);
		hipFree(d_boxcar_values);
		hipFree(d_decimated);
		hipFree(d_output_SNR);
		hipFree(d_output_taps);
		hipFree(gmem_peak_pos);
		hipFree(d_MSD_DIT);
		hipFree(d_MSD_interpolated);

	}
	else printf("Error not enough memory to search for pulses\n");

	total_timer.Stop();
	total_time = total_timer.Elapsed();
	#ifdef GPU_TIMER
	printf("\n  TOTAL TIME OF SPS:%f ms\n", total_time);
	printf("  MSD_time: %f ms; SPDT time: %f ms; Candidate selection time: %f ms;\n", MSD_time, SPDT_time, PF_time);
	printf("----------<\n\n");
	#endif
	//----------> GPU part
	//---------------------------------------------------------------------------
	
}
