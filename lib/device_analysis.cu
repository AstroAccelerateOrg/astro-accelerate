#include "hip/hip_runtime.h"
//#define GPU_ANALYSIS_DEBUG

#include <vector>
#include <stdio.h>
#include <stdlib.h>
#include "headers/params.h"

#include "headers/device_BC_plan.h"
#include "headers/device_peak_find.h"
#include "headers/device_MSD_Configuration.h"
#include "headers/device_MSD.h"
#include "headers/device_MSD_legacy.h"
#include "headers/device_SPS_long.h"
#include "headers/device_threshold.h"
#include "headers/device_single_FIR.h"

#include "timer.h"

struct MSD_Data {
	int taps;
	double mean;
	double sd;
};

//---------------------------------------------------------------------------------
//-------> Kahan MSD
void d_kahan_summation(float *signal, int nDMs, int nTimesamples, int offset, float *result, float *error){
	double sum;
	double sum_error;
	double a,b;
	
	sum=0;
	sum_error=0;
	for(int d=0;d<nDMs; d++){
		for(int s=0; s<(nTimesamples-offset); s++){
			a=signal[(size_t) (d*nTimesamples + s)]-sum_error;
			b=sum+a;
			sum_error=(b-sum);
			sum_error=sum_error-a;
			sum=b;
		}
	}
	*result=sum;
	*error=sum_error;
}

void d_kahan_sd(float *signal, int nDMs, int nTimesamples, int offset, double mean, float *result, float *error){
	double sum;
	double sum_error;
	double a,b,dtemp;
	
	sum=0;
	sum_error=0;
	for(int d=0;d<nDMs; d++){
		for(int s=0; s<(nTimesamples-offset); s++){
			dtemp=(signal[(size_t) (d*nTimesamples + s)]-sum_error - mean);
			a=dtemp*dtemp;
			b=sum+a;
			sum_error=(b-sum);
			sum_error=sum_error-a;
			sum=b;
		}
	}
	*result=sum;
	*error=sum_error;
}


void MSD_Kahan(float *h_input, int nDMs, int nTimesamples, int offset, double *mean, double *sd){
	float error, signal_mean, signal_sd;
	int nElements=nDMs*(nTimesamples-offset);
	
	d_kahan_summation(h_input, nDMs, nTimesamples, offset, &signal_mean, &error);
	signal_mean=signal_mean/nElements;
	
	d_kahan_sd(h_input, nDMs, nTimesamples, offset, signal_mean, &signal_sd, &error);
	signal_sd=sqrt(signal_sd/nElements);

	*mean=signal_mean;
	*sd=signal_sd;
}

void MSD_on_GPU(float *h_input, float *d_input, float *d_MSD, float *signal_mean, float *signal_sd, float *signal_mean_bln, float *signal_sd_bln, float *signal_mean_bl_bln, float *signal_sd_bl_bln, int nDMs, int nTimesamples, int offset, float sigma_constant, float *MSD_limited_time, float *MSD_BLN_pw_time, float *MSD_BLN_grid_time){
	GpuTimer timer;
	float h_MSD[3];
	hipMemcpy( d_input, h_input, ((size_t) nDMs*nTimesamples)*sizeof(float), hipMemcpyHostToDevice);
	
	timer.Start();
	MSD_normal(d_input, d_MSD, nDMs, nTimesamples, offset);
	timer.Stop();
	(*MSD_limited_time) = timer.Elapsed();
	hipMemcpy(h_MSD, d_MSD, 3*sizeof(float), hipMemcpyDeviceToHost); 
	(*signal_mean) = h_MSD[0];
	(*signal_sd)   = h_MSD[1];
	
	
	timer.Start();
	MSD_outlier_rejection(d_input, d_MSD, nDMs, nTimesamples, offset, sigma_constant);
	timer.Stop();
	(*MSD_BLN_pw_time) = timer.Elapsed();
	hipMemcpy(h_MSD, d_MSD, 3*sizeof(float), hipMemcpyDeviceToHost); 
	(*signal_mean_bln) = h_MSD[0];
	(*signal_sd_bln)   = h_MSD[1];
	
	/*
	timer.Start();
	MSD_BLN_grid(d_input, d_MSD, 32, 32, nDMs, nTimesamples, offset, sigma_constant);
	timer.Stop();
	(*MSD_BLN_grid_time) = timer.Elapsed();
	hipMemcpy(h_MSD, d_MSD, 3*sizeof(float), hipMemcpyDeviceToHost); 
	(*signal_mean_bl_bln) = h_MSD[0];
	(*signal_sd_bl_bln)   = h_MSD[1];
	*/
}

void MSD_on_GPU_LA(float *h_input, float *d_input, float *d_MSD, float *h_MSD_LA, float *h_MSD_BLN_LA, int nDMs, int nTimesamples, int offset, float sigma_constant){
	hipMemcpy( d_input, h_input, ((size_t) nDMs*nTimesamples)*sizeof(float), hipMemcpyHostToDevice);
	
	MSD_linear_approximation(d_input, d_MSD, 32, nDMs, nTimesamples, offset);
	hipMemcpy(h_MSD_LA, d_MSD, 3*sizeof(float), hipMemcpyDeviceToHost);
	
	
	MSD_BLN_LA_pw_normal(d_input, d_MSD, 32, nDMs, nTimesamples, offset, sigma_constant);
	hipMemcpy(h_MSD_BLN_LA, d_MSD, 3*sizeof(float), hipMemcpyDeviceToHost); 
}

void MSD_on_GPU_halfed(float *h_input, float *d_input, float *d_MSD, float *signal_mean, float *signal_sd, float *signal_mean_bln, float *signal_sd_bln, int nDMs, int nTimesamples, int offset, float sigma_constant){
	float h_MSD[3];
	float *h_temp;
	int dt=nTimesamples/2;
	h_temp = new float[nDMs*dt];
	
	for(int d=0; d<nDMs; d++){
		for(int s=0; s<dt; s++){
			h_temp[d*dt + s] = h_input[d*nTimesamples + 2*s];
		}
	}
	
	hipMemcpy( d_input, h_temp, ((size_t) nDMs*dt)*sizeof(float), hipMemcpyHostToDevice);
	
	MSD_normal(d_input, d_MSD, nDMs, dt, offset/2);
	hipMemcpy(h_MSD, d_MSD, 3*sizeof(float), hipMemcpyDeviceToHost); 
	(*signal_mean) = h_MSD[0];
	(*signal_sd)   = h_MSD[1];
	
	
	MSD_outlier_rejection(d_input, d_MSD, nDMs, dt, offset/2, sigma_constant);
	hipMemcpy(h_MSD, d_MSD, 3*sizeof(float), hipMemcpyDeviceToHost); 
	(*signal_mean_bln) = h_MSD[0];
	(*signal_sd_bln)   = h_MSD[1];
	
	delete [] h_temp;
}
//-------> Kahan MSD
//---------------------------------------------------------------------------------


void Calculate_FIR(float *input, float *output, int nTaps, int nDMs, int nTimesamples, int ut) {
	int d,s,t;
	float ftemp;
	
	for(d=0; d<nDMs; d++){
		for(s=0; s<nTimesamples-ut; s++){
			ftemp=0;
			for(t=0; t<nTaps; t++){
				ftemp+=input[d*nTimesamples + s + t];
			}
			output[d*nTimesamples + s]=ftemp;
		}
	}	
}

void Decimate_in_time(float *h_input, float *h_CPU_decimate, int DIT_value, int DIT_factor, int nDMs, int nTimesamples, int offset){
	float ftemp;
	int decimated_timesamples;
	
	decimated_timesamples=nTimesamples/(DIT_value*DIT_factor);
	for(int d=0; d<nDMs; d++){
		for(int s=0; s<decimated_timesamples; s++){
			ftemp=0;
			for(int t=0; t<DIT_factor; t++){
				ftemp = ftemp + h_input[d*decimated_timesamples*DIT_factor + s*DIT_factor + t];
			}
			h_CPU_decimate[d*decimated_timesamples + s]=ftemp;
		}
	}
}

void Export_data(float *input, size_t nDMs, size_t nTimesamples, char *filename){
	FILE *fp_out;
	char mod_filename[200];
	
	sprintf(mod_filename,"%s.dat",filename);
	if (( fp_out = fopen(filename, "wb") ) == NULL)	{
		fprintf(stderr, "Error opening output file!\n");
		exit(0);
	}
	fwrite(input, (nDMs*nTimesamples)*sizeof(float), 4, fp_out);
	fclose(fp_out);
	
	for(int d=0; d<nDMs; d++){
		sprintf(mod_filename,"%s_dm%d.dat",filename,d);
		if (( fp_out = fopen(filename, "wb") ) == NULL)	{
			fprintf(stderr, "Error opening output file!\n");
			exit(0);
		}
		fwrite(&input[d*nTimesamples], nTimesamples*sizeof(float), 4, fp_out);
		fclose(fp_out);		
	}
}

void export_file_nDM_nTimesamples(float *data, int nDMs, int nTimesamples, char *filename) {
	FILE *file_out;
	char str[200];

	sprintf(str, "%s_DM.dat", filename);
	if (( file_out = fopen(str, "w") ) == NULL)	{
		fprintf(stderr, "Error opening output file!\n");
		exit(0);
	}

	printf("export nDMs\n");
	for (int s = 0; s < nTimesamples; s++) {
		for (int d = 0; d < nDMs; d++) {
			fprintf(file_out, "%f ", data[d*nTimesamples + s]);
		}
		fprintf(file_out, "\n");
	}

	fclose(file_out);

	sprintf(str, "%s_Time.dat", filename);
	if (( file_out = fopen(str, "w") ) == NULL)	{
		fprintf(stderr, "Error opening output file!\n");
		exit(0);
	}

	printf("export nTimesamples\n");
	for (int d = 0; d < nDMs; d++) {
		for (int s = 0; s < nTimesamples; s++) {
			fprintf(file_out, "%f ", data[d*nTimesamples + s]);
		}
		fprintf(file_out, "\n");
	}

	fclose(file_out);
}


//---------------------------------------------------------------------------------
//-------> Calculating MSD for whole plane

void Create_dit_MSD(float *d_data, size_t nTimesamples, size_t nDMs, std::vector<MSD_Data> *dit_MSD, std::vector<MSD_Data> *dit_MSD_BLN, int max_DIT_value, const char *filename, float sigma_constant){
	GpuTimer timer, total_timer;
	double total_time=0, dit_time=0, MSD_time=0, MSD_BLN_time=0;
	int nRest;
	MSD_Data mdtemp;
	size_t decimated_timesamples;
	int DIT_value;
	float *d_sudy, *d_lichy, *d_MSD;
	float h_MSD[MSD_RESULTS_SIZE];
	char str[200];
	hipMalloc((void **) &d_lichy, (nTimesamples>>1)*nDMs*sizeof(float));
	hipMalloc((void **) &d_sudy, (nTimesamples>>2)*nDMs*sizeof(float));
	hipMalloc((void **) &d_MSD, MSD_RESULTS_SIZE*sizeof(float));
	
	total_timer.Start();
	
	
	//----------------------------------------------------------------------------------------
	DIT_value = 1;
	printf("DiT:%d; nTimesamples:%d; decimated_timesamples:%d\n", (int) DIT_value, (int) nTimesamples, (int) (nTimesamples>>1));
	timer.Start();
	MSD_normal(d_data, d_MSD, nTimesamples, nDMs, 0);
	timer.Stop();	MSD_time += timer.Elapsed();
	hipMemcpy(h_MSD, d_MSD, MSD_RESULTS_SIZE*sizeof(float), hipMemcpyDeviceToHost);
	mdtemp.taps = DIT_value; mdtemp.mean = h_MSD[0]; mdtemp.sd = h_MSD[1];
	dit_MSD->push_back(mdtemp);
	
	timer.Start();
	MSD_outlier_rejection(d_data, d_MSD, nDMs, nTimesamples, 0, sigma_constant);
	timer.Stop();	MSD_BLN_time += timer.Elapsed();
	hipMemcpy(h_MSD, d_MSD, MSD_RESULTS_SIZE*sizeof(float), hipMemcpyDeviceToHost);
	mdtemp.taps = DIT_value; mdtemp.mean = h_MSD[0]; mdtemp.sd = h_MSD[1];
	dit_MSD_BLN->push_back(mdtemp);
	
	timer.Stop();
	total_time +=timer.Elapsed();
	//----------------------------------------------------------------------------------------
	
	//----------------------------------------------------------------------------------------
	timer.Start();
	DIT_value = DIT_value*2;
	printf("DiT:%d; nTimesamples:%d; decimated_timesamples:%d\n", (int) DIT_value, (int) nTimesamples, (int) (nTimesamples>>1));
	nRest = GPU_DiT_v2_wrapper(d_data, d_lichy, nDMs, nTimesamples);
	decimated_timesamples = (nTimesamples>>1);
	timer.Stop();	dit_time += timer.Elapsed();
	
	timer.Start();
	MSD_normal(d_lichy, d_MSD, decimated_timesamples, nDMs, nRest);
	timer.Stop();	MSD_time += timer.Elapsed();
	hipMemcpy(h_MSD, d_MSD, MSD_RESULTS_SIZE*sizeof(float), hipMemcpyDeviceToHost);
	mdtemp.taps = DIT_value; mdtemp.mean = h_MSD[0]; mdtemp.sd = h_MSD[1];
	dit_MSD->push_back(mdtemp);
	
	timer.Start();
	MSD_outlier_rejection(d_lichy, d_MSD, nDMs, decimated_timesamples, nRest, sigma_constant);
	timer.Stop();	MSD_BLN_time += timer.Elapsed();
	hipMemcpy(h_MSD, d_MSD, MSD_RESULTS_SIZE*sizeof(float), hipMemcpyDeviceToHost);
	mdtemp.taps = DIT_value; mdtemp.mean = h_MSD[0]; mdtemp.sd = h_MSD[1];
	dit_MSD_BLN->push_back(mdtemp);	
	timer.Stop();
	total_time +=timer.Elapsed();
	//----------------------------------------------------------------------------------------
	
	checkCudaErrors(hipGetLastError());
	
	for(size_t f=2; f<=max_DIT_value; f++){
		timer.Start();
		DIT_value = DIT_value*2;
		printf("DiT:%d; nTimesamples:%d; decimated_timesamples:%d\n", (int) DIT_value, (int) decimated_timesamples, (int) (decimated_timesamples>>1));
		sprintf(str,"%s_%d", filename, DIT_value);
		if(f%2==0){
			timer.Start();
			nRest = GPU_DiT_v2_wrapper(d_lichy, d_sudy, nDMs, decimated_timesamples);
			timer.Stop();	dit_time += timer.Elapsed();
			if(nRest<0) break;
			decimated_timesamples = (decimated_timesamples>>1);
			
			timer.Start();
			MSD_normal(d_sudy, d_MSD, decimated_timesamples, nDMs, nRest);
			timer.Stop();	MSD_time += timer.Elapsed();
			hipMemcpy(h_MSD, d_MSD, MSD_RESULTS_SIZE*sizeof(float), hipMemcpyDeviceToHost);
			mdtemp.taps = DIT_value; mdtemp.mean = h_MSD[0]; mdtemp.sd = h_MSD[1];
			dit_MSD->push_back(mdtemp);
			
			timer.Start();
			MSD_outlier_rejection(d_sudy, d_MSD, nDMs, decimated_timesamples, nRest, sigma_constant);
			timer.Stop();	MSD_BLN_time += timer.Elapsed();
			hipMemcpy(h_MSD, d_MSD, MSD_RESULTS_SIZE*sizeof(float), hipMemcpyDeviceToHost);
			mdtemp.taps = DIT_value; mdtemp.mean = h_MSD[0]; mdtemp.sd = h_MSD[1];
			dit_MSD_BLN->push_back(mdtemp);	
		}
		else {
			timer.Start();
			nRest = GPU_DiT_v2_wrapper(d_sudy, d_lichy, nDMs, decimated_timesamples);
			timer.Stop();	dit_time += timer.Elapsed();
			if(nRest<0) break;
			decimated_timesamples = (decimated_timesamples>>1);
			
			timer.Start();
			MSD_normal(d_lichy, d_MSD, decimated_timesamples, nDMs, nRest);
			timer.Stop();	MSD_time += timer.Elapsed();
			hipMemcpy(h_MSD, d_MSD, MSD_RESULTS_SIZE*sizeof(float), hipMemcpyDeviceToHost);
			mdtemp.taps = DIT_value; mdtemp.mean = h_MSD[0]; mdtemp.sd = h_MSD[1];
			dit_MSD->push_back(mdtemp);
			
			timer.Start();
			MSD_outlier_rejection(d_lichy, d_MSD, nDMs, decimated_timesamples, nRest, sigma_constant);
			timer.Stop();	MSD_BLN_time += timer.Elapsed();
			hipMemcpy(h_MSD, d_MSD, MSD_RESULTS_SIZE*sizeof(float), hipMemcpyDeviceToHost);
			mdtemp.taps = DIT_value; mdtemp.mean = h_MSD[0]; mdtemp.sd = h_MSD[1];
			dit_MSD_BLN->push_back(mdtemp);	
		}
		checkCudaErrors(hipGetLastError());
	}
	
	total_timer.Stop();
	total_time = total_timer.Elapsed();
	
	printf("Total time: %f; DiT time: %f; MSD time: %f; MSD BLN time: %f;\n", total_time, dit_time, MSD_time, MSD_BLN_time);
	
	checkCudaErrors(hipFree(d_sudy));
	checkCudaErrors(hipFree(d_lichy));
	checkCudaErrors(hipFree(d_MSD));
}

void Create_boxcar_MSD(float *d_data, size_t nTimesamples, size_t nDMs, std::vector<MSD_Data> *boxcar_MSD, std::vector<MSD_Data> *boxcar_MSD_BLN, int max_nTaps, float sigma_constant){
	GpuTimer timer;
	double total_time = 0;
	int nRest;
	MSD_Data mdtemp;
	float *d_boxcar, *d_MSD;
	float h_MSD[MSD_RESULTS_SIZE];
	hipMalloc((void **) &d_boxcar, nTimesamples*nDMs*sizeof(float));
	hipMalloc((void **) &d_MSD, MSD_RESULTS_SIZE*sizeof(float));
	
	timer.Start();
	
	MSD_normal(d_data, d_MSD, nDMs, nTimesamples, 0);
	hipMemcpy(h_MSD, d_MSD, MSD_RESULTS_SIZE*sizeof(float), hipMemcpyDeviceToHost);
	mdtemp.taps = 1; mdtemp.mean = h_MSD[0]; mdtemp.sd = h_MSD[1];
	boxcar_MSD->push_back(mdtemp);
	
	MSD_outlier_rejection(d_data, d_MSD, nDMs, nTimesamples, 0, sigma_constant);
	hipMemcpy(h_MSD, d_MSD, MSD_RESULTS_SIZE*sizeof(float), hipMemcpyDeviceToHost);
	mdtemp.taps = 1; mdtemp.mean = h_MSD[0]; mdtemp.sd = h_MSD[1];
	boxcar_MSD_BLN->push_back(mdtemp);
	
	timer.Stop();
	total_time = total_time + timer.Elapsed();
	printf("DIT value: %d; took %f ms; Total time %fms\n", 1, timer.Elapsed(), total_time);
	
	for(size_t f=2; f<=max_nTaps; f++){
		if( (nTimesamples-f+1)>0 ) {
			timer.Start();
			
			nRest = PD_FIR(d_data, d_boxcar, f, nDMs, nTimesamples);
			
			MSD_normal(d_boxcar, d_MSD, nDMs, nTimesamples, nRest);
			hipMemcpy(h_MSD, d_MSD, MSD_RESULTS_SIZE*sizeof(float), hipMemcpyDeviceToHost);
			mdtemp.taps = f; mdtemp.mean = h_MSD[0]; mdtemp.sd = h_MSD[1];
			boxcar_MSD->push_back(mdtemp);
			
			MSD_outlier_rejection(d_boxcar, d_MSD, nDMs, nTimesamples, nRest, sigma_constant);
			hipMemcpy(h_MSD, d_MSD, MSD_RESULTS_SIZE*sizeof(float), hipMemcpyDeviceToHost);
			mdtemp.taps = f; mdtemp.mean = h_MSD[0]; mdtemp.sd = h_MSD[1];
			boxcar_MSD_BLN->push_back(mdtemp);
			
			timer.Stop();
			total_time = total_time + timer.Elapsed();
			printf("DIT value: %d; took %f ms; Total time %fms\n", (int) f, timer.Elapsed(), total_time);
		}
	}
	
	checkCudaErrors(hipGetLastError());
	
	for(size_t f=130; f<=256; f+=4){
		printf("nTimesamples: %d; f: %d; %d\n", nTimesamples, f, nTimesamples-f+1);
		int itemp = (int) (nTimesamples-f+1);
		if( itemp>0 ) {
			timer.Start();
			
			nRest=PPF_L1(d_data, d_boxcar, nDMs, nTimesamples, f);
			
			if(nRest>0){
				MSD_normal(d_boxcar, d_MSD, nDMs, nTimesamples, nRest);
				hipMemcpy(h_MSD, d_MSD, MSD_RESULTS_SIZE*sizeof(float), hipMemcpyDeviceToHost);
				mdtemp.taps = f; mdtemp.mean = h_MSD[0]; mdtemp.sd = h_MSD[1];
				boxcar_MSD->push_back(mdtemp);
				
				MSD_outlier_rejection(d_boxcar, d_MSD, nDMs, nTimesamples, nRest, sigma_constant);
				hipMemcpy(h_MSD, d_MSD, MSD_RESULTS_SIZE*sizeof(float), hipMemcpyDeviceToHost);
				mdtemp.taps = f; mdtemp.mean = h_MSD[0]; mdtemp.sd = h_MSD[1];
				boxcar_MSD_BLN->push_back(mdtemp);
			}

			timer.Stop();
			total_time = total_time + timer.Elapsed();
			printf("DIT value: %d; took %f ms; Total time %fms\n", (int) f, timer.Elapsed(), total_time);
		}
		checkCudaErrors(hipGetLastError());
	}
	
	checkCudaErrors(hipGetLastError());
	
	for(size_t f=272; f<=512; f+=16){
		printf("nTimesamples: %d; f: %d; %d\n", nTimesamples, f, nTimesamples-f+1);
		int itemp = (int) (nTimesamples-f+1);
		if( itemp>0 ) {
			timer.Start();
			
			nRest=PPF_L1(d_data, d_boxcar, nDMs, nTimesamples, f);
			
			if(nRest>0){
				MSD_normal(d_boxcar, d_MSD, nDMs, nTimesamples, nRest);
				hipMemcpy(h_MSD, d_MSD, MSD_RESULTS_SIZE*sizeof(float), hipMemcpyDeviceToHost);
				mdtemp.taps = f; mdtemp.mean = h_MSD[0]; mdtemp.sd = h_MSD[1];
				boxcar_MSD->push_back(mdtemp);
				
				MSD_outlier_rejection(d_boxcar, d_MSD, nDMs, nTimesamples, nRest, sigma_constant);
				hipMemcpy(h_MSD, d_MSD, MSD_RESULTS_SIZE*sizeof(float), hipMemcpyDeviceToHost);
				mdtemp.taps = f; mdtemp.mean = h_MSD[0]; mdtemp.sd = h_MSD[1];
				boxcar_MSD_BLN->push_back(mdtemp);
			}
			
			timer.Stop();
			total_time = total_time + timer.Elapsed();
			printf("DIT value: %d; took %f ms; Total time %fms\n", (int) f, timer.Elapsed(), total_time);
		}
		checkCudaErrors(hipGetLastError());
	}
	
	checkCudaErrors(hipGetLastError());
	
	for(size_t f=544; f<=1024; f+=32){
		printf("nTimesamples: %d; f: %d; %d\n", nTimesamples, f, nTimesamples-f+1);
		int itemp = (int) (nTimesamples-f+1);
		if( itemp>0 ) {
			timer.Start();
			
			nRest=PPF_L1(d_data, d_boxcar, nDMs, nTimesamples, f);

			if(nRest>0){
				MSD_normal(d_boxcar, d_MSD, nDMs, nTimesamples, nRest);
				hipMemcpy(h_MSD, d_MSD, MSD_RESULTS_SIZE*sizeof(float), hipMemcpyDeviceToHost);
				mdtemp.taps = f; mdtemp.mean = h_MSD[0]; mdtemp.sd = h_MSD[1];
				boxcar_MSD->push_back(mdtemp);
				
				MSD_outlier_rejection(d_boxcar, d_MSD, nDMs, nTimesamples, nRest, sigma_constant);
				hipMemcpy(h_MSD, d_MSD, MSD_RESULTS_SIZE*sizeof(float), hipMemcpyDeviceToHost);
				mdtemp.taps = f; mdtemp.mean = h_MSD[0]; mdtemp.sd = h_MSD[1];
				boxcar_MSD_BLN->push_back(mdtemp);
			}
			
			timer.Stop();
			total_time = total_time + timer.Elapsed();
			printf("DIT value: %d; took %f ms; Total time %fms\n", (int) f, timer.Elapsed(), total_time);
		}
		checkCudaErrors(hipGetLastError());
	}
	
	checkCudaErrors(hipGetLastError());

	for(size_t f=1088; f<=2048; f+=64){
		printf("nTimesamples: %d; f: %d; %d\n", nTimesamples, f, nTimesamples-f+1);
		int itemp = (int) (nTimesamples-f+1);
		if( itemp>0 ) {
			timer.Start();
			
			nRest=PPF_L1(d_data, d_boxcar, nDMs, nTimesamples, f);

			if(nRest>0){
				MSD_normal(d_boxcar, d_MSD, nDMs, nTimesamples, nRest);
				hipMemcpy(h_MSD, d_MSD, MSD_RESULTS_SIZE*sizeof(float), hipMemcpyDeviceToHost);
				mdtemp.taps = f; mdtemp.mean = h_MSD[0]; mdtemp.sd = h_MSD[1];
				boxcar_MSD->push_back(mdtemp);
				
				MSD_outlier_rejection(d_boxcar, d_MSD, nDMs, nTimesamples, nRest, sigma_constant);
				hipMemcpy(h_MSD, d_MSD, MSD_RESULTS_SIZE*sizeof(float), hipMemcpyDeviceToHost);
				mdtemp.taps = f; mdtemp.mean = h_MSD[0]; mdtemp.sd = h_MSD[1];
				boxcar_MSD_BLN->push_back(mdtemp);
			}
			
			timer.Stop();
			total_time = total_time + timer.Elapsed();
			printf("DIT value: %d; took %f ms; Total time %fms\n", (int) f, timer.Elapsed(), total_time);
		}
		checkCudaErrors(hipGetLastError());
	}
	
	checkCudaErrors(hipGetLastError());

	for(size_t f=2176; f<=4096; f+=128){
		printf("nTimesamples: %d; f: %d; %d\n", nTimesamples, f, nTimesamples-f+1);
		int itemp = (int) (nTimesamples-f+1);
		if( itemp>0 ) {
			timer.Start();
			
			nRest=PPF_L1(d_data, d_boxcar, nDMs, nTimesamples, f);

			if(nRest>0){		
				MSD_normal(d_boxcar, d_MSD, nDMs, nTimesamples, nRest);
				hipMemcpy(h_MSD, d_MSD, MSD_RESULTS_SIZE*sizeof(float), hipMemcpyDeviceToHost);
				mdtemp.taps = f; mdtemp.mean = h_MSD[0]; mdtemp.sd = h_MSD[1];
				boxcar_MSD->push_back(mdtemp);
				
				MSD_outlier_rejection(d_boxcar, d_MSD, nDMs, nTimesamples, nRest, sigma_constant);
				hipMemcpy(h_MSD, d_MSD, MSD_RESULTS_SIZE*sizeof(float), hipMemcpyDeviceToHost);
				mdtemp.taps = f; mdtemp.mean = h_MSD[0]; mdtemp.sd = h_MSD[1];
				boxcar_MSD_BLN->push_back(mdtemp);
			}
			
			timer.Stop();
			total_time = total_time + timer.Elapsed();
			printf("DIT value: %d; took %f ms; Total time %fms\n", (int) f, timer.Elapsed(), total_time);
		}
		checkCudaErrors(hipGetLastError());
	}
	
	checkCudaErrors(hipGetLastError());
	
	for(size_t f=4352; f<=8192; f+=256){
		printf("nTimesamples: %d; f: %d; %d\n", nTimesamples, f, nTimesamples-f+1);
		int itemp = (int) (nTimesamples-f+1);
		if( itemp>0 ) {
			timer.Start();
			
			nRest=PPF_L1(d_data, d_boxcar, nDMs, nTimesamples, f);
			
			if(nRest>0){
				MSD_normal(d_boxcar, d_MSD, nDMs, nTimesamples, nRest);
				hipMemcpy(h_MSD, d_MSD, MSD_RESULTS_SIZE*sizeof(float), hipMemcpyDeviceToHost);
				mdtemp.taps = f; mdtemp.mean = h_MSD[0]; mdtemp.sd = h_MSD[1];
				boxcar_MSD->push_back(mdtemp);
				
				MSD_outlier_rejection(d_boxcar, d_MSD, nDMs, nTimesamples, nRest, sigma_constant);
				hipMemcpy(h_MSD, d_MSD, MSD_RESULTS_SIZE*sizeof(float), hipMemcpyDeviceToHost);
				mdtemp.taps = f; mdtemp.mean = h_MSD[0]; mdtemp.sd = h_MSD[1];
				boxcar_MSD_BLN->push_back(mdtemp);
			}
			
			timer.Stop();
			total_time = total_time + timer.Elapsed();
			printf("DIT value: %d; took %f ms; Total time %fms\n", (int) f, timer.Elapsed(), total_time);
		}
		checkCudaErrors(hipGetLastError());
	}
	
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipFree(d_boxcar));
	checkCudaErrors(hipFree(d_MSD));
}

void Export_MSD_data(std::vector<MSD_Data> h_dit_MSD, std::vector<MSD_Data> h_dit_MSD_BLN, std::vector<MSD_Data> h_boxcar_MSD, std::vector<MSD_Data> h_boxcar_MSD_BLN, char *filename){
	std::ofstream FILEOUT;
	FILEOUT.open (filename, std::ofstream::out);

	for(size_t f=0; f<h_dit_MSD.size(); f++){
		FILEOUT << (int) h_dit_MSD[f].taps << " " << h_dit_MSD[f].mean << " " << h_dit_MSD[f].sd << " " << "1" << std::endl;
	}
	FILEOUT << std::endl;
	FILEOUT << std::endl;
	for(size_t f=0; f<h_dit_MSD_BLN.size(); f++){
		FILEOUT << (int) h_dit_MSD_BLN[f].taps << " " << h_dit_MSD_BLN[f].mean << " " << h_dit_MSD_BLN[f].sd << " " << "2" << std::endl;
	}
	FILEOUT << std::endl;
	FILEOUT << std::endl;
	
	for(size_t f=0; f<h_boxcar_MSD.size(); f++){
		FILEOUT << (int) h_boxcar_MSD[f].taps << " " << h_boxcar_MSD[f].mean << " " << h_boxcar_MSD[f].sd << " " << "3" << std::endl;
	}
	FILEOUT << std::endl;
	FILEOUT << std::endl;
	for(size_t f=0; f<h_boxcar_MSD_BLN.size(); f++){
		FILEOUT << (int) h_boxcar_MSD_BLN[f].taps << " " << h_boxcar_MSD_BLN[f].mean << " " << h_boxcar_MSD_BLN[f].sd << " " << "4" << std::endl;
	}
	FILEOUT << std::endl;
	FILEOUT << std::endl;	
	
	FILEOUT.close();
}

void Calculate_MSD_data(float *output_buffer, size_t nTimesamples, size_t nDMs, float sigma_constant, int inBin, float dm_low, float dm_high, float tstart){
	char filename[200];
	int max_DIT_value = 13;
	int max_nTaps = 128;
	std::vector<MSD_Data> h_dit_MSD;
	std::vector<MSD_Data> h_dit_MSD_BLN;
	std::vector<MSD_Data> h_boxcar_MSD;
	std::vector<MSD_Data> h_boxcar_MSD_BLN;
	
	size_t free_mem,total_mem;
	hipMemGetInfo(&free_mem,&total_mem);
	printf("Memory available: %f; output_buffer size: %f;\n", (double) free_mem/(1024.0*1024.0), ((double) nDMs*nTimesamples*sizeof(float))/(1024.0*1024.0));
	
	Create_dit_MSD(output_buffer, nTimesamples/inBin, nDMs, &h_dit_MSD, &h_dit_MSD_BLN, max_DIT_value, filename, sigma_constant);
	Create_boxcar_MSD(output_buffer, nTimesamples/inBin, nDMs, &h_boxcar_MSD, &h_boxcar_MSD_BLN, max_nTaps, sigma_constant);
	
	
	sprintf(filename,"MSD_test-t_%.2f-dm_%.2f-%.2f.dat", tstart, dm_low, dm_high);
	Export_MSD_data(h_dit_MSD, h_dit_MSD_BLN, h_boxcar_MSD, h_boxcar_MSD_BLN, filename);
}

//-------> Calculating MSD for whole plane
//---------------------------------------------------------------------------------



// Extend this to arbitrary size plans
void Create_PD_plan(std::vector<PulseDetection_plan> *PD_plan, std::vector<int> *BC_widths, int nDMs, int nTimesamples){
	int Elements_per_block, itemp, nRest;
	PulseDetection_plan PDmp;
	
	if(BC_widths->size()>0){
		PDmp.shift        = 0;
		PDmp.output_shift = 0;
		PDmp.startTaps    = 0;
		PDmp.iteration    = 0;
		
		PDmp.decimated_timesamples = nTimesamples;
		PDmp.dtm = (nTimesamples>>(PDmp.iteration+1));
		PDmp.dtm = PDmp.dtm - (PDmp.dtm&1);
		
		PDmp.nBoxcars = BC_widths->operator[](0);
		Elements_per_block = PD_NTHREADS*2 - PDmp.nBoxcars;
		itemp = PDmp.decimated_timesamples;
		PDmp.nBlocks = itemp/Elements_per_block;
		nRest = itemp - PDmp.nBlocks*Elements_per_block;
		if(nRest>0) PDmp.nBlocks++;
		PDmp.unprocessed_samples = PDmp.nBoxcars + 6;
		if(PDmp.decimated_timesamples<PDmp.unprocessed_samples) PDmp.nBlocks=0;
		PDmp.total_ut = PDmp.unprocessed_samples;
		
		
		PD_plan->push_back(PDmp);
		
		for(int f=1; f< (int) BC_widths->size(); f++){
			// These are based on previous values of PDmp
			PDmp.shift        = PDmp.nBoxcars/2;
			PDmp.output_shift = PDmp.output_shift + PDmp.decimated_timesamples;
			PDmp.startTaps    = PDmp.startTaps + PDmp.nBoxcars*(1<<PDmp.iteration);
			PDmp.iteration    = PDmp.iteration + 1;
			
			// Definition of new PDmp values
			PDmp.decimated_timesamples = PDmp.dtm;
			PDmp.dtm = (nTimesamples>>(PDmp.iteration+1));
			PDmp.dtm = PDmp.dtm - (PDmp.dtm&1);
			
			PDmp.nBoxcars = BC_widths->operator[](f);
			Elements_per_block=PD_NTHREADS*2 - PDmp.nBoxcars;
			itemp = PDmp.decimated_timesamples;
			PDmp.nBlocks = itemp/Elements_per_block;
			nRest = itemp - PDmp.nBlocks*Elements_per_block;
			if(nRest>0) PDmp.nBlocks++;
			PDmp.unprocessed_samples = PDmp.unprocessed_samples/2 + PDmp.nBoxcars + 6; //
			if(PDmp.decimated_timesamples<PDmp.unprocessed_samples) PDmp.nBlocks=0;
			PDmp.total_ut = PDmp.unprocessed_samples*(1<<PDmp.iteration);
			
			PD_plan->push_back(PDmp);
		}
	}
}


int Get_max_iteration(int max_boxcar_width, std::vector<int> *BC_widths, int *max_width_performed){
	int startTaps, iteration;
	
	startTaps = 0;
	iteration = 0;
	for(int f=0; f<(int) BC_widths->size(); f++){
		startTaps = startTaps + BC_widths->operator[](f)*(1<<f);
		if(startTaps>=max_boxcar_width) {
			iteration = f+1;
			break;
		}
	}
	
	if(max_boxcar_width>startTaps) {
		iteration=(int) BC_widths->size();
	}
	
	*max_width_performed=startTaps;
	return(iteration);
}


void analysis_GPU(float *h_peak_list, size_t *peak_pos, size_t max_peak_size, int i, float tstart, int t_processed, int inBin, int outBin, int *maxshift, int max_ndms, int *ndms, float cutoff, float sigma_constant, float max_boxcar_width_in_sec, float *output_buffer, float *dm_low, float *dm_high, float *dm_step, float tsamp, int candidate_algorithm, int enable_sps_baselinenoise){
	int max_boxcar_width = (int) (max_boxcar_width_in_sec/tsamp);
	int max_width_performed=0;
	//unsigned long int j;
	unsigned long int vals;
	int nTimesamples = t_processed;
	int nDMs = ndms[i];
	int temp_peak_pos;
	
	//----------------------------------------------
	//--- MSD profile of the data
	//Calculate_MSD_data(output_buffer, nTimesamples, nDMs, sigma_constant, inBin, dm_low[i], dm_high[i], tstart);
	//---------------------------------------------<
	

	// Calculate the total number of values
	vals = (unsigned long int) ( nDMs*nTimesamples );
	

	double total_time=0, partial_time=0, MSD_time=0, SPDT_time=0, PF_time=0;
	
	
	//float max, min, threshold;
	int max_iteration;
	int t_BC_widths[10]={PD_MAXTAPS,16,16,16,8,8,8,8,8,8};
	std::vector<int> BC_widths(t_BC_widths,t_BC_widths+sizeof(t_BC_widths)/sizeof(int));
	std::vector<PulseDetection_plan> PD_plan;

	//---------------------------------------------------------------------------
	//----------> GPU part
	printf("\n----------> GPU analysis part\n");
	printf("     Dimensions nDMs:%d; nTimesamples:%d; inBin:%d; outBin:%d; maxshift:%d; \n", ndms[i], t_processed, inBin, outBin, *maxshift);
	GpuTimer total_timer, timer;
	total_timer.Start();
	
	//float h_MSD[3];
	float *d_MSD;
	checkCudaErrors(hipGetLastError());
	if ( hipSuccess != hipMalloc((void**) &d_MSD, sizeof(float)*3)) {printf("Allocation error!\n"); exit(201);}
	
	/*
	//-------------- CPU check
	float *h_temp, *h_boxcar;
	double signal_mean, signal_sd;
	h_temp = (float *)malloc( ((size_t) nDMs*nTimesamples)*sizeof(float));
	h_boxcar = (float *)malloc( ((size_t) nDMs*nTimesamples)*sizeof(float));
	memset(h_temp, 0.0, ((size_t) nDMs*nTimesamples)*sizeof(float));
	memset(h_boxcar, 0.0, ((size_t) nDMs*nTimesamples)*sizeof(float));
	hipMemcpy( h_temp, output_buffer, ((size_t) nDMs*nTimesamples)*sizeof(float), hipMemcpyDeviceToHost);
	MSD_Kahan(h_temp, nDMs, nTimesamples, 0, &signal_mean, &signal_sd);
	printf("MSD_kahan: after 1 tap   Mean: %e, Standard deviation: %e;\n",signal_mean, signal_sd);
	Calculate_FIR(h_temp, h_boxcar, 2, nDMs, nTimesamples, 0);
	MSD_Kahan(h_boxcar, nDMs, nTimesamples, 0, &signal_mean, &signal_sd);
	printf("MSD_kahan: after 1 tap   Mean: %e, Standard deviation: %e;\n",signal_mean, signal_sd);	
	free(h_temp);
	free(h_boxcar);
	//-------------- CPU check
	*/
	
	/*
	//-------------- One Call linear approximation
	timer.Start(); 
	MSD_linear_approximation(output_buffer, d_MSD, PD_MAXTAPS, nDMs, nTimesamples, 0);
	timer.Stop();
	partial_time = timer.Elapsed(); 
	hipMemcpy(h_MSD, d_MSD, 3*sizeof(float), hipMemcpyDeviceToHost); 
	printf("     MSD linear approximation: Mean: %f, Stddev: %f, modifier: %f\n", h_MSD[0], h_MSD[1], h_MSD[2]);
	#ifdef GPU_ANALYSIS_DEBUG
	printf("     One kernel took:%f ms\n", partial_time); 
	#endif
	//-------------- One Call linear approximation
	*/
	
	/*
	//-------------- Base level noise point-wise
	timer.Start(); 
	MSD_outlier_rejection(output_buffer, d_MSD, nDMs, nTimesamples, 0, sigma_constant);
	timer.Stop();
	partial_time = timer.Elapsed(); 
	hipMemcpy(h_MSD, d_MSD, 3*sizeof(float), hipMemcpyDeviceToHost); 
	printf("     MSD BLN point-wise: Mean: %f, Stddev: %f, modifier: %f\n", h_MSD[0], h_MSD[1], h_MSD[2]);
	#ifdef GPU_ANALYSIS_DEBUG
	printf("     MSD BLN point-wise kernel took:%f ms\n", partial_time); 
	#endif
	//-------------- Base level noise point-wise
	*/

	/*
	//-------------- BLN_LA
	timer.Start(); 
	MSD_BLN_LA_pw_normal(output_buffer, d_MSD, nDMs, nTimesamples, PD_MAXTAPS, 0, sigma_constant);
	timer.Stop();
	partial_time = timer.Elapsed(); 
	hipMemcpy(h_MSD, d_MSD, 3*sizeof(float), hipMemcpyDeviceToHost); 
	printf("     MSD BLN linear approximation: Mean: %f, Stddev: %f, modifier: %f\n", h_MSD[0], h_MSD[1], h_MSD[2]);
	#ifdef GPU_ANALYSIS_DEBUG
	printf("     BLN LA took:%f ms\n", partial_time); 
	#endif
	//-------------- BLN_LA
	*/
	
	/*
	//-------------- Base level noise grid
	timer.Start(); 
	MSD_BLN_grid(output_buffer, d_MSD, 32, 32, nDMs, nTimesamples, 0, sigma_constant);
	timer.Stop();
	partial_time = timer.Elapsed(); 
	hipMemcpy(h_MSD, d_MSD, 3*sizeof(float), hipMemcpyDeviceToHost); 
	printf("     MSD BLN grid: Mean: %f, Stddev: %f, modifier: %f\n", h_MSD[0], h_MSD[1], h_MSD[2]);
	#ifdef GPU_ANALYSIS_DEBUG
	printf("     MSD BLN grid kernel took:%f ms\n", partial_time); 
	#endif
	//-------------- Base level noise grid
	*/
	
	size_t free_mem,total_mem;
	hipMemGetInfo(&free_mem,&total_mem);
	printf("     Memory required by boxcar filters:%0.3f MB\n",(4.5*vals*sizeof(float) + 2*vals*sizeof(ushort))/(1024.0*1024) );
	printf("     Memory available:%0.3f MB \n", ((float) free_mem)/(1024.0*1024.0) );
	
	std::vector<int> DM_list;
	unsigned long int max_timesamples=(free_mem*0.95)/(5.5*sizeof(float) + 2*sizeof(ushort));
	int DMs_per_cycle = max_timesamples/nTimesamples;
	int nRepeats, nRest, DM_shift, itemp, local_max_list_size;//BC_shift,
	
	itemp = (int) (DMs_per_cycle/THR_WARPS_PER_BLOCK);
	DMs_per_cycle = itemp*THR_WARPS_PER_BLOCK;
	
	nRepeats = nDMs/DMs_per_cycle;
	nRest = nDMs - nRepeats*DMs_per_cycle;
	local_max_list_size = (DMs_per_cycle*nTimesamples)/4;
	
	for(int f=0; f<nRepeats; f++) DM_list.push_back(DMs_per_cycle);
	if(nRest>0) DM_list.push_back(nRest);
	
	printf("     SPS will run %d batches each containing %d DM trials. Remainder %d DM trials\n", (int) DM_list.size(), DMs_per_cycle, nRest);
	
	
	max_iteration = Get_max_iteration(max_boxcar_width/inBin, &BC_widths, &max_width_performed);
	printf("     Selected iteration:%d; maximum boxcar width requested:%d; maximum boxcar width performed:%d;\n", max_iteration, max_boxcar_width/inBin, max_width_performed);
	Create_PD_plan(&PD_plan, &BC_widths, 1, nTimesamples);
	
	if(DM_list.size()>0){
		DMs_per_cycle = DM_list[0];
		
		float *d_peak_list;
		if ( hipSuccess != hipMalloc((void**) &d_peak_list, sizeof(float)*DMs_per_cycle*nTimesamples)) printf("Allocation error! peaks\n");
		
		float *d_decimated;
		if ( hipSuccess != hipMalloc((void **) &d_decimated,  sizeof(float)*(((DMs_per_cycle*nTimesamples)/2)+PD_MAXTAPS) )) printf("Allocation error! dedispered\n");
		
		float *d_boxcar_values;
		if ( hipSuccess != hipMalloc((void **) &d_boxcar_values,  sizeof(float)*DMs_per_cycle*nTimesamples)) printf("Allocation error! boxcars\n");
		
		float *d_output_SNR;
		if ( hipSuccess != hipMalloc((void **) &d_output_SNR, sizeof(float)*2*DMs_per_cycle*nTimesamples)) printf("Allocation error! SNR\n");
		
		ushort *d_output_taps;
		if ( hipSuccess != hipMalloc((void **) &d_output_taps, sizeof(ushort)*2*DMs_per_cycle*nTimesamples)) printf("Allocation error! taps\n");
		
		int *gmem_peak_pos;
		hipMalloc((void**) &gmem_peak_pos, 1*sizeof(int));
		hipMemset((void*) gmem_peak_pos, 0, sizeof(int));
		
		DM_shift = 0;
		for(int f=0; f<DM_list.size(); f++) {
			//-------------- SPS BLN
			timer.Start();
			//PD_SEARCH_LONG_BLN(&output_buffer[DM_shift*nTimesamples], d_boxcar_values, d_decimated, d_output_SNR, d_output_taps, d_MSD, &PD_plan, max_iteration, DM_list[f], nTimesamples);
			//PD_SEARCH_LONG_BLN_EACH(&output_buffer[DM_shift*nTimesamples], d_boxcar_values, d_decimated, d_output_SNR, d_output_taps, &PD_plan, max_iteration, DM_list[f], nTimesamples, sigma_constant);
			//PD_SEARCH_LONG_LINAPPROX(&output_buffer[DM_shift*nTimesamples], d_boxcar_values, d_decimated, d_output_SNR, d_output_taps, d_MSD, &PD_plan, max_iteration, DM_list[f], nTimesamples);
			if(enable_sps_baselinenoise){
				PD_SEARCH_LONG_BLN_LINAPPROX_EACH(&output_buffer[DM_shift*nTimesamples], d_boxcar_values, d_decimated, d_output_SNR, d_output_taps, &PD_plan, max_iteration, DM_list[f], nTimesamples, sigma_constant, &MSD_time, &SPDT_time);
			}
			else {
				PD_SEARCH_LONG_LINAPPROX_EACH(&output_buffer[DM_shift*nTimesamples], d_boxcar_values, d_decimated, d_output_SNR, d_output_taps, &PD_plan, max_iteration, DM_list[f], nTimesamples, &MSD_time, &SPDT_time);
			}
			//
			timer.Stop();
			partial_time = timer.Elapsed();
			#ifdef GPU_ANALYSIS_DEBUG
			printf("PD_SEARCH took:%f ms\n", partial_time);
			#endif
			//-------------- SPS BLN
			
			checkCudaErrors(hipGetLastError());
			
			#ifdef GPU_ANALYSIS_DEBUG
			printf("BC_shift:%d; DMs_per_cycle:%d; f*DMs_per_cycle:%d; max_iteration:%d;\n", DM_shift*nTimesamples, DM_list[f], DM_shift, max_iteration);
			#endif
			
			if(candidate_algorithm==1){
				//-------------- Thresholding
				timer.Start();
				THRESHOLD(d_output_SNR, d_output_taps, d_peak_list, gmem_peak_pos, cutoff, DM_list[f], nTimesamples, DM_shift, &PD_plan, max_iteration, local_max_list_size);
				timer.Stop();
				partial_time = timer.Elapsed();
				PF_time = timer.Elapsed();
				#ifdef GPU_ANALYSIS_DEBUG
				printf("THR_WARP took:%f ms\n", partial_time);
				#endif
				//-------------- Thresholding
			}
			else {
				//-------------- Peak finding
				timer.Start();
				PEAK_FIND(d_output_SNR, d_output_taps, d_peak_list, DM_list[f], nTimesamples, cutoff, local_max_list_size, gmem_peak_pos, DM_shift, &PD_plan, max_iteration);
				timer.Stop();
				partial_time = timer.Elapsed();
				PF_time = timer.Elapsed();
				#ifdef GPU_ANALYSIS_DEBUG
				printf("PEAK_FIND took:%f ms\n", partial_time);
				#endif
				//-------------- Peak finding
			}
			
			checkCudaErrors(hipGetLastError());
			
			checkCudaErrors(hipMemcpy(&temp_peak_pos, gmem_peak_pos, sizeof(int), hipMemcpyDeviceToHost));
			#ifdef GPU_ANALYSIS_DEBUG
			printf("temp_peak_pos:%d; host_pos:%zu; max:%zu; local_max:%d;\n", temp_peak_pos, (*peak_pos), max_peak_size, local_max_list_size);
			#endif
			if( temp_peak_pos>=local_max_list_size ) {
				printf("     Maximum list size reached! Increase list size or increase sigma cutoff.\n");
				temp_peak_pos=local_max_list_size;
			}
			if( ((*peak_pos) + temp_peak_pos)<max_peak_size){
				checkCudaErrors(hipMemcpy(&h_peak_list[(*peak_pos)*4], d_peak_list, temp_peak_pos*4*sizeof(float), hipMemcpyDeviceToHost));
				*peak_pos = (*peak_pos) + temp_peak_pos;
			}
			else printf("Error peak list is too small!\n");
			

			//---------> Old thresholding code.
			//#ifdef OLD_THRESHOLD
			//#endif
			//---------> Old thresholding code.

			DM_shift = DM_shift + DM_list[f];
			hipMemset((void*) gmem_peak_pos, 0, sizeof(int));
		}
		
		//------------------------> Output
		#pragma omp parallel for
		for (int count = 0; count < (*peak_pos); count++){
			h_peak_list[4*count]     = h_peak_list[4*count]*dm_step[i] + dm_low[i];
			h_peak_list[4*count + 1] = h_peak_list[4*count + 1]*tsamp + tstart;
		}
        
		FILE *fp_out;
		char filename[200];
		
		if(candidate_algorithm==1){
			if((*peak_pos)>0){
				sprintf(filename, "analysed-t_%.2f-dm_%.2f-%.2f.dat", tstart, dm_low[i], dm_high[i]);
				if (( fp_out = fopen(filename, "wb") ) == NULL)	{
					fprintf(stderr, "Error opening output file!\n");
					exit(0);
				}
				fwrite(h_peak_list, (*peak_pos)*sizeof(float), 4, fp_out);
				fclose(fp_out);
			}
		}
		else {
			if((*peak_pos)>0){
				sprintf(filename, "peak_analysed-t_%.2f-dm_%.2f-%.2f.dat", tstart, dm_low[i], dm_high[i]);
				if (( fp_out = fopen(filename, "wb") ) == NULL)	{
					fprintf(stderr, "Error opening output file!\n");
					exit(0);
				}
				fwrite(h_peak_list, (*peak_pos)*sizeof(float), 4, fp_out);
				fclose(fp_out);
			}
		}
		//------------------------> Output
		
		hipFree(d_peak_list);
		hipFree(d_boxcar_values);
		hipFree(d_decimated);
		hipFree(d_output_SNR);
		hipFree(d_output_taps);
		hipFree(gmem_peak_pos);

	}
	else printf("Error not enough memory to search for pulses\n");

	total_timer.Stop();
	total_time = total_timer.Elapsed();
	printf("\n     TOTAL TIME OF SPS:%f ms\n", total_time);
	printf("\n     MSD_time: %f ms; SPDT time: %f ms; Peak find time: %f ms;\n", MSD_time, SPDT_time, PF_time);
	printf("----------<\n\n");
	total_time += partial_time;

	hipFree(d_MSD);
	//----------> GPU part
	//---------------------------------------------------------------------------
	
}
