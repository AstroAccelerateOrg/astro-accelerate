#include "hip/hip_runtime.h"
//#define GPU_ANALYSIS_DEBUG

#include <vector>
#include <stdio.h>
#include <stdlib.h>
#include "headers/params.h"

#include "headers/device_BC_plan.h"
#include "headers/device_peak_find.h"
#include "headers/device_BLN.h"
#include "headers/device_MSD_limited.h"
#include "headers/device_SPS_long.h"
#include "headers/device_threshold.h"
#include "headers/device_single_FIR.h"

#include "timer.h"

void Create_PD_plan(std::vector<PulseDetection_plan> *PD_plan, std::vector<int> *BC_widths, int nDMs, int nTimesamples){
	int Elements_per_block, itemp, nRest;
	PulseDetection_plan PDmp;
	
	if(BC_widths->size()>0){
		PDmp.shift        = 0;
		PDmp.output_shift = 0;
		PDmp.startTaps    = 0;
		PDmp.iteration    = 0;
		
		PDmp.decimated_timesamples = nTimesamples;
		PDmp.dtm = (nTimesamples>>(PDmp.iteration+1));
		PDmp.dtm = PDmp.dtm - (PDmp.dtm&1);
		
		PDmp.nBoxcars = BC_widths->operator[](0);
		Elements_per_block = PD_NTHREADS*2 - PDmp.nBoxcars;
		itemp = PDmp.decimated_timesamples;
		PDmp.nBlocks = itemp/Elements_per_block;
		nRest = itemp - PDmp.nBlocks*Elements_per_block;
		if(nRest>0) PDmp.nBlocks++;
		PDmp.unprocessed_samples = PDmp.nBoxcars + 6;
		if(PDmp.decimated_timesamples<PDmp.unprocessed_samples) PDmp.nBlocks=0;
		PDmp.total_ut = PDmp.unprocessed_samples;
		
		
		PD_plan->push_back(PDmp);
		
		for(int f=1; f< (int) BC_widths->size(); f++){
			// These are based on previous values of PDmp
			PDmp.shift        = PDmp.nBoxcars/2;
			PDmp.output_shift = PDmp.output_shift + PDmp.decimated_timesamples;
			PDmp.startTaps    = PDmp.startTaps + PDmp.nBoxcars*(1<<PDmp.iteration);
			PDmp.iteration    = PDmp.iteration + 1;
			
			// Definition of new PDmp values
			PDmp.decimated_timesamples = PDmp.dtm;
			PDmp.dtm = (nTimesamples>>(PDmp.iteration+1));
			PDmp.dtm = PDmp.dtm - (PDmp.dtm&1);
			
			PDmp.nBoxcars = BC_widths->operator[](f);
			Elements_per_block=PD_NTHREADS*2 - PDmp.nBoxcars;
			itemp = PDmp.decimated_timesamples;
			PDmp.nBlocks = itemp/Elements_per_block;
			nRest = itemp - PDmp.nBlocks*Elements_per_block;
			if(nRest>0) PDmp.nBlocks++;
			PDmp.unprocessed_samples = PDmp.unprocessed_samples/2 + PDmp.nBoxcars + 6; //
			if(PDmp.decimated_timesamples<PDmp.unprocessed_samples) PDmp.nBlocks=0;
			PDmp.total_ut = PDmp.unprocessed_samples*(1<<PDmp.iteration);
			
			PD_plan->push_back(PDmp);
		}
	}
}


int Get_max_iteration(int max_boxcar_width, std::vector<int> *BC_widths){
	int startTaps, iteration;
	
	startTaps = 0;
	iteration = 0;
	for(int f=0; f<(int) BC_widths->size(); f++){
		startTaps = startTaps + BC_widths->operator[](f)*(1<<f);
		if(startTaps>=max_boxcar_width) {
			iteration = f+1;
			break;
		}
	}
	
	if(max_boxcar_width>startTaps) iteration=(int) BC_widths->size();
	
	return(1);
}

void analysis_GPU(float *h_peak_list, size_t *peak_pos, size_t max_peak_size, int i, float tstart, int t_processed, int inBin, int outBin, int *maxshift, int max_ndms, int *ndms, float cutoff, float sigma_constant, float max_boxcar_width_in_sec, float *output_buffer, float *dm_low, float *dm_high, float *dm_step, float tsamp, int candidate_algorithm){
	int max_boxcar_width = (int) (max_boxcar_width_in_sec/tsamp);
	//unsigned long int j;
	unsigned long int vals;
	int nTimesamples = t_processed;
	int nDMs = ndms[i];
	int  temp_peak_pos;
	//double total;

	// Calculate the total number of values
	vals = (unsigned long int) ( nDMs*nTimesamples );
	

	double total_time, partial_time;
	
	//float max, min, threshold;
	int offset, max_iteration;
	int t_BC_widths[10]={PD_MAXTAPS,16,16,16,8,8,8,8,8,8};
	//int t_BC_widths[10]={PD_MAXTAPS,32,32,32,32,32,32,32,32,32};
	std::vector<int> BC_widths(t_BC_widths,t_BC_widths+sizeof(t_BC_widths)/sizeof(int));
	std::vector<PulseDetection_plan> PD_plan;

	//---------------------------------------------------------------------------
	//----------> GPU part
	printf("\n----------> GPU analysis part\n\n");
	printf("     Dimensions nDMs:%d; nTimesamples:%d; inBin:%d; outBin:%d; maxshift:%d; \n", ndms[i], t_processed, inBin, outBin, *maxshift);
	GpuTimer timer;
	
	float h_MSD[3];
	float *d_MSD;
	checkCudaErrors(hipGetLastError());
	if ( hipSuccess != hipMalloc((void**) &d_MSD, sizeof(float)*3)) {printf("Allocation error!\n"); exit(201);}
	

	total_time = 0;
	
	//-------------- Calculating base level noise using outlier rejection
	//timer.Start();
	//BLN(output_buffer, d_MSD, 32, 32, nDMs, nTimesamples, 0, sigma_constant); // Those 128 are there because there was a problem with data, I'm not sure if it is still the case.
	//timer.Stop();
	//partial_time = timer.Elapsed();
	//total_time += partial_time;
	//printf("MSD limited took:%f ms\n", partial_time);
	//
	//hipMemcpy(h_MSD, d_MSD, 3*sizeof(float), hipMemcpyDeviceToHost);
	//signal_mean_1 = h_MSD[0];
	//signal_sd_1 = h_MSD[1];
	//printf("Bin: %d, Mean: %f, Stddev: %f\n", 1, signal_mean_1, signal_sd_1);
	//-------------- Calculating base level noise using outlier rejection
	
	/*
	//-------------- Linear approximation
	float signal_mean_1, signal_sd_1;
	float *d_list;
	size_t mem_size;
	mem_size =sizeof(float)*(size_t)nDMs*(size_t)nTimesamples;
	if ( hipSuccess != hipMalloc((void **) &d_list, mem_size)) printf("Allocation error! SNR\n");
	
	float signal_mean_16, signal_sd_16;
	timer.Start(); 
	MSD_limited(output_buffer, d_MSD, nDMs, nTimesamples, 128); 
	timer.Stop(); 
	partial_time = timer.Elapsed(); 
	total_time += partial_time; 
	printf("MSD limited took:%f ms\n", partial_time); 
	
	
	timer.Start(); 
	hipMemcpy(h_MSD, d_MSD, 3*sizeof(float), hipMemcpyDeviceToHost); 
	signal_mean_1 = h_MSD[0]; 
	signal_sd_1 = h_MSD[1]; 
	printf("MSD Bin: %d, Mean: %f, Stddev: %f\n", 1, signal_mean_1, signal_sd_1); 
	
	
	offset = PD_FIR(output_buffer, d_list, PD_MAXTAPS, nDMs, nTimesamples); 
	MSD_limited(d_list, d_MSD, nDMs, nTimesamples, offset); 
	hipMemcpy(h_MSD, d_MSD, 3*sizeof(float), hipMemcpyDeviceToHost); 
	signal_mean_16 = h_MSD[0]; 
	signal_sd_16 = h_MSD[1]; 
	printf("MSD Bin: %d, Mean: %f, Stddev: %f\n", PD_MAXTAPS, signal_mean_16, signal_sd_16); 
	
	
	h_MSD[0] = signal_mean_1; 
	h_MSD[2] = ( signal_sd_16 - signal_sd_1 )/( (float) ( PD_MAXTAPS - 1 ) ); 
	h_MSD[1] = signal_sd_1; 
	printf("Final: Mean: %f, Stddev: %f, modifier: %f\n", h_MSD[0], h_MSD[1], h_MSD[2]);
	hipMemcpy(d_MSD, h_MSD, 3*sizeof(float), hipMemcpyHostToDevice); 
	
	timer.Stop(); 
	partial_time = timer.Elapsed(); 
	total_time += partial_time; 
	printf("Linear sd took:%f ms\n", total_time); 
	
	hipFree(d_list);
	//-------------- Linear approximation
	*/
	
	//-------------- One Call linear approximation
	timer.Start(); 
	MSD_linear_approximation(output_buffer, d_MSD, PD_MAXTAPS, nDMs, nTimesamples, 0);
	timer.Stop();
	partial_time = timer.Elapsed(); 
	total_time += partial_time; 
	hipMemcpy(h_MSD, d_MSD, 3*sizeof(float), hipMemcpyDeviceToHost); 
	printf("     MSD linear approximation: Mean: %f, Stddev: %f, modifier: %f\n", h_MSD[0], h_MSD[1], h_MSD[2]);
	#ifdef GPU_ANALYSIS_DEBUG
	printf("     One kernel took:%f ms\n", partial_time); 
	#endif
	//-------------- One Call linear approximation
	
	
	
	
	size_t free_mem,total_mem;
	hipMemGetInfo(&free_mem,&total_mem);
	printf("     Memory required by boxcar filters:%0.3f MB\n",(4.5*vals*sizeof(float) + 2*vals*sizeof(ushort))/(1024.0*1024) );
	printf("     Memory available:%0.3f MB \n", ((float) free_mem)/(1024.0*1024.0) );
	
	std::vector<int> DM_list;
	unsigned long int max_timesamples=(free_mem*0.95)/(5.5*sizeof(float) + 2*sizeof(ushort));
	int DMs_per_cycle = max_timesamples/nTimesamples;
	int nRepeats, nRest, DM_shift, itemp, local_max_list_size;//BC_shift,
	
	itemp = (int) (DMs_per_cycle/THR_WARPS_PER_BLOCK);
	DMs_per_cycle = itemp*THR_WARPS_PER_BLOCK;
	
	nRepeats = nDMs/DMs_per_cycle;
	nRest = nDMs - nRepeats*DMs_per_cycle;
	local_max_list_size = (DMs_per_cycle*nTimesamples)/4;
	
	for(int f=0; f<nRepeats; f++) DM_list.push_back(DMs_per_cycle);
	if(nRest>0) DM_list.push_back(nRest);
	
	printf("     SPS will run %d batches each containing %d DM trials. Remainder %d DM trials\n", (int) DM_list.size(), DMs_per_cycle, nRest);
	
	
	max_iteration = Get_max_iteration(max_boxcar_width/inBin, &BC_widths);
	printf("     Selected iteration:%d; for maximum boxcar width:%d;\n", max_iteration, max_boxcar_width/inBin);
	Create_PD_plan(&PD_plan, &BC_widths, 1, nTimesamples);
	
	if(DM_list.size()>0){
		DMs_per_cycle = DM_list[0];
		
		float *d_peak_list;
		if ( hipSuccess != hipMalloc((void**) &d_peak_list, sizeof(float)*DMs_per_cycle*nTimesamples)) printf("Allocation error! peaks\n");
		
		float *d_decimated;
		if ( hipSuccess != hipMalloc((void **) &d_decimated,  sizeof(float)*((DMs_per_cycle*nTimesamples)/2))) printf("Allocation error! dedispered\n");
		
		float *d_boxcar_values;
		if ( hipSuccess != hipMalloc((void **) &d_boxcar_values,  sizeof(float)*DMs_per_cycle*nTimesamples)) printf("Allocation error! boxcars\n");
		
		float *d_output_SNR;
		if ( hipSuccess != hipMalloc((void **) &d_output_SNR, sizeof(float)*2*DMs_per_cycle*nTimesamples)) printf("Allocation error! SNR\n");
		
		ushort *d_output_taps;
		if ( hipSuccess != hipMalloc((void **) &d_output_taps, sizeof(ushort)*2*DMs_per_cycle*nTimesamples)) printf("Allocation error! taps\n");
		
		int *gmem_peak_pos;
		hipMalloc((void**) &gmem_peak_pos, 1*sizeof(int));
		hipMemset((void*) gmem_peak_pos, 0, sizeof(int));
		
		DM_shift = 0;
		for(int f=0; f<DM_list.size(); f++) {
			//-------------- SPS BLN
			timer.Start();
			//PD_SEARCH_LONG_BLN_IF(&output_buffer[DM_shift*nTimesamples], d_boxcar_values, d_decimated, d_output_SNR, d_output_taps, d_MSD, &PD_plan, max_iteration, DM_list[f], nTimesamples);
			PD_SEARCH_LONG_BLN_IF_LINAPPROX(&output_buffer[DM_shift*nTimesamples], d_boxcar_values, d_decimated, d_output_SNR, d_output_taps, d_MSD, &PD_plan, max_iteration, DM_list[f], nTimesamples);
			timer.Stop();
			partial_time = timer.Elapsed();
			total_time += partial_time;
			#ifdef GPU_ANALYSIS_DEBUG
			printf("PD_SEARCH took:%f ms\n", partial_time);
			#endif
			//-------------- SPS BLN

			#ifdef GPU_ANALYSIS_DEBUG
			printf("BC_shift:%d; DMs_per_cycle:%d; f*DMs_per_cycle:%d; max_iteration:%d; offset:%d;\n", DM_shift*nTimesamples, DM_list[f], DM_shift, max_iteration, offset);
			#endif
			
			if(candidate_algorithm==1){
				//-------------- Thresholding
				timer.Start();
				THRESHOLD(d_output_SNR, d_output_taps, d_peak_list, gmem_peak_pos, cutoff, DM_list[f], nTimesamples, DM_shift, &PD_plan, max_iteration, local_max_list_size);
				timer.Stop();
				partial_time = timer.Elapsed();
				total_time += partial_time;
				#ifdef GPU_ANALYSIS_DEBUG
				printf("THR_WARP took:%f ms\n", partial_time);
				#endif
				//-------------- Thresholding
			}
			else {
				//-------------- Peak finding
				timer.Start();
				PEAK_FIND(d_output_SNR, d_output_taps, d_peak_list, DM_list[f], nTimesamples, cutoff, local_max_list_size, gmem_peak_pos, DM_shift, &PD_plan, max_iteration);
				timer.Stop();
				partial_time = timer.Elapsed();
				total_time += partial_time;
				#ifdef GPU_ANALYSIS_DEBUG
				printf("PEAK_FIND took:%f ms\n", partial_time);
				#endif
				//-------------- Peak finding
			}
			
			hipMemcpy(&temp_peak_pos, gmem_peak_pos, sizeof(int), hipMemcpyDeviceToHost);
			#ifdef GPU_ANALYSIS_DEBUG
			printf("temp_peak_pos:%d; host_pos:%d; max:%d;\n", temp_peak_pos, (*peak_pos), (int) max_peak_size);
			#endif
			if( ((*peak_pos) + temp_peak_pos)<max_peak_size){
				hipMemcpy(&h_peak_list[(*peak_pos)*4], d_peak_list, temp_peak_pos*4*sizeof(float), hipMemcpyDeviceToHost);
				*peak_pos = (*peak_pos) + temp_peak_pos;
			}
			else printf("Error peak list is too small!\n");
			

			//---------> Old thresholding code.
			//#ifdef OLD_THRESHOLD
			//#endif
			//---------> Old thresholding code.

			DM_shift = DM_shift + DM_list[f];
			hipMemset((void*) gmem_peak_pos, 0, sizeof(int));
		}
		
		//------------------------> Output
		#pragma omp parallel for
		for (int count = 0; count < (*peak_pos); count++){
			h_peak_list[4*count]     = h_peak_list[4*count]*dm_step[i] + dm_low[i];
			h_peak_list[4*count + 1] = h_peak_list[4*count + 1]*tsamp + tstart;
		}
        
		FILE *fp_out;
		char filename[200];
		
		if(candidate_algorithm==1){
			if((*peak_pos)>0){
				sprintf(filename, "analysed-t_%.2f-dm_%.2f-%.2f.dat", tstart, dm_low[i], dm_high[i]);
				if (( fp_out = fopen(filename, "wb") ) == NULL)	{
					fprintf(stderr, "Error opening output file!\n");
					exit(0);
				}
				fwrite(h_peak_list, (*peak_pos)*sizeof(float), 4, fp_out);
				fclose(fp_out);
			}
		}
		else {
			if((*peak_pos)>0){
				sprintf(filename, "peak_analysed-t_%.2f-dm_%.2f-%.2f.dat", tstart, dm_low[i], dm_high[i]);
				if (( fp_out = fopen(filename, "wb") ) == NULL)	{
					fprintf(stderr, "Error opening output file!\n");
					exit(0);
				}
				fwrite(h_peak_list, (*peak_pos)*sizeof(float), 4, fp_out);
				fclose(fp_out);
			}
		}
		//------------------------> Output
		
		hipFree(d_peak_list);
		hipFree(d_boxcar_values);
		hipFree(d_decimated);
		hipFree(d_output_SNR);
		hipFree(d_output_taps);
		hipFree(gmem_peak_pos);

	}
	else printf("Error not enough memory to search for pulses\n");

	
	printf("\n====> TOTAL TIME OF SPS:%f\n\n", total_time);

	hipFree(d_MSD);
	//----------> GPU part
	//---------------------------------------------------------------------------
	
}
