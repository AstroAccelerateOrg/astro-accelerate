//Added by Karel Adamek
<<<<<<< HEAD

#include "AstroAccelerate/params.h"
#include "device_MSD_grid_kernel.cu"


void MSD_grid_init(void){
=======
#include "AstroAccelerate/params.h"
#include "device_MSD_grid_kernel.cu"

void MSD_grid_init(void)
{
>>>>>>> fe80b9c735d1c898047cbb64bcf8da05cd6a21da
	//---------> Specific nVidia stuff
	hipDeviceSetCacheConfig(hipFuncCachePreferNone);
	hipDeviceSetSharedMemConfig(hipSharedMemBankSizeFourByte);
}

<<<<<<< HEAD

int MSD_grid(float *d_input, float *d_output, int CellDim_x, int CellDim_y, int nDMs, int nTimesamples){
	//---------> Task specific
	int GridSize_x, GridSize_y, x_steps, y_steps, nThreads;
	GridSize_x=nTimesamples/CellDim_x;
	GridSize_y=nDMs/CellDim_y;
	x_steps=CellDim_x/WARP;
	if(CellDim_y<16) {
		y_steps=1;
		nThreads=WARP*CellDim_y;
	}
	else {
		nThreads=WARP*16;
		y_steps=CellDim_y/16;
	}
		
	//---------> Initial phase
	int nCUDAblocks_x=GridSize_x;
	int nCUDAblocks_y=GridSize_y;
=======
int MSD_grid(float *d_input, float *d_output, int CellDim_x, int CellDim_y, int nDMs, int nTimesamples)
{
	//---------> Task specific
	int GridSize_x, GridSize_y, x_steps, y_steps, nThreads;
	GridSize_x = nTimesamples/CellDim_x;
	GridSize_y = nDMs/CellDim_y;
	x_steps = CellDim_x/WARP;
	if(CellDim_y<16)
	{
		y_steps  = 1;
		nThreads = WARP*CellDim_y;
	}
	else
	{
		nThreads = WARP*16;
		y_steps  = CellDim_y/16;
	}
		
	//---------> Initial phase
	int nCUDAblocks_x = GridSize_x;
	int nCUDAblocks_y = GridSize_y;
>>>>>>> fe80b9c735d1c898047cbb64bcf8da05cd6a21da
	
	dim3 gridSize(nCUDAblocks_x, nCUDAblocks_y, 1);
	dim3 blockSize(nThreads, 1, 1);
	
<<<<<<< HEAD
	
=======
>>>>>>> fe80b9c735d1c898047cbb64bcf8da05cd6a21da
	//---------> Pulse detection FIR
	MSD_grid_init();
	MSD_GPU_grid<<<gridSize,blockSize,nThreads*8>>>(d_input, d_output, x_steps, y_steps, nTimesamples, 1);
		
	return(1);
}
