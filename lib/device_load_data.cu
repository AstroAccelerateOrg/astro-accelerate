#include "hip/hip_runtime.h"
//#include "headers/device_dedispersion_kernel.h"

//extern "C" void load_data(int i, float *device_pointer, float *host_pointer, size_t size, int nsamp, int maxshift, int nchans, int t_processed_s, int t_processed_c, float *dmshifts);

//{{{ load_data_from_host_to_device

void load_data(int i, int *inBin, unsigned short *device_pointer, unsigned short *host_pointer, int t_processed, int maxshift, int nchans, float *dmshifts, hipStream_t stream)
{

	//{{{ Copy data and set up the GPU constants/variables.
	if (i == -1)
	{
		long int length = ( t_processed + maxshift );
		size_t size = nchans * length * sizeof(unsigned short);
		hipMemcpyToSymbolAsync(HIP_SYMBOL(dm_shifts), dmshifts, nchans * sizeof(float),0,hipMemcpyHostToDevice,stream);
//		hipMemcpy(device_pointer, host_pointer, size, hipMemcpyHostToDevice);
		hipMemcpyAsync(device_pointer, host_pointer, size, hipMemcpyHostToDevice,stream);
		hipMemcpyToSymbolAsync(HIP_SYMBOL(i_nchans), &nchans, sizeof(int),0,hipMemcpyHostToDevice,stream);
		hipMemcpyToSymbolAsync(HIP_SYMBOL(i_nsamp), &length, sizeof(int),0,hipMemcpyHostToDevice,stream);
		hipMemcpyToSymbolAsync(HIP_SYMBOL(i_t_processed_s), &t_processed, sizeof(int),0,hipMemcpyHostToDevice,stream);
	}
	else if (i > 0)
	{
		long int length = ( t_processed + maxshift );
//		hipHostRegister(i_nsamp,sizeof(int),0);
		hipMemcpyToSymbolAsync(HIP_SYMBOL(i_nsamp), &length, sizeof(int),0,hipMemcpyHostToDevice,stream);
		hipMemcpyToSymbolAsync(HIP_SYMBOL(i_t_processed_s), &t_processed, sizeof(int),0,hipMemcpyHostToDevice,stream);
	}
	//}}}

	float h_sqrt_taps[PD_MAXTAPS + 1];
//	float *h_sqrt_taps = NULL;
//	hipHostMalloc((void **) &h_sqrt_taps,(PD_MAXTAPS+1)*sizeof(float));
	for (int f = 0; f <= PD_MAXTAPS; f++)
		h_sqrt_taps[f] = (float) sqrt((double) f);
	hipMemcpyToSymbolAsync(HIP_SYMBOL(c_sqrt_taps), h_sqrt_taps, ( PD_MAXTAPS + 1 ) * sizeof(float),0,hipMemcpyHostToDevice,stream);
//	hipHostFree(h_sqrt_taps);

}

//}}}
