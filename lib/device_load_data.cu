#include "hip/hip_runtime.h"
//#include "AstroAccelerate/device_dedispersion_kernel.h"

//extern "C" void load_data(int i, float *device_pointer, float *host_pointer, size_t size, int nsamp, int maxshift, int nchans, int t_processed_s, int t_processed_c, float *dmshifts);

<<<<<<< HEAD
//{{{ load_data_from_host_to_device

void load_data(int i, int *inBin, unsigned short *device_pointer, unsigned short *host_pointer, int t_processed, int maxshift, int nchans, float *dmshifts) {

	//{{{ Copy data and set up the GPU constants/variables.
	if(i==-1) {
		int length=(t_processed+maxshift);
		size_t size=nchans*length*sizeof(unsigned short);
=======
void load_data(int i, int *inBin, unsigned short *device_pointer, unsigned short *host_pointer, int t_processed, int maxshift, int nchans, float *dmshifts)
{
	// Copy data and set up the GPU constants/variables.
	if(i == -1)
	{
		int length = (t_processed+maxshift);
		size_t size = nchans*length*sizeof(unsigned short);
>>>>>>> fe80b9c735d1c898047cbb64bcf8da05cd6a21da
		hipMemcpyToSymbol(HIP_SYMBOL(dm_shifts), dmshifts, nchans * sizeof(float));
		hipMemcpy(device_pointer, host_pointer, size, hipMemcpyHostToDevice);
		hipMemcpyToSymbol(HIP_SYMBOL(i_nchans), &nchans, sizeof(int));
		hipMemcpyToSymbol(HIP_SYMBOL(i_nsamp), &length, sizeof(int));
		hipMemcpyToSymbol(HIP_SYMBOL(i_t_processed_s), &t_processed, sizeof(int));
<<<<<<< HEAD
	} else if (i > 0) {
		int length=(t_processed+maxshift);
		hipMemcpyToSymbol(HIP_SYMBOL(i_nsamp), &length, sizeof(int));
		hipMemcpyToSymbol(HIP_SYMBOL(i_t_processed_s), &t_processed, sizeof(int));
	}
	//}}}
=======
	}
	else if (i > 0)
	{
		int length = (t_processed+maxshift);
		hipMemcpyToSymbol(HIP_SYMBOL(i_nsamp), &length, sizeof(int));
		hipMemcpyToSymbol(HIP_SYMBOL(i_t_processed_s), &t_processed, sizeof(int));
	}
>>>>>>> fe80b9c735d1c898047cbb64bcf8da05cd6a21da
	
	float h_sqrt_taps[PD_MAXTAPS+1];
	for(int f=0; f<=PD_MAXTAPS; f++) h_sqrt_taps[f]=(float) sqrt((double) f);
	hipMemcpyToSymbol(HIP_SYMBOL(c_sqrt_taps), h_sqrt_taps, (PD_MAXTAPS+1)*sizeof(float));

<<<<<<< HEAD
}

//}}}
=======
}
>>>>>>> fe80b9c735d1c898047cbb64bcf8da05cd6a21da
