
#include <hip/hip_runtime.h>
#include <stdio.h>

/* Note we send in a pointer to the file pointer becuase this function needs to update the position of the file pointer
 */

void get_file_data(FILE **fp, int *nchans, int *nsamples, int *nsamp, int *nifs, int *nbits, float *tsamp, float *tstart, float *fch1, float *foff)
{

	fpos_t file_loc;

	char *string = (char *) malloc(80 * sizeof(char));

	int nchar;
	int nbytes = sizeof(int);

	long int total_data;

	double temp;

	while (1)
	{

		strcpy(string, "ERROR");
		fread(&nchar, sizeof(int), 1, *fp);

		if (feof(*fp))
			exit(0);

		if (nchar > 1 && nchar < 80)
		{

			fread(string, nchar, 1, *fp);
			string[nchar] = '\0';
			// For debugging only
			printf("\n%d\t%s", nchar, string), fflush(stdout);
			nbytes += nchar;

			if (strcmp(string, "HEADER_END") == 0)
				break;

			if (strcmp(string, "tsamp") == 0)
			{
				fread(&temp, sizeof(double), 1, *fp);
				*tsamp = (float) temp;
			}
			else if (strcmp(string, "tstart") == 0)
			{
				fread(&temp, sizeof(double), 1, *fp);
				*tstart = (float) temp;
			}
			else if (strcmp(string, "fch1") == 0)
			{
				fread(&temp, sizeof(double), 1, *fp);
				*fch1 = (float) temp;
			}
			else if (strcmp(string, "foff") == 0)
			{
				fread(&temp, sizeof(double), 1, *fp);
				*foff = (float) temp;
			}
			else if (strcmp(string, "nchans") == 0)
			{
				fread(nchans, sizeof(int), 1, *fp);
			}
			else if (strcmp(string, "nifs") == 0)
			{
				fread(nifs, sizeof(int), 1, *fp);
			}
			else if (strcmp(string, "nbits") == 0)
			{
				fread(nbits, sizeof(int), 1, *fp);
			}
			else if (strcmp(string, "nsamples") == 0)
			{
				fread(nsamples, sizeof(int), 1, *fp);
			}
		}
	}

	// Check that we are working with one IF channel
	if (*nifs != 1)
	{
		printf("\nERROR!! Can only work with one IF channel!\n");
		exit(1);
	}

	fgetpos(*fp, &file_loc);
        long data_start = ftell(*fp);
	if (fseek(*fp, 0, SEEK_END) != 0) 
 	{
		printf("\nERROR!! Failed to seek to end of data file\n");
		exit(1);
	}
        total_data = ftell(*fp);
	if (total_data == -1)
	{
		printf("\nERROR!! Failed to seek to end of data file\n");
		exit(1);
	}
	total_data -= data_start;

	if (( *nbits ) == 32)
	{
		*nsamp = (total_data/sizeof(float)) - 1;
	}
	else if (( *nbits ) == 8)
	{
		*nsamp = (total_data/sizeof(unsigned char)) - 1;
	}
	else
	{
		printf("\n\n======================= ERROR =======================\n");
		printf(" Currently this code only runs with 8 and 32 bit data\n");
		printf("\n=====================================================\n");
	}

	// Move the file pointer back to the end of the header
	fsetpos(*fp, &file_loc);

}
