#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "AstroAccelerate/params.h"
#include "AstroAccelerate/host_periods.h"
<<<<<<< HEAD
#include "AstroAccelerate/device_MSD_plane.h"
#include "AstroAccelerate/device_MSD_limited.h"
#include "AstroAccelerate/device_SNR_limited.h"
#include "AstroAccelerate/device_single_pulse_search.h"
#include "AstroAccelerate/device_threshold.h"
#include "AstroAccelerate/device_single_FIR.h"
#include "timer.h"

//---------------------------------------------------------------------------------
//-------> Kahan MSD
void d_kahan_summation(float *signal, int nDMs, int nTimesamples, int offset, float *result, float *error)
{
	double sum;
	double sum_error;
	double a,b;
	
	sum = 0;
	sum_error = 0;
	for(int d=0; d<nDMs; d++)
	{
		for(int s=0; s<(nTimesamples-offset); s++)
		{
			a = signal[d*nTimesamples + s]-sum_error;
			b = sum+a;
			sum_error = (b-sum);
			sum_error = sum_error-a;
			sum = b;
		}
	}
	*result = sum;
	*error = sum_error;
}

void d_kahan_sd(float *signal, int nDMs, int nTimesamples, int offset, double mean, float *result, float *error)
{
	double sum;
	double sum_error;
	double a,b,dtemp;
	
	sum = 0;
	sum_error = 0;
	for(int d=0; d<nDMs; d++)
	{
		for(int s=0; s<(nTimesamples-offset); s++)
		{
			dtemp = (signal[d*nTimesamples + s]-sum_error - mean);
			a = dtemp*dtemp;
			b = sum+a;
			sum_error = (b-sum);
			sum_error = sum_error-a;
			sum = b;
		}
	}
	*result = sum;
	*error = sum_error;
}

void MSD_Kahan(float *h_input, int nDMs, int nTimesamples, int offset, double *mean, double *sd)
{
	float error, signal_mean, signal_sd;
	int nElements = nDMs*(nTimesamples-offset);
	
	d_kahan_summation(h_input, nDMs, nTimesamples, offset, &signal_mean, &error);
	signal_mean = signal_mean/nElements;
	
	d_kahan_sd(h_input, nDMs, nTimesamples, offset, signal_mean, &signal_sd, &error);
	signal_sd = sqrt(signal_sd/nElements);

	*mean = signal_mean;
	*sd = signal_sd;
}

//-------> Kahan MSD
//---------------------------------------------------------------------------------
void find_min_i_max(float *h_temp, int nDMs, int nTimesamples, int offset, float *max, float *min)
{
	float signal_max, signal_min;
	signal_max = h_temp[0]; 
	signal_min = h_temp[0];
	for(int d=0; d<nDMs; d++)
	{
		for(int s=0; s<(nTimesamples-offset); s++)
		{
			if(h_temp[d*nTimesamples + s]>signal_max) signal_max = h_temp[d*nTimesamples + s];
			if(h_temp[d*nTimesamples + s]<signal_min) signal_min = h_temp[d*nTimesamples + s];
		}
	}
	*max = signal_max;
	*min = signal_min;
}

void print_to_file(float *list, int size, float tsamp, float start_time, float dm_low, float dm_step,  char *filename)
{
	FILE	*file_out;
	
	if ((file_out=fopen(filename, "w")) == NULL)
	{
		fprintf(stderr, "Error opening output file!\n");
		exit(0);
	}
	
	for (int f=0; f<size; f++)
		fprintf(file_out, "%f, %f, %f, %f\n", list[4*f+1]*tsamp+start_time, dm_low + list[4*f]*dm_step, list[4*f+2], list[4*f+3]);
	
	fclose(file_out);
}

void export_file_nDM_nTimesamples(float *data, int nDMs, int nTimesamples, char *filename)
{
	FILE	*file_out;
	char str[200];
		
	sprintf(str,"%s_DM.dat",filename);
	if ((file_out=fopen(str, "w")) == NULL)
	{
		fprintf(stderr, "Error opening output file!\n");
		exit(0);
	}
	
	printf("export nDMs\n");
	for (int s=0; s<nTimesamples; s++)
	{
		for(int d=0; d<nDMs; d++)
			fprintf(file_out, "%f ", data[d*nTimesamples + s]);
		fprintf(file_out, "\n");
	}
	
	fclose(file_out);
	
	sprintf(str,"%s_Time.dat",filename);
	if ((file_out=fopen(str, "w")) == NULL)
	{
		fprintf(stderr, "Error opening output file!\n");
		exit(0);
	}
	
	printf("export nTimesamples\n");
	for(int d=0; d<nDMs; d++)
	{
		for (int s=0; s<nTimesamples; s++)
			fprintf(file_out, "%f ", data[d*nTimesamples + s]);
		fprintf(file_out, "\n");
	}
	
	fclose(file_out);	
}


void analysis(int i, float tstart, int t_processed, int nsamp, int nchans, int maxshift, int max_ndms, int *ndms, int *outBin, float cutoff, float *output_buffer, float *dm_low, float *dm_high, float *dm_step, float tsamp)
{
	FILE	*fp_out;
	char	filename[200];

	int remaining_time;

	float	start_time;

	unsigned long int vals;
	int nTimesamples=t_processed;
	int nDMs=ndms[i];
	
	float mean, stddev_orig;

	// Calculate the total number of values
	vals = (unsigned long int)(nDMs*nTimesamples);

=======

void analysis(int i, float tstart, int t_processed, int nsamp, int nchans, int maxshift, int max_ndms, int *ndms, int *outBin, float cutoff, float *output_buffer, float *dm_low, float *dm_high, float *dm_step, float tsamp) {

	FILE	*fp_out;
	char	filename[200];

	int	k, dm_count, remaining_time, bin_factor, counter;

	float	start_time;

	unsigned long int j;
	unsigned long int vals;
	
	float mean, stddev, stddev_orig;

	float *exchange_ptr;
	float *binned_output = (float *)malloc(max_ndms*(t_processed)*sizeof(float)/2+1);
	float *binned_output_next = (float *)malloc(max_ndms*(t_processed)*sizeof(float)/4);

	double	total;

	//printf("\n\n%f\t%f\t%f\t%d", dm_low[i], dm_high[i], dm_step[i], ndms[i]), fflush(stdout);

	// Calculate the total number of values
	vals = (unsigned long int)(t_processed*ndms[i]);

	//chunk=(int)(vals/24);

	//start_time = ((input_increment/nchans)*tsamp);
>>>>>>> 0ec19baf405fa311d6a7ea91dbb146bcccf88229
	start_time=tstart;
	remaining_time = (t_processed);

	sprintf(filename, "analysed-t_%.2f-dm_%.2f-%.2f.dat", start_time, dm_low[i], dm_high[i]);
<<<<<<< HEAD
	if ((fp_out=fopen(filename, "w")) == NULL)
	{
=======
	if ((fp_out=fopen(filename, "w")) == NULL) {
>>>>>>> 0ec19baf405fa311d6a7ea91dbb146bcccf88229
		fprintf(stderr, "Error opening output file!\n");
		exit(0);
	}

<<<<<<< HEAD
	float *h_temp=(float*)malloc(vals*sizeof(float));
	float *h_output_list;
	
	//---------------------------------------------------------------------------
	//----------> GPU part
	printf("\n GPU analysis part\n\n");
	printf("Dimensions nDMs:%d; nTimesamples:%d;\n",ndms[i],t_processed);
	GpuTimer timer;
	
	float *d_MSD;			hipMalloc((void**)&d_MSD, 3*sizeof(float));
	float *d_SNR_MSD;		hipMalloc((void**)&d_SNR_MSD, 3*sizeof(float));
	float *d_FIR_values;	hipMalloc((void**)&d_FIR_values, vals*sizeof(float));
	float *d_SNR_values;	hipMalloc((void**)&d_SNR_values, vals*sizeof(float));		hipMemset((void*)d_SNR_values, 0 , vals*sizeof(float));
	float *d_SNR_taps;		hipMalloc((void**)&d_SNR_taps, vals*sizeof(float));		hipMemset((void*)d_SNR_taps, 0 , vals*sizeof(float));
	int *gmem_pos;			hipMalloc((void**)&gmem_pos, 1*sizeof(int));				hipMemset((void*)gmem_pos, 0 , sizeof(int));
	float h_MSD[3];

	int h_list_size;
			
	// ----------------------------------------------------------------------------------------------------	
	// ---------> Mean and standard deviation is calculated once, higher taps are ignored
	MSD_limited(output_buffer, d_MSD, nDMs, nTimesamples, 0);
	
	hipMemcpy(h_MSD, d_MSD, 3*sizeof(float), hipMemcpyDeviceToHost);
	mean = h_MSD[0]; 
	stddev_orig = h_MSD[1];
	printf("Bin: %d, Mean: %f, Stddev: %f, nPoints: %f \n", 1, mean, stddev_orig,h_MSD[2]);

	timer.Start();
	PD_SEARCH(output_buffer, d_SNR_values, d_SNR_taps, d_MSD, PD_MAXTAPS-1,  nDMs, nTimesamples);
	timer.Stop();
	printf("PD_SEARCH took:%f ms\n", timer.Elapsed());
	
	timer.Start();
	THRESHOLD_ignore(d_SNR_values, d_SNR_taps, output_buffer, gmem_pos, 10.0, PD_MAXTAPS-1, nDMs, nTimesamples, vals/4);
	timer.Stop();
	printf("THR_WARP took:%f ms\n", timer.Elapsed());
	// ---------> Mean and standard deviation is calculated once, higher taps are ignored
	// ----------------------------------------------------------------------------------------------------	
	
	hipMemcpy(&h_list_size, gmem_pos, sizeof(int), hipMemcpyDeviceToHost);
	h_output_list = (float*)malloc(h_list_size*4*sizeof(float));
	hipMemcpy(h_output_list, output_buffer, h_list_size*4*sizeof(float), hipMemcpyDeviceToHost);
	
	
	for (int count = 0; count < h_list_size; count++)
		fprintf(fp_out, "%f, %f, %f, %f\n", h_output_list[4*count+1]*tsamp+start_time, dm_low[i] + h_output_list[4*count]*dm_step[i], h_output_list[4*count+2], h_output_list[4*count+3]);
	
	hipFree(d_MSD);
	hipFree(d_FIR_values);
	hipFree(d_SNR_values);
	hipFree(d_SNR_taps);
	hipFree(gmem_pos);
	//----------> GPU part
	//---------------------------------------------------------------------------
	
	free(h_temp);
	free(h_output_list);
	
	fclose(fp_out);
}
/*
void export_sps()
{

}*/
=======
	// Calculate the mean
	total  = 0.0;
	#pragma omp parallel for default(shared) private(j) reduction(+:total)
	for(j = 0; j < vals; j++) {
		total += (double)output_buffer[j];
//		printf("\nout:\t%f", output_buffer[j]);
	}
	mean = (float)(total/(double)vals);  // Mean for data sample
/*
	for (dm_count = 0; dm_count < ndms[i]; dm_count++) {
		for(k = 0; k < remaining_time; k++) {
			total = total+(double)output_buffer[remaining_time*dm_count + k];
		}
		//printf("\n%f %d %d", (float)(total / (double)(dm_count*remaining_time)), dm_count, k);
		//printf("\n%f %d %d", output_buffer[remaining_time*dm_count + k], dm_count, k);
		//printf("\n%llf %d %d", total, dm_count, k);
	}
	mean = (float)(total/(double)(ndms[i]*remaining_time));  // Mean for data sample
*/
	// Calculate standard deviation
	total = 0;

	#pragma omp parallel for default(shared) private(j) reduction(+:total)
	for(j = 0; j < vals; j++) {
		total += (double)((output_buffer[j] - mean)*(output_buffer[j] - mean));
	}
	stddev_orig = (float)sqrt(total / (double)vals); // Stddev for data sample
/*
	for (dm_count = 0; dm_count < ndms[i]; dm_count++) {
		for(k = 0; k < remaining_time; k++) {
			total = total+(double)pow((output_buffer[remaining_time*dm_count + k]-mean),2.0);
		}
	//	printf("\n%f %d %d", (float)sqrt(total / (double)(dm_count*remaining_time)), dm_count, k);
	}
	stddev_orig = (float)sqrt(total / (double)(ndms[i]*remaining_time)); // Stddev for data sample
*/
	//printf("\ni: %d, dm_low: %f, dm_high: %f, dm_step: %f", i, dm_low[i], dm_high[i], dm_step[i]);
	//printf("\nnsamp: %d, tsamp: %f, remaining: %d, maxshift: %d", nsamp, tsamp, remaining_time, maxshift);
	// Print mean and stddev
	printf("\nBin: %d, Mean: %f, Stddev: %f", 1, mean, stddev_orig), fflush(stdout);

	// Apply threshold
	for (dm_count = 0; dm_count < ndms[i]; dm_count++) {
		for(k = 0; k < remaining_time; k++) {
			//if((output_buffer[remaining_time*dm_count + k]-mean)/(stddev_orig) >= cutoff && output_buffer[remaining_time*dm_count + k]+(mean)>0.0f) {
			if((output_buffer[remaining_time*dm_count + k]-mean)/(stddev_orig) >= cutoff) {
				//fprintf(fp_out, "%f, %f, %f, %d, %d\n", ((float)k)*tsamp+start_time, dm_low[i] + ((float)dm_count)*dm_step[i], (output_buffer[remaining_time*dm_count + k])/stddev_orig, i, 1);
				fprintf(fp_out, "%f, %f, %f, %d, %d\n", ((float)k)*tsamp+start_time, dm_low[i] + ((float)dm_count)*dm_step[i], (output_buffer[remaining_time*dm_count + k]-mean)/stddev_orig, i, 1);
			}	
		}
	}

	#pragma omp parallel for private(dm_count,k)
	for (dm_count = 0; dm_count < ndms[i]; dm_count++) {
		int shift=(remaining_time/2)*dm_count;
		int shift2=(remaining_time)*dm_count;
		for(k = 0; k < remaining_time/2; k++) {
			int shift3=2*k+shift2;
			binned_output[shift + k] = ((output_buffer[shift3]) + (output_buffer[shift3 + 1]))*0.5f;
		}
	}

	bin_factor = outBin[i];
	remaining_time=remaining_time/2;
	vals=vals/2;
	counter=1;

	while(bin_factor > 1) {

		// Check the recalculation of the mean
		//total  = 0.0;
		//for(k = 0; k < vals; k++) {
		//	total += (double)binned_output[k];
		//}
		//mean = (float)(total/(double)vals);  // Mean for data sample

		// Calculate standard deviation
		total = 0;

/*		#pragma omp parallel for default(shared) private(k) reduction(+:total)
		for(k = 0; k < vals; k++) {
			total += (double)((binned_output[k] - mean)*(binned_output[k] - mean));
		}
		stddev = (float)sqrt(total / (double)vals); // Stddev for data sample
*/

		stddev = stddev_orig/((float)sqrt(2.0f*powf(2,(counter-1)))); // Stddev for data sample
		// Print mean and stddev
//		printf("\nBin: %d, Mean: %f, Stddev: %f", (int)powf(2,counter), mean, stddev), fflush(stdout);

		// Apply threshold
		for (dm_count = 0; dm_count < ndms[i]; dm_count++) {
			for(k = 0; k < remaining_time; k++) {
				if((binned_output[remaining_time*dm_count + k]-mean)/(stddev) >= cutoff && binned_output[remaining_time*dm_count + k]+(mean)>0.0f) {
					//fprintf(fp_out, "%f, %f, %f, %d, %d\n", k*powf(2,counter)*tsamp+start_time, dm_low[i] + dm_count*dm_step[i], (binned_output[remaining_time*dm_count + k])/stddev, i, (int)powf(2,counter));
					fprintf(fp_out, "%f, %f, %f, %d, %d\n", k*powf(2,counter)*tsamp+start_time, dm_low[i] + dm_count*dm_step[i], (binned_output[remaining_time*dm_count + k]-mean)/stddev, i, (int)powf(2,counter));
				}
			}
		}

		#pragma omp parallel for private(dm_count,k) 
		for (dm_count = 0; dm_count < ndms[i]; dm_count++) {
			int shift=(remaining_time/2)*dm_count;
			int shift2=(remaining_time)*dm_count;
			for(k = 0; k < remaining_time/2; k++) {
				int shift3=2*k+shift2;
				binned_output_next[shift + k] = ((binned_output[shift3]) + (binned_output[shift3 + 1]))*0.5f;
			}
		}
	
		remaining_time=remaining_time/2;
		bin_factor=bin_factor/2;
		vals=vals/2;
		exchange_ptr=binned_output;
		binned_output=binned_output_next;
		binned_output_next=exchange_ptr;
		counter++;
	}
	
	free(binned_output);
	free(binned_output_next);
	fclose(fp_out);
}
>>>>>>> 0ec19baf405fa311d6a7ea91dbb146bcccf88229
