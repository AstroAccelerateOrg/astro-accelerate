#include "hip/hip_runtime.h"
//Added by Karel Adamek

#include "AstroAccelerate/params.h"
#include "device_MSD_plane_kernel.cu"

<<<<<<< HEAD
void MSD_init(void){
=======
void MSD_init(void)
{
>>>>>>> fe80b9c735d1c898047cbb64bcf8da05cd6a21da
	//---------> Specific nVidia stuff
	hipDeviceSetCacheConfig(hipFuncCachePreferShared);
	hipDeviceSetSharedMemConfig(hipSharedMemBankSizeFourByte);
}

<<<<<<< HEAD
int MSD(float *d_input, float *d_MSD, int nDMs, int nTimesamples){
=======
int MSD(float *d_input, float *d_MSD, int nDMs, int nTimesamples)
{
>>>>>>> fe80b9c735d1c898047cbb64bcf8da05cd6a21da
	//---------> Task specific
	int nBlocks, nRest, Elements_per_block, nElements, nThreads;
	float *d_output;

<<<<<<< HEAD
	nElements=nDMs*nTimesamples;
	Elements_per_block=2*WARP*MSD_ELEM_PER_THREAD*MSD_WARPS_PER_BLOCK;
	nBlocks=nElements/Elements_per_block;
	nRest=nElements - nBlocks*Elements_per_block;
	
	//---------> CUDA block and CUDA grid parameters
	int nCUDAblocks_x=nBlocks;
	int nCUDAblocks_y=1;
=======
	nElements = nDMs*nTimesamples;
	Elements_per_block = 2*WARP*MSD_ELEM_PER_THREAD*MSD_WARPS_PER_BLOCK;
	nBlocks = nElements/Elements_per_block;
	nRest = nElements - nBlocks*Elements_per_block;
	
	//---------> CUDA block and CUDA grid parameters
	int nCUDAblocks_x = nBlocks;
	int nCUDAblocks_y = 1;
>>>>>>> fe80b9c735d1c898047cbb64bcf8da05cd6a21da
	
	dim3 gridSize(nCUDAblocks_x, nCUDAblocks_y, 1);
	dim3 blockSize(WARP*MSD_WARPS_PER_BLOCK, 1, 1);
	
<<<<<<< HEAD
	if( nRest<128 ) nThreads=32;
	else nThreads=128;
	
=======
	if( nRest<128 ) nThreads = 32;
	else nThreads = 128;
	 
>>>>>>> fe80b9c735d1c898047cbb64bcf8da05cd6a21da
	dim3 remainder_gridSize(1, 1, 1);
	dim3 remainder_blockSize(nThreads, 1, 1);
	
	dim3 final_gridSize(1, 1, 1);
	dim3 final_blockSize(WARP*MSD_WARPS_PER_BLOCK, 1, 1);
	
	//---------> Allocation of temporary memory
	hipMalloc((void **) &d_output, (nBlocks*2 + 3)*sizeof(float));
	
	//---------> Pulse detection FIR
	MSD_init();
	MSD_GPU<<<gridSize,blockSize>>>((float2 *) d_input, d_output);
<<<<<<< HEAD
	if(nRest>0) {
		MSD_GPU_remainder<<<remainder_gridSize, remainder_blockSize>>>(&d_input[nBlocks*2*WARP*MSD_ELEM_PER_THREAD*MSD_WARPS_PER_BLOCK], &d_output[2*nBlocks], nRest);
	}
=======
	if(nRest>0)
		MSD_GPU_remainder<<<remainder_gridSize, remainder_blockSize>>>(&d_input[nBlocks*2*WARP*MSD_ELEM_PER_THREAD*MSD_WARPS_PER_BLOCK], &d_output[2*nBlocks], nRest);
>>>>>>> fe80b9c735d1c898047cbb64bcf8da05cd6a21da
	MSD_GPU_final<<<final_gridSize, final_blockSize>>>(d_output, d_MSD, nBlocks, nRest, (float) nElements);
	
	//---------> De-allocation of temporary memory
	hipFree(d_output);
	
	// Unprocessed samples depends on whether kernel MSD_GPU_one_reduction_remainder is launched or not. If not then unprocessed samples are nRest BUT only at the end since this treats data as 1d array! 
	return(0);
<<<<<<< HEAD
}

=======
}
>>>>>>> fe80b9c735d1c898047cbb64bcf8da05cd6a21da
