#include <stdio.h>
#include "AstroAccelerate/params.h"

// CUDA-C includes
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

//extern "C" void init_gpu(int argc, char **argv, int enable_debug, size_t *gpu_memory);

//{{{ init_gpu

void init_gpu(int argc, char **arg, int enable_debug, size_t *gpu_memory)
{

	int deviceCount = 0;
	hipError_t error_id = hipGetDeviceCount(&deviceCount);

	if (error_id != hipSuccess)
	{
		printf("hipGetDeviceCount returned %d\n-> %s\n", (int) error_id, hipGetErrorString(error_id));
		printf("Result = FAIL\n");
		exit(EXIT_FAILURE);
	}

	// This function call returns 0 if there are no CUDA capable devices.
	if (deviceCount == 0)
	{
		printf("There are no available device(s) that support CUDA\n");
	}
	else
	{
		printf("Detected %d CUDA Capable device(s)\n", deviceCount);
	}

	int dev, driverVersion = 0, runtimeVersion = 0;
	dev = CARD;

	hipSetDevice(dev);
	size_t free, total;

	hipMemGetInfo(&free, &total);
	*gpu_memory = ( free/4 );
}

//}}}

