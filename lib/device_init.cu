#include "hip/hip_runtime.h"
#include <stdio.h>
#include "headers/params.h"

// CUDA-C includes
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

//extern "C" void init_gpu(int argc, char **argv, int enable_debug, size_t *gpu_memory);

//{{{ init_gpu

void init_gpu(int argc, char **arg, int enable_debug, size_t *gpu_memory)
{

	int deviceCount = 0;
	hipError_t error_id = hipGetDeviceCount(&deviceCount);

	if (error_id != hipSuccess)
	{
		printf("hipGetDeviceCount returned %d\n-> %s\n", (int) error_id, hipGetErrorString(error_id));
		printf("Result = FAIL\n");
		exit(EXIT_FAILURE);
	}

	// This function call returns 0 if there are no CUDA capable devices.
	if (deviceCount == 0)
	{
		printf("There are no available device(s) that support CUDA\n");
	}
	else
	{
		printf("Detected %d CUDA Capable device(s)\n", deviceCount);
	}

	int dev, driverVersion = 0, runtimeVersion = 0;
	dev = CARD;

	hipSetDevice(dev);
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, dev);

	printf("\nDevice %d: \"%s\"\n", dev, deviceProp.name);

	// Console log
	hipDriverGetVersion(&driverVersion);
	hipRuntimeGetVersion(&runtimeVersion);
	printf("  CUDA Driver Version / Runtime Version          %d.%d / %d.%d\n", driverVersion / 1000, ( driverVersion % 100 ) / 10, runtimeVersion / 1000, ( runtimeVersion % 100 ) / 10);
	printf("  CUDA Capability Major/Minor version number:    %d.%d\n", deviceProp.major, deviceProp.minor);
	printf("  Total amount of global memory:                 %.0f MBytes (%llu bytes)\n", (float) deviceProp.totalGlobalMem / 1048576.0f, (unsigned long long) deviceProp.totalGlobalMem);
	printf("  GPU Clock rate:                                %.0f MHz (%0.2f GHz)\n", deviceProp.clockRate * 1e-3f, deviceProp.clockRate * 1e-6f);
	printf("  Memory Clock rate:                             %.0f Mhz\n", deviceProp.memoryClockRate * 1e-3f);
	printf("  Memory Bus Width:                              %d-bit\n", deviceProp.memoryBusWidth);
	if (deviceProp.l2CacheSize)
	{
		printf("  L2 Cache Size:                                 %d bytes\n", deviceProp.l2CacheSize);
	}
	printf("  Maximum Texture Dimension Size (x,y,z)         1D=(%d), 2D=(%d, %d), 3D=(%d, %d, %d)\n", deviceProp.maxTexture1D, deviceProp.maxTexture2D[0], deviceProp.maxTexture2D[1], deviceProp.maxTexture3D[0], deviceProp.maxTexture3D[1], deviceProp.maxTexture3D[2]);
	printf("  Maximum Layered 1D Texture Size, (num) layers  1D=(%d), %d layers\n", deviceProp.maxTexture1DLayered[0], deviceProp.maxTexture1DLayered[1]);
	printf("  Maximum Layered 2D Texture Size, (num) layers  2D=(%d, %d), %d layers\n", deviceProp.maxTexture2DLayered[0], deviceProp.maxTexture2DLayered[1], deviceProp.maxTexture2DLayered[2]);
	printf("  Total amount of constant memory:               %lu bytes\n", deviceProp.totalConstMem);
	printf("  Total amount of shared memory per block:       %lu bytes\n", deviceProp.sharedMemPerBlock);
	printf("  Total number of registers available per block: %d\n", deviceProp.regsPerBlock);
	printf("  Warp size:                                     %d\n", deviceProp.warpSize);
	printf("  Maximum number of threads per multiprocessor:  %d\n", deviceProp.maxThreadsPerMultiProcessor);
	printf("  Maximum number of threads per block:           %d\n", deviceProp.maxThreadsPerBlock);
	printf("  Max dimension size of a thread block (x,y,z): (%d, %d, %d)\n", deviceProp.maxThreadsDim[0], deviceProp.maxThreadsDim[1], deviceProp.maxThreadsDim[2]);
	printf("  Max dimension size of a grid size    (x,y,z): (%d, %d, %d)\n", deviceProp.maxGridSize[0], deviceProp.maxGridSize[1], deviceProp.maxGridSize[2]);
	printf("  Maximum memory pitch:                          %lu bytes\n", deviceProp.memPitch);
	printf("  Texture alignment:                             %lu bytes\n", deviceProp.textureAlignment);
	printf("  Concurrent copy and kernel execution:          %s with %d copy engine(s)\n", (
	    deviceProp.deviceOverlap ? "Yes" : "No" ), deviceProp.asyncEngineCount);
	printf("  Run time limit on kernels:                     %s\n",
	    deviceProp.kernelExecTimeoutEnabled ? "Yes" : "No");
	printf("  Integrated GPU sharing Host Memory:            %s\n",
	    deviceProp.integrated ? "Yes" : "No");
	printf("  Support host page-locked memory mapping:       %s\n",
	    deviceProp.canMapHostMemory ? "Yes" : "No");
	printf("  Alignment requirement for Surfaces:            %s\n",
	    deviceProp.surfaceAlignment ? "Yes" : "No");
	printf("  Device has ECC support:                        %s\n",
	    deviceProp.ECCEnabled ? "Enabled" : "Disabled");
	printf("  CUDA Device Driver Mode (TCC or WDDM):         %s\n",
	    deviceProp.tccDriver ? "TCC (Tesla Compute Cluster Driver)" : "WDDM (Windows Display Driver Model)");
	printf("  Device supports Unified Addressing (UVA):      %s\n",
	    deviceProp.unifiedAddressing ? "Yes" : "No");
	printf("  Device PCI Bus ID / PCI location ID:           %d / %d\n", deviceProp.pciBusID, deviceProp.pciDeviceID);

	size_t free, total;

	hipMemGetInfo(&free, &total);
	*gpu_memory = ( free );
	//*gpu_memory = ( free*(6.0/7.0) );
}

//}}}

