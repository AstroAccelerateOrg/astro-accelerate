#include "hip/hip_runtime.h"
// Added by Karel Adamek 

#ifndef SINGLE_FIR_KERNEL_H_
#define SINGLE_FIR_KERNEL_H_

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "headers/params.h"

__global__ void PD_FIR_GPU(float const* __restrict__ d_input, float *d_output, int nTaps, int nLoops, int nTimesamples)
{
	extern __shared__ float s_input[];

	int itemp, pos;
	float sum[PD_FIR_NWINDOWS];

	//----------------------------------------------
	//---- Reading data
	itemp = PD_FIR_ACTIVE_WARPS * WARP * PD_FIR_NWINDOWS + nTaps - 1;
	for (int i = 0; i < nLoops; i++)
	{
		pos = i * PD_FIR_ACTIVE_WARPS * WARP + threadIdx.x;
		if (pos < itemp)
		{
			s_input[pos] = d_input[blockIdx.y * nTimesamples + blockIdx.x * PD_FIR_ACTIVE_WARPS * WARP * PD_FIR_NWINDOWS + pos];
		}
	}

	__syncthreads();

	//----------------------------------------------
	//---- Calculating FIR version 2

	pos = PD_FIR_NWINDOWS * threadIdx.x;
	sum[0] = 0;
	for (int t = 0; t < nTaps; t++)
	{
		sum[0] += s_input[pos + t];
	}
	for (int i = 1; i < PD_FIR_NWINDOWS; i++)
	{
		pos = PD_FIR_NWINDOWS * threadIdx.x + i - 1;
		sum[i] = sum[i - 1] - s_input[pos] + s_input[pos + nTaps];
	}

	//----------------------------------------------
	//---- Writing data	
	for (int i = 0; i < PD_FIR_NWINDOWS; i++)
	{
		pos = PD_FIR_NWINDOWS * threadIdx.x + i;
		d_output[blockIdx.y * nTimesamples + blockIdx.x * PD_FIR_ACTIVE_WARPS * WARP * PD_FIR_NWINDOWS + pos] = sum[i];
	}
}

#endif
