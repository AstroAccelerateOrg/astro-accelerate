#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hipfft/hipfft.h>
#include <math.h>
#include "headers/params.h"

void periodicity(int range, int nsamp, int max_ndms, int processed, int nboots, int num_trial_bins, int navdms, float narrow, float wide, int nsearch, float aggression, float cutoff, float ***output_buffer, int *ndms, int *inBin, float *dm_low, float *dm_high, float *dm_step, float tsamp)
{

	// Example FFT....

	printf("\n");

	printf("[1DCUFFT] is starting...\n");

	//FILE	*fp_c, *fp_dm, *fp_harm;
	FILE *fp_c, *fp_dm;
	char filename[200];

	int number_of_candidates = 10;

	float** h_top_list = (float**) malloc(sizeof(float*) * 5);
	for (int a = 0; a < 5; a++)
	{
		h_top_list[a] = (float*) malloc(sizeof(float) * number_of_candidates);
	}
	for (int a = 0; a < 5; a++)
	{
		for (int b = 0; b < number_of_candidates; b++)
		{
			h_top_list[a][b] = 0.0f;
		}
	}

	for (int i = 0; i < range; i++)
	{
		int samps = processed / inBin[i];

		// Allocate memory for signal
		hipfftReal* d_signal_in;
		hipMalloc((void**) &d_signal_in, samps * sizeof(hipfftReal));

		hipfftComplex* d_signal_out;
		hipMalloc((void**) &d_signal_out, ( samps / 2 + 1 ) * sizeof(hipfftComplex));

		hipfftComplex* h_signal = (hipfftComplex*) malloc(( samps / 2 + 1 ) * sizeof(hipfftComplex));
		float* h_signal_x = (float*) malloc(sizeof(float) * ( samps / 2 + 1 ) * ndms[i]);
		float* h_signal_y = (float*) malloc(sizeof(float) * ( samps / 2 + 1 ) * ndms[i]);
		float* h_signal_p = (float*) malloc(sizeof(float) * ( samps / 2 + 1 ) * ndms[i]);
		float* h_harm = (float*) malloc(sizeof(float) * ( samps / 2 + 1 ) * ndms[i]);
		float* h_signal_inter = (float*) malloc(sizeof(float) * 2 * ( samps / 2 + 1 ) * ndms[i]);

		float** h_candidates = (float**) malloc(sizeof(float*) * ndms[i]);
		for (int a = 0; a < ndms[i]; a++)
		{
			h_candidates[a] = (float*) malloc(sizeof(float) * ( samps / 2 + 1 ));
		}
		for (int a = 0; a < ndms[i]; a++)
		{
			for (int b = 0; b < samps / 2 + 1; b++)
			{
				h_candidates[a][b] = 0.0f;
			}
		}

		sprintf(filename, "fourier-%d.dat", i);
		if (( fp_c = fopen(filename, "w") ) == NULL)
		{
			fprintf(stderr, "Error opening output file!\n");
			exit(0);
		}
		sprintf(filename, "fourier_inter-%d.dat", i);
		if (( fp_dm = fopen(filename, "w") ) == NULL)
		{
			fprintf(stderr, "Error opening output file!\n");
			exit(0);
		}

		// CUFFT plan
		hipfftHandle plan;
		hipfftPlan1d(&plan, samps, HIPFFT_R2C, 1);

		for (int dm_count = 0; dm_count < ndms[i]; dm_count++)
		{

			hipMemcpy(d_signal_in, output_buffer[i][dm_count], samps * sizeof(float), hipMemcpyHostToDevice);

			// Transform signal 
			//printf("\nTransforming dm: %f using hipfftExecR2C\n", dm);
			hipfftExecR2C(plan, (hipfftReal *) d_signal_in, (hipfftComplex *) d_signal_out);

			// Copy device memory to host
			hipMemcpy(h_signal, d_signal_out, sizeof(hipfftComplex) * ( samps / 2 + 1 ), hipMemcpyDeviceToHost);

			h_signal_p[0 + dm_count * ( samps / 2 )] = 0.0;
#pragma omp parallel for
			for (int j = 1; j < samps / 2; j++)
			{
				//	h_signal[j].x = h_signal[j].x-h_signal[0].x;
				//	h_signal[j].y = h_signal[j].y-h_signal[0].y;
				h_signal_x[j + dm_count * ( samps / 2 )] = h_signal[j].x;
				h_signal_y[j + dm_count * ( samps / 2 )] = h_signal[j].y;
				h_signal_p[j + dm_count * ( samps / 2 )] = ( ( h_signal[j].x * h_signal[j].x + h_signal[j].y * h_signal[j].y ) );
				h_signal_inter[2 * j + dm_count * samps] = h_signal_p[j + dm_count * ( samps / 2 )];
				h_signal_inter[2 * j + 1 + dm_count * samps] = 0.616850275 * ( ( h_signal[j].x - h_signal[j + 1].x ) * ( h_signal[j].x - h_signal[j + 1].x ) + ( h_signal[j].y - h_signal[j + 1].y ) * ( h_signal[j].y - h_signal[j + 1].y ) );
			}
		}

		//Destroy CUFFT context
		hipfftDestroy(plan);

		// cleanup memory
		free(h_signal);
		hipFree(d_signal_in);
		hipFree(d_signal_out);

		double mean, stddev;

		double total = 0.0;

		// Calculate the mean
		for (int dm_count = 0; dm_count < ndms[i]; dm_count++)
		{
			for (int j = 0; j < ( samps / 2 ); j++)
			{
				total += ( (double) ( h_signal_p[j + dm_count * ( samps / 2 )] ) );
			}
		}
		mean = ( total / (double) ( ( samps / 2 ) * ndms[i] ) ); // Mean for data sample

		// Calculate standard deviation
		total = 0.0;
		for (int dm_count = 0; dm_count < ndms[i]; dm_count++)
		{
			for (int j = 0; j < ( samps / 2 ); j++)
			{
				total += (double) ( ( h_signal_p[j + dm_count * ( samps / 2 )] - (float) mean ) * ( h_signal_p[j + dm_count * ( samps / 2 )] - (float) mean ) );
			}
		}
		stddev = sqrt(abs(total) / (double) ( ( samps / 2 ) * ndms[i] )); // Stddev for data sample

		for (int dm_count = 0; dm_count < ndms[i]; dm_count++)
		{
			float dm = dm_low[i] + dm_step[i] * dm_count;
			for (int j = 0; j < ( samps / 2 ); j++)
			{
				if ((float) ( ( h_signal_p[j + dm_count * ( samps / 2 )] - mean ) / stddev ) > cutoff)
				{
					fprintf(fp_c, "\n%f\t%f\t%f", dm, j * ( ( 1.0f / tsamp ) / ( samps ) ), (float) ( ( (double) h_signal_p[j + dm_count * ( samps / 2 )] - mean ) / stddev ));

				}
			}
			fprintf(fp_c, "\n");
			for (int j = 0; j < samps; j++)
			{
				if ((float) ( ( h_signal_inter[j + dm_count * samps] - mean ) / stddev ) > cutoff)
				{
					fprintf(fp_dm, "\n%f\t%lf\t%f", dm, j * ( ( 1.0 / tsamp ) / ( 2 * samps ) ), (float) ( ( (double) h_signal_inter[j + dm_count * samps] - mean ) / stddev ));

				}
			}
			fprintf(fp_dm, "\n");
		}
		/*
		 int harm_max=32;
		 for (int dm_count = 0; dm_count < ndms[i]; dm_count++) {
		 for(int j=0;j< (samps/2);j++){
		 h_harm[j+dm_count*(samps/2)] = (h_signal_p[j+dm_count*(samps/2)]);
		 }
		 }
		 int harm=1;
		 sprintf(filename, "harmonic-%d-%d.dat", i, harm);
		 if ((fp_harm=fopen(filename, "w")) == NULL) {
		 fprintf(stderr, "Error opening output file!\n");
		 exit(0);
		 }

		 // Calculate the mean
		 total=0.0;
		 for (int dm_count = 0; dm_count < ndms[i]; dm_count++) {
		 for(int j=0;j< (samps/2);j++){
		 total += ((double)(h_harm[j+dm_count*(samps/2)]));
		 }
		 }
		 mean = (total/(double)((samps/2)*ndms[i]));  // Mean for data sample

		 // Calculate standard deviation
		 total = 0.0;
		 for (int dm_count = 0; dm_count < ndms[i]; dm_count++) {
		 for(int j=0;j< (samps/2); j++){
		 total += (double)((h_harm[j+dm_count*(samps/2)]-(float)mean)*(h_harm[j+dm_count*(samps/2)]-(float)mean));
		 }
		 }
		 stddev = sqrt(abs(total) / (double)((samps/2)*ndms[i])); // Stddev for data sample

		 for (int dm_count = 0; dm_count < ndms[i]; dm_count++) {
		 float dm = dm_low[i]+dm_step[i]*dm_count;
		 for(int j=0;j< samps/2; j++){
		 float candidate = (float)(((double)h_harm[j+dm_count*(samps/2)]-mean)/stddev);
		 if(candidate > cutoff) {
		 fprintf(fp_harm, "\n%f\t%f\t%f", dm, j*1*((1.0f/tsamp)/(samps)), candidate);

		 for(int c = 0; c < number_of_candidates; c++) {
		 if(candidate > h_top_list[4][c]) {
		 for(int d = number_of_candidates - 1; d > c; d--) {
		 h_top_list[0][d] = h_top_list[0][d-1];
		 h_top_list[1][d] = h_top_list[1][d-1];
		 h_top_list[2][d] = h_top_list[2][d-1];
		 h_top_list[3][d] = h_top_list[3][d-1];
		 h_top_list[4][d] = h_top_list[4][d-1];
		 }
		 h_top_list[0][c] = dm;
		 h_top_list[1][c] = j*1*((1.0f/tsamp)/(samps));
		 h_top_list[2][c] = harm;
		 h_top_list[3][c] = j;
		 h_top_list[4][c] = candidate;
		 c=number_of_candidates;
		 }
		 }
		 }
		 h_candidates[dm_count][j] = candidate;
		 }
		 fprintf(fp_harm, "\n");
		 }
		 fclose(fp_harm);

		 for(harm = 2; harm <= harm_max; harm=2*harm) {

		 sprintf(filename, "harmonic-%d-%d.dat", i, harm);
		 if ((fp_harm=fopen(filename, "w")) == NULL) {
		 fprintf(stderr, "Error opening output file!\n");
		 exit(0);
		 }

		 for (int dm_count = 0; dm_count < ndms[i]; dm_count++) {
		 for(int j=0;j < (samps/(2*harm))-harm; j++){
		 h_harm[j*harm+dm_count*(samps/2)] += (h_signal_p[j+dm_count*(samps/2)]);
		 for(int lerp = j+1; lerp < j+harm; lerp++) h_harm[lerp+dm_count*(samps/2)] += (h_signal_p[j+dm_count*(samps/2)] +
		 (h_signal_p[j+1+dm_count*(samps/2)]-h_signal_p[j+dm_count*(samps/2)])*((lerp-j)/harm));
		 }
		 }

		 // Calculate the mean
		 total=0.0;
		 for (int dm_count = 0; dm_count < ndms[i]; dm_count++) {
		 for(int j=0;j< (samps/2);j++){
		 total += ((double)(h_harm[j+dm_count*(samps/2)]));
		 }
		 }
		 mean = (total/(double)((samps/2)*ndms[i]));  // Mean for data sample

		 // Calculate standard deviation
		 total = 0.0;
		 for (int dm_count = 0; dm_count < ndms[i]; dm_count++) {
		 for(int j=0;j< (samps/2); j++){
		 total += (double)((h_harm[j+dm_count*(samps/2)]-(float)mean)*(h_harm[j+dm_count*(samps/2)]-(float)mean));
		 }
		 }
		 stddev = sqrt(abs(total) / (double)((samps/2)*ndms[i])); // Stddev for data sample

		 for (int dm_count = 0; dm_count < ndms[i]; dm_count++) {
		 float dm = dm_low[i]+dm_step[i]*dm_count;
		 for(int j=0;j< samps/2; j++){
		 float candidate = (float)(((double)h_harm[j+dm_count*(samps/2)]-mean)/stddev);
		 if(candidate > sqrt(harm)*cutoff) {
		 fprintf(fp_harm, "\n%f\t%f\t%f", dm, j*harm*((1.0f/tsamp)/(samps)), candidate);
		 for(int c = 0; c < number_of_candidates; c++) {
		 if(candidate > h_top_list[4][c]) {
		 for(int d = number_of_candidates - 1; d > c; d--) {
		 h_top_list[0][d] = h_top_list[0][d-1];
		 h_top_list[1][d] = h_top_list[1][d-1];
		 h_top_list[2][d] = h_top_list[2][d-1];
		 h_top_list[3][d] = h_top_list[3][d-1];
		 h_top_list[4][d] = h_top_list[4][d-1];
		 }
		 h_top_list[0][c] = dm;
		 h_top_list[1][c] = j*harm*((1.0f/tsamp)/(samps));
		 h_top_list[2][c] = harm;
		 h_top_list[3][c] = harm;
		 h_top_list[4][c] = candidate;
		 c=number_of_candidates;
		 }
		 }
		 }
		 h_candidates[dm_count][j] = (float)(((double)h_harm[j+dm_count*(samps/2)]-mean)/stddev);
		 }
		 fprintf(fp_harm, "\n");
		 }
		 fclose(fp_harm);
		 }
		 */
	}

//	for (int c = 0 ; c < ( number_of_candidates - 1 ); c++) {
//		printf("\nCandidate: %d, DM: %f, PERIOD: %f, HARMONIC: %f, PxH: %f, SNR: %f", c, h_top_list[0][c], 1.0f/h_top_list[1][c], h_top_list[2][c], (1.0f/h_top_list[1][c] * h_top_list[2][c]), h_top_list[4][c]);
//	}
}

