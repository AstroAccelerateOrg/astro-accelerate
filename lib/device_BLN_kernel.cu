#include "hip/hip_runtime.h"
// Added by Karel Adamek 

#ifndef BLN_KERNEL_H_
#define BLN_KERNEL_H_

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "headers/params.h"

__global__ void BLN_MSD_GPU_grid(float const* __restrict__ d_input, float *d_output, int x_steps, int y_steps, int nColumns, int msd) {
	extern __shared__ float Ms_Ss[];
	
	int warp_id, local_id, dim_y, pos;
	float x; // current element
	float M; // streaming mean
	float S; // streaming sum of squares (x_i-\bar{x})
	float j;
	float ftemp;
	
	local_id = threadIdx.x & (WARP - 1);
	warp_id = threadIdx.x>>5;
	dim_y = blockDim.x>>5;
	
	//----------------------------------------------
	//---- Calculating of streaming mean and sum of squares
	pos = (blockIdx.y*dim_y + warp_id)*y_steps*nColumns + blockIdx.x*WARP*x_steps + local_id;
	M=__ldg(&d_input[pos]);
	S=0;
	j=1.0f;
	for(int xf=1; xf<x_steps; xf++){
		pos = pos + WARP;
		x = __ldg(&d_input[pos]);
		j = j+1.0f;
		M = M + x;
		ftemp = (j*x - M);
		S = S + 1.0f/(j*(j-1.0f))*ftemp*ftemp;			
	}
	
	pos = pos + nColumns - (x_steps-1)*WARP;
	for(int yf=1; yf<y_steps; yf++){
		for(int xf=0; xf<x_steps; xf++){
			x = __ldg(&d_input[pos]);
			j = j+1.0f;
			M = M + x;
			ftemp = (j*x - M);
			S = S + 1.0f/(j*(j-1.0f))*ftemp*ftemp;
			pos = pos + WARP;
		}
		pos = pos + nColumns - x_steps*WARP;
	}
	
	Ms_Ss[threadIdx.x]=M;
	Ms_Ss[blockDim.x + threadIdx.x]=S;
	
	__syncthreads();
	
	// now all threads had saved their work, reduction follows
	
	// first we must load initial values
	//j=Neco;
	for(int i=(blockDim.x>>1); i>HALF_WARP; i=i>>1){
		if(threadIdx.x<i){
			j=j*2;
			ftemp = (M - Ms_Ss[i + threadIdx.x]);
			S = S + Ms_Ss[blockDim.x + i + threadIdx.x] + (1.0f/j)*ftemp*ftemp;
			M = M + Ms_Ss[i + threadIdx.x];
			
			Ms_Ss[threadIdx.x]=M;
			Ms_Ss[blockDim.x + threadIdx.x]=S;
		}
		// in the last iteration we do not need to save the results... or synchronize threads...
		__syncthreads();
	}
	
	// by now we should have only 32 partial results. shuffle reduction follows
	for(int q=HALF_WARP; q>0; q=q>>1){
		j=j*2;
		ftemp = (M - __shfl_down(M, q));
		S = S + __shfl_down(S, q) + (1.0f/j)*ftemp*ftemp;
		M = M + __shfl_down(M, q);
	}
	
	//----------------------------------------------
	//---- Writing data
	if(threadIdx.x==0){
		pos = blockIdx.y*gridDim.x + blockIdx.x;
		if(msd) {
			// produce mean and sd instead of T and S
			d_output[3*pos] = M/j;
			d_output[3*pos + 1] = sqrt(S/j);
		}
		else {
			d_output[3*pos] = M;
			d_output[3*pos + 1] = S;
		}
	}
}

__global__ void BLN_outlier_rejection(float const* __restrict__ d_input, float *d_output, int size, float nElements, float multiplier) {
	__shared__ float Ms[WARP*WARP];
	__shared__ float Ss[WARP*WARP];
	__shared__ float js[WARP*WARP];
	__shared__ float s_signal_mean;
	__shared__ float s_signal_sd;
	
	
	int  pos; //warp_id,
	float M, Mt, S, j, jv;
	float ftemp;
	float signal_mean, signal_sd;
	
	//warp_id = threadIdx.x>>5;
	
	//----------------------------------------------
	//---- Calculation of the initial MSD
	pos=threadIdx.x;
	if(size>blockDim.x){
		M=__ldg(&d_input[3*pos]);
		S=__ldg(&d_input[3*pos+1]);
		j=nElements;
		pos = pos + blockDim.x;
		while (pos<size){
			jv=nElements;
			ftemp = ( jv/j*M - __ldg(&d_input[3*pos]) );
			S = S + __ldg(&d_input[3*pos+1]) + (j/(jv*(j+jv)))*ftemp*ftemp;
			M = M + __ldg(&d_input[3*pos]);
			j=j+jv;
			pos = pos + blockDim.x;
		}
		
		__syncthreads();
		
		Ms[threadIdx.x]=M;
		Ss[threadIdx.x]=S;
		js[threadIdx.x]=j;
		// now all threads had saved their work, reduction follows		
		// first we must load initial values
		for(int i=(blockDim.x>>1); i>HALF_WARP; i=i>>1){
			if(threadIdx.x<i){
				jv=js[i + threadIdx.x];
				ftemp = (jv/j*M - Ms[i + threadIdx.x]);
				S = S + Ss[i + threadIdx.x] + (j/(jv*(j+jv)))*ftemp*ftemp;
				M = M + Ms[i + threadIdx.x];
				j=j+jv;
				
				Ms[threadIdx.x]=M;
				Ss[threadIdx.x]=S;
				js[threadIdx.x]=j;
			}
			__syncthreads();
		}
		
		// by now we should have only 32 partial results. shuffle reduction follows
		for(int q=HALF_WARP; q>0; q=q>>1){
			jv=__shfl_down(j, q);
			ftemp = (jv/j*M - __shfl_down(M, q));
			S = S + __shfl_down(S, q) + (j/(jv*(j+jv)))*ftemp*ftemp;
			M = M + __shfl_down(M, q);
			j=j+jv;
		}
		
	}
	else {
		if(threadIdx.x==0){
			pos=0;
			M=__ldg(&d_input[3*pos]);
			S=__ldg(&d_input[3*pos+1]);
			j=nElements;
			for(pos=1; pos<size; pos++){
				jv=__ldg(&d_input[3*pos+2]);
				ftemp = ( jv/j*M - __ldg(&d_input[3*pos]) );
				S = S + __ldg(&d_input[3*pos+1]) + (j/(jv*(j+jv)))*ftemp*ftemp;
				M = M + __ldg(&d_input[3*pos]);
				j=j+jv;
			}
		}
	}
	
	if(threadIdx.x==0){
		s_signal_mean = M/j;
		s_signal_sd   = sqrt(S/j);
	}
	
	__syncthreads();
	
	signal_mean = s_signal_mean;
	signal_sd   = s_signal_sd;
	//---- Calculation of the initial MSD
	//----------------------------------------------
	
	//if(threadIdx.x==0) printf("Initial mean:%f; and standard deviation:%f;\n", signal_mean, signal_sd);

	//----------------------------------------------
	//---- Iterations with outlier rejection
	for(int f=0; f<5; f++){
		pos=threadIdx.x;
		if(size>blockDim.x){
			M=0;
			S=0;
			j=0;
			while (pos<size){
				Mt=__ldg(&d_input[3*pos]);
				if( (Mt/nElements > (signal_mean - multiplier*signal_sd)) && (Mt/nElements < (signal_mean + multiplier*signal_sd)) ){
					if(j==0){
						M = Mt;
						S = __ldg(&d_input[3*pos+1]);
						j = nElements;
					}
					else{
						jv=nElements;
						ftemp = ( jv/j*M - Mt);
						S = S + __ldg(&d_input[3*pos+1]) + (j/(jv*(j+jv)))*ftemp*ftemp;
						M = M + Mt;
						j=j+jv;
					}
				}
				pos = pos + blockDim.x;
			}
			
			__syncthreads();
			
			Ms[threadIdx.x]=M;
			Ss[threadIdx.x]=S;
			js[threadIdx.x]=j;
			// now all threads had saved their work, reduction follows		
			// first we must load initial values
			for(int i=(blockDim.x>>1); i>HALF_WARP; i=i>>1){
				if(threadIdx.x<i){
					jv=js[i + threadIdx.x];
					if(jv!=0){
						if(j==0){
							S = Ss[i + threadIdx.x];
							M = Ms[i + threadIdx.x];
							j = jv;
						}
						else {
							ftemp = (jv/j*M - Ms[i + threadIdx.x]);
							S = S + Ss[i + threadIdx.x] + (j/(jv*(j+jv)))*ftemp*ftemp;
							M = M + Ms[i + threadIdx.x];
							j=j+jv;
						}
					}
					
					Ms[threadIdx.x]=M;
					Ss[threadIdx.x]=S;
					js[threadIdx.x]=j;
				}
				__syncthreads();
			}
			
			// by now we should have only 32 partial results. shuffle reduction follows
			for(int q=HALF_WARP; q>0; q=q>>1){
				jv=__shfl_down(j, q);
				if(jv!=0){
					if(j==0) {
						S = __shfl_down(S, q);
						M = __shfl_down(M, q);
						j = jv;
					}
					else {
						ftemp = (jv/j*M - __shfl_down(M, q));
						S = S + __shfl_down(S, q) + (j/(jv*(j+jv)))*ftemp*ftemp;
						M = M + __shfl_down(M, q);
						j=j+jv;						
					}

				}
			}
			
		}
		else {
			if(threadIdx.x==0){
				M=0;
				S=0;
				j=0;
				for(pos=0; pos<size; pos++){
					Mt=__ldg(&d_input[3*pos]);
					if( (Mt/nElements > (signal_mean - multiplier*signal_sd)) && (Mt/nElements < (signal_mean + multiplier*signal_sd)) ){
						if(j==0){
							M=Mt;
							S=__ldg(&d_input[3*pos+1]);
							j=nElements;							
						}
						else{
							jv=nElements;
							ftemp = ( jv/j*M - __ldg(&d_input[3*pos]) );
							S = S + __ldg(&d_input[3*pos+1]) + (j/(jv*(j+jv)))*ftemp*ftemp;
							M = M + __ldg(&d_input[3*pos]);
							j=j+jv;
						}
					}
				}
			}
		}
		
		if(threadIdx.x==0){
			s_signal_mean = M/j;
			s_signal_sd   = sqrt(S/j);
		}
		
		__syncthreads();
		
		signal_mean = s_signal_mean;
		signal_sd   = s_signal_sd;
		
		//if(threadIdx.x==0) printf("Corrected mean:%f; and standard deviation:%f;\n", signal_mean, signal_sd);
	}
	//---- Iterations with outlier rejection
	//----------------------------------------------
	
	
	
	//----------------------------------------------
	//---- Writing data
	if(threadIdx.x==0){
		d_output[0] = signal_mean;
		d_output[1] = signal_sd;
		d_output[2] = j;
	}
	//---- Writing data
	//----------------------------------------------
}

#endif
