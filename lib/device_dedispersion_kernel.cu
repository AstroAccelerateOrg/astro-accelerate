#include "hip/hip_runtime.h"
#ifndef DEDISPERSE_KERNEL_H_
#define DEDISPERSE_KERNEL_H_

#define ARRAYSIZE SDIVINT * SDIVINDM

#include "float.h"
#include "headers/kernel_params.h"

// Stores temporary shift values
__device__ __constant__ float dm_shifts[8192];
__device__ __constant__ int i_nsamp, i_nchans, i_t_processed_s;
//__device__  __shared__ ushort2 f_line[UNROLLS][ARRAYSIZE + 1];

//{{{ shared_dedisperse_loop

__global__ void shared_dedisperse_kernel_range_0(int bin, unsigned short *d_input, float *d_output, float mstartdm, float mdmstep)
{
	extern __shared__ ushort2 f_line[];
	int i, j, c;
	int shift[UNROLLS_0];

	ushort temp_f;
	int local, unroll;

	float findex = ( threadIdx.x * 2 );
	float local_kernel_one[SNUMREG_0];
	float local_kernel_two[SNUMREG_0];

	for (i = 0; i < SNUMREG_0; i++)
	{
		local_kernel_one[i] = 0.0f;
		local_kernel_two[i] = 0.0f;
	}

	int idx = ( threadIdx.x + ( threadIdx.y * SDIVINT_0 ) );
	int nsamp_counter = ( idx + ( blockIdx.x * ( 2 * SNUMREG_0 * SDIVINT_0 ) ) );

	float shift_two = ( mstartdm + ( __int2float_rz(blockIdx.y) * SFDIVINDM_0 * mdmstep ) );
	float shift_one = ( __int2float_rz(threadIdx.y) * mdmstep );

	for (c = 0; c < i_nchans; c += UNROLLS_0)
	{

		__syncthreads();

		for (j = 0; j < UNROLLS_0; j++)
		{
			temp_f = ( __ldg(( d_input + ( __float2int_rz(dm_shifts[c + j] * shift_two) ) ) + ( nsamp_counter + ( j * i_nsamp ) )) );

			f_line[j*UNROLLS_0 + idx].x = temp_f;
			if (idx > 0)
			{
				f_line[j*UNROLLS_0 + idx - 1].y = temp_f;
			}
			shift[j] = __float2int_rz(shift_one * dm_shifts[c + j] + findex);
		}

		nsamp_counter = ( nsamp_counter + ( UNROLLS_0 * i_nsamp ) );

		__syncthreads();

		for (i = 0; i < SNUMREG_0; i++)
		{
			local = 0;
			unroll = ( i * 2 * SDIVINT_0 );
			for (j = 0; j < UNROLLS_0; j++)
			{
				local += *(int*) &f_line[j*UNROLLS_0 + ( shift[j] + unroll )];
			}
			local_kernel_one[i] += ( (ushort2*) ( &local ) )->x;
			local_kernel_two[i] += ( (ushort2*) ( &local ) )->y;
		}
	}

	// Write the accumulators to the output array. 
	local = ( ( ( ( blockIdx.y * SDIVINDM_0 ) + threadIdx.y ) * ( i_t_processed_s ) ) + ( blockIdx.x * 2 * SNUMREG_0 * SDIVINT_0 ) ) + 2 * threadIdx.x;

#pragma unroll
	for (i = 0; i < SNUMREG_0; i++)
	{
		*( (float2*) ( d_output + local + ( i * 2 * SDIVINT_0 ) ) ) = make_float2(local_kernel_one[i] / i_nchans / bin, local_kernel_two[i] / i_nchans / bin);
//		d_output[local + (i*2*SDIVINT_0)    ] = (local_kernel_one[i])/i_nchans;
//		d_output[local + (i*2*SDIVINT_0) + 1] = (local_kernel_two[i])/i_nchans;
	}
}


__global__ void shared_dedisperse_kernel_range_1(int bin, unsigned short *d_input, float *d_output, float mstartdm, float mdmstep)
{
	extern __shared__ ushort2 f_line[];
	int i, j, c;
	int shift[UNROLLS_1];

	ushort temp_f;
	int local, unroll;

	float findex = ( threadIdx.x * 2 );
	float local_kernel_one[SNUMREG_1];
	float local_kernel_two[SNUMREG_1];

	for (i = 0; i < SNUMREG_1; i++)
	{
		local_kernel_one[i] = 0.0f;
		local_kernel_two[i] = 0.0f;
	}

	int idx = ( threadIdx.x + ( threadIdx.y * SDIVINT_1 ) );
	int nsamp_counter = ( idx + ( blockIdx.x * ( 2 * SNUMREG_1 * SDIVINT_1 ) ) );

	float shift_two = ( mstartdm + ( __int2float_rz(blockIdx.y) * SFDIVINDM_1 * mdmstep ) );
	float shift_one = ( __int2float_rz(threadIdx.y) * mdmstep );

	for (c = 0; c < i_nchans; c += UNROLLS_1)
	{

		__syncthreads();

		for (j = 0; j < UNROLLS_1; j++)
		{
			temp_f = ( __ldg(( d_input + ( __float2int_rz(dm_shifts[c + j] * shift_two) ) ) + ( nsamp_counter + ( j * i_nsamp ) )) );

			f_line[j*UNROLLS_1 + idx].x = temp_f;
			if (idx > 0)
			{
				f_line[j*UNROLLS_1 + idx - 1].y = temp_f;
			}
			shift[j] = __float2int_rz(shift_one * dm_shifts[c + j] + findex);
		}

		nsamp_counter = ( nsamp_counter + ( UNROLLS_1 * i_nsamp ) );

		__syncthreads();

		for (i = 0; i < SNUMREG_1; i++)
		{
			local = 0;
			unroll = ( i * 2 * SDIVINT_1 );
			for (j = 0; j < UNROLLS_1; j++)
			{
				local += *(int*) &f_line[j*UNROLLS_1 + ( shift[j] + unroll )];
			}
			local_kernel_one[i] += ( (ushort2*) ( &local ) )->x;
			local_kernel_two[i] += ( (ushort2*) ( &local ) )->y;
		}
	}

	// Write the accumulators to the output array. 
	local = ( ( ( ( blockIdx.y * SDIVINDM_1 ) + threadIdx.y ) * ( i_t_processed_s ) ) + ( blockIdx.x * 2 * SNUMREG_1 * SDIVINT_1 ) ) + 2 * threadIdx.x;

#pragma unroll
	for (i = 0; i < SNUMREG_1; i++)
	{
		*( (float2*) ( d_output + local + ( i * 2 * SDIVINT_1 ) ) ) = make_float2(local_kernel_one[i] / i_nchans / bin, local_kernel_two[i] / i_nchans / bin);
//		d_output[local + (i*2*SDIVINT_1)    ] = (local_kernel_one[i])/i_nchans;
//		d_output[local + (i*2*SDIVINT_1) + 1] = (local_kernel_two[i])/i_nchans;
	}
}


__global__ void shared_dedisperse_kernel_range_2(int bin, unsigned short *d_input, float *d_output, float mstartdm, float mdmstep)
{
	extern __shared__ ushort2 f_line[];
	int i, j, c;
	int shift[UNROLLS_2];

	ushort temp_f;
	int local, unroll;

	float findex = ( threadIdx.x * 2 );
	float local_kernel_one[SNUMREG_2];
	float local_kernel_two[SNUMREG_2];

	for (i = 0; i < SNUMREG_2; i++)
	{
		local_kernel_one[i] = 0.0f;
		local_kernel_two[i] = 0.0f;
	}

	int idx = ( threadIdx.x + ( threadIdx.y * SDIVINT_2 ) );
	int nsamp_counter = ( idx + ( blockIdx.x * ( 2 * SNUMREG_2 * SDIVINT_2 ) ) );

	float shift_two = ( mstartdm + ( __int2float_rz(blockIdx.y) * SFDIVINDM_2 * mdmstep ) );
	float shift_one = ( __int2float_rz(threadIdx.y) * mdmstep );

	for (c = 0; c < i_nchans; c += UNROLLS_2)
	{

		__syncthreads();

		for (j = 0; j < UNROLLS_2; j++)
		{
			temp_f = ( __ldg(( d_input + ( __float2int_rz(dm_shifts[c + j] * shift_two) ) ) + ( nsamp_counter + ( j * i_nsamp ) )) );

			f_line[j*UNROLLS_2 + idx].x = temp_f;
			if (idx > 0)
			{
				f_line[j*UNROLLS_2 + idx - 1].y = temp_f;
			}
			shift[j] = __float2int_rz(shift_one * dm_shifts[c + j] + findex);
		}

		nsamp_counter = ( nsamp_counter + ( UNROLLS_2 * i_nsamp ) );

		__syncthreads();

		for (i = 0; i < SNUMREG_2; i++)
		{
			local = 0;
			unroll = ( i * 2 * SDIVINT_2 );
			for (j = 0; j < UNROLLS_2; j++)
			{
				local += *(int*) &f_line[j*UNROLLS_2 + ( shift[j] + unroll )];
			}
			local_kernel_one[i] += ( (ushort2*) ( &local ) )->x;
			local_kernel_two[i] += ( (ushort2*) ( &local ) )->y;
		}
	}

	// Write the accumulators to the output array. 
	local = ( ( ( ( blockIdx.y * SDIVINDM_2 ) + threadIdx.y ) * ( i_t_processed_s ) ) + ( blockIdx.x * 2 * SNUMREG_2 * SDIVINT_2 ) ) + 2 * threadIdx.x;

#pragma unroll
	for (i = 0; i < SNUMREG_2; i++)
	{
		*( (float2*) ( d_output + local + ( i * 2 * SDIVINT_2 ) ) ) = make_float2(local_kernel_one[i] / i_nchans / bin, local_kernel_two[i] / i_nchans / bin);
//		d_output[local + (i*2*SDIVINT_2)    ] = (local_kernel_one[i])/i_nchans;
//		d_output[local + (i*2*SDIVINT_2) + 1] = (local_kernel_two[i])/i_nchans;
	}
}


__global__ void shared_dedisperse_kernel_range_3(int bin, unsigned short *d_input, float *d_output, float mstartdm, float mdmstep)
{
	extern __shared__ ushort2 f_line[];
	int i, j, c;
	int shift[UNROLLS_3];

	ushort temp_f;
	int local, unroll;

	float findex = ( threadIdx.x * 2 );
	float local_kernel_one[SNUMREG_3];
	float local_kernel_two[SNUMREG_3];

	for (i = 0; i < SNUMREG_3; i++)
	{
		local_kernel_one[i] = 0.0f;
		local_kernel_two[i] = 0.0f;
	}

	int idx = ( threadIdx.x + ( threadIdx.y * SDIVINT_3 ) );
	int nsamp_counter = ( idx + ( blockIdx.x * ( 2 * SNUMREG_3 * SDIVINT_3 ) ) );

	float shift_two = ( mstartdm + ( __int2float_rz(blockIdx.y) * SFDIVINDM_3 * mdmstep ) );
	float shift_one = ( __int2float_rz(threadIdx.y) * mdmstep );

	for (c = 0; c < i_nchans; c += UNROLLS_3)
	{

		__syncthreads();

		for (j = 0; j < UNROLLS_3; j++)
		{
			temp_f = ( __ldg(( d_input + ( __float2int_rz(dm_shifts[c + j] * shift_two) ) ) + ( nsamp_counter + ( j * i_nsamp ) )) );

			f_line[j*UNROLLS_3 + idx].x = temp_f;
			if (idx > 0)
			{
				f_line[j*UNROLLS_3 + idx - 1].y = temp_f;
			}
			shift[j] = __float2int_rz(shift_one * dm_shifts[c + j] + findex);
		}

		nsamp_counter = ( nsamp_counter + ( UNROLLS_3 * i_nsamp ) );

		__syncthreads();

		for (i = 0; i < SNUMREG_3; i++)
		{
			local = 0;
			unroll = ( i * 2 * SDIVINT_3 );
			for (j = 0; j < UNROLLS_3; j++)
			{
				local += *(int*) &f_line[j*UNROLLS_3 + ( shift[j] + unroll )];
			}
			local_kernel_one[i] += ( (ushort2*) ( &local ) )->x;
			local_kernel_two[i] += ( (ushort2*) ( &local ) )->y;
		}
	}

	// Write the accumulators to the output array. 
	local = ( ( ( ( blockIdx.y * SDIVINDM_3 ) + threadIdx.y ) * ( i_t_processed_s ) ) + ( blockIdx.x * 2 * SNUMREG_3 * SDIVINT_3 ) ) + 2 * threadIdx.x;

#pragma unroll
	for (i = 0; i < SNUMREG_3; i++)
	{
		*( (float2*) ( d_output + local + ( i * 2 * SDIVINT_3 ) ) ) = make_float2(local_kernel_one[i] / i_nchans / bin, local_kernel_two[i] / i_nchans / bin);
//		d_output[local + (i*2*SDIVINT_3)    ] = (local_kernel_one[i])/i_nchans;
//		d_output[local + (i*2*SDIVINT_3) + 1] = (local_kernel_two[i])/i_nchans;
	}
}


__global__ void shared_dedisperse_kernel_range_4(int bin, unsigned short *d_input, float *d_output, float mstartdm, float mdmstep)
{
	extern __shared__ ushort2 f_line[];
	int i, j, c;
	int shift[UNROLLS_4];

	ushort temp_f;
	int local, unroll;

	float findex = ( threadIdx.x * 2 );
	float local_kernel_one[SNUMREG_4];
	float local_kernel_two[SNUMREG_4];

	for (i = 0; i < SNUMREG_4; i++)
	{
		local_kernel_one[i] = 0.0f;
		local_kernel_two[i] = 0.0f;
	}

	int idx = ( threadIdx.x + ( threadIdx.y * SDIVINT_4 ) );
	int nsamp_counter = ( idx + ( blockIdx.x * ( 2 * SNUMREG_4 * SDIVINT_4 ) ) );

	float shift_two = ( mstartdm + ( __int2float_rz(blockIdx.y) * SFDIVINDM_4 * mdmstep ) );
	float shift_one = ( __int2float_rz(threadIdx.y) * mdmstep );

	for (c = 0; c < i_nchans; c += UNROLLS_4)
	{

		__syncthreads();

		for (j = 0; j < UNROLLS_4; j++)
		{
			temp_f = ( __ldg(( d_input + ( __float2int_rz(dm_shifts[c + j] * shift_two) ) ) + ( nsamp_counter + ( j * i_nsamp ) )) );

			f_line[j*UNROLLS_4 + idx].x = temp_f;
			if (idx > 0)
			{
				f_line[j*UNROLLS_4 + idx - 1].y = temp_f;
			}
			shift[j] = __float2int_rz(shift_one * dm_shifts[c + j] + findex);
		}

		nsamp_counter = ( nsamp_counter + ( UNROLLS_4 * i_nsamp ) );

		__syncthreads();

		for (i = 0; i < SNUMREG_4; i++)
		{
			local = 0;
			unroll = ( i * 2 * SDIVINT_4 );
			for (j = 0; j < UNROLLS_4; j++)
			{
				local += *(int*) &f_line[j*UNROLLS_4 + ( shift[j] + unroll )];
			}
			local_kernel_one[i] += ( (ushort2*) ( &local ) )->x;
			local_kernel_two[i] += ( (ushort2*) ( &local ) )->y;
		}
	}

	// Write the accumulators to the output array. 
	local = ( ( ( ( blockIdx.y * SDIVINDM_4 ) + threadIdx.y ) * ( i_t_processed_s ) ) + ( blockIdx.x * 2 * SNUMREG_4 * SDIVINT_4 ) ) + 2 * threadIdx.x;

#pragma unroll
	for (i = 0; i < SNUMREG_4; i++)
	{
		*( (float2*) ( d_output + local + ( i * 2 * SDIVINT_4 ) ) ) = make_float2(local_kernel_one[i] / i_nchans / bin, local_kernel_two[i] / i_nchans / bin);
//		d_output[local + (i*2*SDIVINT_4)    ] = (local_kernel_one[i])/i_nchans;
//		d_output[local + (i*2*SDIVINT_4) + 1] = (local_kernel_two[i])/i_nchans;
	}
}


__global__ void shared_dedisperse_kernel_range_5(int bin, unsigned short *d_input, float *d_output, float mstartdm, float mdmstep)
{
	extern __shared__ ushort2 f_line[];
	int i, j, c;
	int shift[UNROLLS_5];

	ushort temp_f;
	int local, unroll;

	float findex = ( threadIdx.x * 2 );
	float local_kernel_one[SNUMREG_5];
	float local_kernel_two[SNUMREG_5];

	for (i = 0; i < SNUMREG_5; i++)
	{
		local_kernel_one[i] = 0.0f;
		local_kernel_two[i] = 0.0f;
	}

	int idx = ( threadIdx.x + ( threadIdx.y * SDIVINT_5 ) );
	int nsamp_counter = ( idx + ( blockIdx.x * ( 2 * SNUMREG_5 * SDIVINT_5 ) ) );

	float shift_two = ( mstartdm + ( __int2float_rz(blockIdx.y) * SFDIVINDM_5 * mdmstep ) );
	float shift_one = ( __int2float_rz(threadIdx.y) * mdmstep );

	for (c = 0; c < i_nchans; c += UNROLLS_5)
	{

		__syncthreads();

		for (j = 0; j < UNROLLS_5; j++)
		{
			temp_f = ( __ldg(( d_input + ( __float2int_rz(dm_shifts[c + j] * shift_two) ) ) + ( nsamp_counter + ( j * i_nsamp ) )) );

			f_line[j*UNROLLS_5 + idx].x = temp_f;
			if (idx > 0)
			{
				f_line[j*UNROLLS_5 + idx - 1].y = temp_f;
			}
			shift[j] = __float2int_rz(shift_one * dm_shifts[c + j] + findex);
		}

		nsamp_counter = ( nsamp_counter + ( UNROLLS_5 * i_nsamp ) );

		__syncthreads();

		for (i = 0; i < SNUMREG_5; i++)
		{
			local = 0;
			unroll = ( i * 2 * SDIVINT_5 );
			for (j = 0; j < UNROLLS_5; j++)
			{
				local += *(int*) &f_line[j*UNROLLS_5 + ( shift[j] + unroll )];
			}
			local_kernel_one[i] += ( (ushort2*) ( &local ) )->x;
			local_kernel_two[i] += ( (ushort2*) ( &local ) )->y;
		}
	}

	// Write the accumulators to the output array. 
	local = ( ( ( ( blockIdx.y * SDIVINDM_5 ) + threadIdx.y ) * ( i_t_processed_s ) ) + ( blockIdx.x * 2 * SNUMREG_5 * SDIVINT_5 ) ) + 2 * threadIdx.x;

#pragma unroll
	for (i = 0; i < SNUMREG_5; i++)
	{
		*( (float2*) ( d_output + local + ( i * 2 * SDIVINT_5 ) ) ) = make_float2(local_kernel_one[i] / i_nchans / bin, local_kernel_two[i] / i_nchans / bin);
//		d_output[local + (i*2*SDIVINT_5)    ] = (local_kernel_one[i])/i_nchans;
//		d_output[local + (i*2*SDIVINT_5) + 1] = (local_kernel_two[i])/i_nchans;
	}
}


__global__ void shared_dedisperse_kernel_range_6(int bin, unsigned short *d_input, float *d_output, float mstartdm, float mdmstep)
{
	extern __shared__ ushort2 f_line[];
	int i, j, c;
	int shift[UNROLLS_6];

	ushort temp_f;
	int local, unroll;

	float findex = ( threadIdx.x * 2 );
	float local_kernel_one[SNUMREG_6];
	float local_kernel_two[SNUMREG_6];

	for (i = 0; i < SNUMREG_6; i++)
	{
		local_kernel_one[i] = 0.0f;
		local_kernel_two[i] = 0.0f;
	}

	int idx = ( threadIdx.x + ( threadIdx.y * SDIVINT_6 ) );
	int nsamp_counter = ( idx + ( blockIdx.x * ( 2 * SNUMREG_6 * SDIVINT_6 ) ) );

	float shift_two = ( mstartdm + ( __int2float_rz(blockIdx.y) * SFDIVINDM_6 * mdmstep ) );
	float shift_one = ( __int2float_rz(threadIdx.y) * mdmstep );

	for (c = 0; c < i_nchans; c += UNROLLS_6)
	{

		__syncthreads();

		for (j = 0; j < UNROLLS_6; j++)
		{
			temp_f = ( __ldg(( d_input + ( __float2int_rz(dm_shifts[c + j] * shift_two) ) ) + ( nsamp_counter + ( j * i_nsamp ) )) );

			f_line[j*UNROLLS_6 + idx].x = temp_f;
			if (idx > 0)
			{
				f_line[j*UNROLLS_6 + idx - 1].y = temp_f;
			}
			shift[j] = __float2int_rz(shift_one * dm_shifts[c + j] + findex);
		}

		nsamp_counter = ( nsamp_counter + ( UNROLLS_6 * i_nsamp ) );

		__syncthreads();

		for (i = 0; i < SNUMREG_6; i++)
		{
			local = 0;
			unroll = ( i * 2 * SDIVINT_6 );
			for (j = 0; j < UNROLLS_6; j++)
			{
				local += *(int*) &f_line[j*UNROLLS_6 + ( shift[j] + unroll )];
			}
			local_kernel_one[i] += ( (ushort2*) ( &local ) )->x;
			local_kernel_two[i] += ( (ushort2*) ( &local ) )->y;
		}
	}

	// Write the accumulators to the output array. 
	local = ( ( ( ( blockIdx.y * SDIVINDM_6 ) + threadIdx.y ) * ( i_t_processed_s ) ) + ( blockIdx.x * 2 * SNUMREG_6 * SDIVINT_6 ) ) + 2 * threadIdx.x;

#pragma unroll
	for (i = 0; i < SNUMREG_6; i++)
	{
		*( (float2*) ( d_output + local + ( i * 2 * SDIVINT_6 ) ) ) = make_float2(local_kernel_one[i] / i_nchans / bin, local_kernel_two[i] / i_nchans / bin);
//		d_output[local + (i*2*SDIVINT_6)    ] = (local_kernel_one[i])/i_nchans;
//		d_output[local + (i*2*SDIVINT_6) + 1] = (local_kernel_two[i])/i_nchans;
	}
}


__global__ void shared_dedisperse_kernel_range_7(int bin, unsigned short *d_input, float *d_output, float mstartdm, float mdmstep)
{
	extern __shared__ ushort2 f_line[];
	int i, j, c;
	int shift[UNROLLS_7];

	ushort temp_f;
	int local, unroll;

	float findex = ( threadIdx.x * 2 );
	float local_kernel_one[SNUMREG_7];
	float local_kernel_two[SNUMREG_7];

	for (i = 0; i < SNUMREG_7; i++)
	{
		local_kernel_one[i] = 0.0f;
		local_kernel_two[i] = 0.0f;
	}

	int idx = ( threadIdx.x + ( threadIdx.y * SDIVINT_7 ) );
	int nsamp_counter = ( idx + ( blockIdx.x * ( 2 * SNUMREG_7 * SDIVINT_7 ) ) );

	float shift_two = ( mstartdm + ( __int2float_rz(blockIdx.y) * SFDIVINDM_7 * mdmstep ) );
	float shift_one = ( __int2float_rz(threadIdx.y) * mdmstep );

	for (c = 0; c < i_nchans; c += UNROLLS_7)
	{

		__syncthreads();

		for (j = 0; j < UNROLLS_7; j++)
		{
			temp_f = ( __ldg(( d_input + ( __float2int_rz(dm_shifts[c + j] * shift_two) ) ) + ( nsamp_counter + ( j * i_nsamp ) )) );

			f_line[j*UNROLLS_7 + idx].x = temp_f;
			if (idx > 0)
			{
				f_line[j*UNROLLS_7 + idx - 1].y = temp_f;
			}
			shift[j] = __float2int_rz(shift_one * dm_shifts[c + j] + findex);
		}

		nsamp_counter = ( nsamp_counter + ( UNROLLS_7 * i_nsamp ) );

		__syncthreads();

		for (i = 0; i < SNUMREG_7; i++)
		{
			local = 0;
			unroll = ( i * 2 * SDIVINT_7 );
			for (j = 0; j < UNROLLS_7; j++)
			{
				local += *(int*) &f_line[j*UNROLLS_7 + ( shift[j] + unroll )];
			}
			local_kernel_one[i] += ( (ushort2*) ( &local ) )->x;
			local_kernel_two[i] += ( (ushort2*) ( &local ) )->y;
		}
	}

	// Write the accumulators to the output array. 
	local = ( ( ( ( blockIdx.y * SDIVINDM_7 ) + threadIdx.y ) * ( i_t_processed_s ) ) + ( blockIdx.x * 2 * SNUMREG_7 * SDIVINT_7 ) ) + 2 * threadIdx.x;

#pragma unroll
	for (i = 0; i < SNUMREG_7; i++)
	{
		*( (float2*) ( d_output + local + ( i * 2 * SDIVINT_7 ) ) ) = make_float2(local_kernel_one[i] / i_nchans / bin, local_kernel_two[i] / i_nchans / bin);
//		d_output[local + (i*2*SDIVINT_7)    ] = (local_kernel_one[i])/i_nchans;
//		d_output[local + (i*2*SDIVINT_7) + 1] = (local_kernel_two[i])/i_nchans;
	}
}


#endif

