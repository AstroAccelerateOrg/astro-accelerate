#include <cutil_inline.h>

extern "C" float *malloc_gpu(size_t size, int zero_mem);

//{{{ malloc_gpu

float *malloc_gpu(size_t size, int zero_mem) {

	//{{{ Allocate GPU memory
	
//	hipEvent_t start, stop;
//	float time;
//	hipEventCreate(&start);
//	hipEventCreate(&stop);

//	printf("\n\n\tallocStart"),fflush(stdout);
//	hipEventRecord(start,0);

	float *device_pointer;

	cutilSafeCall( hipMalloc((void **) &device_pointer, size));

	if(zero_mem == 0) cutilSafeCall( hipMemset(device_pointer, 0, size));

//	hipEventRecord(stop, 0);
//	hipEventSynchronize(stop);
//	hipEventElapsedTime(&time, start, stop);
//	printf("\n\tallocStop"),fflush(stdout);        
//	printf("\n\tAllocated memory on GPU:\t\t\t%lf ms\n", time);

	//}}}
	
	return device_pointer;
	
}

//}}}
