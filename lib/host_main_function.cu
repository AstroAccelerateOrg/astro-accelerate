#include "headers/headers_mains.h"
#include "headers/device_bin.h"
#include "headers/device_init.h"
#include "headers/device_dedisperse.h"
#include "headers/device_dedispersion_kernel.h"
#include "headers/device_zero_dm.h"
#include "headers/device_zero_dm_outliers.h"
#include "headers/device_rfi.h"

#include "headers/device_BLN.h" //Added by KA
#include "headers/device_SPS_inplace_kernel.h" //Added by KA
#include "headers/device_SPS_inplace.h" //Added by KA
#include "headers/device_MSD_grid.h" //Added by KA
#include "headers/device_MSD_plane.h" //Added by KA
#include "headers/device_MSD_limited.h" //Added by KA
#include "headers/device_SNR_limited.h" //Added by KA
#include "headers/device_threshold.h" //Added by KA
#include "headers/device_single_FIR.h" //Added by KA
#include "headers/device_analysis.h" //Added by KA

#include "headers/device_peak_find.h" //Added by KA

#include "headers/device_load_data.h"
#include "headers/device_corner_turn.h"
#include "headers/device_save_data.h"
#include "headers/host_acceleration.h"
#include "headers/host_allocate_memory.h"
#include "headers/host_analysis.h"
#include "headers/host_periods.h"
#include "headers/host_debug.h"
#include "headers/host_get_file_data.h"
#include "headers/host_get_recorded_data.h"
#include "headers/host_get_user_input.h"
#include "headers/host_help.h"
#include "headers/host_rfi.h"
#include "headers/host_stratagy.h"
#include "headers/host_write_file.h"

// fdas
#include "headers/device_acceleration_fdas.h"

#include "headers/host_main_function.h"

#include "headers/params.h"

#include "timer.h"

void main_function
	(
	int argc,
	char* argv[],
	// Internal code variables
	// File pointers
	FILE *fp,
	// Counters and flags
	int i,
	int t,
	int dm_range,
	int range,
	int enable_debug,
	int enable_analysis,
	int enable_acceleration,
	int enable_periodicity,
	int output_dmt,
	int enable_zero_dm,
	int enable_zero_dm_with_outliers,
	int enable_rfi,
	int enable_fdas_custom_fft,
	int enable_fdas_inbin,
	int enable_fdas_norm,
	int *inBin,
	int *outBin,
	int *ndms,
	int maxshift,
	int max_ndms,
	int max_samps,
	int num_tchunks,
	int total_ndms,
	int multi_file,
	float max_dm,
	// Memory sizes and pointers
  size_t inputsize,
  size_t outputsize,
	size_t gpu_inputsize,
	size_t gpu_outputsize,
	size_t gpu_memory,
  unsigned short  *input_buffer,
	float ***output_buffer,
	unsigned short  *d_input,
	float *d_output,
	float *dmshifts,
	float *user_dm_low,
	float *user_dm_high,
	float *user_dm_step,
	float *dm_low,
	float *dm_high,
	float *dm_step,
	// Telescope parameters
	int nchans,
	int nsamp,
	int nbits,
	int nsamples,
	int nifs,
	int **t_processed,
	int nboots,
	int ntrial_bins,
	int navdms,
	int nsearch,
	float aggression,
	float narrow,
	float wide,
	int	maxshift_original,
	double	tsamp_original,
	long int inc,
	float tstart,
	float tstart_local,
	float tsamp,
	float fch1,
	float foff,
	// Analysis variables
	float power,
	float sigma_cutoff,
	float sigma_constant,
	float max_boxcar_width_in_sec,
	clock_t start_time
	)
{
	// Initialise the GPU.	
	init_gpu(argc, argv, enable_debug, &gpu_memory);
	if(enable_debug == 1) debug(2, start_time, range, outBin, enable_debug, enable_analysis, output_dmt, multi_file, sigma_cutoff, power, max_ndms, user_dm_low, user_dm_high,
	user_dm_step, dm_low, dm_high, dm_step, ndms, nchans, nsamples, nifs, nbits, tsamp, tstart, fch1, foff, maxshift, max_dm, nsamp, gpu_inputsize, gpu_outputsize, inputsize, outputsize);

	// Calculate the dedispersion stratagy.
	stratagy(&maxshift, &max_samps, &num_tchunks, &max_ndms, &total_ndms, &max_dm, power, nchans, nsamp, fch1, foff, tsamp, range, user_dm_low, user_dm_high, user_dm_step,
                 &dm_low, &dm_high, &dm_step, &ndms, &dmshifts, inBin, &t_processed, &gpu_memory);
	if(enable_debug == 1) debug(4, start_time, range, outBin, enable_debug, enable_analysis, output_dmt, multi_file, sigma_cutoff, power, max_ndms, user_dm_low, user_dm_high,
	user_dm_step, dm_low, dm_high, dm_step, ndms, nchans, nsamples, nifs, nbits, tsamp, tstart, fch1, foff, maxshift, max_dm, nsamp, gpu_inputsize, gpu_outputsize, inputsize, outputsize);

	// Allocate memory on host and device.
	allocate_memory_cpu_output(&fp, gpu_memory, maxshift, num_tchunks, max_ndms, total_ndms, nsamp, nchans, nbits, range, ndms, t_processed, &input_buffer, &output_buffer, &d_input, &d_output,
                        &gpu_inputsize, &gpu_outputsize, &inputsize, &outputsize);
	if(enable_debug == 1) debug(5, start_time, range, outBin, enable_debug, enable_analysis, output_dmt, multi_file, sigma_cutoff, power, max_ndms, user_dm_low, user_dm_high,
	user_dm_step, dm_low, dm_high, dm_step, ndms, nchans, nsamples, nifs, nbits, tsamp, tstart, fch1, foff, maxshift, max_dm, nsamp, gpu_inputsize, gpu_outputsize, inputsize, outputsize);

	// Allocate memory on host and device.
	allocate_memory_gpu(&fp, gpu_memory, maxshift, num_tchunks, max_ndms, total_ndms, nsamp, nchans, nbits, range, ndms, t_processed, &input_buffer, &output_buffer, &d_input, &d_output,
                        &gpu_inputsize, &gpu_outputsize, &inputsize, &outputsize);
	if(enable_debug == 1) debug(5, start_time, range, outBin, enable_debug, enable_analysis, output_dmt, multi_file, sigma_cutoff, power, max_ndms, user_dm_low, user_dm_high,
	user_dm_step, dm_low, dm_high, dm_step, ndms, nchans, nsamples, nifs, nbits, tsamp, tstart, fch1, foff, maxshift, max_dm, nsamp, gpu_inputsize, gpu_outputsize, inputsize, outputsize);

	// Clip RFI

	//rfi(nsamp, nchans, &input_buffer);
	/*
	 FILE	*fp_o;

	 if ((fp_o=fopen("rfi_clipped.dat", "wb")) == NULL) {
	 fprintf(stderr, "Error opening output file!\n");
	 exit(0);
	 }
	 fwrite(input_buffer, nchans*nsamp*sizeof(unsigned short), 1, fp_o);
	 */

	printf("\nDe-dispersing...");
	GpuTimer timer;
	timer.Start();


	tsamp_original = tsamp;
	maxshift_original = maxshift;

	GpuTimer range_timer;
	double *time_for_range;
	time_for_range = (double *) malloc(range*sizeof(time_for_range));
	for (dm_range=0; dm_range < range; dm_range++) time_for_range[dm_range]=0;

	for (t = 0; t < num_tchunks; t++)
	{
		printf("\nt_processed:\t%d, %d", t_processed[0][t], t);

		load_data(-1, inBin, d_input, &input_buffer[(long int) ( inc * nchans )], t_processed[0][t], maxshift, nchans, dmshifts);

		
		if (enable_zero_dm)
		{
			zero_dm(d_input, nchans, t_processed[0][t]+maxshift);
		}
		
		if (enable_zero_dm_with_outliers)
		{
			zero_dm_outliers(d_input, nchans, t_processed[0][t]+maxshift);
	 	}
	
		corner_turn(d_input, d_output, nchans, t_processed[0][t] + maxshift);
		
		if (enable_rfi)
		{
 			rfi_gpu(d_input, nchans, t_processed[0][t]+maxshift);
		}

		int oldBin = 1;
		for (dm_range = 0; dm_range < range; dm_range++)
		{
			// AB -- this is so that later ranges don't break if an early range is skipped
			if (FILTER_OUT_RANGES && dm_range!=RANGE_TO_KEEP) {
				if (inBin[dm_range] > oldBin)
				{
					bin_gpu(d_input, d_output, nchans, t_processed[dm_range - 1][t] + maxshift * inBin[dm_range]);
					( tsamp ) = ( tsamp ) * 2.0f;
				}
				continue;
			}
			// END AB

			printf("\n\n%f\t%f\t%f\t%d", dm_low[dm_range], dm_high[dm_range], dm_step[dm_range], ndms[dm_range]), fflush(stdout);
			printf("\nAmount of telescope time processed: %f", tstart_local);
			maxshift = maxshift_original / inBin[dm_range];

			hipDeviceSynchronize();
			range_timer.Start();
			load_data(dm_range, inBin, d_input, &input_buffer[(long int) ( inc * nchans )], t_processed[dm_range][t], maxshift, nchans, dmshifts);

			if (inBin[dm_range] > oldBin)
			{
				bin_gpu(d_input, d_output, nchans, t_processed[dm_range - 1][t] + maxshift * inBin[dm_range]);
				( tsamp ) = ( tsamp ) * 2.0f;
			}

			dedisperse(dm_range, t_processed[dm_range][t], inBin, dmshifts, d_input, d_output, nchans, ( t_processed[dm_range][t] + maxshift ), maxshift, &tsamp, dm_low, dm_high, dm_step, ndms);

			if (enable_acceleration == 1)
			{
				// gpu_outputsize = ndms[dm_range] * ( t_processed[dm_range][t] ) * sizeof(float);
				//save_data(d_output, out_tmp, gpu_outputsize);

				//#pragma omp parallel for
				for (int k = 0; k < ndms[dm_range]; k++)
				{
					//memcpy(&output_buffer[dm_range][k][inc / inBin[dm_range]], &out_tmp[k * t_processed[dm_range][t]], sizeof(float) * t_processed[dm_range][t]);

					save_data_offset(d_output, k * t_processed[dm_range][t], output_buffer[dm_range][k], inc / inBin[dm_range], sizeof(float) * t_processed[dm_range][t]);
				}
			//	save_data(d_output, &output_buffer[dm_range][0][((long int)inc)/inBin[dm_range]], gpu_outputsize);
			}
			hipDeviceSynchronize();
			range_timer.Stop();
			time_for_range[dm_range] += range_timer.Elapsed()/1000.0;;
			if (output_dmt == 1)
			{
				//for (int k = 0; k < ndms[dm_range]; k++)
				//	write_output(dm_range, t_processed[dm_range][t], ndms[dm_range], gpu_memory, output_buffer[dm_range][k], gpu_outputsize, dm_low, dm_high);
				//write_output(dm_range, t_processed[dm_range][t], ndms[dm_range], gpu_memory, out_tmp, gpu_outputsize, dm_low, dm_high);
			}
			if (enable_analysis == 1) {
				// TODO: put the file export back to analysis I leaving it here at the moment since for interface we need to output from the analysis.
				float *h_output_list;
				float *h_peak_list;
				size_t max_list_size, max_peak_size;
				size_t list_pos, peak_pos;
				max_list_size = (size_t) ( ndms[dm_range]*t_processed[dm_range][t]/2 ); // we can store 1/2 of the input plane
				max_peak_size = (size_t) ( ndms[dm_range]*t_processed[dm_range][t]/2 );
				h_output_list = (float*) malloc(max_list_size*4*sizeof(float)); // Allocations
				h_peak_list   = (float*) malloc(max_list_size*4*sizeof(float));
				
				list_pos=0;
				peak_pos=0;
				
				analysis_GPU(h_output_list, &list_pos, max_list_size, h_peak_list, &peak_pos, max_peak_size, dm_range, tstart_local, t_processed[dm_range][t], inBin[dm_range], outBin[dm_range], &maxshift, max_ndms, ndms, sigma_cutoff, sigma_constant, max_boxcar_width_in_sec, d_output, dm_low, dm_high, dm_step, tsamp);
				
				
				printf("-------> list_pos:%zu; \n", list_pos);
				#pragma omp parallel for
				for (int count = 0; count < list_pos; count++){
					h_output_list[4*count]     = h_output_list[4*count]*dm_step[dm_range] + dm_low[dm_range];
					h_output_list[4*count + 1] = h_output_list[4*count + 1]*tsamp + tstart_local;
					//h_output_list[4*count + 2] = h_output_list[4*count + 2];
					//h_output_list[4*count + 3] = h_output_list[4*count + 3];
					
				}
				
				#pragma omp parallel for
				for (int count = 0; count < peak_pos; count++){
					h_peak_list[4*count]     = h_peak_list[4*count]*dm_step[dm_range] + dm_low[dm_range];
					h_peak_list[4*count + 1] = h_peak_list[4*count + 1]*tsamp + tstart_local;
					//h_output_list[4*count + 2] = h_output_list[4*count + 2];
					//h_output_list[4*count + 3] = h_output_list[4*count + 3];
				}

				FILE *fp_out;
				char filename[200];
				
				if(list_pos>0){
					sprintf(filename, "analysed-t_%.2f-dm_%.2f-%.2f.dat", tstart_local, dm_low[dm_range], dm_high[dm_range]);
					//if ((fp_out=fopen(filename, "w")) == NULL) {
					if (( fp_out = fopen(filename, "wb") ) == NULL)	{
						fprintf(stderr, "Error opening output file!\n");
						exit(0);
					}
					fwrite(h_output_list, list_pos*sizeof(float), 4, fp_out);
					fclose(fp_out);
				}
				
				if(peak_pos>0){
					sprintf(filename, "peak_analysed-t_%.2f-dm_%.2f-%.2f.dat", tstart_local, dm_low[dm_range], dm_high[dm_range]);
					//if ((fp_out=fopen(filename, "w")) == NULL) {
					if (( fp_out = fopen(filename, "wb") ) == NULL)	{
						fprintf(stderr, "Error opening output file!\n");
						exit(0);
					}
					fwrite(h_peak_list, peak_pos*sizeof(float), 4, fp_out);
					fclose(fp_out);
				}
				
				
				free(h_peak_list);
				free(h_output_list);
				
				
				// This is for testing purposes and should be removed or commented out
				//analysis_CPU(dm_range, tstart_local, t_processed[dm_range][t], (t_processed[dm_range][t]+maxshift), nchans, maxshift, max_ndms, ndms, outBin, sigma_cutoff, out_tmp,dm_low, dm_high, dm_step, tsamp);
			}
			oldBin = inBin[dm_range];
		}

		//memset(out_tmp, 0.0f, t_processed[0][0] + maxshift * max_ndms * sizeof(float));

		inc = inc + t_processed[0][t];
		printf("\nINC:\t%ld", inc);
		tstart_local = ( tsamp_original * inc );
		tsamp = tsamp_original;
		maxshift = maxshift_original;
	}

	timer.Stop();
	float time = timer.Elapsed() / 1000;

	printf("\n\n === OVERALL DEDISPERSION THROUGHPUT INCLUDING SYNCS AND DATA TRANSFERS ===\n");

	printf("\n(Performed Brute-Force Dedispersion: %g (GPU estimate)",  time);
	printf("\nAmount of telescope time processed: %f", tstart_local);
	printf("\nNumber of samples processed: %ld", inc);
	printf("\nReal-time speedup factor: %lf", ( tstart_local ) / time);

	for (dm_range=0; dm_range<range; dm_range++){
		if (FILTER_OUT_RANGES && dm_range!=RANGE_TO_KEEP) continue;
		printf("\n%d SPEEDUP FACTOR (t processed/sec): %.8f time: %.8f\n", dm_range, tstart_local/time_for_range[dm_range], time_for_range[dm_range]);
	}
	free (time_for_range);

	hipFree(d_input);
	hipFree(d_output);
	//free(out_tmp);
	free(input_buffer);

	double time_processed = ( tstart_local ) / tsamp_original;
	double dm_t_processed = time_processed * total_ndms;
	double all_processed = dm_t_processed * nchans;
	printf("\nGops based on %.2lf ops per channel per tsamp: %f", NOPS, ( ( NOPS * all_processed ) / ( time ) ) / 1000000000.0);
	int num_reg = SNUMREG;
	float num_threads = total_ndms * ( t_processed[0][0] ) / ( num_reg );
	float data_size_loaded = ( num_threads * nchans * sizeof(ushort) ) / 1000000000;
	float time_in_sec = time;
	float bandwidth = data_size_loaded / time_in_sec;
	printf("\nDevice global memory bandwidth in GB/s: %f", bandwidth);
	printf("\nDevice shared memory bandwidth in GB/s: %f", bandwidth * ( num_reg ));
	float size_gb = ( nchans * ( t_processed[0][0] ) * sizeof(float) * 8 ) / 1000000000.0;
	printf("\nTelescope data throughput in Gb/s: %f", size_gb / time_in_sec);

	if (enable_periodicity == 1)
	{
		//
		GpuTimer timer;
		timer.Start();
		//
		periodicity(range, nsamp, max_ndms, inc, nboots, ntrial_bins, navdms, narrow, wide, nsearch, aggression, sigma_cutoff, output_buffer, ndms, inBin, dm_low, dm_high, dm_step, tsamp_original);
		//
		timer.Stop();
		float time = timer.Elapsed()/1000;
		printf("\n\n === OVERALL PERIODICITY THROUGHPUT INCLUDING SYNCS AND DATA TRANSFERS ===\n");

		printf("\nPerformed Peroidicity Location: %f (GPU estimate)", time);
		printf("\nAmount of telescope time processed: %f", tstart_local);
		printf("\nNumber of samples processed: %ld", inc);
		printf("\nReal-time speedup factor: %f", ( tstart_local ) / ( time ));
	}

	if (enable_acceleration == 1)
	{
		// Input needed for fdas is output_buffer which is DDPlan
		// Assumption: gpu memory is free and available
		//
		GpuTimer timer;
		timer.Start();
		// acceleration(range, nsamp, max_ndms, inc, nboots, ntrial_bins, navdms, narrow, wide, nsearch, aggression, sigma_cutoff, output_buffer, ndms, inBin, dm_low, dm_high, dm_step, tsamp_original);
		acceleration_fdas(range, nsamp, max_ndms, inc, nboots, ntrial_bins, navdms, narrow, wide, nsearch, aggression, sigma_cutoff,
						  output_buffer, ndms, inBin, dm_low, dm_high, dm_step, tsamp_original, enable_fdas_custom_fft, enable_fdas_inbin, enable_fdas_norm, sigma_constant);
		//
		timer.Stop();
		float time = timer.Elapsed()/1000;
		printf("\n\n === OVERALL TDAS THROUGHPUT INCLUDING SYNCS AND DATA TRANSFERS ===\n");

		printf("\nPerformed Acceleration Location: %lf (GPU estimate)", time);
		printf("\nAmount of telescope time processed: %f", tstart_local);
		printf("\nNumber of samples processed: %ld", inc);
		printf("\nReal-time speedup factor: %lf", ( tstart_local ) / ( time ));
	}
}
