#include "hip/hip_runtime.h"
#include "headers/headers_mains.h"

#include <hip/hip_runtime_api.h>

#include "headers/device_bin.h"
#include "headers/device_init.h"
#include "headers/device_dedisperse.h"
#include "headers/device_dedispersion_kernel.h"
#include "headers/device_zero_dm.h"
#include "headers/device_zero_dm_outliers.h"
#include "headers/device_rfi.h"

// MSD
#include "headers/device_MSD_Configuration.h"
#include "headers/device_MSD.h"
#include "headers/device_MSD_plane_profile.h"

#include "headers/device_SPS_inplace_kernel.h" //Added by KA
#include "headers/device_SPS_inplace.h" //Added by KA
#include "headers/device_SNR_limited.h" //Added by KA
#include "headers/device_SPS_long.h" //Added by KA
#include "headers/device_threshold.h" //Added by KA
#include "headers/device_single_FIR.h" //Added by KA
#include "headers/device_analysis.h" //Added by KA
#include "headers/device_periods.h" //Added by KA
#include "headers/device_peak_find.h" //Added by KA
#include "headers/device_power.h"
#include "headers/device_harmonic_summing.h"



#include "headers/device_load_data.h"
#include "headers/device_corner_turn.h"
#include "headers/device_save_data.h"
#include "headers/host_acceleration.h"
#include "headers/host_allocate_memory.h"
#include "headers/host_analysis.h"
#include "headers/host_export.h"
#include "headers/host_periods.h"
#include "headers/host_debug.h"
#include "headers/host_get_file_data.h"
#include "headers/host_get_recorded_data.h"
#include "headers/host_get_user_input.h"
#include "headers/host_help.h"
#include "headers/host_rfi.h"
#include "headers/host_stratagy.h"
#include "headers/host_MSD_stratagy.h"
#include "headers/host_write_file.h"

// fdas
#include "headers/device_acceleration_fdas.h"

#include "headers/host_main_function.h"

#include "headers/params.h"

#include "timer.h"


//#define EXPORT_DD_DATA

void main_function
	(
	int argc,
	char* argv[],
	// Internal code variables
	// File pointers
	FILE *fp,
	// Counters and flags
	int i,
	int t,
	int dm_range,
	int range,
	int enable_debug,
	int enable_analysis,
	int enable_acceleration,
	int enable_output_ffdot_plan,
	int enable_output_fdas_list,
	int enable_periodicity,
	int output_dmt,
	int enable_zero_dm,
	int enable_zero_dm_with_outliers,
	int enable_rfi,
	int enable_sps_baselinenoise,
	int enable_fdas_custom_fft,
	int enable_fdas_inbin,
	int enable_fdas_norm,
	int *inBin,
	int *outBin,
	int *ndms,
	int maxshift,
	int max_ndms,
	int max_samps,
	int num_tchunks,
	int total_ndms,
	int multi_file,
	float max_dm,
	// Memory sizes and pointers
  size_t inputsize,
  size_t outputsize,
	size_t gpu_inputsize,
	size_t gpu_outputsize,
	size_t gpu_memory,
  unsigned short  *input_buffer,
	float ***output_buffer,
	unsigned short  *d_input,
	float *d_output,
	float *dmshifts,
	float *user_dm_low,
	float *user_dm_high,
	float *user_dm_step,
	float *dm_low,
	float *dm_high,
	float *dm_step,
	// Telescope parameters
	int nchans,
	int nsamp,
	int nbits,
	int nsamples,
	int nifs,
	int **t_processed,
	int nboots,
	int ntrial_bins,
	int navdms,
	int nsearch,
	float aggression,
	float narrow,
	float wide,
	int	maxshift_original,
	double	tsamp_original,
	long int inc,
	float tstart,
	float tstart_local,
	float tsamp,
	float fch1,
	float foff,
	// Analysis variables
	float power,
	float sigma_cutoff,
	float sigma_constant,
	float max_boxcar_width_in_sec,
	clock_t start_time,
	int candidate_algorithm,
	int nb_selected_dm,
	float *selected_dm_low,
	float *selected_dm_high,
	int analysis_debug,
	int failsafe,
	float periodicity_sigma_cutoff,
	int periodicity_nHarmonics
	)
{

	// Initialise the GPU.	
	init_gpu(argc, argv, enable_debug, &gpu_memory);
	if(enable_debug == 1) debug(2, start_time, range, outBin, enable_debug, enable_analysis, output_dmt, multi_file, sigma_cutoff, power, max_ndms, user_dm_low, user_dm_high,
	user_dm_step, dm_low, dm_high, dm_step, ndms, nchans, nsamples, nifs, nbits, tsamp, tstart, fch1, foff, maxshift, max_dm, nsamp, gpu_inputsize, gpu_outputsize, inputsize, outputsize);

	checkCudaErrors(hipGetLastError());
	
	// Calculate the dedispersion stratagy.
	stratagy(&maxshift, &max_samps, &num_tchunks, &max_ndms, &total_ndms, &max_dm, power, nchans, nsamp, fch1, foff, tsamp, range, user_dm_low, user_dm_high, user_dm_step,
                 &dm_low, &dm_high, &dm_step, &ndms, &dmshifts, inBin, &t_processed, &gpu_memory, enable_analysis);
	if(enable_debug == 1) debug(4, start_time, range, outBin, enable_debug, enable_analysis, output_dmt, multi_file, sigma_cutoff, power, max_ndms, user_dm_low, user_dm_high,
	user_dm_step, dm_low, dm_high, dm_step, ndms, nchans, nsamples, nifs, nbits, tsamp, tstart, fch1, foff, maxshift, max_dm, nsamp, gpu_inputsize, gpu_outputsize, inputsize, outputsize);
//	printf("\n\n GPU_memory:\t %zu gpu: %zu",gpu_memory/1024/1024,gpu_inputsize);

	checkCudaErrors(hipGetLastError());
	
	// Allocate memory on host and device.
	printf("\nAllocate memory CPU ...\n");
	allocate_memory_cpu_output(&fp, gpu_memory, maxshift, num_tchunks, max_ndms, total_ndms, nsamp, nchans, nbits, range, ndms, t_processed, &input_buffer, &output_buffer, &d_input, &d_output,&gpu_inputsize, &gpu_outputsize, &inputsize, &outputsize);
	if(enable_debug == 1) debug(5, start_time, range, outBin, enable_debug, enable_analysis, output_dmt, multi_file, sigma_cutoff, power, max_ndms, user_dm_low, user_dm_high,
	user_dm_step, dm_low, dm_high, dm_step, ndms, nchans, nsamples, nifs, nbits, tsamp, tstart, fch1, foff, maxshift, max_dm, nsamp, gpu_inputsize, gpu_outputsize, inputsize, outputsize);
	printf("\ndone memory CPU ...\n");

	checkCudaErrors(hipGetLastError());
	
	// Allocate memory on host and device.
	printf("\nAllocate memory GPU ...\n");
	allocate_memory_gpu(&fp, gpu_memory, maxshift, num_tchunks, max_ndms, total_ndms, nsamp, nchans, nbits, range, ndms, t_processed, &input_buffer, &output_buffer, &d_input, &d_output,
                        &gpu_inputsize, &gpu_outputsize, &inputsize, &outputsize);
	if(enable_debug == 1) debug(5, start_time, range, outBin, enable_debug, enable_analysis, output_dmt, multi_file, sigma_cutoff, power, max_ndms, user_dm_low, user_dm_high,
	user_dm_step, dm_low, dm_high, dm_step, ndms, nchans, nsamples, nifs, nbits, tsamp, tstart, fch1, foff, maxshift, max_dm, nsamp, gpu_inputsize, gpu_outputsize, inputsize, outputsize);
	printf("\ndone memory GPU ...\n");

	checkCudaErrors(hipGetLastError());


        unsigned long int MSD_data_info;
	size_t MSD_profile_size_in_bytes;
        float *d_MSD_workarea = NULL;
	float *d_MSD_interpolated = NULL; 	
	ushort *d_MSD_output_taps = NULL;
	int *gmem_peak = NULL;
	int h_MSD_DIT_width;
	float *h_peak_list;
	float *h_MSD_interpolated, *h_MSD_DIT;
        size_t max_peak_size;
        size_t peak_pos;
        max_peak_size = (size_t) ( max_ndms*t_processed[0][0]/2 );
        h_peak_list   = (float*) malloc(max_peak_size*4*sizeof(float));
//	hipHostMalloc((void **) &h_peak_list, sizeof(float)*max_peak_size*4);

	checkCudaErrors(hipGetLastError());

	printf("\nStratagy MSD ...\n");
        stratagy_MSD(max_ndms,max_boxcar_width_in_sec, tsamp, t_processed[0][0], &MSD_data_info, &MSD_profile_size_in_bytes, &h_MSD_DIT_width);
	checkCudaErrors(hipGetLastError());
        
//	printf("Test data: %lld \t MSD_profile: %zu", MSD_data_info, MSD_profile_size_in_bytes);

	printf("\nAllocate memory MSD ...\n");
	allocate_memory_MSD(&d_MSD_workarea, &d_MSD_output_taps, &d_MSD_interpolated, &h_MSD_interpolated, &h_MSD_DIT, &gmem_peak, MSD_data_info, h_MSD_DIT_width, t_processed[0][0], MSD_profile_size_in_bytes);
	checkCudaErrors(hipGetLastError());


	printf("\n\n GPU_memory:\t %zu gpu: %zu\n\n",gpu_memory/1024/1024,gpu_inputsize);


	hipStream_t streams[NUM_STREAMS];
	for (int s=0; s < NUM_STREAMS;s++)
		hipStreamCreate(&streams[s]);

	
	// Clip RFI
	if (enable_rfi) {
		rfi(nsamp, nchans, &input_buffer);
	}
	/*
	 FILE	*fp_o;

	 if ((fp_o=fopen("rfi_clipped.dat", "wb")) == NULL) {
	 fprintf(stderr, "Error opening output file!\n");
	 exit(0);
	 }
	 fwrite(input_buffer, nchans*nsamp*sizeof(unsigned short), 1, fp_o);
	 */
		checkCudaErrors(hipGetLastError());

	printf("\nDe-dispersing...");
	GpuTimer timer;
	timer.Start();


	tsamp_original = tsamp;
	maxshift_original = maxshift;
	int t_pos;


	//float *out_tmp;
	//out_tmp = (float *) malloc(( t_processed[0][0] + maxshift ) * max_ndms * sizeof(float));
	//memset(out_tmp, 0.0f, t_processed[0][0] + maxshift * max_ndms * sizeof(float));

	// need to add remainder for num_tchunks
	for (t = 0; t < num_tchunks/NUM_STREAMS; t++) {
		for (int s = 0; s < NUM_STREAMS; s++){
	
		t_pos = t*NUM_STREAMS+s;
		printf("\nt_processed:\t%d, %d", t_processed[0][t_pos], t_pos);
		
		checkCudaErrors(hipGetLastError());

			load_data(-1, inBin, &d_input[(unsigned short)s*(t_processed[0][0]+maxshift)*nchans], &input_buffer[(long int) ( inc * nchans )], t_processed[0][t_pos], maxshift, nchans, dmshifts,streams[s]);
//			load_data(-1, inBin, &d_input[1], &input_buffer[(long int) ( inc * nchans )], t_processed[0][t], maxshift, nchans, dmshifts,streams[s]);

	

		checkCudaErrors(hipGetLastError());
		
		if (enable_zero_dm) {
			zero_dm(d_input, nchans, t_processed[0][t_pos]+maxshift);
		}
		
		checkCudaErrors(hipGetLastError());
		
		if (enable_zero_dm_with_outliers) {
			zero_dm_outliers(d_input, nchans, t_processed[0][t_pos]+maxshift);
	 	}
		
		checkCudaErrors(hipGetLastError());
	
		corner_turn(&d_input[(unsigned short)s*(t_processed[0][0]+maxshift)*nchans], &d_output[s*(t_processed[0][0]+maxshift)*nchans], nchans, t_processed[0][t_pos] + maxshift,streams[s]);
		
	
		checkCudaErrors(hipGetLastError());
		
		//if (enable_rfi) {
 		//	rfi_gpu(d_input, nchans, t_processed[0][t]+maxshift);
		//}
		
		checkCudaErrors(hipGetLastError());
		
		int oldBin = 1;
		for (dm_range = 0; dm_range < range; dm_range++) {
			printf("\n\n%f\t%f\t%f\t%d", dm_low[dm_range], dm_high[dm_range], dm_step[dm_range], ndms[dm_range]), fflush(stdout);
			printf("\nAmount of telescope time processed: %f", tstart_local);
			maxshift = maxshift_original / inBin[dm_range];

			checkCudaErrors(hipGetLastError());
			
			load_data(dm_range, inBin, &d_input[(unsigned short)(s*(t_processed[0][0]+maxshift)*nchans)], &input_buffer[(long int) ( inc * nchans )], t_processed[dm_range][t_pos], maxshift, nchans, dmshifts,streams[s]);
			
			checkCudaErrors(hipGetLastError());
			
			if (inBin[dm_range] > oldBin) {
				printf("\nBin process.....\n");
				bin_gpu(d_input, d_output, nchans, t_processed[dm_range - 1][t_pos] + maxshift * inBin[dm_range], streams[s]);
				( tsamp ) = ( tsamp ) * 2.0f;
			}
			
			checkCudaErrors(hipGetLastError());
			
	dedisperse(dm_range, t_processed[dm_range][t_pos], inBin, dmshifts, &d_input[(unsigned short)s*(t_processed[0][0]+maxshift)*nchans], &d_output[s*(t_processed[0][0]+maxshift)*nchans], nchans, ( t_processed[dm_range][t_pos] + maxshift ), maxshift, &tsamp, dm_low, dm_high, dm_step, ndms, nbits, streams[s], failsafe);
//	dedisperse(dm_range, t_processed[dm_range][t], inBin, dmshifts, d_input, d_output, nchans, ( t_processed[dm_range][t] + maxshift ), maxshift, &tsamp, dm_low, dm_high, dm_step, ndms, nbits, failsafe);
	
			checkCudaErrors(hipGetLastError());
//			hipStreamSynchronize(streams[s]);

//			if ( (enable_acceleration == 1) || (enable_periodicity == 1) || (analysis_debug ==1) ) {
			if ( (enable_acceleration == 1) || (1 == 1) || (analysis_debug ==1) ) {
				// gpu_outputsize = ndms[dm_range] * ( t_processed[dm_range][t] ) * sizeof(float);
				//save_data(d_output, out_tmp, gpu_outputsize);

				//#pragma omp parallel for
				for (int k = 0; k < ndms[dm_range]; k++) {
					//memcpy(&output_buffer[dm_range][k][inc / inBin[dm_range]], &out_tmp[k * t_processed[dm_range][t]], sizeof(float) * t_processed[dm_range][t]);
//					hipHostRegister(output_buffer[dm_range][k],sizeof(float)*(t_processed[dm_range][t_pos]),0);
					save_data_offset(&d_output[s*(t_processed[0][0]+maxshift)*nchans], k * t_processed[dm_range][t_pos], output_buffer[dm_range][k], inc / inBin[dm_range], sizeof(float) * t_processed[dm_range][t_pos], streams[s]);

				}
//				save_data(d_output, &output_buffer[dm_range][0][((long int)inc)/inBin[dm_range]], gpu_outputsize);
			}

			if (output_dmt == 1)
			{
				//for (int k = 0; k < ndms[dm_range]; k++)
				//	write_output(dm_range, t_processed[dm_range][t], ndms[dm_range], gpu_memory, output_buffer[dm_range][k], gpu_outputsize, dm_low, dm_high);
				//write_output(dm_range, t_processed[dm_range][t], ndms[dm_range], gpu_memory, out_tmp, gpu_outputsize, dm_low, dm_high);
			}
			
			checkCudaErrors(hipGetLastError());
			
			if (enable_analysis == 1) {
				
				printf("\n VALUE OF ANALYSIS DEBUG IS %d\n", analysis_debug);

				if (analysis_debug == 1) {
					float *out_tmp;
					gpu_outputsize = ndms[dm_range] * ( t_processed[dm_range][t_pos] ) * sizeof(float);
					out_tmp = (float *) malloc(( t_processed[0][0] + maxshift ) * max_ndms * sizeof(float));
					memset(out_tmp, 0.0f, t_processed[0][0] + maxshift * max_ndms * sizeof(float));
					save_data(d_output, out_tmp, gpu_outputsize);
					analysis_CPU(dm_range, tstart_local, t_processed[dm_range][t_pos], (t_processed[dm_range][t_pos]+maxshift), nchans, maxshift, max_ndms, ndms, outBin, sigma_cutoff, out_tmp,dm_low, dm_high, dm_step, tsamp, max_boxcar_width_in_sec);
					free(out_tmp);
				}
				else {
//					float *h_peak_list;
//					size_t max_peak_size;
//					size_t peak_pos;
//					max_peak_size = (size_t) ( ndms[dm_range]*t_processed[dm_range][t_pos]/2 );
//					h_peak_list   = (float*) malloc(max_peak_size*4*sizeof(float));
//					hipHostMalloc((void **)&h_peak_list,max_peak_size*4*sizeof(float));

					peak_pos=0;
//					analysis_GPU(h_peak_list, &peak_pos, max_peak_size, dm_range, tstart_local, t_processed[dm_range][t_pos], inBin[dm_range], outBin[dm_range], &maxshift, max_ndms, ndms, sigma_cutoff, sigma_constant, max_boxcar_width_in_sec, &d_output[s*(t_processed[0][0]+maxshift)*nchans], dm_low, dm_high, dm_step, tsamp, streams[s], candidate_algorithm, enable_sps_baselinenoise, d_MSD_workarea, d_MSD_output_taps, d_MSD_interpolated, h_MSD_DIT, h_MSD_interpolated, gmem_peak, MSD_data_info);
//					analysis_GPU(h_peak_list, &peak_pos, max_peak_size, dm_range, tstart_local, t_processed[dm_range][t_pos], inBin[dm_range], outBin[dm_range], &maxshift, max_ndms, ndms, sigma_cutoff, sigma_constant, max_boxcar_width_in_sec, &d_output[s*(t_processed[0][0]+maxshift)*nchans], dm_low, dm_high, dm_step, tsamp, streams[s], candidate_algorithm, enable_sps_baselinenoise, d_MSD_workarea, d_MSD_output_taps, d_MSD_interpolated, gmem_peak, MSD_data_info);

//					int old_s;
//					old_s == (s+1)%NUM_STREAMS;
//					if (s == 0) old_s=1;
//					else old_s=0;
//					hipStreamSynchronize(streams[old_s]);
//					free(h_peak_list);
//					hipHostFree(h_peak_list);
				}

				// This is for testing purposes and should be removed or commented out
				//analysis_CPU(dm_range, tstart_local, t_processed[dm_range][t], (t_processed[dm_range][t]+maxshift), nchans, maxshift, max_ndms, ndms, outBin, sigma_cutoff, out_tmp,dm_low, dm_high, dm_step, tsamp);
			}
			oldBin = inBin[dm_range];
		}

		//memset(out_tmp, 0.0f, t_processed[0][0] + maxshift * max_ndms * sizeof(float));

		inc = inc + t_processed[0][t_pos];
		printf("\nINC:\t%ld", inc);
		tstart_local = ( tsamp_original * inc );
		tsamp = tsamp_original;
		maxshift = maxshift_original;
	}
}

	timer.Stop();
	float time = timer.Elapsed() / 1000;

	printf("\n\n === OVERALL DEDISPERSION THROUGHPUT INCLUDING SYNCS AND DATA TRANSFERS ===\n");

	printf("\n(Performed Brute-Force Dedispersion: %g (GPU estimate)",  time);
	printf("\nAmount of telescope time processed: %f", tstart_local);
	printf("\nNumber of samples processed: %ld", inc);
	printf("\nReal-time speedup factor: %lf", ( tstart_local ) / time);

	hipFree(d_input);
	hipFree(d_output);
	hipFree(d_MSD_workarea);
	hipFree(d_MSD_output_taps);
	hipFree(d_MSD_interpolated);
	hipHostFree(input_buffer);
//	hipHostFree(h_peak_list);

	checkCudaErrors(hipGetLastError());
	
	#ifdef EXPORT_DD_DATA
		size_t DMs_per_file;
		int *ranges_to_export;
		ranges_to_export = new int[range];
		for(int f=0; f<range; f++) ranges_to_export[f]=1;
		printf("\n\n");
		printf("Exporting dedispersion data...\n");
		DMs_per_file = Calculate_sd_per_file_from_file_size(1000, inc, 1);
		printf("  DM per file: %d;\n", DMs_per_file);
		Export_DD_data(range, output_buffer, inc, ndms, inBin, dm_low, dm_high, dm_step, "DD_data", ranges_to_export, DMs_per_file);
		delete[] ranges_to_export;
	#endif

	double time_processed = ( tstart_local ) / tsamp_original;
	double dm_t_processed = time_processed * total_ndms;
	double all_processed = dm_t_processed * nchans;
	printf("\nGops based on %.2lf ops per channel per tsamp: %f", NOPS, ( ( NOPS * all_processed ) / ( time ) ) / 1000000000.0);
	int num_reg = SNUMREG;
	float num_threads = total_ndms * ( t_processed[0][0] ) / ( num_reg );
	float data_size_loaded = ( num_threads * nchans * sizeof(ushort) ) / 1000000000;
	float time_in_sec = time;
	float bandwidth = data_size_loaded / time_in_sec;
	printf("\nDevice shared memory bandwidth in GB/s: %f", bandwidth * ( num_reg ));
	float size_gb = ( nchans * ( t_processed[0][0] ) * sizeof(float) * 8 ) / 1000000000.0;
	printf("\nTelescope data throughput in Gb/s: %f", size_gb / time_in_sec);

	checkCudaErrors(hipGetLastError());

	if (enable_periodicity == 1) {
		//
		GpuTimer timer;
		timer.Start();
		//
//		GPU_periodicity(range, nsamp, max_ndms, inc, periodicity_sigma_cutoff, output_buffer, ndms, inBin, dm_low, dm_high, dm_step, tsamp_original, periodicity_nHarmonics, candidate_algorithm, enable_sps_baselinenoise, sigma_constant, h_MSD_DIT, h_MSD_interpolated, 0);
		GPU_periodicity(range, nsamp, max_ndms, inc, periodicity_sigma_cutoff, output_buffer, ndms, inBin, dm_low, dm_high, dm_step, tsamp_original, periodicity_nHarmonics, candidate_algorithm, enable_sps_baselinenoise, sigma_constant, 0);
		//
		timer.Stop();
		float time = timer.Elapsed()/1000;
		printf("\n\n === OVERALL PERIODICITY THROUGHPUT INCLUDING SYNCS AND DATA TRANSFERS ===\n");

		printf("\nPerformed Peroidicity Location: %f (GPU estimate)", time);
		printf("\nAmount of telescope time processed: %f", tstart_local);
		printf("\nNumber of samples processed: %ld", inc);
		printf("\nReal-time speedup factor: %f", ( tstart_local ) / ( time ));
	}

	if (enable_acceleration == 1) {
		// Input needed for fdas is output_buffer which is DDPlan
		// Assumption: gpu memory is free and available
		//
		GpuTimer timer;
		timer.Start();
		// acceleration(range, nsamp, max_ndms, inc, nboots, ntrial_bins, navdms, narrow, wide, nsearch, aggression, sigma_cutoff, output_buffer, ndms, inBin, dm_low, dm_high, dm_step, tsamp_original);
		acceleration_fdas(range, nsamp, max_ndms, inc, nboots, ntrial_bins, navdms, narrow, wide, nsearch, aggression, sigma_cutoff,
						  output_buffer, ndms, inBin, dm_low, dm_high, dm_step, tsamp_original, enable_fdas_custom_fft, enable_fdas_inbin, enable_fdas_norm, sigma_constant, enable_output_ffdot_plan, enable_output_fdas_list);
		//
		timer.Stop();
		float time = timer.Elapsed()/1000;
		printf("\n\n === OVERALL TDAS THROUGHPUT INCLUDING SYNCS AND DATA TRANSFERS ===\n");

		printf("\nPerformed Acceleration Location: %lf (GPU estimate)", time);
		printf("\nAmount of telescope time processed: %f", tstart_local);
		printf("\nNumber of samples processed: %ld", inc);
		printf("\nReal-time speedup factor: %lf", ( tstart_local ) / ( time ));
	}

hipStreamDestroy(streams[NUM_STREAMS-1]);
//	hipHostFree(h_MSD_DIT);
//	hipHostFree(h_MSD_interpolated);
}
