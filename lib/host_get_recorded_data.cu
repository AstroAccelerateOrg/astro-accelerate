#include <stdio.h>
#include <hip/hip_runtime.h>
//#include <omp.h>

/* Again this funtion uses a pointer to the file pointer so that it can update the position of the file pointer :)
 * Note the brakets surrounding the input_buffer, these are needed due to operator percidence...
 * (*input_buffer_odd)[(c*(*nsamp))  + total_data] = (float)temp_buffer[c];
 */

void get_recorded_data(FILE **fp, int nsamp, int nchans, int nbits, unsigned short **input_buffer, size_t *inputsize)
{

	int c;

	unsigned long int total_data;

	//{{{ Load in the raw data from the input file and transpose
	if (nbits == 8)
	{

		// Allocate a tempory buffer to store a line of frequency data
		unsigned char *temp_buffer = (unsigned char *) malloc(nchans * sizeof(unsigned char));

		// Read in the data, transpose it and store it in the input buffer
		total_data = 0;
		while (!feof(*fp))
		{

			if (fread(temp_buffer, sizeof(unsigned char), nchans, *fp) != nchans)
				break;
			for (c = 0; c < nchans; c++)
			{
				( *input_buffer )[c + total_data * ( nchans )] = (unsigned short) temp_buffer[c];
			}
			total_data++;

		}
	}
	else
	{
		printf("\n\n========================= ERROR =========================\n");
		printf(" This is a SKA prototype code and only runs with 8 bit data\n");
		printf("\n=========================================================\n");
	}

	//}}}
}
