
#include <omp.h>
#include <stdio.h>
#include "AstroAccelerate/params.h"
#include "device_rfi_kernel.cu"

//{{{ rfi_gpu

void rfi_gpu(unsigned short *d_input, int nchans, int nsamp) {

	int divisions_in_f  = 32;
	int num_blocks_f    = nchans/divisions_in_f;

	printf("\nCORNER TURN!");
	printf("\n%d %d", nsamp, nchans);
	printf("\n%d %d", divisions_in_f, 1);
	printf("\n%d %d", num_blocks_f, 1);

	dim3 threads_per_block(divisions_in_f, 1);
	dim3 num_blocks(num_blocks_f,1);

	double start_t, end_t;
	start_t = omp_get_wtime();

	rfi_gpu_kernel<<< num_blocks, threads_per_block >>>(d_input, nchans, nsamp);
	hipDeviceSynchronize();

	end_t = omp_get_wtime();
	float time = (float)(end_t-start_t);
	printf("\nPerformed RFI: %f (GPU estimate)", time);

	//}}}

}

//}}}

