#include "hip/hip_runtime.h"
#ifndef STATS_KERNEL_H_
#define STATS_KERNEL_H_

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include "headers/params.h"

//{{{ Set stats
__global__ void stats_kernel(int half_samps, float *d_sum, float *d_sum_square, float *d_signal_power)
{

	int t = blockIdx.x * blockDim.x * STATSLOOP + threadIdx.x;

	float local = 0.0;
	float sum = 0.0;
	float sum_square = 0.0;

	for (int i = t; i < t + STATSLOOP * blockDim.x; i += blockDim.x)
	{
		local = d_signal_power[i];
		sum += local;
		sum_square += local * local;
	}
	d_sum[blockIdx.x * blockDim.x + threadIdx.x] = sum;
	d_sum_square[blockIdx.x * blockDim.x + threadIdx.x] = sum_square;
}
#endif

