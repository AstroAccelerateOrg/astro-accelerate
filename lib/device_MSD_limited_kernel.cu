#include "hip/hip_runtime.h"
// Added by Karel Adamek 

#ifndef MSD_LIMITED_KERNEL_H_
#define MSD_LIMITED_KERNEL_H_

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "headers/params.h"



//----------------------------------------------------------------------------------------
//------------- Device functions
__device__ __inline__ void Initiate(float *M, float *S, float *j, float element){
	*M = element;
	*S = 0;
	*j = 1.0f;
}

__device__ __inline__ void Add_one(float *M, float *S, float *j, float element){
	float ftemp;
	*j = (*j) + 1.0f;
	*M = (*M) + element;
	ftemp = ( (*j)*element - (*M) );
	*S = (*S) + 1.0f / ( (*j)*( (*j) - 1.0f ) )*ftemp*ftemp;
}

__device__ __inline__ void Merge(float *A_M, float *A_S, float *A_j, float B_M, float B_S, float B_j){
	float ftemp;
	
	ftemp = ( B_j / (*A_j)*(*A_M) - B_M );
	(*A_S) = (*A_S) + B_S + ( (*A_j) / ( B_j*( (*A_j) + B_j ) ) )*ftemp*ftemp;
	(*A_M) = (*A_M) + B_M;
	(*A_j) = (*A_j) + B_j;
}

__device__ __inline__ void Reduce_SM(float *M, float *S, float *j, float *s_input){
	float jv;
	
	(*M)=s_input[threadIdx.x];
	(*S)=s_input[blockDim.x + threadIdx.x];
	(*j)=s_input[2*blockDim.x + threadIdx.x];
	
	for (int i = ( blockDim.x >> 1 ); i > HALF_WARP; i = i >> 1) {
		if (threadIdx.x < i) {
			jv = s_input[2*blockDim.x + i + threadIdx.x];
			if( ((int) jv)!=0){
				if( (*j)==0 ){
					(*S) = s_input[blockDim.x + i + threadIdx.x];
					(*M) = s_input[i + threadIdx.x];
					(*j) = jv;
				}
				else {
					Merge(M, S, j, s_input[i + threadIdx.x], s_input[blockDim.x + i + threadIdx.x], jv);
				}
			}
			
			s_input[threadIdx.x] = (*M);
			s_input[blockDim.x + threadIdx.x] = (*S);
			s_input[2*blockDim.x + threadIdx.x] = (*j);
		}
		__syncthreads();
	}
}

__device__ __inline__ void Reduce_SM_regular(float *M, float *S, float *j, float *s_input){
	(*M)=s_input[threadIdx.x];
	(*S)=s_input[blockDim.x + threadIdx.x];
	(*j)=s_input[2*blockDim.x + threadIdx.x];
	
	for (int i = ( blockDim.x >> 1 ); i > HALF_WARP; i = i >> 1) {
		if (threadIdx.x < i) {
			Merge(M, S, j, s_input[i + threadIdx.x], s_input[blockDim.x + i + threadIdx.x], s_input[2*blockDim.x + i + threadIdx.x]);
			
			s_input[threadIdx.x] = (*M);
			s_input[blockDim.x + threadIdx.x] = (*S);
			s_input[2*blockDim.x + threadIdx.x] = (*j);
		}
		__syncthreads();
	}
}

__device__ __inline__ void Reduce_WARP(float *M, float *S, float *j){
	float B_M, B_S, B_j;
	
	for (int q = HALF_WARP; q > 0; q = q >> 1) {
		B_M = __shfl_down((*M), q);
		B_S = __shfl_down((*S), q);
		B_j = __shfl_down((*j), q);
		
		if(B_j>0){
			if( (*j)==0 ) {
				(*S) = B_S;
				(*M) = B_M;
				(*j) = B_j;
			}
			else {
				Merge(M, S, j, B_M, B_S, B_j);
			}
		}
	}
}

__device__ __inline__ void Reduce_WARP_regular(float *M, float *S, float *j){
	for (int q = HALF_WARP; q > 0; q = q >> 1) {
		Merge(M, S, j, __shfl_down((*M), q), __shfl_down((*S), q), __shfl_down((*j), q));
	}
}

__device__ void Sum_partials_regular(float *M, float *S, float *j, float *d_input, float *s_input, int size){
	int pos;
	
	//----------------------------------------------
	//---- Summing partials
	pos = threadIdx.x;
	if (size > blockDim.x) {
		(*M) = d_input[3*pos];
		(*S) = d_input[3*pos + 1];
		(*j) = d_input[3*pos + 2];
		
		pos = pos + blockDim.x;
		while (pos < size) {
			Merge( M, S, j, d_input[3*pos], d_input[3*pos + 1], d_input[3*pos + 2]);
			pos = pos + blockDim.x;
		}

		s_input[threadIdx.x] = (*M);
		s_input[blockDim.x + threadIdx.x] = (*S);
		s_input[2*blockDim.x + threadIdx.x] = (*j);
		
		__syncthreads();

		Reduce_SM_regular( M, S, j, s_input);
		Reduce_WARP_regular(M, S, j);
	}
	else {
		if (threadIdx.x == 0) {
			pos = 0;
			(*M) = d_input[3*pos];
			(*S) = d_input[3*pos + 1];
			(*j) = d_input[3*pos + 2];
			
			for (pos = 1; pos < size; pos++) {
				Merge( M, S, j, d_input[3*pos], d_input[3*pos + 1], d_input[3*pos + 2]);
			}
		}
	}
	//---- Summing partials
	//----------------------------------------------
}

__device__ void Sum_partials_nonregular(float *M, float *S, float *j, float *d_input, float *s_input, int size){
	int pos;
	float jv;
	
	//----------------------------------------------
	//---- Summing partials
	pos = threadIdx.x;
	if (size > blockDim.x) {
		(*M) = 0;	(*S) = 0;	(*j) = 0;
		while (pos < size) {
			jv = d_input[3*pos + 2];
			if( ((int) jv)>0 ){
				if( (int) (*j)==0 ){
					(*M) = d_input[3*pos]; 
					(*S) = d_input[3*pos + 1];
					(*j) = jv;
				}
				else {
					Merge( M, S, j, d_input[3*pos], d_input[3*pos + 1], jv);
				}
			}
			pos = pos + blockDim.x;
		}

		s_input[threadIdx.x] = (*M);
		s_input[blockDim.x + threadIdx.x] = (*S);
		s_input[2*blockDim.x + threadIdx.x] = (*j);
		
		__syncthreads();

		Reduce_SM( M, S, j, s_input);
		Reduce_WARP(M, S, j);
	}
	else {
		if (threadIdx.x == 0) {
			pos = 0;
			(*M) = 0;	(*S) = 0;	(*j) = 0;
			for (pos = 1; pos < size; pos++) {
				jv = d_input[3*pos + 2];
				if( ((int) jv)!=0 ){
					if( (int) (*j)==0 ){
						(*M) = d_input[3*pos]; 
						(*S) = d_input[3*pos + 1];
						(*j) = jv;
					}
					else {
						Merge( M, S, j, d_input[3*pos], d_input[3*pos + 1], jv);
					}
				}
			}
		}
	}
	//---- Summing partials
	//----------------------------------------------
}

//------------- Device functions
//----------------------------------------------------------------------------------------


// Computes partials for mean and standard deviation of the data with offset at the end
// PD_THREADS could be replaced it is not required to be #defined
__global__ void MSD_GPU_limited(float const* __restrict__ d_input, float *d_output, int y_steps, int nTimesamples, int offset) {
	__shared__ float s_input[3*PD_NTHREADS];
	float M, S, j, ftemp;
	
	int spos = blockIdx.x*PD_NTHREADS + threadIdx.x;
	int gpos = blockIdx.y*y_steps*nTimesamples + spos;
	M=0;	S=0;	j=0;
	if( spos<(nTimesamples-offset) ){
		
		ftemp=__ldg(&d_input[gpos]);
		Initiate( &M, &S, &j, ftemp);
		
		gpos = gpos + nTimesamples;
		for (int yf = 1; yf < y_steps; yf++) {
			ftemp=__ldg(&d_input[gpos]);
			Add_one( &M, &S, &j, ftemp);
			gpos = gpos + nTimesamples;
		}
	}
	
	s_input[threadIdx.x] = M;
	s_input[blockDim.x + threadIdx.x] = S;
	s_input[2*blockDim.x + threadIdx.x] = j;
	
	__syncthreads();
	
	Reduce_SM( &M, &S, &j, s_input );
	Reduce_WARP( &M, &S, &j);
	
	//----------------------------------------------
	//---- Writing data
	if (threadIdx.x == 0) {
		gpos = blockIdx.y*gridDim.x + blockIdx.x;
		d_output[3*gpos] = M;
		d_output[3*gpos + 1] = S;
		d_output[3*gpos + 2] = j;
	}
}

// Computes mean and standard deviation from partial
__global__ void MSD_GPU_final_regular(float *d_input, float *d_output, int size) {
	__shared__ float s_input[3*WARP*WARP];

	float M, S, j;
	
	Sum_partials_regular( &M, &S, &j, d_input, s_input, size);

	//----------------------------------------------
	//---- Writing data
	if (threadIdx.x == 0) {
		d_output[0] = M / j;
		d_output[1] = sqrt(S / j);
		d_output[2] = j;
	}
}

__global__ void MSD_GPU_final_regular(float *d_input, float *d_MSD, float *d_pp, int size) {
	__shared__ float s_input[3*WARP*WARP];

	float M, S, j;
	
	Sum_partials_regular( &M, &S, &j, d_input, s_input, size);

	if(d_pp[2]>0){
		Merge(&M, &S, &j, d_pp[0], d_pp[1], d_pp[2]);
	}
	
	//----------------------------------------------
	//---- Writing data
	if (threadIdx.x == 0) {
		d_MSD[0] = M / j;
		d_MSD[1] = sqrt(S / j);
		d_MSD[2] = j;
		d_pp[0] = M;
		d_pp[1] = S;
		d_pp[2] = j;
	}
}


__global__ void MSD_GPU_final_nonregular(float *d_input, float *d_MSD, int size) {
	__shared__ float s_input[3*WARP*WARP];
	
	float M, S, j;

	Sum_partials_nonregular( &M, &S, &j, d_input, s_input, size);
	
	//----------------------------------------------
	//---- Writing data
	if (threadIdx.x == 0) {
		d_MSD[0] = M / j;
		d_MSD[1] = sqrt(S / j);
		d_MSD[2] = j;
	}
}

__global__ void MSD_GPU_final_nonregular(float *d_input, float *d_MSD, float *d_pp, int size) {
	__shared__ float s_input[3*WARP*WARP];
	
	float M, S, j;

	Sum_partials_nonregular( &M, &S, &j, d_input, s_input, size);
	
	if(d_pp[2]>0){
		Merge(&M, &S, &j, d_pp[0], d_pp[1], d_pp[2]);
	}
	
	//----------------------------------------------
	//---- Writing data
	if (threadIdx.x == 0) {
		d_MSD[0] = M / j;
		d_MSD[1] = sqrt(S / j);
		d_MSD[2] = j;
		d_pp[0] = M;
		d_pp[1] = S;
		d_pp[2] = j;
	}
}


__global__ void MSD_GPU_LA_ALL(float const* __restrict__ d_input, float *d_output, float *d_output_taps, int y_steps, int nTaps, int nTimesamples, int offset) {
	__shared__ float s_input[3*PD_NTHREADS];
	__shared__ float s_base[3*PD_NTHREADS];
	
	// MSD variables
	float M, S, j;
	float M_b, S_b, j_b;
	// FIR variables
	int d, gpos, spos, local_id;
	ushort EpT, limit;
	float2 ftemp1, ftemp2, ftemp3;
	float Bw[2];
	
	EpT = 2*PD_NTHREADS-nTaps+4;
	limit = blockDim.x - (nTaps>>2) - 1;

	// First y coordinate is separated
	//-------------------> FIR
	spos = blockIdx.x*(EpT) + 2*threadIdx.x;
	gpos = blockIdx.y*y_steps*nTimesamples + spos;
	Bw[0]=0; Bw[1]=0; j=0; j_b=0;
	if( (spos+4)<(nTimesamples-offset) ){
		// loading data for FIR filter. Each thread calculates two samples
		ftemp1.x= __ldg(&d_input[gpos]);	
		ftemp1.y= __ldg(&d_input[gpos+1]);
		ftemp2.x= __ldg(&d_input[gpos+2]);
		ftemp2.y= __ldg(&d_input[gpos+3]);
		ftemp3.x= __ldg(&d_input[gpos+4]);
		
		// Calculate FIR of 4 taps
		Bw[0]=ftemp1.x + ftemp1.y + ftemp2.x + ftemp2.y;
		Bw[1]=ftemp1.y + ftemp2.x + ftemp2.y + ftemp3.x;
		
		// Initialization of MSD variables for non-processed StrDev
		Initiate( &M_b, &S_b, &j_b, ftemp1.x );
		// First addition (second actually, but first done this way) non-processed StrDev
		Add_one( &M_b, &S_b, &j_b, ftemp1.y );
	}
	
	s_input[2*threadIdx.x] = Bw[0];
	s_input[2*threadIdx.x+1] = Bw[1];
	
	__syncthreads();
	
	// Calculating FIR up to nTaps
	for(d=4; d<nTaps; d=d+4){
		local_id = threadIdx.x+(d>>1);
		if( local_id<=limit ){
			Bw[0] = Bw[0] + s_input[2*local_id]; Bw[1] = Bw[1] + s_input[2*local_id+1];
		}
	}
	
	// Note: threads with local_id<0 which have wrong result create sums as well but are removed from final results later
	//       same is for base values as these would be included twice. First time here and next time in threadblock next to it
	//       this is due to halo needed for FIR filter	
	Initiate( &M, &S, &j, Bw[0] ); // Initialization of MSD variables for processed StrDev
	Add_one( &M, &S, &j, Bw[1] ); // First addition (second actually, but first done this way) processed StrDev
	
	
	// Rest of the iteration in y direction	
	for (int yf = 1; yf < y_steps; yf++) {
		__syncthreads();
		//-------------------> FIR
		spos = blockIdx.x*(EpT) + 2*threadIdx.x;
		gpos = blockIdx.y*y_steps*nTimesamples + yf*nTimesamples + spos;
		Bw[0]=0; Bw[1]=0;
		if( (spos+4)<(nTimesamples-offset) ){
			ftemp1.x= __ldg(&d_input[gpos]);	
			ftemp1.y= __ldg(&d_input[gpos+1]);
			ftemp2.x= __ldg(&d_input[gpos+2]);
			ftemp2.y= __ldg(&d_input[gpos+3]);
			ftemp3.x= __ldg(&d_input[gpos+4]);

			Bw[0]=ftemp1.x + ftemp1.y + ftemp2.x + ftemp2.y;
			Bw[1]=ftemp1.y + ftemp2.x + ftemp2.y + ftemp3.x;
			
			Add_one( &M_b, &S_b, &j_b, ftemp1.x );
			Add_one( &M_b, &S_b, &j_b, ftemp1.y );
		}
		
		s_input[2*threadIdx.x] = Bw[0];
		s_input[2*threadIdx.x+1] = Bw[1];
	
		__syncthreads();
	
		for(d=4; d<nTaps; d=d+4){	
			local_id = threadIdx.x+(d>>1);
			if( local_id<=limit ){
				Bw[0] = Bw[0] + s_input[2*local_id]; Bw[1] = Bw[1] + s_input[2*local_id+1];
			}
		}
		
		Add_one( &M, &S, &j, Bw[0] );
		Add_one( &M, &S, &j, Bw[1] );
	}
	
	__syncthreads();
	
	s_input[threadIdx.x] = 0;
	s_input[blockDim.x + threadIdx.x] = 0;
	s_input[2*blockDim.x + threadIdx.x] = 0;
	
	s_base[threadIdx.x] = 0;
	s_base[blockDim.x + threadIdx.x] = 0;
	s_base[2*blockDim.x + threadIdx.x] = 0;
	
	__syncthreads();

	spos=blockIdx.x*(EpT) + 2*threadIdx.x;	
	if( local_id<=limit ) {
		// Note: ommited number of samples in the last trailing threadblocks is due to -nTaps which is here. 
		//       Missing data should be contained in local_id. Thus this code is missing some time sample even it it does not need to. 
		//       When removed it produces different number of added time samples in j and j_b which is wierd
		if( spos<(nTimesamples-offset-nTaps) ) { // -nTaps might not be necessary
			s_input[local_id] = M;
			s_input[blockDim.x + local_id] = S;
			s_input[2*blockDim.x + local_id] = j;
			
			s_base[local_id] = M_b;
			s_base[blockDim.x + local_id] = S_b;
			s_base[2*blockDim.x + local_id] = j_b;
		}

	}
	__syncthreads();
	
	//------------------------------------------------------------------------------------
	//---------> StrDev of processed input
	Reduce_SM( &M, &S, &j, s_input );
	Reduce_WARP( &M, &S, &j);
	
	//------------------------------------------------------------------------------------
	//---------> StrDev of unprocessed input
	Reduce_SM( &M_b, &S_b, &j_b, s_base );
	Reduce_WARP( &M_b, &S_b, &j_b);

	//----------------------------------------------
	//---- Writing data
	if (threadIdx.x == 0) {
		gpos = blockIdx.y*gridDim.x + blockIdx.x;
		d_output_taps[3*gpos] = M;
		d_output_taps[3*gpos + 1] = S;
		d_output_taps[3*gpos + 2] = j;
		
		d_output[3*gpos] = M_b;
		d_output[3*gpos + 1] = S_b;
		d_output[3*gpos + 2] = j_b;
	}
}


__global__ void MSD_GPU_LA_ALL_Nth(float const* __restrict__ d_input, float const* __restrict   d_bv_in, float *d_output, float *d_output_taps, int y_steps, int nTaps, int nTimesamples, int offset) {
	__shared__ float s_input[3*PD_NTHREADS];
	__shared__ float s_base[3*PD_NTHREADS];
	
	// MSD variables
	float M, S, j;
	float M_b, S_b, j_b;
	// FIR variables
	int d, gpos, spos, local_id;
	ushort EpT, limit;
	float2 ftemp1, ftemp2, ftemp3;
	float Bw[2];
	
	EpT = 2*PD_NTHREADS-nTaps+4;
	limit = blockDim.x - (nTaps>>2) - 1;

	// First y coordinate is separated
	//-------------------> FIR
	spos = blockIdx.x*(EpT) + 2*threadIdx.x;
	gpos = blockIdx.y*y_steps*nTimesamples + spos;
	Bw[0]=0; Bw[1]=0; j=0; j_b=0;
	if( (spos+4)<(nTimesamples-offset) ){
		// loading data for FIR filter. Each thread calculates two samples
		ftemp1.x= __ldg(&d_input[gpos]);	
		ftemp1.y= __ldg(&d_input[gpos+1]);
		ftemp2.x= __ldg(&d_input[gpos+2]);
		ftemp2.y= __ldg(&d_input[gpos+3]);
		ftemp3.x= __ldg(&d_input[gpos+4]);
		
		// Calculate FIR of 4 taps
		Bw[0]=ftemp1.x + ftemp1.y + ftemp2.x + ftemp2.y;
		Bw[1]=ftemp1.y + ftemp2.x + ftemp2.y + ftemp3.x;
		
		Initiate( &M_b, &S_b, &j_b, __ldg(&d_bv_in[gpos]) );
		Add_one( &M_b, &S_b, &j_b, __ldg(&d_bv_in[gpos+1]) );
	}
	
	s_input[2*threadIdx.x] = Bw[0];
	s_input[2*threadIdx.x+1] = Bw[1];
	
	__syncthreads();
	
	// Calculating FIR up to nTaps
	for(d=4; d<nTaps; d=d+4){
		local_id = threadIdx.x+(d>>1);
		if( local_id<=limit ){
			Bw[0] = Bw[0] + s_input[2*local_id]; Bw[1] = Bw[1] + s_input[2*local_id+1];
		}
	}
	
	// Note: threads with local_id<0 which have wrong result create sums as well but are removed from final results later
	//       same is for base values as these would be included twice. First time here and next time in threadblock next to it
	//       this is due to halo needed for FIR filter
	Initiate( &M, &S, &j, __ldg(&d_bv_in[gpos]) + Bw[0] );
	Add_one( &M, &S, &j, __ldg(&d_bv_in[gpos+1]) + Bw[1] );
	
	
	// Rest of the iteration in y direction	
	for (int yf = 1; yf < y_steps; yf++) {
		__syncthreads();
		//-------------------> FIR
		spos = blockIdx.x*(EpT) + 2*threadIdx.x;
		gpos = blockIdx.y*y_steps*nTimesamples + yf*nTimesamples + spos;
		Bw[0]=0; Bw[1]=0;
		if( (spos+4)<(nTimesamples-offset) ){
			ftemp1.x= __ldg(&d_input[gpos]);	
			ftemp1.y= __ldg(&d_input[gpos+1]);
			ftemp2.x= __ldg(&d_input[gpos+2]);
			ftemp2.y= __ldg(&d_input[gpos+3]);
			ftemp3.x= __ldg(&d_input[gpos+4]);

			Bw[0]=ftemp1.x + ftemp1.y + ftemp2.x + ftemp2.y;
			Bw[1]=ftemp1.y + ftemp2.x + ftemp2.y + ftemp3.x;
			
			Add_one( &M_b, &S_b, &j_b, __ldg(&d_bv_in[gpos]) );
			Add_one( &M_b, &S_b, &j_b, __ldg(&d_bv_in[gpos+1]) );
		}
		
		s_input[2*threadIdx.x] = Bw[0];
		s_input[2*threadIdx.x+1] = Bw[1];
	
		__syncthreads();
	
		for(d=4; d<nTaps; d=d+4){	
			local_id = threadIdx.x+(d>>1);
			if( local_id<=limit ){
				Bw[0] = Bw[0] + s_input[2*local_id]; Bw[1] = Bw[1] + s_input[2*local_id+1];
			}
		}
		
		Add_one( &M, &S, &j, __ldg(&d_bv_in[gpos]) + Bw[0] );
		Add_one( &M, &S, &j, __ldg(&d_bv_in[gpos+1]) + Bw[1] );
	}
	
	__syncthreads();
	
	s_input[threadIdx.x] = 0;
	s_input[blockDim.x + threadIdx.x] = 0;
	s_input[2*blockDim.x + threadIdx.x] = 0;
	
	s_base[threadIdx.x] = 0;
	s_base[blockDim.x + threadIdx.x] = 0;
	s_base[2*blockDim.x + threadIdx.x] = 0;
	
	__syncthreads();

	spos=blockIdx.x*(EpT) + 2*threadIdx.x;	
	if( local_id<=limit ) {		
		// Note: ommited number of samples in the last trailing threadblocks is due to -nTaps which is here. 
		//       Missing data should be contained in local_id. Thus this code is missing some time sample even it it does not need to. 
		//       When removed it produces different number of added time samples in j and j_b which is wierd
		if( spos<(nTimesamples-offset-nTaps) ) { // -nTaps might not be necessary
			s_input[local_id] = M;
			s_input[blockDim.x + local_id] = S;
			s_input[2*blockDim.x + local_id] = j;
			
			s_base[local_id] = M_b;
			s_base[blockDim.x + local_id] = S_b;
			s_base[2*blockDim.x + local_id] = j_b;
		}

	}
	__syncthreads();
	
	//------------------------------------------------------------------------------------
	//---------> StrDev of processed input
	Reduce_SM( &M, &S, &j, s_input );
	Reduce_WARP( &M, &S, &j);
	
	//------------------------------------------------------------------------------------
	//---------> StrDev of unprocessed input
	Reduce_SM( &M_b, &S_b, &j_b, s_base );
	Reduce_WARP( &M_b, &S_b, &j_b);

	//----------------------------------------------
	//---- Writing data
	if (threadIdx.x == 0) {
		gpos = blockIdx.y*gridDim.x + blockIdx.x;
		d_output_taps[3*gpos] = M;
		d_output_taps[3*gpos + 1] = S;
		d_output_taps[3*gpos + 2] = j;
		
		d_output[3*gpos] = M_b;
		d_output[3*gpos + 1] = S_b;
		d_output[3*gpos + 2] = j_b;
	}
}


__global__ void MSD_GPU_final_create_LA(float *d_input, float *d_output, float *d_MSD_base, int nTaps, int size) {
	__shared__ float s_input[3*WARP*WARP];

	float M, S, j;

	Sum_partials_regular( &M, &S, &j, d_input, s_input, size);
	
	//----------------------------------------------
	//---- Writing data
	if (threadIdx.x == 0) {
		d_output[0] = d_MSD_base[0];
		d_output[1] = d_MSD_base[1];
		d_output[2] = (sqrt(S / j) - d_MSD_base[1])/( (float) (nTaps-1));
	}
}


__global__ void MSD_GPU_final_create_LA_Nth(float *d_input, float *d_output, float *d_MSD_base, float *d_MSD_DIT, int nTaps, int size, int DIT_value) {
	__shared__ float s_input[3*WARP*WARP];

	float M, S, j;

	Sum_partials_regular( &M, &S, &j, d_input, s_input, size);

	//----------------------------------------------
	//---- Writing data
	if (threadIdx.x == 0) {
		d_output[0] = d_MSD_base[0];
		d_output[1] = d_MSD_base[1];
		d_output[2] = (sqrt(S / j) - d_MSD_base[1])/( (float) nTaps);
		d_output[3] = d_MSD_DIT[0]*DIT_value; 
	}
}


#endif
