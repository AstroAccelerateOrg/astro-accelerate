#include "hip/hip_runtime.h"
//Added by Karel Adamek
//#define MSD_DEBUG

#include "headers/params.h"
#include "device_MSD_limited_kernel.cu"

int Choose_x_dim(int grid_dim){
	int seive[15] =	{ 32, 31, 29, 23, 19, 17, 16, 13, 11, 8, 7, 5, 4, 3, 2 };

	int f, nRest, nBlocks, N, N_accepted;

	N = 1;
	N_accepted = 1;
	for (int i = 0; i < 4; i++)	{
		for (f = 0; f < 15; f++) {
			nBlocks = grid_dim / seive[f];
			nRest = grid_dim - nBlocks*seive[f];
			if (nRest == 0) {
				N_accepted = N_accepted*N;
				N = seive[f];
				break;
			}
		}
		if (( N_accepted*N ) > 32 || N == 1)
			return ( N_accepted );
		grid_dim = grid_dim / N;
	}

	return ( N_accepted );
}

int Choose_y_dim(int grid_dim){
	int seive[5] = { 32, 16, 8, 4, 2 };

	int f, nRest, nBlocks, N;

	N = 1;
	for (f = 0; f < 5; f++) {
		nBlocks = grid_dim / seive[f];
		nRest = grid_dim - nBlocks*seive[f];
		if (nRest == 0) {
			N = seive[f];
			break;
		}
	}

	return ( N );
}

/*
int Choose_divider(int number, int max_divider){
	int seive[12]={2, 3, 4, 5, 7, 11, 13, 17, 19, 23, 29, 31};
	int f, nRest, nBlocks, N, N_accepted;
	
	N=1;N_accepted=1;
	do {
		N=1;
		for(f=0; f<12; f++){
			nBlocks=number/seive[f];
			nRest=number - nBlocks*seive[f];
			if(nRest==0) {
				N=seive[f];
				N_accepted=N_accepted*N;
				break;
			}
		}
		number=number/N;
	} while ( (N_accepted)<=max_divider && N>1 );
	
	return(N_accepted/N);
}
*/

void MSD_limited_init() {
	//---------> Specific nVidia stuff
	hipDeviceSetCacheConfig (hipFuncCachePreferShared);
	hipDeviceSetSharedMemConfig (hipSharedMemBankSizeFourByte);
}


int MSD_limited(float *d_input, float *d_MSD, int nDMs, int nTimesamples, int offset) {
	//---------> Task specific
	ushort nBlocks_x, nBlocks_y;
	int	nBlocks_total, nSteps_x, nSteps_y, nRest;
	float *d_output;
	
	//---------> determining data block size per kernel
	nSteps_x  = PD_NTHREADS;
	nBlocks_x = (int) ((nTimesamples-offset)/nSteps_x);
	nRest     = nTimesamples - offset - nBlocks_x*nSteps_x;
	if(nRest>32) nBlocks_x++;
		
	nSteps_y  = Choose_divider(nDMs,64);
	nBlocks_y = nDMs/nSteps_y;
	nBlocks_total=nBlocks_x*nBlocks_y;
	
	//---------> determining number of threads for final kernel
	int nThreads=2048;
	int itemp=0;
	while(itemp==0 && nThreads>32){
		nThreads=(nThreads>>1);
		itemp=(int) (nBlocks_total/(nThreads*32));
	}
	if(nThreads<32) nThreads=32;
	
	dim3 gridSize(nBlocks_x, nBlocks_y, 1);
	dim3 blockSize(PD_NTHREADS, 1, 1);
	dim3 final_gridSize(1, 1, 1);
	dim3 final_blockSize(nThreads, 1, 1);
	
	#ifdef MSD_DEBUG
	printf("\n\n");
	printf("----------------> MSD debug: (MSD_limited)\n");
	printf("Kernel for calculating partials:\n");
	printf("ThreadBlocks (TB) in x:%d; Elements processed by TB in x:%d; Remainder in x:%d is processed\n", nBlocks_x, nSteps_x, nRest);
	printf("ThreadBlocks (TB) in y:%d; Elements processed by TB in y:%d; Remainder in y:%d is processed\n", nBlocks_y, nSteps_y, 0);
	printf("gridSize=(%d,%d,%d)\n", gridSize.x, gridSize.y, gridSize.z);
	printf("blockSize=(%d,%d,%d)\n", blockSize.x, blockSize.y, blockSize.z);
	printf("Shared memory required: %0.3f B\n", (float) (PD_NTHREADS*3*4));
	printf("Kernel for final calculation of mean and standard deviation:\n");
	size_t free_mem,total_mem;
	hipMemGetInfo(&free_mem,&total_mem);
	printf("Memory required for temporary storage:%0.3f MB which is %d floats\n",(nBlocks_total*3*sizeof(float))/(1024.0*1024),  nBlocks_total*3);
	printf("Memory available:%0.3f MB \n", ((float) free_mem)/(1024.0*1024.0) );
	printf("gridSize=(%d,%d,%d)\n", final_gridSize.x, final_gridSize.y, final_gridSize.z);
	printf("blockSize=(%d,%d,%d)\n", final_blockSize.x, final_blockSize.y, final_blockSize.z);	
	printf("---------------------------<\n");
	#endif

	//---------> Allocation of temporary memory
	hipMalloc((void **) &d_output, nBlocks_total*3*sizeof(float));

	MSD_init();
	MSD_GPU_limited<<<gridSize,blockSize>>>(d_input, d_output, nDMs/nBlocks_y, nTimesamples, offset);
	MSD_GPU_final_regular<<<final_gridSize,final_blockSize>>>(d_output, d_MSD, nBlocks_total);

	hipFree(d_output);
	
	#ifdef MSD_DEBUG
	float h_MSD[3];
	hipMemcpy(h_MSD, d_MSD, 3*sizeof(float), hipMemcpyDeviceToHost); 
	printf("Output: Mean: %e, Standard deviation: %e; Elements:%zu;\n", h_MSD[0], h_MSD[1], (size_t) h_MSD[2]);
	printf("---------------------------<\n");
	#endif

	if (nRest < 32)	return ( nRest );
	else return ( 0 );
}


int MSD_linear_approximation(float *d_input, float *d_MSD_T, int nTaps, int nDMs, int nTimesamples, int offset){
	//---------> Task specific
	int nBlocks_x, nBlocks_y, nBlocks_total, nSteps_x, nSteps_y, nRest, nThreads, itemp; //epw = elements per warp 32 for float 64 for float2
	float *d_output;
	float *d_output_taps;
	float *d_MSD_T_base;

	//---------> determining data block size per kernel
	nSteps_x  = 2*PD_NTHREADS-nTaps+4;
	nBlocks_x = (int) ((nTimesamples-offset)/nSteps_x);
	nRest     = nTimesamples - offset - nBlocks_x*nSteps_x;
	if(nRest>128) nBlocks_x++;
	
	nSteps_y = Choose_divider(nDMs,64);
	nBlocks_y=nDMs/nSteps_y;
	nBlocks_total=nBlocks_x*nBlocks_y;
	
	dim3 gridSize(nBlocks_x, nBlocks_y, 1);
	dim3 blockSize(PD_NTHREADS, 1, 1);
	
	//---------> determining number of threads for final kernel
	nThreads=2048;
	itemp=0;
	while(itemp==0 && nThreads>32){
		nThreads=(nThreads>>1);
		itemp=(int) (nBlocks_total/(nThreads*32));
	}
	if(nThreads<32) nThreads=32;
	
	dim3 final_gridSize(1, 1, 1);
	dim3 final_blockSize(nThreads, 1, 1);

	#ifdef MSD_DEBUG
	printf("\n\n");
	printf("----------------> MSD debug:\n");
	printf("Kernel for calculating partials:\n");
	printf("ThreadBlocks (TB) in x:%d; Elements processed by TB in x:%d; Remainder in x:%d", nBlocks_x, nSteps_x, nRest);
	if(nRest>3*nTaps)//printf(" is processed\n");
	else//printf(" is not processed\n");
	printf("ThreadBlocks (TB) in y:%d; Elements processed by TB in y:%d; Remainder in y:%d is processed\n", nBlocks_y, nSteps_y, 0);
	printf("gridSize=(%d,%d,%d)\n", gridSize.x, gridSize.y, gridSize.z);
	printf("blockSize=(%d,%d,%d)\n", blockSize.x, blockSize.y, blockSize.z);
	printf("Shared memory required: %0.3f B\n", (float) (PD_NTHREADS*24));
	printf("Kernel for final calculation of mean and standard deviation:\n");
	size_t free_mem,total_mem;
	hipMemGetInfo(&free_mem,&total_mem);
	printf("Memory required for temporary storage:%0.3f MB which is %d floats\n",(nBlocks_total*3*2*sizeof(float))/(1024.0*1024),  nBlocks_total*3*2);
	printf("Memory available:%0.3f MB \n", ((float) free_mem)/(1024.0*1024.0) );
	printf("gridSize=(%d,%d,%d)\n", final_gridSize.x, final_gridSize.y, final_gridSize.z);
	printf("blockSize=(%d,%d,%d)\n", final_blockSize.x, final_blockSize.y, final_blockSize.z);	
	printf("\n");
	#endif
	
	if(nBlocks_total>0){
		//---------> Allocation of temporary memory
		if ( hipSuccess != hipMalloc((void **) &d_output, nBlocks_total*3*sizeof(float))) {printf("Allocation error!\n"); exit(1001);}
		if ( hipSuccess != hipMalloc((void **) &d_output_taps, nBlocks_total*3*sizeof(float))) {printf("Allocation error!\n"); exit(1001);}
		if ( hipSuccess != hipMalloc((void **) &d_MSD_T_base, sizeof(float)*3)) {printf("Allocation error!\n"); exit(1001);}
		
		//---------> MSD
		MSD_init();
		MSD_GPU_LA_ALL<<<gridSize,blockSize>>>(d_input, d_output, d_output_taps, nSteps_y, nTaps, nTimesamples, offset);
		MSD_GPU_final_regular<<<final_gridSize, final_blockSize>>>(d_output, d_MSD_T_base, nBlocks_total);
		MSD_GPU_final_create_LA<<<final_gridSize, final_blockSize>>>(d_output_taps, d_MSD_T, d_MSD_T_base, nTaps, nBlocks_total);
		
		#ifdef MSD_DEBUG
		float h_MSD_T[3], h_MSD_T_base[3];
		hipMemcpy(h_MSD_T, d_MSD_T, 3*sizeof(float), hipMemcpyDeviceToHost); 
		hipMemcpy(h_MSD_T_base, d_MSD_T_base, 3*sizeof(float), hipMemcpyDeviceToHost);
		printf("Output: Mean: %e, Standard deviation: %e; modifier:%e;\n", h_MSD_T[0], h_MSD_T[1], h_MSD_T[2]);
		printf("GPU results after 1 taps: Mean: %e, Standard deviation: %e; Number of elements:%d;\n", h_MSD_T_base[0], h_MSD_T_base[1], (int) h_MSD_T_base[2]);
		printf("---------------------------<\n");
		#endif
		
		//---------> De-allocation of temporary memory
		hipFree(d_output);
		hipFree(d_output_taps);
		hipFree(d_MSD_T_base);
	}
	else {
		printf("Number of time samples is too small! Increase number of samples send to the boxcar filters. (MSD_linear_approximation)\n");
		exit(1002);
	}
	
	if(nRest<64) return(nRest);
	else return(0);	
}


int MSD_LA_Nth(float *d_input, float *d_bv_in, float *d_MSD_T, float *d_MSD_DIT, int nTaps, int nDMs, int nTimesamples, int offset, int DIT_value){
	//---------> Task specific
	int nBlocks_x, nBlocks_y, nBlocks_total, nSteps_x, nSteps_y, nRest, nThreads, itemp; //epw = elements per warp 32 for float 64 for float2
	float *d_output;
	float *d_output_FIR;
	float *d_MSD_T_base;

	//---------> determining data block size per kernel
	nSteps_x  = 2*PD_NTHREADS-nTaps+4;
	nBlocks_x = (int) ((nTimesamples-offset)/nSteps_x);
	nRest     = nTimesamples - offset - nBlocks_x*nSteps_x;
	if(nRest>0) nBlocks_x++;
	
	nSteps_y = Choose_divider(nDMs,64);
	nBlocks_y=nDMs/nSteps_y;
	nBlocks_total=nBlocks_x*nBlocks_y;
	
	dim3 gridSize(nBlocks_x, nBlocks_y, 1);
	dim3 blockSize(PD_NTHREADS, 1, 1);
	
	//---------> determining number of threads for final kernel
	nThreads=2048;
	itemp=0;
	while(itemp==0 && nThreads>32){
		nThreads=(nThreads>>1);
		itemp=(int) (nBlocks_total/(nThreads*32));
	}
	if(nThreads<32) nThreads=32;
	
	dim3 final_gridSize(1, 1, 1);
	dim3 final_blockSize(nThreads, 1, 1);
	
	#ifdef MSD_DEBUG
	printf("\n\n");
	printf("----------------> MSD debug:\n");
	printf("Kernel for calculating partials: (MSD_LA_Nth)\n");
	printf("nTimesamples:%d; offset:%d, nDMs:%d;\n", nTimesamples, offset, nDMs);
	printf("ThreadBlocks (TB) in x:%d; Elements processed by TB in x:%d; Remainder in x:%d", nBlocks_x, nSteps_x, nRest);
	if(nRest>3*nTaps)//printf(" is processed\n");
	else//printf(" is not processed\n");
	printf("ThreadBlocks (TB) in y:%d; Elements processed by TB in y:%d; Remainder in y:%d is processed\n", nBlocks_y, nSteps_y, 0);
	printf("gridSize=(%d,%d,%d)\n", gridSize.x, gridSize.y, gridSize.z);
	printf("blockSize=(%d,%d,%d)\n", blockSize.x, blockSize.y, blockSize.z);
	printf("Shared memory required: %0.3f B\n", (float) (PD_NTHREADS*24));
	printf("Kernel for final calculation of mean and standard deviation:\n");
	size_t free_mem,total_mem;
	hipMemGetInfo(&free_mem,&total_mem);
	printf("Memory required for temporary storage:%0.3f MB which is %d floats\n",(nBlocks_total*3*2*sizeof(float))/(1024.0*1024),  nBlocks_total*3*2);
	printf("Memory available:%0.3f MB \n", ((float) free_mem)/(1024.0*1024.0) );
	printf("gridSize=(%d,%d,%d)\n", final_gridSize.x, final_gridSize.y, final_gridSize.z);
	printf("blockSize=(%d,%d,%d)\n", final_blockSize.x, final_blockSize.y, final_blockSize.z);	
	printf("\n");
	#endif
	
	if(nBlocks_total>0){
		//---------> Allocation of temporary memory
		if ( hipSuccess != hipMalloc((void **) &d_output, nBlocks_total*3*sizeof(float))) {printf("Allocation error!\n"); exit(1001);}
		if ( hipSuccess != hipMalloc((void **) &d_output_FIR, nBlocks_total*3*sizeof(float))) {printf("Allocation error!\n"); exit(1001);}
		if ( hipSuccess != hipMalloc((void **) &d_MSD_T_base, sizeof(float)*3)) {printf("Allocation error!\n"); exit(1001);}
		
		//---------> MSD
		MSD_init();
		MSD_GPU_LA_ALL_Nth<<<gridSize,blockSize>>>(d_input, d_bv_in, d_output, d_output_FIR, nSteps_y, nTaps, nTimesamples, offset);
		MSD_GPU_final_regular<<<final_gridSize, final_blockSize>>>(d_output, d_MSD_T_base, nBlocks_total);
		MSD_GPU_final_create_LA_Nth<<<final_gridSize, final_blockSize>>>(d_output_FIR, d_MSD_T, d_MSD_T_base, d_MSD_DIT, nTaps, nBlocks_total, DIT_value);
		
		#ifdef MSD_DEBUG
		float h_MSD_T[4], h_MSD_T_base[3], h_MSD_DIT[3];
		hipMemcpy(h_MSD_T, d_MSD_T, 4*sizeof(float), hipMemcpyDeviceToHost); 
		hipMemcpy(h_MSD_T_base, d_MSD_T_base, 3*sizeof(float), hipMemcpyDeviceToHost);
		hipMemcpy(h_MSD_DIT, d_MSD_DIT, 3*sizeof(float), hipMemcpyDeviceToHost);
		printf("d_MSD_T: BV Mean: %f, Standard deviation: %f; modifier:%f; DIT Mean:%f\n", h_MSD_T[0], h_MSD_T[1], h_MSD_T[2], h_MSD_T[3]);
		printf("MSD for d_bv_in: Mean: %f, Standard deviation: %f; Number of elements:%d;\n", h_MSD_T_base[0], h_MSD_T_base[1], (int) h_MSD_T_base[2]);
		printf("MSD for DIT: Mean: %f, Standard deviation: %f; Number of elements:%d;\n", h_MSD_DIT[0], h_MSD_DIT[1], (int) h_MSD_DIT[2]);
		printf("---------------------------<\n");
		#endif
		
		//---------> De-allocation of temporary memory
		hipFree(d_output);
		hipFree(d_output_FIR);
		hipFree(d_MSD_T_base);
	}
	else {
		printf("WARNING: Number of time samples is too small! Increase number of samples send to the boxcar filters. (MSD_LA_Nth)\n");
		return(1);
	}

	return(0);
}

