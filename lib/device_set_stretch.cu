#include <omp.h>
#include <stdio.h>
#include "headers/params.h"
#include "device_set_stretch_kernel.cu"
#include "hip/hip_runtime_api.h"

//{{{ Dopler Stretch 

void set_stretch_gpu(hipEvent_t event, hipStream_t stream, int samps, float mean, float *d_input)
{

	int divisions_in_t = 32;
	int num_blocks_t = samps / divisions_in_t;

	dim3 threads_per_block(divisions_in_t);
	dim3 num_blocks(num_blocks_t);

	hipStreamWaitEvent(stream, event, 0);
	set_stretch_kernel<<<num_blocks, threads_per_block, 0, stream>>>(samps, mean, d_input);
	getLastCudaError("stretch_kernel failed");
	hipEventRecord(event, stream);
}

//}}}

