#include "hip/hip_runtime.h"
/*************************************************************************
    This is GPU implementation of a polyphase filter. 
    Copyright (C) 2015  Adamek Karel, Novotny Jan, Armour Wes

    This file is part of Astro-Accelerate PolyPhase Filter (AAPPF).

    AAPPF is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    AAPPF is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with AAPPF.  If not, see <http://www.gnu.org/licenses/>.

**************************************************************************/

#include <hipfft/hipfft.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "../timer.h"
#include "../utils_cuda.h"
#include "../utils_file.h"

#include "params.h"
#include "../debug.h"

int device=0;

// --------> maxwell and kepler
__global__ void Fir_L1(short2 const* __restrict__ d_data, float2* d_spectra, float const* __restrict__ d_coeff, int nTaps, int nChannels, int yshift) {
	int t = 0;
	int bl= SPECTRA_PER_BLOCK*blockIdx.x*nChannels;
	float temp;
	short2 utemp;
	float2 ftemp[SPECTRA_PER_BLOCK];
	
	for (int i = 0; i<SPECTRA_PER_BLOCK;i++){
	  ftemp[i].x = 0.0f;
	  ftemp[i].y = 0.0f;
	}

	for(t=threadIdx.x + yshift; t<(nTaps)*nChannels; t+=nChannels){
	  temp = __ldg(&d_coeff[t]); 
	  for(int i=0;i<SPECTRA_PER_BLOCK;i++){
	    utemp = __ldg(&d_data[bl+i*nChannels + t]);
	    ftemp[i].x += temp*((float)(utemp.x));
	    ftemp[i].y += temp*((float)(utemp.y));
	  }
	}

	t=bl + threadIdx.x + yshift;
	for (int i=0;i<SPECTRA_PER_BLOCK;i++){
		d_spectra[t + i*nChannels]=ftemp[i];
	}

	return;
}

// --------------> fermi generation
/*
__global__ void Fir_L1(short2 const* __restrict__ d_data, float2* d_spectra, float const* __restrict__ d_coeff, int nTaps, int nChannels, int yshift) {
	int t = 0;
	int bl= SPECTRA_PER_BLOCK*blockIdx.x*nChannels;
	float temp;
	float2 ftemp[SPECTRA_PER_BLOCK];
	
	for (int i = 0; i<SPECTRA_PER_BLOCK;i++){
	  ftemp[i].x = 0.0f;
	  ftemp[i].y = 0.0f;
	}

	for(t=threadIdx.x + yshift; t<(nTaps)*nChannels; t+=nChannels){
  	  temp = d_coeff[t]; 
	  for(int i=0;i<SPECTRA_PER_BLOCK;i++){
	    ftemp[i].x += temp*((float)d_data[bl+i*nChannels + t].x);
	    ftemp[i].y += temp*((float)d_data[bl+i*nChannels + t].y);
	  }
	}

	t=bl + threadIdx.x + yshift;
	for (int i=0;i<SPECTRA_PER_BLOCK;i++){
		d_spectra[t + i*nChannels]=ftemp[i];
	}

	return;
}
*/

int Check_parameters(int nTaps, int nChannels, int nSpectra) {
	long int nColumns, maxgrid_x,itemp;
	size_t free_mem,total_mem;
	hipDeviceProp_t devProp;
	
	checkCudaErrors(hipSetDevice(device));
	checkCudaErrors(hipGetDeviceProperties(&devProp,device));
	maxgrid_x = devProp.maxGridSize[0];
	hipMemGetInfo(&free_mem,&total_mem);
	
	nColumns=(free_mem-nChannels*sizeof(float)*nTaps -(nTaps-1)*nChannels*sizeof(float2))/(2.0*sizeof(float2)*nChannels);
	if(maxgrid_x*SPECTRA_PER_BLOCK<nColumns) nColumns=maxgrid_x*SPECTRA_PER_BLOCK;
	nColumns=(int) nColumns*0.9;
	itemp=(int) nColumns/SPECTRA_PER_BLOCK;
	nColumns=itemp*SPECTRA_PER_BLOCK;
	return(nColumns);
}


void GPU_Polyphase(short2 *input, float2 *output, float *coeff, int nChannels, int nTaps, int nSpectra){
	int maxgrid_x;
	
	//---------> Initial nVidia stuff
	int devCount;
	hipDeviceProp_t devProp;
	size_t free_mem,total_mem;

	checkCudaErrors(hipGetDeviceCount(&devCount));

	if (DEBUG) {
		printf("\nThere are %d devices.", devCount);
		for (int i = 0; i < devCount; i++){
			checkCudaErrors(hipGetDeviceProperties(&devProp,i));
			printf("\n\t Using device:\t\t\t%s\n", devProp.name);
			printf("\n\t Max grid size:\t\t\t%d\n", devProp.maxGridSize[1]);
		}
	}

	checkCudaErrors(hipSetDevice(device));
	checkCudaErrors(hipGetDeviceProperties(&devProp,device));

	maxgrid_x = devProp.maxGridSize[1];

	hipMemGetInfo(&free_mem,&total_mem);
	printf("\nDevice has %f MiB of total memory, which %f MiB is available.\n", (double) total_mem/(1024.0*1024.0), (double) free_mem/(1024.0*1024.0));
	
	//---------> Measurements
	double coeff_transport_in=0.0, transfer_in=0.0, transfer_out=0.0, fir_time=0.0, fft_time=0.0;
	GpuTimer timer;
	
	//---------> Specific nVidia stuff
	hipDeviceSetCacheConfig(hipFuncCachePreferL1);
	
	//---------> Spectra
	long int nColumns;
	int nCUDAblocks,Sremainder,nRepeats,itemp;
	nColumns=(free_mem-nChannels*sizeof(float)*nTaps -(nTaps-1)*nChannels*sizeof(float2))/(2.0*sizeof(float2)*nChannels);
	if(maxgrid_x*SPECTRA_PER_BLOCK<nColumns) nColumns=maxgrid_x*SPECTRA_PER_BLOCK;
	nColumns=(int) nColumns*0.9;
	itemp=(int) nColumns/SPECTRA_PER_BLOCK;
	nColumns=itemp*SPECTRA_PER_BLOCK;
	nRepeats=(int) (nSpectra/nColumns);
	Sremainder=nSpectra-nRepeats*nColumns;
	if (DEBUG) printf("Maximum number of spectra in memory is %d which is %e MB \n",nColumns, (double) (nColumns*nChannels*sizeof(float2)/(1024.0*1024.0))   );
	//if (DEBUG) printf("nSpectra is split into %d chunks. Sremainder: %d\n",nRepeats,Sremainder);
	
	//---------> Channels
	int nKernels=(int) nChannels/THREADS_PER_BLOCK; //Head size
	int Kremainder=nChannels-nKernels*THREADS_PER_BLOCK; //Tail size
	
	//---------> Memory allocation
	if (DEBUG) printf("\nDevice memory allocation...: \t\t");
	int input_size=nChannels*(nColumns+nTaps-1);
	int output_size=nChannels*nColumns;
	int coeff_size=nChannels*nTaps;
	float2 *d_output;
	float *d_coeff;
	short2 *d_input;
	timer.Start();
	checkCudaErrors(hipMalloc((void **) &d_input,  sizeof(short2)*input_size));
	checkCudaErrors(hipMalloc((void **) &d_output, sizeof(float2)*output_size));
	checkCudaErrors(hipMalloc((void **) &d_coeff,  sizeof(float)*coeff_size));
	timer.Stop();
	if (DEBUG) printf("done in %g ms.", timer.Elapsed());
	
	
	//---------> Transfer to the device
	if (DEBUG) printf("\nCopy coefficients from host to device...\t");
	timer.Start();
	checkCudaErrors(hipMemcpy(d_coeff, coeff, coeff_size*sizeof(float), hipMemcpyHostToDevice));
	timer.Stop();
	coeff_transport_in=timer.Elapsed();
	if (DEBUG) printf("done in %g ms.\n", coeff_transport_in);
		
	//---------> CUDA block and CUDA grid parameters
	nCUDAblocks=(int) nColumns/SPECTRA_PER_BLOCK;
	dim3 GridSize(nCUDAblocks, 1, 1);
	dim3 BlockSize(THREADS_PER_BLOCK, 1, 1);
	
	//---------> Transfer to the device
	if (DEBUG) printf("\nCopy data from host to device...\t");
	timer.Start();
	if(nRepeats>0){
		checkCudaErrors(hipMemcpy(d_input, input, input_size*sizeof(short2), hipMemcpyHostToDevice));
	}
	else {
		checkCudaErrors(hipMemcpy(d_input, &input[output_size*nRepeats], (Sremainder+nTaps-1)*nChannels*sizeof(short2), hipMemcpyHostToDevice));
	}
	timer.Stop();
	transfer_in+=timer.Elapsed();
	if (DEBUG) printf("done in %g ms.\n", timer.Elapsed());
		
	// ---------------> Polyphase filter
	for (int r = 0; r < nRepeats; r++){
		//---------> FIR part
		BlockSize.x=THREADS_PER_BLOCK;
		timer.Start();
		for (int nutak=0;nutak<nKernels;nutak++){	
			Fir_L1<<<GridSize, BlockSize>>>((short2*) d_input, (float2*) d_output, d_coeff, nTaps, nChannels, nutak*THREADS_PER_BLOCK);
		}
		if (Kremainder>0){
			BlockSize.x=Kremainder;
			Fir_L1<<<GridSize, BlockSize>>>((short2*) d_input, (float2*) d_output, d_coeff, nTaps, nChannels, nKernels*THREADS_PER_BLOCK);
		}
		timer.Stop();
	 	fir_time+=timer.Elapsed();
	}
	if (Sremainder>0){
		itemp=(int) Sremainder/SPECTRA_PER_BLOCK;
		itemp=(itemp+1)*(SPECTRA_PER_BLOCK);
		nCUDAblocks=itemp/SPECTRA_PER_BLOCK;
		GridSize.x=nCUDAblocks;BlockSize.x=THREADS_PER_BLOCK;
		//---------> FIR
		BlockSize.x=THREADS_PER_BLOCK;
		timer.Start();
		for (int nutak=0;nutak<nKernels;nutak++){	
			Fir_L1<<<GridSize, BlockSize>>>((short2*) d_input, (float2*) d_output, d_coeff, nTaps, nChannels, nutak*THREADS_PER_BLOCK);
		}
		if (Kremainder>0){
			BlockSize.x=Kremainder;
			Fir_L1<<<GridSize, BlockSize>>>((short2*) d_input, (float2*) d_output, d_coeff, nTaps, nChannels, nKernels*THREADS_PER_BLOCK);
		}
		timer.Stop();
		fir_time+=timer.Elapsed();
	}
	
	//----- error check -----
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipFree(d_input));
	checkCudaErrors(hipFree(d_coeff));

	//---------> FFT
	for (int r = 0; r < nRepeats; r++){
		hipfftHandle plan;
		hipfftResult error;
		error = hipfftPlan1d(&plan, nChannels, HIPFFT_C2C, nColumns);
		if (HIPFFT_SUCCESS != error){
			printf("CUFFT error: %d", error);
		}
		
		//execute plan and copy back to host
			timer.Start();
				hipfftExecC2C(plan, (hipfftComplex *)d_output, (hipfftComplex *)d_output, HIPFFT_FORWARD);
			timer.Stop();
			fft_time += timer.Elapsed();
		
		//Destroy the cuFFT plan
		hipfftDestroy(plan);
	}
	if (Sremainder>0){
		//---------> FFT
		hipfftHandle plan;
		hipfftResult error;
		error = hipfftPlan1d(&plan, nChannels, HIPFFT_C2C, Sremainder);
		if (HIPFFT_SUCCESS != error){
			printf("CUFFT error: %d", error);
		}
		
		//execute plan and copy back to host
			timer.Start();
				hipfftExecC2C(plan, (hipfftComplex *)d_output, (hipfftComplex *)d_output, HIPFFT_FORWARD);
			timer.Stop();
			fft_time += timer.Elapsed();
		
		//Destroy the cuFFT plan
		hipfftDestroy(plan);
	}
	
	//---------> Transfer to the host
	if (DEBUG) printf("Copy data from device to host \t");
	timer.Start();
	if(nRepeats>0){
		checkCudaErrors(hipMemcpy(output,d_output,output_size*sizeof(float2), hipMemcpyDeviceToHost));
	}
	else {
		checkCudaErrors(hipMemcpy(&output[output_size*nRepeats],d_output,Sremainder*nChannels*sizeof(float2), hipMemcpyDeviceToHost));	
	}
	timer.Stop();
	if (DEBUG) printf("done in %g ms.\n", timer.Elapsed());
	transfer_out+=timer.Elapsed();
	
	//----- error check -----
	checkCudaErrors(hipGetLastError());
	//checkCudaErrors(hipDeviceSynchronize());
	//-----------------------
	
	if (DEBUG) printf("Number of spectra: %d;\nNumber of Channels: %d;\nNumber of Taps: %d;\nFIR filter execution time: %0.3f ms;\ncuFFT execution time: %0.3f ms;\nPolyphase execution time: %0.3f ms;\nData transfer time %0.3f ms\n",nSpectra,nChannels,nTaps, fir_time, fft_time, fir_time + fft_time, transfer_in + transfer_out);
	
	if (DEBUG && WRITE){ 
		char str[200];
		sprintf(str,"GPU-polyphase.dat");
		printf("\n Write results into file...\t");
		save_time(str, nSpectra, fir_time, fft_time, transfer_in, transfer_out, nChannels, nTaps, SPECTRA_PER_BLOCK, THREADS_PER_BLOCK, 1);
		printf("\t done.\n-------------------------------------\n");
	}
	
	//---------> Cleanup
	if (DEBUG) printf("Free device data.....\t");
	checkCudaErrors(hipFree(d_output));
	if (DEBUG) printf("\t done.\n-----------------------------------\n");

}
