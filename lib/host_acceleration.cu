#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hipfft/hipfft.h>
<<<<<<< HEAD
#include <omp.h>
#include "AstroAccelerate/params.h"
#include "AstroAccelerate/device_stats.h"
#include "AstroAccelerate/device_stretch.h"
#include "AstroAccelerate/device_set_stretch.h"
#include "AstroAccelerate/device_power.h"
#include "hip/hip_runtime_api.h"

void acceleration(int range, int nsamp, int max_ndms, int processed, int nboots, int num_trial_bins, int navdms, float narrow, float wide, int nsearch, float aggression, float cutoff, float ***output_buffer, int *ndms, int *inBin, float *dm_low, float *dm_high, float *dm_step, float tsamp)
{
	// Example FFT....
	printf("\n[1DCUFFT] is starting...\n");

	size_t	size;
	int	a;
	float mean, stddev;

	int chunk = omp_get_num_procs();
	
	for(int i=0; i<range; i++)
	{
		hipStream_t stream_e;
		hipError_t result_e;
		result_e = hipStreamCreate(&stream_e);

		hipEvent_t event_e;
		hipEventCreate(&event_e);

		hipStream_t stream_o;
		hipError_t result_o;
		result_o = hipStreamCreate(&stream_o);

		hipEvent_t event_o;
		hipEventCreate(&event_o);

		int samps = processed/inBin[i];

		printf("\nsamps:\t%d", samps);
		int nearest = (int)floorf(log2f((float)samps));
		printf("\nnearest:\t%d", nearest);
		samps = (int)powf(2.0, nearest);
		printf("\nsamps:\t%d", samps);


		// Allocate memory for signal even
		float* d_signal_in_e;
		size = samps*sizeof(float);
		printf("\nSize of GPU input signal:\t%u MB", size/1024/1024);
		checkCudaErrors(hipMalloc((void**)&d_signal_in_e, size));

		float* d_signal_transformed_e;
		size = samps*sizeof(float);
		printf("\nSize of GPU stretched signal:\t%u MB", size/1024/1024);
		checkCudaErrors(hipMalloc((void**)&d_signal_transformed_e, size));

		hipfftComplex* d_signal_fft_e;
		size = (samps/2 + 1)*sizeof(hipfftComplex);
		printf("\nSize of GPU output signal:\t%u MB", size/1024/1024);
		checkCudaErrors(hipMalloc((void**)&d_signal_fft_e, size));

		float* d_signal_power_e;
		size = sizeof(float) * (samps/2) * (2*ACCMAX + ACCSTEP)/ACCSTEP;
		printf("\nSize of GPU power signal:\t%u MB", size/1024/1024);
		checkCudaErrors(hipMalloc((void**)&d_signal_power_e, size));

		float2* h_signal_e;
		size = (samps)*sizeof(float2);
		printf("\nSize of host output signal:\t%u MB", size/1024/1024);
		checkCudaErrors(hipHostMalloc((void**)&h_signal_e, size));

		float* h_signal_transformed_e;
		size = samps*sizeof(float);
		printf("\nSize of GPU stretched signal:\t%u MB", size/1024/1024);
		checkCudaErrors(hipHostMalloc((void**)&h_signal_transformed_e, size));
	
		float* h_signal_power_e;
		size = sizeof(float) * (samps/2) * (2*ACCMAX + ACCSTEP)/ACCSTEP;
		printf("\nSize of total host power signal:\t%u MB", size/1024/1024), fflush(stdout);
		checkCudaErrors(hipHostMalloc((void**)&h_signal_power_e, size));

		// Allocate memory for signal odd
		float* d_signal_in_o;
		size = samps*sizeof(float);
		printf("\nSize of GPU input signal:\t%u MB", size/1024/1024);
		checkCudaErrors(hipMalloc((void**)&d_signal_in_o, size));

		float* d_signal_transformed_o;
		size = samps*sizeof(float);
		printf("\nSize of GPU stretched signal:\t%u MB", size/1024/1024);
		checkCudaErrors(hipMalloc((void**)&d_signal_transformed_o, size));

		hipfftComplex* d_signal_fft_o;
		size = (samps/2 + 1)*sizeof(hipfftComplex);
		printf("\nSize of GPU output signal:\t%u MB", size/1024/1024);
		checkCudaErrors(hipMalloc((void**)&d_signal_fft_o, size));

		float* d_signal_power_o;
		size = sizeof(float) * (samps/2) * (2*ACCMAX + ACCSTEP)/ACCSTEP;
		printf("\nSize of GPU power signal:\t%u MB", size/1024/1024);
		checkCudaErrors(hipMalloc((void**)&d_signal_power_o, size));

		float2* h_signal_o;
		size = (samps)*sizeof(float2);
		printf("\nSize of host output signal:\t%u MB", size/1024/1024);
		checkCudaErrors(hipHostMalloc((void**)&h_signal_o, size));

		float* h_signal_transformed_o;
		size = samps*sizeof(float);
		printf("\nSize of GPU stretched signal:\t%u MB", size/1024/1024);
		checkCudaErrors(hipHostMalloc((void**)&h_signal_transformed_o, size));
	
		float* h_signal_power_o;
		size = sizeof(float) * (samps/2) * (2*ACCMAX + ACCSTEP)/ACCSTEP;
		printf("\nSize of total host power signal:\t%u MB", size/1024/1024), fflush(stdout);
		checkCudaErrors(hipHostMalloc((void**)&h_signal_power_o, size));

		// CUFFT plan even
		hipfftHandle plan_e;
		hipfftPlan1d(&plan_e, samps, HIPFFT_R2C, 1);
		hipfftSetStream(plan_e, stream_e);
		
		// CUFFT plan odd
		hipfftHandle plan_o;
		hipfftPlan1d(&plan_o, samps, HIPFFT_R2C, 1);
		hipfftSetStream(plan_o, stream_o);
	
		int trials = (2*ACCMAX +ACCSTEP)/ACCSTEP;

		// Transfer even memory asynchronously
		checkCudaErrors(hipMemcpyAsync(d_signal_in_e, output_buffer[i][0],   samps*sizeof(float), hipMemcpyHostToDevice, stream_e));
		hipEventRecord(event_e, stream_e);

		// Cacluclate even dm
		for(a = 0; a < trials; a++)
		{
			int acc = -ACCMAX + a*ACCSTEP;
			float mean = 127.959f;
			set_stretch_gpu(event_e, stream_e, samps, mean, d_signal_transformed_e);
			stretch_gpu(event_e, stream_e, acc, samps, tsamp, d_signal_in_e, d_signal_transformed_e);
			hipStreamWaitEvent(stream_e, event_e, 0);
			checkCudaErrors(hipfftExecR2C(plan_e, (float *)d_signal_transformed_e, (hipfftComplex *)d_signal_fft_e));
			power_gpu(event_e, stream_e, samps, a, d_signal_fft_e, d_signal_power_e);
		}

		for (int dm_count = 1; dm_count < ndms[i]-1; dm_count+=2)
		{
			// Transfer odd memory asynchronously
			hipStreamWaitEvent(stream_o, event_o, 0);
			checkCudaErrors(hipMemcpyAsync(d_signal_in_o, output_buffer[i][dm_count], samps*sizeof(float), hipMemcpyHostToDevice, stream_o));
			hipEventRecord(event_o, stream_o);

			for(a = 0; a < trials; a++) {
				int acc = -ACCMAX + a*ACCSTEP;
				float mean=127.959f;
				set_stretch_gpu(event_o, stream_o, samps, mean, d_signal_transformed_o);
				stretch_gpu(event_o, stream_o, acc, samps, tsamp, d_signal_in_o, d_signal_transformed_o);
				checkCudaErrors(hipfftExecR2C(plan_o, (float *)d_signal_transformed_o, (hipfftComplex *)d_signal_fft_o));
				hipStreamWaitEvent(stream_o, event_o, 0);
				power_gpu(event_o, stream_o, samps, a, d_signal_fft_o, d_signal_power_o);
			}

			// Threshold even f-fdot plane
			hipStreamSynchronize(stream_e);
			stats_gpu(event_e, stream_e, samps, &mean, &stddev, h_signal_power_e, d_signal_power_e);

			// Transfer even memory asynchronously
			checkCudaErrors(hipMemcpyAsync(d_signal_in_e, output_buffer[i][dm_count+1],   samps*sizeof(float), hipMemcpyHostToDevice, stream_e));
			hipEventRecord(event_e, stream_e);
			
			// Cacluclate even dm
			for(a = 0; a < trials; a++) {
				int acc = -ACCMAX + a*ACCSTEP;
				float mean=127.959f;
				set_stretch_gpu(event_e, stream_e, samps, mean, d_signal_transformed_e);
				stretch_gpu(event_e, stream_e, acc, samps, tsamp, d_signal_in_e, d_signal_transformed_e);
				hipStreamWaitEvent(stream_e, event_e, 0);
				checkCudaErrors(hipfftExecR2C(plan_e, (float *)d_signal_transformed_e, (hipfftComplex *)d_signal_fft_e));
				power_gpu(event_e, stream_e, samps, a, d_signal_fft_e, d_signal_power_e);
			}

			// Threshold odd f-fdot plane
			hipStreamSynchronize(stream_o);
			stats_gpu(event_o, stream_o, samps, &mean, &stddev, h_signal_power_o, d_signal_power_o);
		}
		
		//Destroy CUFFT context
		hipfftDestroy(plan_e);
		hipfftDestroy(plan_o);

		//Destroy streams
		result_e = hipStreamDestroy(stream_e);
		result_o = hipStreamDestroy(stream_o);

		// cleanup even memory
		hipHostFree(h_signal_e);
		hipHostFree(h_signal_power_e);
		hipFree(d_signal_in_e);
		hipFree(d_signal_fft_e);
		hipFree(d_signal_power_e);
		hipFree(d_signal_transformed_e);

		// cleanup odd memory
		hipHostFree(h_signal_o);
		hipHostFree(h_signal_power_o);
		hipFree(d_signal_in_o);
		hipFree(d_signal_fft_o);
		hipFree(d_signal_power_o);
		hipFree(d_signal_transformed_o);
	}
}
=======
#include <math.h>
#include "AstroAccelerate/params.h"

void acceleration(int range, int nsamp, int max_ndms, int processed, int nboots, int num_trial_bins, int navdms, float narrow, float wide, int nsearch, float aggression, float cutoff, float ***output_buffer, int *ndms, int *inBin, float *dm_low, float *dm_high, float *dm_step, float tsamp) {

	// Example FFT....

	printf("\n");

	printf("[1DCUFFT] is starting...\n");

	FILE	*fp_c;
	char	filename[200];

	for(int i=0; i<range; i++) {
		int samps = processed/inBin[i];

		// Allocate memory for signal
		hipfftReal* d_signal_in;
		hipMalloc((void**)&d_signal_in, samps*sizeof(hipfftReal));

		hipfftComplex* d_signal_out;
		hipMalloc((void**)&d_signal_out, (samps/2 + 1)*sizeof(hipfftComplex));

		hipfftComplex* h_signal = (hipfftComplex*)malloc((samps/2 + 1)*sizeof(hipfftComplex));
		float* h_signal_x = (float*)malloc(sizeof(float) * (samps/2 + 1) * ndms[i]);
		float* h_signal_y = (float*)malloc(sizeof(float) * (samps/2 + 1) * ndms[i]);
		float* h_signal_inter_x = (float*)malloc(sizeof(float) * 2*(samps/2 + 1) * ndms[i]);
		float* h_signal_inter_y = (float*)malloc(sizeof(float) * 2*(samps/2 + 1) * ndms[i]);
		
		// CUFFT plan
		hipfftHandle plan;
		hipfftPlan1d(&plan, samps, HIPFFT_R2C, 1);

		sprintf(filename, "acceleration-%d.dat", i);
		if ((fp_c=fopen(filename, "w")) == NULL) {
			fprintf(stderr, "Error opening output file!\n");
			exit(0);
		}
	
		for (int dm_count = 0; dm_count < ndms[i]; dm_count++) {
	
			hipMemcpy(d_signal_in, output_buffer[i][dm_count], samps*sizeof(float), hipMemcpyHostToDevice);

			// Transform signal 
			//printf("\nTransforming dm: %f using hipfftExecR2C\n", dm);
			hipfftExecR2C(plan, (hipfftReal *)d_signal_in, (hipfftComplex *)d_signal_out);

			// Copy device memory to host
			hipMemcpy(h_signal, d_signal_out, sizeof(hipfftComplex) * (samps/2 + 1) , hipMemcpyDeviceToHost);

			// Set the DC offset to zero
			//h_signal_p[0+dm_count*(samps/2)] = 0.0; 
			// Store the real and complex parts as floats
			#pragma omp parallel for
			for(int j=1;j< samps/2;j++){
			//	h_signal[j].x = h_signal[j].x-h_signal[0].x;
			//	h_signal[j].y = h_signal[j].y-h_signal[0].y;
				h_signal_x[j+dm_count*(samps/2)] = h_signal[j].x;
				h_signal_y[j+dm_count*(samps/2)] = h_signal[j].y;
				h_signal_inter_x[2*j+dm_count*samps]= h_signal[j+dm_count*(samps/2)].x;
				h_signal_inter_x[2*j+1+dm_count*samps]=0.785398163*((h_signal[j].x-h_signal[j+1].x));
				h_signal_inter_y[2*j+dm_count*samps]= h_signal[j+dm_count*(samps/2)].y;
				h_signal_inter_y[2*j+1+dm_count*samps]=0.785398163*((h_signal[j].y-h_signal[j+1].y));
			}
			int acc_max=0;
			for(int acc=0; acc < acc_max; acc++) {

				// Convolve templates here.....

			}
			for(int j=0;j< samps/2; j++){
				fprintf(fp_c, "\n%d\t%f\t%f", j, h_signal[j].x, h_signal[j].y);
			}
			fprintf(fp_c, "\n");
		}

		//Destroy CUFFT context
		hipfftDestroy(plan);

		// cleanup memory
		free(h_signal);
		free(h_signal_x);
		free(h_signal_y);
		free(h_signal_inter_x);
		free(h_signal_inter_y);
		hipFree(d_signal_in);
		hipFree(d_signal_out);
	}
}


>>>>>>> 0ec19baf405fa311d6a7ea91dbb146bcccf88229
