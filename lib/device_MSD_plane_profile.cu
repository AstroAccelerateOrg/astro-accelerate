#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>

#include "headers/device_single_FIR.h"
#include "headers/device_bin.h"
#include "headers/device_MSD.h"

//#define MSD_PLANE_DEBUG
//#define MSD_PLANE_EXPORT



struct MSD_Data {
	int width;
	double mean;
	double sd;
};



//---------------------------------------------------------------
//------------- MSD plane profile

void Do_MSD_normal(float *d_MSD, float *d_input, float *d_MSD_workarea, int nTimesamples, int nDMs, int offset, float OR_sigma_multiplier, int enable_outlier_rejection, hipStream_t streams){
	MSD_Configuration conf(nTimesamples, nDMs, offset, 0);
	if(enable_outlier_rejection){
		MSD_outlier_rejection(d_MSD, d_input, d_MSD_workarea, &conf, OR_sigma_multiplier, streams);
	}
	else {
		MSD_normal(d_MSD, d_input, d_MSD_workarea, &conf, streams);
	}
}

void Do_MSD_continuous(float *d_MSD, float *d_input, float *d_previous_partials, float *d_MSD_workarea, int nTimesamples, int nDMs, int offset, float OR_sigma_multiplier, int enable_outlier_rejection, hipStream_t streams){
	MSD_Configuration conf(nTimesamples, nDMs, offset, 0);
	if(enable_outlier_rejection){
		MSD_outlier_rejection_continuous(d_MSD, d_input, d_previous_partials, d_MSD_workarea, &conf, OR_sigma_multiplier, streams);
	}
	else {
		MSD_normal_continuous(d_MSD, d_input, d_previous_partials, d_MSD_workarea, &conf, streams);
	}
}

inline void Do_MSD(float *d_MSD, float *d_input, float *d_previous_partials, float *d_MSD_workarea, int nTimesamples, int nDMs, int offset, float OR_sigma_multiplier, int enable_outlier_rejection, bool perform_continuous, hipStream_t streams) {
	if(perform_continuous) Do_MSD_continuous(d_MSD, d_input, d_previous_partials, d_MSD_workarea, nTimesamples, nDMs, offset, OR_sigma_multiplier, enable_outlier_rejection, streams);
	else Do_MSD_normal(d_MSD, d_input, d_MSD_workarea, nTimesamples, nDMs, offset, OR_sigma_multiplier, enable_outlier_rejection, streams);
}


void MSD_plane_profile_debug(float *d_MSD, int DIT_value, int nTimesamples){
	float h_MSD[MSD_RESULTS_SIZE];
	checkCudaErrors(hipMemcpy(h_MSD, d_MSD, MSD_RESULTS_SIZE*sizeof(float), hipMemcpyDeviceToHost));
	printf("    DiT:%d; nTimesamples:%d; decimated_timesamples:%d; MSD:[%f; %f; %f]\n", (int) DIT_value, (int) nTimesamples, (int) (nTimesamples>>1), h_MSD[0], h_MSD[1], h_MSD[2]);
}


void MSD_of_input_plane(float *d_MSD_DIT, std::vector<int> *h_MSD_DIT_widths, float *d_input_data, float *d_MSD_DIT_previous, float *d_sudy, float *d_lichy, float *d_MSD_workarea, size_t nTimesamples, size_t nDMs, int nDecimations, int max_width_performed, float OR_sigma_multiplier, int enable_outlier_rejection, bool high_memory, bool perform_continuous, double *total_time, double *dit_time, double *MSD_time, hipStream_t streams){
//	GpuTimer timer, total_timer;
	double t_dit_time=0, t_MSD_time=0;
	int nRest;
	size_t decimated_timesamples;
	int DIT_value;

//	total_timer.Start();
	//----------------------------------------------------------------------------------------
	//-------- DIT = 1
	DIT_value = 1;
	printf("\nnTimesamples: %zu nDMs: %zu\n", nTimesamples, nDMs);
	Do_MSD(d_MSD_DIT, d_input_data, d_MSD_DIT_previous, d_MSD_workarea, nTimesamples, nDMs, 0, OR_sigma_multiplier, enable_outlier_rejection, perform_continuous, streams);
	h_MSD_DIT_widths->push_back(DIT_value);

	#ifdef MSD_PLANE_DEBUG
	printf("    MSD format: [ mean ; StDev ; nElements ]\n");
	MSD_plane_profile_debug(d_MSD_DIT, DIT_value, nTimesamples);
	#endif
	//----------------------------------------------------------------------------------------
	
	checkCudaErrors(hipGetLastError());
	
	//----------------------------------------------------------------------------------------
	//-------- DIT = 2
	DIT_value = DIT_value*2;
	
	if(high_memory){
		//printf("High memory: DIT=2 is not split\n");
//		timer.StartWithStream(streams);
		nRest = GPU_DiT_v2_wrapper(d_input_data, d_lichy, nDMs, nTimesamples, streams);
		decimated_timesamples = (nTimesamples>>1);
//		timer.StopWithStream(streams);	t_dit_time += timer.ElapsedWithStream(streams);
		
//		timer.StartWithStream(streams);
		Do_MSD(&d_MSD_DIT[MSD_RESULTS_SIZE], d_lichy, &d_MSD_DIT_previous[MSD_RESULTS_SIZE], d_MSD_workarea, decimated_timesamples, nDMs, nRest, OR_sigma_multiplier, enable_outlier_rejection, perform_continuous, streams);
//		timer.StopWithStream(streams);	t_MSD_time += timer.ElapsedWithStream(streams);
		h_MSD_DIT_widths->push_back(DIT_value);
		
		#ifdef MSD_PLANE_DEBUG
		MSD_plane_profile_debug(&d_MSD_DIT[MSD_RESULTS_SIZE], DIT_value, decimated_timesamples);
		#endif
		
//		timer.Start();
		nRest = GPU_DiT_v2_wrapper(d_lichy, d_sudy, nDMs, decimated_timesamples, streams);
//		timer.Stop();	t_dit_time += timer.Elapsed();
	}
	else {
		//printf("Low memory: DIT=2 is split in two\n");
		// First decimation is split into two parts, that way we can lower the memory requirements for MSD_plane_profile
		// First half of the decimation
		int nDMs_half = (nDMs>>1);
//		timer.Start();
		nRest = GPU_DiT_v2_wrapper(d_input_data, d_lichy, nDMs_half, nTimesamples, streams);
		decimated_timesamples = (nTimesamples>>1);
//		timer.Stop();	t_dit_time += timer.Elapsed();

//		timer.Start();
		Do_MSD_continuous(&d_MSD_DIT[MSD_RESULTS_SIZE], d_lichy, &d_MSD_DIT[2*MSD_RESULTS_SIZE], d_MSD_workarea, decimated_timesamples, nDMs_half, nRest, OR_sigma_multiplier, enable_outlier_rejection, streams);
//		timer.Stop();	t_MSD_time += timer.Elapsed();
		
//		timer.Start();
		nRest = GPU_DiT_v2_wrapper(d_lichy, d_sudy, nDMs_half, decimated_timesamples,streams);
//		timer.Stop();	t_dit_time += timer.Elapsed();
		
		// second half of the decimation
//		timer.Start();
		nRest = GPU_DiT_v2_wrapper(&d_input_data[nDMs_half*nTimesamples], d_lichy, nDMs_half, nTimesamples, streams);
		decimated_timesamples = (nTimesamples>>1);
//		timer.Stop();	t_dit_time += timer.Elapsed();

//		timer.Start();
		Do_MSD_continuous(&d_MSD_DIT[MSD_RESULTS_SIZE], d_lichy, &d_MSD_DIT[2*MSD_RESULTS_SIZE], d_MSD_workarea, decimated_timesamples, nDMs_half, nRest, OR_sigma_multiplier, enable_outlier_rejection, streams);
//		timer.Stop();	t_MSD_time += timer.Elapsed();
		h_MSD_DIT_widths->push_back(DIT_value);
		
//		timer.Start();
		nRest = GPU_DiT_v2_wrapper(d_lichy, &d_sudy[nDMs_half*(decimated_timesamples>>1)], nDMs_half, decimated_timesamples, streams);
//		timer.Stop();	t_dit_time += timer.Elapsed();
		
		#ifdef MSD_PLANE_DEBUG
		MSD_plane_profile_debug(&d_MSD_DIT[MSD_RESULTS_SIZE], DIT_value, decimated_timesamples);
		#endif
	}
	
	decimated_timesamples = (nTimesamples>>2);
	DIT_value = DIT_value*2;
	
//	timer.Start();
	Do_MSD(&d_MSD_DIT[2*MSD_RESULTS_SIZE], d_sudy, &d_MSD_DIT_previous[2*MSD_RESULTS_SIZE], d_MSD_workarea, decimated_timesamples, nDMs, nRest, OR_sigma_multiplier, enable_outlier_rejection, perform_continuous, streams);
//	timer.Stop();	t_MSD_time += timer.Elapsed();
	h_MSD_DIT_widths->push_back(DIT_value);	
	
	#ifdef MSD_PLANE_DEBUG
	MSD_plane_profile_debug(&d_MSD_DIT[2*MSD_RESULTS_SIZE], DIT_value, decimated_timesamples);
	#endif
	//----------------------------------------------------------------------------------------
	
	checkCudaErrors(hipGetLastError());
	
	//----------------------------------------------------------------------------------------
	//-------- DIT > 3
	for(size_t f=3; f<=nDecimations; f++){
//		timer.Start();
		DIT_value = DIT_value*2;
		//if(DIT_value<=max_width_performed){
			if(f%2==0){
//				timer.Start();
				nRest = GPU_DiT_v2_wrapper(d_lichy, d_sudy, nDMs, decimated_timesamples, streams);
//				timer.Stop();	t_dit_time += timer.Elapsed();
				if(nRest<0) break;
				decimated_timesamples = (decimated_timesamples>>1);

//				timer.Start();
				Do_MSD(&d_MSD_DIT[f*MSD_RESULTS_SIZE], d_sudy, &d_MSD_DIT_previous[f*MSD_RESULTS_SIZE], d_MSD_workarea, decimated_timesamples, nDMs, nRest, OR_sigma_multiplier, enable_outlier_rejection, perform_continuous, streams);
//				timer.Stop();	t_MSD_time += timer.Elapsed();
			}
			else {
//				timer.Start();
				nRest = GPU_DiT_v2_wrapper(d_sudy, d_lichy, nDMs, decimated_timesamples, streams);
//				timer.Stop();	t_dit_time += timer.Elapsed();
				if(nRest<0) break;
				decimated_timesamples = (decimated_timesamples>>1);

//				timer.Start();
				Do_MSD(&d_MSD_DIT[f*MSD_RESULTS_SIZE], d_lichy, &d_MSD_DIT_previous[f*MSD_RESULTS_SIZE], d_MSD_workarea, decimated_timesamples, nDMs, nRest, OR_sigma_multiplier, enable_outlier_rejection, perform_continuous, streams);
//				timer.Stop();	t_MSD_time += timer.Elapsed();
			}
			h_MSD_DIT_widths->push_back(DIT_value);
			
			#ifdef MSD_PLANE_DEBUG
				MSD_plane_profile_debug(&d_MSD_DIT[f*MSD_RESULTS_SIZE], DIT_value, decimated_timesamples);
			#endif
	//	}
		checkCudaErrors(hipGetLastError());
	}
	//----------------------------------------------------------------------------------------
	
	checkCudaErrors(hipGetLastError());
	
	//----------------------------------------------------------------------------------------
	//-------- Boxcar for last boxcar width if needed
	/*
	if(DIT_value<max_width_performed){
		DIT_value = (DIT_value>>1);
		decimated_timesamples = nTimesamples/DIT_value;
		int nTaps = max_width_performed/DIT_value;
		if(max_width_performed%DIT_value!=0) nTaps++;
		
		if(nDecimations%2==0){
			nRest = PPF_L1(d_lichy, d_sudy, nDMs, decimated_timesamples, nTaps);

			checkCudaErrors(hipGetLastError());
			
			timer.Start();
			Do_MSD(&d_MSD_DIT[(nDecimations+1)*MSD_RESULTS_SIZE], d_sudy, decimated_timesamples, nDMs, nRest, OR_sigma_multiplier, MSD_type);
			timer.Stop();	MSD_time += timer.Elapsed();
		}
		else {
			nRest = PPF_L1(d_sudy, d_lichy, nDMs, decimated_timesamples, nTaps);
			
			checkCudaErrors(hipGetLastError());
			
			timer.Start();
			Do_MSD(&d_MSD_DIT[(nDecimations+1)*MSD_RESULTS_SIZE], d_lichy, decimated_timesamples, nDMs, nRest, OR_sigma_multiplier, MSD_type);
			timer.Stop();	MSD_time += timer.Elapsed();
		}
		h_MSD_DIT_widths->push_back(DIT_value*nTaps);

		#ifdef GPU_ANALYSIS_DEBUG
			printf("    Performing additional boxcar: nTaps: %d; max_width_performed: %d; DIT_value/2: %d;\n", nTaps, max_width_performed, DIT_value);
			checkCudaErrors(hipMemcpy(h_MSD, &d_MSD_DIT[(nDecimations+1)*MSD_RESULTS_SIZE], MSD_RESULTS_SIZE*sizeof(float), hipMemcpyDeviceToHost));
			printf("    DIT: %d; MSD:[%f; %f; %f]\n", DIT_value*nTaps, h_MSD[0], h_MSD[1], h_MSD[2]);
		#endif		
	}
	*/
	//----------------------------------------------------------------------------------------
	
	checkCudaErrors(hipGetLastError());
	
//	total_timer.Stop();
//	(*total_time) = total_timer.Elapsed();
	(*dit_time) = t_dit_time;
	(*MSD_time) = t_MSD_time;
	
//	#ifdef GPU_PARTIAL_TIMER
		printf("    MSD of input plane: Total time: %f ms; DiT time: %f ms; MSD time: %f ms;\n", (*total_time), (*dit_time), (*MSD_time));
//	#endif
}


void MSD_Interpolate_linear(float *mean, float *StDev, float desired_width, float *h_MSD_DIT, std::vector<int> *h_MSD_DIT_widths){
	int MSD_DIT_size = h_MSD_DIT_widths->size();
	int position = (int) floorf(log2f((float) desired_width));
	
	float width1 = h_MSD_DIT_widths->operator[](position);
	float mean1 = h_MSD_DIT[(position)*MSD_RESULTS_SIZE];
	float StDev1 = h_MSD_DIT[(position)*MSD_RESULTS_SIZE +1];
	
	if(position == MSD_DIT_size-1 && width1==(int) desired_width) {
		(*mean) = mean1;
		(*StDev) = StDev1;
	}
	else {
		float width2 = h_MSD_DIT_widths->operator[](position+1);
		float distance_in_width = width2 - width1;
		
		float mean2 = h_MSD_DIT[(position+1)*MSD_RESULTS_SIZE];
		float distance_in_mean = mean2 - mean1;
		
		float StDev2 = h_MSD_DIT[(position+1)*MSD_RESULTS_SIZE +1];
		float distance_in_StDev = StDev2 - StDev1;
		
		#ifdef MSD_PLANE_DEBUG
//			printf("Position: \t %i\n", position);
//			printf("width:[%f;%f]; mean:[%f;%f]; sd:[%f;%f]\n",width1, width2, mean1, mean2, StDev1, StDev2);
//			printf("d width %f; d mean: %f; d StDef: %f\n", distance_in_width, distance_in_mean, distance_in_StDev); 
//			printf("\tDesired_width: %f\n", desired_width);
		#endif
		
		(*mean) = mean1 + (distance_in_mean/distance_in_width)*((float) desired_width - width1);
		(*StDev) = StDev1 + (distance_in_StDev/distance_in_width)*((float) desired_width - width1);
	}
}


void MSD_Interpolate_square(float *mean, float *StDev, float desired_width, float *h_MSD_DIT, std::vector<int> *h_MSD_DIT_widths){
	int MSD_DIT_size = h_MSD_DIT_widths->size();
	int position = (int) floorf(log2f((float) desired_width));
	
	if(position == MSD_DIT_size-2) position--;
	if(position == MSD_DIT_size-1 && h_MSD_DIT_widths->operator[](position)==(int) desired_width) {
		(*mean)  = h_MSD_DIT[(position)*MSD_RESULTS_SIZE];
		(*StDev) = h_MSD_DIT[(position)*MSD_RESULTS_SIZE +1];
	}
	else {
		float w = desired_width;
		
		float w0 = h_MSD_DIT_widths->operator[](position);
		float mean0  = h_MSD_DIT[(position)*MSD_RESULTS_SIZE];
		float StDev0 = h_MSD_DIT[(position)*MSD_RESULTS_SIZE +1];
		
		float w1 = h_MSD_DIT_widths->operator[](position+1);
		float mean1  = h_MSD_DIT[(position+1)*MSD_RESULTS_SIZE];
		float StDev1 = h_MSD_DIT[(position+1)*MSD_RESULTS_SIZE +1];
		
		float w2 = h_MSD_DIT_widths->operator[](position+2);
		float mean2  = h_MSD_DIT[(position+2)*MSD_RESULTS_SIZE];
		float StDev2 = h_MSD_DIT[(position+2)*MSD_RESULTS_SIZE +1];
		
		float a0 = ((w - w1)*(w - w2))/((w0 - w1)*(w0 - w2));
		float a1 = ((w - w0)*(w - w2))/((w1 - w0)*(w1 - w2));
		float a2 = ((w - w0)*(w - w1))/((w2 - w0)*(w2 - w1));
		
		(*mean)  = a0*mean0 + a1*mean1 + a2*mean2;
		(*StDev) = a0*StDev0 + a1*StDev1 + a2*StDev2;
	}
}


void MSD_Export_plane(const char *filename, float *h_MSD_DIT, std::vector<int> *h_MSD_DIT_widths, float *h_MSD_interpolated, std::vector<int> *h_boxcar_widths, int max_width_performed) {
	char str[200];
	std::ofstream FILEOUT;
	int MSD_INTER_SIZE = 2;
	
	sprintf(str,"%s_DIT.dat", filename);
	FILEOUT.open (str, std::ofstream::out);
	for(size_t f=0; f<(int) h_MSD_DIT_widths->size(); f++){
		FILEOUT << (int) h_MSD_DIT_widths->operator[](f) << " " << h_MSD_DIT[f*MSD_RESULTS_SIZE] << " " << h_MSD_DIT[f*MSD_RESULTS_SIZE + 1] << std::endl;
	}
	FILEOUT.close();
	
	sprintf(str,"%s_Interpolated.dat", filename);
	FILEOUT.open (str, std::ofstream::out);
	for(size_t f=0; f<(int) h_boxcar_widths->size(); f++){
		if(h_boxcar_widths->operator[](f)<=max_width_performed)
			FILEOUT << (int) h_boxcar_widths->operator[](f) << " " << h_MSD_interpolated[f*MSD_INTER_SIZE] << " " << h_MSD_interpolated[f*MSD_INTER_SIZE + 1] << std::endl;
	}
	FILEOUT.close();
}


   void MSD_Interpolate_values(float *d_MSD_interpolated, float *d_MSD_DIT, std::vector<int> *h_MSD_DIT_widths, float *h_MSD_DIT, float *h_MSD_interpolated2, int nMSDs, std::vector<int> *h_boxcar_widths, int max_width_performed, const char *filename, hipStream_t streams){
//  void MSD_Interpolate_values(float *d_MSD_interpolated, float *d_MSD_DIT, std::vector<int> *h_MSD_DIT_widths, int nMSDs, std::vector<int> *h_boxcar_widths, int max_width_performed, const char *filename, hipStream_t streams){
	#ifdef GPU_PARTIAL_TIMER
	GpuTimer timer;
	timer.Start();
	#endif
	
	int MSD_INTER_SIZE = 2;
//	float *h_MSD_DIT;
//	float *h_MSD_interpolated;
	int nWidths = (int) h_boxcar_widths->size();
//	printf("\n\n\n nWidths: %i\n\n\n", nWidths);
//	h_MSD_DIT = new float[nMSDs*MSD_RESULTS_SIZE];
//	hipHostRegister(h_MSD_DIT,sizeof(float)*nMSDs*MSD_RESULTS_SIZE,0);
//	hipHostMalloc(&h_MSD_DIT,sizeof(float)*nMSDs*MSD_RESULTS_SIZE);
//	hipHostMalloc(&h_MSD_interpolated,sizeof(float)*nWidths*MSD_INTER_SIZE);
//	h_MSD_interpolated = new float[nWidths*MSD_INTER_SIZE];
	
	int MSD_DIT_size = h_MSD_DIT_widths->size();	
	int *d_MSD_DIT_widths;
	checkCudaErrors(hipMalloc((void **) &d_MSD_DIT_widths, sizeof(int)*MSD_DIT_size));
	checkCudaErrors(hipMemcpyAsync(d_MSD_DIT_widths, &h_MSD_DIT_widths->operator[](0), sizeof(int)*MSD_DIT_size,hipMemcpyHostToDevice,streams));
	int *d_boxcar;
	hipMalloc((void **) &d_boxcar, sizeof(int)*nWidths);
	checkCudaErrors(hipMemcpyAsync(d_boxcar, &h_boxcar_widths->operator[](0), sizeof(int)*nWidths,hipMemcpyHostToDevice,streams));

		
//	printf("\n\n nMSDs*: %i size of msd dit: %i %i\n\n", sizeof(float)*nMSDs*MSD_RESULTS_SIZE, MSD_DIT_size, &h_MSD_DIT_widths[0]);	
//	checkCudaErrors(hipMemcpyAsync(h_MSD_DIT, d_MSD_DIT, nMSDs*MSD_RESULTS_SIZE*sizeof(float), hipMemcpyDeviceToHost, streams));
	
//	for(int f=0; f<nWidths; f++){
//		if(h_boxcar_widths->operator[](f)<=max_width_performed) {
//			float mean, StDev;
//			MSD_Interpolate_linear(&mean, &StDev, (float) h_boxcar_widths->operator[](f), h_MSD_DIT, h_MSD_DIT_widths);
			MSD_GPU_Interpolate_linear<<<1,nWidths,0,streams>>>(d_MSD_DIT, d_MSD_interpolated, d_MSD_DIT_widths, h_MSD_DIT_widths->size(), d_boxcar, max_width_performed);
		//	h_MSD_interpolated[f*MSD_INTER_SIZE] = mean;
		//	h_MSD_interpolated[f*MSD_INTER_SIZE+1] = StDev;
//		}
//	}
	
	#ifdef MSD_PLANE_EXPORT
//		MSD_Export_plane(filename, h_MSD_DIT, h_MSD_DIT_widths, h_MSD_interpolated2, h_boxcar_widths, max_width_performed);
	#endif
	
//	checkCudaErrors(hipMemcpyAsync(d_MSD_interpolated, h_MSD_interpolated, nWidths*MSD_INTER_SIZE*sizeof(float), hipMemcpyHostToDevice,streams));
	
//	hipHostUnregister(h_MSD_DIT);
//	delete[] h_MSD_DIT;
//	hipHostFree(h_MSD_DIT);
//	delete[] h_MSD_interpolated;
	
	#ifdef GPU_PARTIAL_TIMER
	timer.Stop();
	printf("    Interpolation step took %f ms;\n", timer.Elapsed());
	#endif
}

//-------------------------------------------------------------------------<

void Get_MSD_plane_profile_memory_requirements(size_t *MSD_profile_size_in_bytes, size_t *MSD_DIT_profile_size_in_bytes, size_t *workarea_size_in_bytes, size_t primary_dimension, size_t secondary_dimension, std::vector<int> *boxcar_widths) {
	// temporary work area for decimations. We need 2*1/4 = 1/2.
	size_t t_wsib = (primary_dimension*secondary_dimension*sizeof(float))/2;
	
	// temporary storage for MSD values of decimated input data
	int max_boxcar_width = boxcar_widths->operator[](boxcar_widths->size()-1);
	int nDecimations = ((int) floorf(log2f((float)max_boxcar_width))) + 2;
	t_wsib = t_wsib + nDecimations*MSD_RESULTS_SIZE*sizeof(float);
	
	// temporary storage for calculation of MSD. We have to choose the maximum from all possible variants.
	size_t decimated_pd = primary_dimension;
	int max_nBlocks = 0;
	for(int f=0; f<nDecimations; f++){
		MSD_Configuration conf(decimated_pd, secondary_dimension, 0, 0);
		if(conf.nBlocks_total>max_nBlocks) max_nBlocks = conf.nBlocks_total;
		decimated_pd = (decimated_pd>>1);
	}
	t_wsib = t_wsib + max_nBlocks*MSD_PARTIAL_SIZE*sizeof(float);

	#ifdef MSD_PLANE_DEBUG
		printf("Data size primary dim: %zu; secondary dim: %zu;\n", primary_dimension, secondary_dimension);
		printf("Temporary storage for data: %zu bytes = %zu floats;\n", t_wsib, t_wsib/4);
		printf("Size of DIT MSDs: %d elements = %d float = %d bytes\n", nDecimations, nDecimations*MSD_RESULTS_SIZE, nDecimations*MSD_RESULTS_SIZE*sizeof(float));
		printf("Max MSD blocks: %d blocks = %d float = %d bytes\n", max_nBlocks, max_nBlocks*MSD_PARTIAL_SIZE, max_nBlocks*MSD_PARTIAL_SIZE*sizeof(float));
	#endif
	
	(*workarea_size_in_bytes) = t_wsib;
	(*MSD_profile_size_in_bytes) = boxcar_widths->size()*2*sizeof(float);
	(*MSD_DIT_profile_size_in_bytes) = nDecimations*MSD_PARTIAL_SIZE*sizeof(float);
}


// TODO:
//		Make it fail reasonably, which means if max_boxcar_width = 1 calculate only MSD for given plane and omit DIT completely
//		Add checks when StDev blows up because of too much DIT
//		Add checks if there is enough timesamples to do DIT.
// Note: By separating DIT = 2 into two parts we slightly decreasing precision if compared to non spit case, because outlier rejection has fewer points to work with. This could be a problem if we have a plane small enough to fit into memory but we still plit it in two.
//		Add branch that would not split DIT=2 if there is enough memory. 
void MSD_plane_profile(float *d_MSD_interpolated, float *d_input_data, float *d_MSD_DIT_previous, float *h_MSD_DIT, float *h_MSD_interpolated, float *workarea, bool high_memory, size_t primary_dimension, size_t secondary_dimension, std::vector<int> *boxcar_widths, float tstart, float dm_low, float dm_high, float OR_sigma_multiplier, int enable_outlier_rejection, bool perform_continuous, double *total_time, double *dit_time, double *MSD_time, hipStream_t streams){
//void MSD_plane_profile(float *d_MSD_interpolated, float *d_input_data, float *d_MSD_DIT_previous, float *workarea, bool high_memory, size_t primary_dimension, size_t secondary_dimension, std::vector<int> *boxcar_widths, float tstart, float dm_low, float dm_high, float OR_sigma_multiplier, int enable_outlier_rejection, bool perform_continuous, double *total_time, double *dit_time, double *MSD_time, hipStream_t streams){
	int boxcar_widths_size = (int) boxcar_widths->size();
	int max_boxcar_width = boxcar_widths->operator[](boxcar_widths_size-1);
	int nDecimations = ((int) floorf(log2f((float)max_boxcar_width))) + 1;
	int nDIT_widths = nDecimations + 1;
	std::vector<int> h_MSD_DIT_widths;
	
	size_t datasize = primary_dimension*secondary_dimension;
	printf("\n nDecimations: %i datasize: %zu results_size: %i\n", nDecimations, datasize, MSD_RESULTS_SIZE);
	float *d_sudy, *d_lichy, *d_MSD_DIT, *d_MSD_workarea;
	d_sudy = workarea;
	d_lichy = &workarea[datasize/4];
	if(high_memory) {
		d_MSD_DIT = &workarea[datasize/4 + datasize/2];
		d_MSD_workarea = &workarea[datasize/4 + datasize/2 + (nDecimations+1)*MSD_RESULTS_SIZE];
	}
	else {
		d_MSD_DIT = &workarea[datasize/2];
		d_MSD_workarea = &workarea[datasize/2 + (nDecimations+1)*MSD_RESULTS_SIZE];
	}
	
//	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipMemsetAsync((void*) d_MSD_DIT, 0, (nDecimations+1)*MSD_RESULTS_SIZE*sizeof(float), streams));
//	checkCudaErrors(hipGetLastError());
	
	MSD_of_input_plane(d_MSD_DIT, &h_MSD_DIT_widths, d_input_data, d_MSD_DIT_previous, d_sudy, d_lichy, d_MSD_workarea, primary_dimension, secondary_dimension, nDecimations, max_boxcar_width, OR_sigma_multiplier, enable_outlier_rejection, high_memory, perform_continuous, total_time, dit_time, MSD_time, streams);
	
	#ifdef MSD_PLANE_DEBUG
		printf("    Number of calculated MSD values: %d; number of interpolated MSD values: %d;\n",nDIT_widths, boxcar_widths_size);
	#endif
	
	char filename[100];
	sprintf(filename,"MSD_plane_profile_i_test-t_%.2f-dm_%.2f-%.2f", tstart, dm_low, dm_high);
	MSD_Interpolate_values(d_MSD_interpolated, d_MSD_DIT, &h_MSD_DIT_widths, h_MSD_DIT, h_MSD_interpolated, nDIT_widths, boxcar_widths, max_boxcar_width, filename, streams);
//	MSD_Interpolate_values(d_MSD_interpolated, d_MSD_DIT, &h_MSD_DIT_widths, nDIT_widths, boxcar_widths, max_boxcar_width, filename, streams);
}

//------------- MSD plane profile
//---------------------------------------------------------------



































//---------------------------------------------------------------
//------------- MSD plane profile boxcars

void Create_boxcar_MSD(float *d_data, size_t nTimesamples, size_t nDMs, std::vector<MSD_Data> *boxcar_MSD, std::vector<MSD_Data> *boxcar_MSD_BLN, int max_nTaps, int max_boxcar_width, float OR_sigma_multiplier, hipStream_t streams){
	GpuTimer timer;
	double total_time = 0;
	int nRest;
	MSD_Data mdtemp;
	float *d_boxcar, *d_MSD;
	float h_MSD[MSD_RESULTS_SIZE];
	hipMalloc((void **) &d_boxcar, nTimesamples*nDMs*sizeof(float));
	hipMalloc((void **) &d_MSD, MSD_RESULTS_SIZE*sizeof(float));
	
	timer.StartWithStream(streams);
	
	MSD_normal(d_MSD, d_data, nTimesamples, nDMs, 0, streams);
	hipMemcpyAsync(h_MSD, d_MSD, MSD_RESULTS_SIZE*sizeof(float), hipMemcpyDeviceToHost, streams);
	mdtemp.width = 1; mdtemp.mean = h_MSD[0]; mdtemp.sd = h_MSD[1];
	boxcar_MSD->push_back(mdtemp);
	
	MSD_outlier_rejection(d_MSD, d_data, nTimesamples, nDMs, 0, OR_sigma_multiplier, streams);
	hipMemcpyAsync(h_MSD, d_MSD, MSD_RESULTS_SIZE*sizeof(float), hipMemcpyDeviceToHost, streams);
	mdtemp.width = 1; mdtemp.mean = h_MSD[0]; mdtemp.sd = h_MSD[1];
	boxcar_MSD_BLN->push_back(mdtemp);
	
	timer.StopWithStream(streams);
	total_time = total_time + timer.Elapsed();
	printf("DIT value: %d; took %f ms; Total time %fms\n", 1, timer.Elapsed(), total_time);
	
	for(size_t f=2; f<=max_nTaps; f++){
		if( (nTimesamples-f+1)>0 ) {
//			timer.Start();
			
			nRest = PD_FIR(d_data, d_boxcar, f, nDMs, nTimesamples);
			
			MSD_normal(d_MSD, d_boxcar, nTimesamples, nDMs, nRest, streams);
			hipMemcpyAsync(h_MSD, d_MSD, MSD_RESULTS_SIZE*sizeof(float), hipMemcpyDeviceToHost, streams);
			mdtemp.width = f; mdtemp.mean = h_MSD[0]; mdtemp.sd = h_MSD[1];
			boxcar_MSD->push_back(mdtemp);
			
			MSD_outlier_rejection(d_MSD, d_boxcar, nTimesamples, nDMs, nRest, OR_sigma_multiplier, streams);
			hipMemcpyAsync(h_MSD, d_MSD, MSD_RESULTS_SIZE*sizeof(float), hipMemcpyDeviceToHost, streams);
			mdtemp.width = f; mdtemp.mean = h_MSD[0]; mdtemp.sd = h_MSD[1];
			boxcar_MSD_BLN->push_back(mdtemp);
			
//			timer.Stop();
			total_time = total_time + timer.Elapsed();
			printf("DIT value: %d; took %f ms; Total time %fms\n", (int) f, timer.Elapsed(), total_time);
		}
	}
	
	checkCudaErrors(hipGetLastError());
	
	for(size_t f=130; f<=256 && f<max_boxcar_width; f+=4){
		printf("nTimesamples: %zu; f: %zu; %zu\n", nTimesamples, f, nTimesamples-f+1);
		int itemp = (int) (nTimesamples-f+1);
		if( itemp>0 ) {
//			timer.Start();
			
			nRest=PPF_L1(d_data, d_boxcar, nDMs, nTimesamples, f);
			
			if(nRest>0){
				MSD_normal(d_MSD, d_boxcar, nTimesamples, nDMs, nRest, streams);
				hipMemcpyAsync(h_MSD, d_MSD, MSD_RESULTS_SIZE*sizeof(float), hipMemcpyDeviceToHost, streams);
				mdtemp.width = f; mdtemp.mean = h_MSD[0]; mdtemp.sd = h_MSD[1];
				boxcar_MSD->push_back(mdtemp);
				
				MSD_outlier_rejection(d_MSD, d_boxcar, nTimesamples, nDMs, nRest, OR_sigma_multiplier, streams);
				hipMemcpyAsync(h_MSD, d_MSD, MSD_RESULTS_SIZE*sizeof(float), hipMemcpyDeviceToHost, streams);
				mdtemp.width = f; mdtemp.mean = h_MSD[0]; mdtemp.sd = h_MSD[1];
				boxcar_MSD_BLN->push_back(mdtemp);
			}

//			timer.Stop();
//			total_time = total_time + timer.Elapsed();
			printf("DIT value: %d; took %f ms; Total time %fms\n", (int) f, timer.Elapsed(), total_time);
		}
		checkCudaErrors(hipGetLastError());
	}
	
	checkCudaErrors(hipGetLastError());
	
	for(size_t f=272; f<=512 && f<max_boxcar_width; f+=16){
		printf("nTimesamples: %zu; f: %zu; %zu\n", nTimesamples, f, nTimesamples-f+1);
		int itemp = (int) (nTimesamples-f+1);
		if( itemp>0 ) {
//			timer.Start();
			
			nRest=PPF_L1(d_data, d_boxcar, nDMs, nTimesamples, f);
			
			if(nRest>0){
				MSD_normal(d_MSD, d_boxcar, nTimesamples, nDMs, nRest, streams);
				hipMemcpyAsync(h_MSD, d_MSD, MSD_RESULTS_SIZE*sizeof(float), hipMemcpyDeviceToHost, streams);
				mdtemp.width = f; mdtemp.mean = h_MSD[0]; mdtemp.sd = h_MSD[1];
				boxcar_MSD->push_back(mdtemp);
				
				MSD_outlier_rejection(d_MSD, d_boxcar, nTimesamples, nDMs, nRest, OR_sigma_multiplier, streams);
				hipMemcpyAsync(h_MSD, d_MSD, MSD_RESULTS_SIZE*sizeof(float), hipMemcpyDeviceToHost, streams);
				mdtemp.width = f; mdtemp.mean = h_MSD[0]; mdtemp.sd = h_MSD[1];
				boxcar_MSD_BLN->push_back(mdtemp);
			}
			
//			timer.Stop();
//			total_time = total_time + timer.Elapsed();
			printf("DIT value: %d; took %f ms; Total time %fms\n", (int) f, timer.Elapsed(), total_time);
		}
		checkCudaErrors(hipGetLastError());
	}
	
	checkCudaErrors(hipGetLastError());
	
	for(size_t f=544; f<=1024 && f<max_boxcar_width; f+=32){
		printf("nTimesamples: %zu; f: %zu; %zu\n", nTimesamples, f, nTimesamples-f+1);
		int itemp = (int) (nTimesamples-f+1);
		if( itemp>0 ) {
//			timer.Start();
			
			nRest=PPF_L1(d_data, d_boxcar, nDMs, nTimesamples, f);

			if(nRest>0){
				MSD_normal(d_MSD, d_boxcar, nTimesamples, nDMs, nRest, streams);
				hipMemcpyAsync(h_MSD, d_MSD, MSD_RESULTS_SIZE*sizeof(float), hipMemcpyDeviceToHost, streams);
				mdtemp.width = f; mdtemp.mean = h_MSD[0]; mdtemp.sd = h_MSD[1];
				boxcar_MSD->push_back(mdtemp);
				
				MSD_outlier_rejection(d_MSD, d_boxcar, nTimesamples, nDMs, nRest, OR_sigma_multiplier, streams);
				hipMemcpyAsync(h_MSD, d_MSD, MSD_RESULTS_SIZE*sizeof(float), hipMemcpyDeviceToHost, streams);
				mdtemp.width = f; mdtemp.mean = h_MSD[0]; mdtemp.sd = h_MSD[1];
				boxcar_MSD_BLN->push_back(mdtemp);
			}
			
//			timer.Stop();
			total_time = total_time + timer.Elapsed();
			printf("DIT value: %d; took %f ms; Total time %fms\n", (int) f, timer.Elapsed(), total_time);
		}
		checkCudaErrors(hipGetLastError());
	}
	
	checkCudaErrors(hipGetLastError());

	for(size_t f=1088; f<=2048 && f<max_boxcar_width; f+=64){
		printf("nTimesamples: %zu; f: %zu; %zu\n", nTimesamples, f, nTimesamples-f+1);
		int itemp = (int) (nTimesamples-f+1);
		if( itemp>0 ) {
//			timer.Start();
			
			nRest=PPF_L1(d_data, d_boxcar, nDMs, nTimesamples, f);

			if(nRest>0){
				MSD_normal(d_MSD, d_boxcar, nTimesamples, nDMs, nRest, streams);
				hipMemcpyAsync(h_MSD, d_MSD, MSD_RESULTS_SIZE*sizeof(float), hipMemcpyDeviceToHost,streams);
				mdtemp.width = f; mdtemp.mean = h_MSD[0]; mdtemp.sd = h_MSD[1];
				boxcar_MSD->push_back(mdtemp);
				
				MSD_outlier_rejection(d_MSD, d_boxcar, nTimesamples, nDMs, nRest, OR_sigma_multiplier, streams);
				hipMemcpyAsync(h_MSD, d_MSD, MSD_RESULTS_SIZE*sizeof(float), hipMemcpyDeviceToHost,streams);
				mdtemp.width = f; mdtemp.mean = h_MSD[0]; mdtemp.sd = h_MSD[1];
				boxcar_MSD_BLN->push_back(mdtemp);
			}
			
//			timer.Stop();
			total_time = total_time + timer.Elapsed();
			printf("DIT value: %d; took %f ms; Total time %fms\n", (int) f, timer.Elapsed(), total_time);
		}
		checkCudaErrors(hipGetLastError());
	}
	
	checkCudaErrors(hipGetLastError());

	for(size_t f=2176; f<=4096 && f<max_boxcar_width; f+=128){
		printf("nTimesamples: %zu; f: %zu; %zu\n", nTimesamples, f, nTimesamples-f+1);
		int itemp = (int) (nTimesamples-f+1);
		if( itemp>0 ) {
//			timer.Start();
			
			nRest=PPF_L1(d_data, d_boxcar, nDMs, nTimesamples, f);

			if(nRest>0){		
				MSD_normal(d_MSD, d_boxcar, nTimesamples, nDMs, nRest, streams);
				hipMemcpyAsync(h_MSD, d_MSD, MSD_RESULTS_SIZE*sizeof(float), hipMemcpyDeviceToHost, streams);
				mdtemp.width = f; mdtemp.mean = h_MSD[0]; mdtemp.sd = h_MSD[1];
				boxcar_MSD->push_back(mdtemp);
				
				MSD_outlier_rejection(d_MSD, d_boxcar, nTimesamples, nDMs, nRest, OR_sigma_multiplier, streams);
				hipMemcpyAsync(h_MSD, d_MSD, MSD_RESULTS_SIZE*sizeof(float), hipMemcpyDeviceToHost, streams);
				mdtemp.width = f; mdtemp.mean = h_MSD[0]; mdtemp.sd = h_MSD[1];
				boxcar_MSD_BLN->push_back(mdtemp);
			}
			
//			timer.Stop();
			total_time = total_time + timer.Elapsed();
			printf("DIT value: %d; took %f ms; Total time %fms\n", (int) f, timer.Elapsed(), total_time);
		}
		checkCudaErrors(hipGetLastError());
	}
	
	checkCudaErrors(hipGetLastError());
	
	for(size_t f=4352; f<=8192 && f<max_boxcar_width; f+=256){
		printf("nTimesamples: %zu; f: %zu; %zu\n", nTimesamples, f, nTimesamples-f+1);
		int itemp = (int) (nTimesamples-f+1);
		if( itemp>0 ) {
//			timer.Start();
			
			nRest=PPF_L1(d_data, d_boxcar, nDMs, nTimesamples, f);
			
			if(nRest>0){
				MSD_normal(d_MSD, d_boxcar, nTimesamples, nDMs, nRest,streams);
				hipMemcpyAsync(h_MSD, d_MSD, MSD_RESULTS_SIZE*sizeof(float), hipMemcpyDeviceToHost, streams);
				mdtemp.width = f; mdtemp.mean = h_MSD[0]; mdtemp.sd = h_MSD[1];
				boxcar_MSD->push_back(mdtemp);
				
				MSD_outlier_rejection(d_MSD, d_boxcar, nTimesamples, nDMs, nRest, OR_sigma_multiplier,streams);
				hipMemcpyAsync(h_MSD, d_MSD, MSD_RESULTS_SIZE*sizeof(float), hipMemcpyDeviceToHost, streams);
				mdtemp.width = f; mdtemp.mean = h_MSD[0]; mdtemp.sd = h_MSD[1];
				boxcar_MSD_BLN->push_back(mdtemp);
			}
			
//			timer.Stop();
			total_time = total_time + timer.Elapsed();
			printf("DIT value: %d; took %f ms; Total time %fms\n", (int) f, timer.Elapsed(), total_time);
		}
		checkCudaErrors(hipGetLastError());
	}
	
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipFree(d_boxcar));
	checkCudaErrors(hipFree(d_MSD));
}



void MSD_plane_profile_boxcars(float *d_input_data, size_t nTimesamples, size_t nDMs, std::vector<int> *boxcar_widths, float OR_sigma_multiplier, float dm_low, float dm_high, float tstart, hipStream_t streams){
	char filename[200];
	std::vector<MSD_Data> h_boxcar_MSD;
	std::vector<MSD_Data> h_boxcar_MSD_OR;
	
	size_t free_mem, total_mem, req_mem;
	hipMemGetInfo(&free_mem,&total_mem);
	req_mem = nTimesamples*nDMs*sizeof(float);
	printf("Memory available: %f; Memory required: %f;\n", (double) free_mem/(1024.0*1024.0), ((double) req_mem)/(1024.0*1024.0));
	if(free_mem<req_mem) {
		printf("Not enough memory to perform the comparison!\n");
		return;
	}
	
	int boxcar_widths_size = boxcar_widths->size();
	int max_boxcar_width = boxcar_widths->operator[](boxcar_widths_size-1);
	
	Create_boxcar_MSD(d_input_data, nTimesamples, nDMs, &h_boxcar_MSD, &h_boxcar_MSD_OR, 128, max_boxcar_width, OR_sigma_multiplier, streams);
	
	sprintf(filename,"MSD_boxcars_OR%f-t_%.2f-dm_%.2f-%.2f.dat", OR_sigma_multiplier, tstart, dm_low, dm_high);
	
	std::ofstream FILEOUT;
	FILEOUT.open (filename, std::ofstream::out);

	for(size_t f=0; f<h_boxcar_MSD.size(); f++){
		FILEOUT << (int) h_boxcar_MSD[f].width << " " << h_boxcar_MSD[f].mean << " " << h_boxcar_MSD[f].sd << " " << "3" << std::endl;
	}
	FILEOUT << std::endl;
	FILEOUT << std::endl;
	for(size_t f=0; f<h_boxcar_MSD_OR.size(); f++){
		FILEOUT << (int) h_boxcar_MSD_OR[f].width << " " << h_boxcar_MSD_OR[f].mean << " " << h_boxcar_MSD_OR[f].sd << " " << "4" << std::endl;
	}
	FILEOUT << std::endl;
	FILEOUT << std::endl;
	
	FILEOUT.close();
}

//------------- MSD plane profile boxcars
//---------------------------------------------------------------




