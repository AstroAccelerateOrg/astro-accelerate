#include "hip/hip_runtime.h"
// Added by Karel Adamek 

#ifndef MSD_LEGACY_KERNEL_H_
#define MSD_LEGACY_KERNEL_H_

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "headers/params.h"


__global__ void MSD_GPU_LA_ALL(float const* __restrict__ d_input, float *d_output, float *d_output_taps, int y_steps, int nTaps, int nTimesamples, int offset) {
	__shared__ float s_input[3*PD_NTHREADS];
	__shared__ float s_base[3*PD_NTHREADS];
	
	// MSD variables
	float M, S, j;
	float M_b, S_b, j_b;
	// FIR variables
	int d, gpos, spos, local_id;
	ushort EpT, limit;
	float2 ftemp1, ftemp2, ftemp3;
	float Bw[2];
	
	EpT = 2*PD_NTHREADS-nTaps+4;
	limit = blockDim.x - (nTaps>>2) - 1;

	// First y coordinate is separated
	//-------------------> FIR
	spos = blockIdx.x*(EpT) + 2*threadIdx.x;
	gpos = blockIdx.y*y_steps*nTimesamples + spos;
	Bw[0]=0; Bw[1]=0; j=0; j_b=0;
	if( (spos+4)<(nTimesamples-offset) ){
		// loading data for FIR filter. Each thread calculates two samples
		ftemp1.x= __ldg(&d_input[gpos]);	
		ftemp1.y= __ldg(&d_input[gpos+1]);
		ftemp2.x= __ldg(&d_input[gpos+2]);
		ftemp2.y= __ldg(&d_input[gpos+3]);
		ftemp3.x= __ldg(&d_input[gpos+4]);
		
		// Calculate FIR of 4 taps
		Bw[0]=ftemp1.x + ftemp1.y + ftemp2.x + ftemp2.y;
		Bw[1]=ftemp1.y + ftemp2.x + ftemp2.y + ftemp3.x;
		
		// Initialization of MSD variables for non-processed StrDev
		Initiate( &M_b, &S_b, &j_b, ftemp1.x );
		// First addition (second actually, but first done this way) non-processed StrDev
		Add_one( &M_b, &S_b, &j_b, ftemp1.y );
	}
	
	s_input[2*threadIdx.x] = Bw[0];
	s_input[2*threadIdx.x+1] = Bw[1];
	
	__syncthreads();
	
	// Calculating FIR up to nTaps
	for(d=4; d<nTaps; d=d+4){
		local_id = threadIdx.x+(d>>1);
		if( local_id<=limit ){
			Bw[0] = Bw[0] + s_input[2*local_id]; Bw[1] = Bw[1] + s_input[2*local_id+1];
		}
	}
	
	// Note: threads with local_id<0 which have wrong result create sums as well but are removed from final results later
	//       same is for base values as these would be included twice. First time here and next time in threadblock next to it
	//       this is due to halo needed for FIR filter	
	Initiate( &M, &S, &j, Bw[0] ); // Initialization of MSD variables for processed StrDev
	Add_one( &M, &S, &j, Bw[1] ); // First addition (second actually, but first done this way) processed StrDev
	
	
	// Rest of the iteration in y direction	
	for (int yf = 1; yf < y_steps; yf++) {
		__syncthreads();
		//-------------------> FIR
		spos = blockIdx.x*(EpT) + 2*threadIdx.x;
		gpos = blockIdx.y*y_steps*nTimesamples + yf*nTimesamples + spos;
		Bw[0]=0; Bw[1]=0;
		if( (spos+4)<(nTimesamples-offset) ){
			ftemp1.x= __ldg(&d_input[gpos]);	
			ftemp1.y= __ldg(&d_input[gpos+1]);
			ftemp2.x= __ldg(&d_input[gpos+2]);
			ftemp2.y= __ldg(&d_input[gpos+3]);
			ftemp3.x= __ldg(&d_input[gpos+4]);

			Bw[0]=ftemp1.x + ftemp1.y + ftemp2.x + ftemp2.y;
			Bw[1]=ftemp1.y + ftemp2.x + ftemp2.y + ftemp3.x;
			
			Add_one( &M_b, &S_b, &j_b, ftemp1.x );
			Add_one( &M_b, &S_b, &j_b, ftemp1.y );
		}
		
		s_input[2*threadIdx.x] = Bw[0];
		s_input[2*threadIdx.x+1] = Bw[1];
	
		__syncthreads();
	
		for(d=4; d<nTaps; d=d+4){	
			local_id = threadIdx.x+(d>>1);
			if( local_id<=limit ){
				Bw[0] = Bw[0] + s_input[2*local_id]; Bw[1] = Bw[1] + s_input[2*local_id+1];
			}
		}
		
		Add_one( &M, &S, &j, Bw[0] );
		Add_one( &M, &S, &j, Bw[1] );
	}
	
	__syncthreads();
	
	s_input[threadIdx.x] = 0;
	s_input[blockDim.x + threadIdx.x] = 0;
	s_input[2*blockDim.x + threadIdx.x] = 0;
	
	s_base[threadIdx.x] = 0;
	s_base[blockDim.x + threadIdx.x] = 0;
	s_base[2*blockDim.x + threadIdx.x] = 0;
	
	__syncthreads();

	spos=blockIdx.x*(EpT) + 2*threadIdx.x;	
	if( local_id<=limit ) {
		// Note: ommited number of samples in the last trailing threadblocks is due to -nTaps which is here. 
		//       Missing data should be contained in local_id. Thus this code is missing some time sample even it it does not need to. 
		//       When removed it produces different number of added time samples in j and j_b which is wierd
		if( spos<(nTimesamples-offset-nTaps) ) { // -nTaps might not be necessary
			s_input[local_id] = M;
			s_input[blockDim.x + local_id] = S;
			s_input[2*blockDim.x + local_id] = j;
			
			s_base[local_id] = M_b;
			s_base[blockDim.x + local_id] = S_b;
			s_base[2*blockDim.x + local_id] = j_b;
		}

	}
	__syncthreads();
	
	//------------------------------------------------------------------------------------
	//---------> StrDev of processed input
	Reduce_SM( &M, &S, &j, s_input );
	Reduce_WARP( &M, &S, &j);
	
	//------------------------------------------------------------------------------------
	//---------> StrDev of unprocessed input
	Reduce_SM( &M_b, &S_b, &j_b, s_base );
	Reduce_WARP( &M_b, &S_b, &j_b);

	//----------------------------------------------
	//---- Writing data
	if (threadIdx.x == 0) {
		gpos = blockIdx.y*gridDim.x + blockIdx.x;
		d_output_taps[3*gpos] = M;
		d_output_taps[3*gpos + 1] = S;
		d_output_taps[3*gpos + 2] = j;
		
		d_output[3*gpos] = M_b;
		d_output[3*gpos + 1] = S_b;
		d_output[3*gpos + 2] = j_b;
	}
}


__global__ void MSD_GPU_LA_ALL_Nth(float const* __restrict__ d_input, float const* __restrict   d_bv_in, float *d_output, float *d_output_taps, int y_steps, int nTaps, int nTimesamples, int offset) {
	__shared__ float s_input[3*PD_NTHREADS];
	__shared__ float s_base[3*PD_NTHREADS];
	
	// MSD variables
	float M, S, j;
	float M_b, S_b, j_b;
	// FIR variables
	int d, gpos, spos, local_id;
	ushort EpT, limit;
	float2 ftemp1, ftemp2, ftemp3;
	float Bw[2];
	
	EpT = 2*PD_NTHREADS-nTaps+4;
	limit = blockDim.x - (nTaps>>2) - 1;

	// First y coordinate is separated
	//-------------------> FIR
	spos = blockIdx.x*(EpT) + 2*threadIdx.x;
	gpos = blockIdx.y*y_steps*nTimesamples + spos;
	Bw[0]=0; Bw[1]=0; j=0; j_b=0;
	if( (spos+4)<(nTimesamples-offset) ){
		// loading data for FIR filter. Each thread calculates two samples
		ftemp1.x= __ldg(&d_input[gpos]);	
		ftemp1.y= __ldg(&d_input[gpos+1]);
		ftemp2.x= __ldg(&d_input[gpos+2]);
		ftemp2.y= __ldg(&d_input[gpos+3]);
		ftemp3.x= __ldg(&d_input[gpos+4]);
		
		// Calculate FIR of 4 taps
		Bw[0]=ftemp1.x + ftemp1.y + ftemp2.x + ftemp2.y;
		Bw[1]=ftemp1.y + ftemp2.x + ftemp2.y + ftemp3.x;
		
		Initiate( &M_b, &S_b, &j_b, __ldg(&d_bv_in[gpos]) );
		Add_one( &M_b, &S_b, &j_b, __ldg(&d_bv_in[gpos+1]) );
	}
	
	s_input[2*threadIdx.x] = Bw[0];
	s_input[2*threadIdx.x+1] = Bw[1];
	
	__syncthreads();
	
	// Calculating FIR up to nTaps
	for(d=4; d<nTaps; d=d+4){
		local_id = threadIdx.x+(d>>1);
		if( local_id<=limit ){
			Bw[0] = Bw[0] + s_input[2*local_id]; Bw[1] = Bw[1] + s_input[2*local_id+1];
		}
	}
	
	// Note: threads with local_id<0 which have wrong result create sums as well but are removed from final results later
	//       same is for base values as these would be included twice. First time here and next time in threadblock next to it
	//       this is due to halo needed for FIR filter
	Initiate( &M, &S, &j, __ldg(&d_bv_in[gpos]) + Bw[0] );
	Add_one( &M, &S, &j, __ldg(&d_bv_in[gpos+1]) + Bw[1] );
	
	
	// Rest of the iteration in y direction	
	for (int yf = 1; yf < y_steps; yf++) {
		__syncthreads();
		//-------------------> FIR
		spos = blockIdx.x*(EpT) + 2*threadIdx.x;
		gpos = blockIdx.y*y_steps*nTimesamples + yf*nTimesamples + spos;
		Bw[0]=0; Bw[1]=0;
		if( (spos+4)<(nTimesamples-offset) ){
			ftemp1.x= __ldg(&d_input[gpos]);	
			ftemp1.y= __ldg(&d_input[gpos+1]);
			ftemp2.x= __ldg(&d_input[gpos+2]);
			ftemp2.y= __ldg(&d_input[gpos+3]);
			ftemp3.x= __ldg(&d_input[gpos+4]);

			Bw[0]=ftemp1.x + ftemp1.y + ftemp2.x + ftemp2.y;
			Bw[1]=ftemp1.y + ftemp2.x + ftemp2.y + ftemp3.x;
			
			Add_one( &M_b, &S_b, &j_b, __ldg(&d_bv_in[gpos]) );
			Add_one( &M_b, &S_b, &j_b, __ldg(&d_bv_in[gpos+1]) );
		}
		
		s_input[2*threadIdx.x] = Bw[0];
		s_input[2*threadIdx.x+1] = Bw[1];
	
		__syncthreads();
	
		for(d=4; d<nTaps; d=d+4){	
			local_id = threadIdx.x+(d>>1);
			if( local_id<=limit ){
				Bw[0] = Bw[0] + s_input[2*local_id]; Bw[1] = Bw[1] + s_input[2*local_id+1];
			}
		}
		
		Add_one( &M, &S, &j, __ldg(&d_bv_in[gpos]) + Bw[0] );
		Add_one( &M, &S, &j, __ldg(&d_bv_in[gpos+1]) + Bw[1] );
	}
	
	__syncthreads();
	
	s_input[threadIdx.x] = 0;
	s_input[blockDim.x + threadIdx.x] = 0;
	s_input[2*blockDim.x + threadIdx.x] = 0;
	
	s_base[threadIdx.x] = 0;
	s_base[blockDim.x + threadIdx.x] = 0;
	s_base[2*blockDim.x + threadIdx.x] = 0;
	
	__syncthreads();

	spos=blockIdx.x*(EpT) + 2*threadIdx.x;	
	if( local_id<=limit ) {		
		// Note: ommited number of samples in the last trailing threadblocks is due to -nTaps which is here. 
		//       Missing data should be contained in local_id. Thus this code is missing some time sample even it it does not need to. 
		//       When removed it produces different number of added time samples in j and j_b which is wierd
		if( spos<(nTimesamples-offset-nTaps) ) { // -nTaps might not be necessary
			s_input[local_id] = M;
			s_input[blockDim.x + local_id] = S;
			s_input[2*blockDim.x + local_id] = j;
			
			s_base[local_id] = M_b;
			s_base[blockDim.x + local_id] = S_b;
			s_base[2*blockDim.x + local_id] = j_b;
		}

	}
	__syncthreads();
	
	//------------------------------------------------------------------------------------
	//---------> StrDev of processed input
	Reduce_SM( &M, &S, &j, s_input );
	Reduce_WARP( &M, &S, &j);
	
	//------------------------------------------------------------------------------------
	//---------> StrDev of unprocessed input
	Reduce_SM( &M_b, &S_b, &j_b, s_base );
	Reduce_WARP( &M_b, &S_b, &j_b);

	//----------------------------------------------
	//---- Writing data
	if (threadIdx.x == 0) {
		gpos = blockIdx.y*gridDim.x + blockIdx.x;
		d_output_taps[3*gpos] = M;
		d_output_taps[3*gpos + 1] = S;
		d_output_taps[3*gpos + 2] = j;
		
		d_output[3*gpos] = M_b;
		d_output[3*gpos + 1] = S_b;
		d_output[3*gpos + 2] = j_b;
	}
}


__global__ void MSD_GPU_final_create_LA(float *d_input, float *d_output, float *d_MSD_base, int nTaps, int size) {
	__shared__ float s_input[3*WARP*WARP];

	float M, S, j;

	Sum_partials_regular( &M, &S, &j, d_input, s_input, size);
	
	//----------------------------------------------
	//---- Writing data
	if (threadIdx.x == 0) {
		d_output[0] = d_MSD_base[0];
		d_output[1] = d_MSD_base[1];
		d_output[2] = (sqrt(S / j) - d_MSD_base[1])/( (float) (nTaps-1));
	}
}


__global__ void MSD_GPU_final_create_LA_Nth(float *d_input, float *d_output, float *d_MSD_base, float *d_MSD_DIT, int nTaps, int size, int DIT_value) {
	__shared__ float s_input[3*WARP*WARP];

	float M, S, j;

	Sum_partials_regular( &M, &S, &j, d_input, s_input, size);

	//----------------------------------------------
	//---- Writing data
	if (threadIdx.x == 0) {
		d_output[0] = d_MSD_base[0];
		d_output[1] = d_MSD_base[1];
		d_output[2] = (sqrt(S / j) - d_MSD_base[1])/( (float) nTaps);
		d_output[3] = d_MSD_DIT[0]*DIT_value; 
	}
}


#endif
