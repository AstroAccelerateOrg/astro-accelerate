#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hipfft/hipfft.h>
#include <math.h>
#include <vector>
#include "headers/params.h"

#include "headers/device_periodicity_parameters.h"
#include "headers/device_peak_find.h"
#include "headers/device_MSD_BLN_grid.h"
#include "headers/device_MSD_BLN_pw.h"
#include "headers/device_MSD_limited.h"
#include "headers/device_power.h"
#include "headers/device_harmonic_summing.h"

// define to see debug info
#define GPU_PERIODICITY_SEARCH_DEBUG

// define to reuse old MSD results to generate a new one (it means new MSD is calculated from more samples)
#define PS_REUSE_MSD

// define to use rescaling of the previous results to the final value of MSD. It is useless without PS_REUSE_MSD defined
//#define PS_RESCALE_AND_THRESHOLD_LIST

class Candidate_list {
public:
	float MSD[3];
	std::vector<float> data;
	int range;
	
	void Allocate(int nElements) {
		data.resize(nElements*4);
	}
	
	void Rescale_and_threshold(float *new_MSD, float sigma_cutoff) {
		float SNR;
		std::vector<float> new_data;
		for(unsigned int f=0; (f<(unsigned int) data.size() && data.size()>0) ; f++) {
			SNR = data[4*f+2]*(MSD[1]*sqrt(data[4*f+3])) + MSD[0];
			data[4*f+2] = (SNR-new_MSD[0])/(sqrt(data[4*f+3])*new_MSD[1]);
			if(data[4*f+2]>sigma_cutoff) {
				new_data.push_back(data[4*f]);
				new_data.push_back(data[4*f+1]);
				new_data.push_back(data[4*f+2]);
				new_data.push_back(data[4*f+3]);
			}
		}
		data = new_data;
	}
};

void Export_data_in_range(float *GPU_data, int nTimesamples, int nDMs, int DM_start, int DM_end, const char *filename, float dm_step, float dm_low) {
	std::ofstream FILEOUT;
	
	float *h_temp;
	h_temp = new float[nTimesamples*nDMs];
	hipMemcpy(h_temp, GPU_data, nTimesamples*nDMs*sizeof(float), hipMemcpyDeviceToHost);
	
	FILEOUT.open (filename, std::ofstream::out);
	if(DM_start==DM_end) DM_end++;
	for(int dm=DM_start; dm<DM_end; dm++){
		for(int t=0; t<nTimesamples; t++){
			FILEOUT << t << " " << (dm*dm_step + dm_low) << " " << GPU_data[dm*nTimesamples + t] << std::endl;
		}
		FILEOUT << std::endl;
		FILEOUT << std::endl;
	}
	FILEOUT.close();
	
	delete [] h_temp;
}

void Export_data_in_range(float2 *GPU_data, int nTimesamples, int nDMs, int DM_start, int DM_end, const char *filename, float dm_step, float dm_low) {
	std::ofstream FILEOUT;

	float *h_temp;
	h_temp = new float[nTimesamples*nDMs];
	hipMemcpy(h_temp, GPU_data, nTimesamples*nDMs*sizeof(float), hipMemcpyDeviceToHost);
	
	FILEOUT.open (filename, std::ofstream::out);
	if(DM_start==DM_end) DM_end++;
	for(int dm=DM_start; dm<DM_end; dm++){
		for(int t=0; t<nTimesamples; t++){
			FILEOUT << t << " " << (dm*dm_step + dm_low) << " " << GPU_data[dm*nTimesamples + t].x << " " << GPU_data[dm*nTimesamples + t].y << std::endl;
		}
		FILEOUT << std::endl;
		FILEOUT << std::endl;
	}
	FILEOUT.close();
	
	delete [] h_temp;
}

void Process_and_export_data_to_file(float *data, int size, const char *filename, int nTimesamples, float sampling_time, float dm_step, float dm_low ){
	if(size>0){
		#pragma omp parallel for
		for (int count = 0; count < size; count++){
			data[4*count]     = data[4*count]*dm_step + dm_low;
			data[4*count + 1] = data[4*count + 1]*(1.0/(sampling_time*nTimesamples));
		}
		
		FILE *fp_out;
		if (( fp_out = fopen(filename, "wb") ) == NULL)	{
			fprintf(stderr, "Error opening output file!\n");
			exit(0);
		}
		fwrite(data, size*sizeof(float), 4, fp_out);
		fclose(fp_out);
	}
}

void GPU_periodicity(int range, int nsamp, int max_ndms, int processed, float sigma_cutoff, float ***output_buffer, int *ndms, int *inBin, float *dm_low, float *dm_high, float *dm_step, float tsamp, int nHarmonics) {
	// processed = maximum number of time-samples through out all ranges
	// nTimesamples = number of time-samples in given range 'i'
	// TODO:
	//     ->Be more clever regarding memory allocations for cuFFT use:
	//			const int NX = 1024;
	//			const int BATCH = 100000;
	//			size_t workSize;
	//			hipfftEstimate1d(NX, HIPFFT_C2C, BATCH, &workSize);
	// 	     or
	//			hipfftHandle plan;
	//			hipfftCreate(&plan);
	//			hipfftGetSize1d(plan, NX, HIPFFT_C2C, BATCH, &workSize);
	//      ->Use callbacks for power calculation
	//      ->Solve peak finding problem from batch to batch (we do not want to find peaks on shared borders)
	//      ->Interbinning which is not performed at the moment
	//      ->max_ndms is possibly the same thing as max_nDMs! Investigate.
	//      ->check is zero-th element does not mess up statistics
	//      ->prepare data on the host before copying them to device
	//      ->try to implement rescaling data in the at the end
	
	//------------------ Temporary for debuging
	int export_data=0;
	
	Periodicity_parameters per_param;
	per_param.assign(sigma_cutoff, nHarmonics);

//--------------------------------------------------------------------
//------> Starting Periodicity from scratch
	printf("\n");
	printf("------------ STARTING PERIODICITY SEARCH ------------\n\n");
	per_param.print_parameters();
	
	//---------> Initial stuff
	int nTimesamples, nDMs, max_nDMs, itemp;
	size_t max_nDMs_in_memory, max_nDMs_in_range;
	GpuTimer timer, periodicity_timer;
	double Total_periodicity_time = 0, Total_calc_time = 0, calc_time_per_range = 0, Total_copy_time = 0, copy_time_per_range = 0;
	
	periodicity_timer.Start();
	
	//---------> Finding nearest lower power of two (because of FFT algorithm)
	int nearest = (int) floorf(log2f((float) processed));
	nTimesamples = (int) powf(2.0, nearest);
	printf("Decreasing number of processed samples to nearest lower power of two, because of FFT algorithm...\n");
	printf("Number of processed timesamples: %d; nearest power of two: %d\n", processed, nTimesamples);
	processed = nTimesamples;
	
	// determining maximum number of DM trials in through all ranges
	if(range>0){
		max_nDMs = ndms[0];
		for(int f=0; f<range; f++){
			if(ndms[f]>max_nDMs) max_nDMs=ndms[f];
		}
	}
	printf("maximum number of DM trials through all ranges is %d\n", max_nDMs);
	
	//--------> Determining maximum number of DM trials we can fit into memory
	size_t free_mem,total_mem;
	hipMemGetInfo(&free_mem,&total_mem);
	printf("     Memory available:%0.3f MB \n", ((float) free_mem)/(1024.0*1024.0) );
	max_nDMs_in_memory = (free_mem*0.95)/( (processed+2)*(5.5*sizeof(float) + 2*sizeof(ushort))); // 1 for real input real, 2 for complex output, 2 for complex cuFFT, 1 for peaks + 1 ushort
	if( (max_nDMs+32)<max_nDMs_in_memory) { //if we can fit all DM trials from largest range into memory then we need to find nearest higher multiple of PHS_NTHREADS
		itemp = (int) (max_nDMs/PHS_NTHREADS);
		if( (max_nDMs%PHS_NTHREADS)>0 ) itemp++;
		max_nDMs_in_memory = itemp*PHS_NTHREADS;
	}
	itemp = (int) (max_nDMs_in_memory/PHS_NTHREADS); // if we cannot fit all DM trials from largest range into memory we find nearest lower multiple of PHS_NTHREADS
	max_nDMs_in_memory = itemp*PHS_NTHREADS;
	printf("     Maximum number of DM trials which fit into memory is %d; Input plane size: %0.2f MB;\n", max_nDMs_in_memory, (((float) max_nDMs_in_memory*processed*sizeof(float))/(1024.0*1024.0)));
	
	
	//--------> Allocation of GPU memory. We allocate such amount of memory as to accommodate maximum number of DM trials from all ranges.
	unsigned int input_plane_size = (processed+2)*max_nDMs_in_memory;
	float *d_one_A; //for input and interbinned values
	if ( hipSuccess != hipMalloc((void **) &d_one_A,  sizeof(float)*input_plane_size )) printf("Periodicity Allocation error! d_one_A\n");
	
	float *d_two_B; //for cuFFT complex output and peaks
	if ( hipSuccess != hipMalloc((void **) &d_two_B,  sizeof(float)*2*input_plane_size )) printf("Periodicity Allocation error! d_two_B\n");
	
	float *d_half_C; // for power values
	if ( hipSuccess != hipMalloc((void **) &d_half_C,  sizeof(float)*input_plane_size/2 )) printf("Periodicity Allocation error! d_spectra_Real\n");
	
	ushort *d_power_harmonics, *d_interbin_harmonics;
	if ( hipSuccess != hipMalloc((void **) &d_power_harmonics, sizeof(ushort)*input_plane_size )) printf("Periodicity Allocation error! d_harmonics\n");
	if ( hipSuccess != hipMalloc((void **) &d_interbin_harmonics, sizeof(ushort)*input_plane_size )) printf("Periodicity Allocation error! d_harmonics\n");
	//-----------------------------------------------------------------------------
	hipMemset((void*) d_power_harmonics, 0, sizeof(ushort)*input_plane_size);
	hipMemset((void*) d_interbin_harmonics, 0, sizeof(ushort)*input_plane_size);
	//-----------------------------------------------------------------------------
	
	float *d_dedispersed_data, *d_FFT_complex_output, *d_frequency_power, *d_frequency_interbin, *d_frequency_power_CT, *d_frequency_interbin_CT, *d_power_SNR, *d_interbin_SNR, *d_power_list, *d_interbin_list;
	d_dedispersed_data      = d_one_A;
	d_FFT_complex_output    = d_two_B;
	d_frequency_power       = d_half_C;
	d_frequency_interbin    = d_one_A;
	d_frequency_power_CT    = &d_two_B[0];
	d_frequency_interbin_CT = &d_two_B[input_plane_size];
	d_power_SNR             = d_half_C;
	d_interbin_SNR          = d_one_A;
	d_power_list            = &d_two_B[0];
	d_interbin_list         = &d_two_B[input_plane_size];
	
	int *gmem_power_peak_pos;
	if ( hipSuccess != hipMalloc((void**) &gmem_power_peak_pos, 1*sizeof(int)) )  printf("Periodicity Allocation error! gmem_power_peak_pos\n");
	int *gmem_interbin_peak_pos;
	if ( hipSuccess != hipMalloc((void**) &gmem_interbin_peak_pos, 1*sizeof(int)) )  printf("Periodicity Allocation error! gmem_interbin_peak_pos\n");
	
	float *d_MSD;
	if ( hipSuccess != hipMalloc((void**) &d_MSD, sizeof(float)*4)) {printf("Periodicity Allocation error! d_MSD\n");}
	#ifdef PS_REUSE_MSD
	float *d_previous_partials;
	if ( hipSuccess != hipMalloc((void**) &d_previous_partials, sizeof(float)*4)) {printf("Periodicity Allocation error! d_previous_partials\n");}
	hipMemset((void*) d_previous_partials, 0, 3*sizeof(float));
	#endif
	
	checkCudaErrors(hipGetLastError());
	
	int local_max_list_size = (input_plane_size)/4; //maximum number of peaks per batch
	
	#ifndef PS_RESCALE_AND_THRESHOLD_LIST
	float *h_all_power_peaks, *h_all_interbin_peaks;
	h_all_power_peaks  = (float *)malloc(input_plane_size*sizeof(float));  // this might be too much, but it is very conservative assumption
	h_all_interbin_peaks  = (float *)malloc(input_plane_size*2*sizeof(float));  // this might be too much, but it is very conservative assumption
	size_t max_host_power_peaks = (input_plane_size)/4;
	size_t max_host_interbin_peaks = (input_plane_size*2)/4;
	size_t host_power_peak_pos;
	size_t host_interbin_peak_pos;
	#endif
	int temp_host_power_peak_pos, temp_host_interbin_peak_pos;

	
	for (int i = 0; i < range; i++) {
		calc_time_per_range = 0; copy_time_per_range = 0;
		#ifdef PS_RESCALE_AND_THRESHOLD_LIST
		std::vector<Candidate_list> power_list;
		std::vector<Candidate_list> interbin_list;
		#else
		host_power_peak_pos = 0; host_interbin_peak_pos = 0;
		#endif
		
		nTimesamples = processed/inBin[i];
		nDMs = ndms[i];
		max_nDMs_in_range = max_nDMs_in_memory*inBin[i];
		printf("Processing de-dispersion range:%f--%f:%f; inBin:%d; Timesamples:%d; DM trials:%d; max_nDMs:%d;\n", dm_low[i], dm_high[i], dm_step[i], inBin[i], nTimesamples, nDMs, max_nDMs_in_range);
		
		int nRepeats, nRest, DM_shift, DMs_per_cycle;
		std::vector<int> DM_list;
		
		//---------> Setting up batches
		nRepeats = nDMs/max_nDMs_in_range;
		nRest = nDMs - nRepeats*max_nDMs_in_range;
		for(int f=0; f<nRepeats; f++) DM_list.push_back(max_nDMs_in_range);
		if(nRest>0) DM_list.push_back(nRest);
		
		if(nRepeats>0) printf("     Periodicity search will run %d batches each containing %d DM trials. Remainder %d DM trials\n", (int) DM_list.size(), max_nDMs_in_range, nRest);
		else printf("     Periodicity search will run 1 batch containing %d DM trials.\n", nRest);
		
		if(DM_list.size()>0){
			DM_shift = 0;
			for(int dm=0; dm<DM_list.size(); dm++) {
				#ifdef PS_RESCALE_AND_THRESHOLD_LIST
				Candidate_list local_power_list;
				Candidate_list local_interbin_list;
				#endif
				
				DMs_per_cycle = DM_list[dm];
				printf("\tBatch %d contains %d DM trials.\n",dm,DMs_per_cycle);
				
				hipMemset((void*) gmem_power_peak_pos, 0, sizeof(int));
				hipMemset((void*) gmem_interbin_peak_pos, 0, sizeof(int));
				
				
				//---------> Copy data from the host
				timer.Start();
				for(int ff=0; ff<DMs_per_cycle; ff++){
					checkCudaErrors( hipMemcpy( &d_one_A[ff*nTimesamples], output_buffer[i][DM_shift + ff], nTimesamples*sizeof(float), hipMemcpyHostToDevice));
				}
				timer.Stop();
				copy_time_per_range = copy_time_per_range + timer.Elapsed();
				//---------<
				
				
				//---------> cuFFT
				timer.Start();
				hipfftHandle plan_input;
				hipfftResult cufft_error;
				cufft_error = hipfftPlan1d(&plan_input, nTimesamples, HIPFFT_R2C, DMs_per_cycle);
				if ( cufft_error != HIPFFT_SUCCESS) printf("CUFFT error: %d", cufft_error);
				//cufft_error = hipfftExecR2C(plan_input, (hipfftReal *)d_one_A, (hipfftComplex *)d_two_B);
				cufft_error = hipfftExecR2C(plan_input, (hipfftReal *)d_dedispersed_data, (hipfftComplex *)d_FFT_complex_output);
				if ( cufft_error != HIPFFT_SUCCESS) printf("CUFFT error: %d", cufft_error);
				hipfftDestroy(plan_input);
				timer.Stop();
				printf("     -> cuFFT took %f ms\n", timer.Elapsed());
				calc_time_per_range = calc_time_per_range + timer.Elapsed();
				//---------<
				
				//-----------------------------------------------------------------------------------
				if(i==0 && dm==0 && export_data) Export_data_in_range(d_one_A, nTimesamples, nDMs, 483, 493, "Input_data.dat", dm_step[i], dm_low[i]);
				//-----------------------------------------------------------------------------------
				
				//-----------------------------------------------------------------------------------
				if(i==0 && dm==0 && export_data) Export_data_in_range(d_two_B, ((nTimesamples>>1)+1), nDMs, 483, 493, "FFT_data.dat", dm_step[i], dm_low[i]);
				//-----------------------------------------------------------------------------------
				
				checkCudaErrors(hipGetLastError());
				
				//---------> Calculate powers and interbinning
				timer.Start();
				//simple_power_and_interbin( (float2 *) d_two_B, d_half_C, d_one_A, nTimesamples, DMs_per_cycle);
				simple_power_and_interbin( (float2 *) d_FFT_complex_output, d_frequency_power, d_frequency_interbin, nTimesamples, DMs_per_cycle);
				timer.Stop();
				printf("     -> Calculation of powers and interbining took %f ms\n", timer.Elapsed());
				calc_time_per_range = calc_time_per_range + timer.Elapsed();
				//---------<
				
				//-----------------------------------------------------------------------------------
				if(i==0 && dm==0 && export_data) Export_data_in_range(d_half_C, nTimesamples/2, nDMs, 483, 493, "power_data.dat", dm_step[i], dm_low[i]);
				//-----------------------------------------------------------------------------------
				
				//-----------------------------------------------------------------------------------
				if(i==0 && dm==0 && export_data) Export_data_in_range(d_one_A, nTimesamples, nDMs, 483, 493, "Interbin_data.dat", dm_step[i], dm_low[i]);
				//-----------------------------------------------------------------------------------
				
				checkCudaErrors(hipGetLastError());
				
				//---------> Mean and StDev on powers
				timer.Start();
				#ifdef PS_REUSE_MSD
				MSD_limited_continuous(d_frequency_power, d_MSD, d_previous_partials, DMs_per_cycle, (nTimesamples>>1), 0);
				#else
				//MSD_limited(d_half_C, d_MSD, DMs_per_cycle, (nTimesamples>>1), 0);
				MSD_limited(d_frequency_power, d_MSD, DMs_per_cycle, (nTimesamples>>1), 0);
				#endif
				timer.Stop();
				printf("     -> MSD took %f ms\n", timer.Elapsed());
				calc_time_per_range = calc_time_per_range + timer.Elapsed();
				//---------<
				
				
				checkCudaErrors(hipGetLastError());
				
				//---------> Corner turn
				timer.Start();
				//corner_turn_SM(d_half_C, &d_two_B[0], (nTimesamples>>1), DMs_per_cycle);
				//corner_turn_SM(d_one_A, &d_two_B[input_plane_size], nTimesamples, DMs_per_cycle);
				corner_turn_SM(d_frequency_power, d_frequency_power_CT, (nTimesamples>>1), DMs_per_cycle);
				corner_turn_SM(d_frequency_interbin, d_frequency_interbin_CT, nTimesamples, DMs_per_cycle);
				timer.Stop();
				printf("     -> corner turn took %f ms\n", timer.Elapsed());
				calc_time_per_range = calc_time_per_range + timer.Elapsed();
				//---------<
				
				checkCudaErrors(hipGetLastError());
				
				//---------> Harmonic summing
				timer.Start();
				//periodicity_simple_harmonic_summing(&d_two_B[0], d_half_C, d_power_harmonics, d_MSD, (nTimesamples>>1), DMs_per_cycle, per_param.nHarmonics);
				//periodicity_simple_harmonic_summing(&d_two_B[input_plane_size], d_one_A, d_interbin_harmonics, d_MSD, nTimesamples, DMs_per_cycle, per_param.nHarmonics);
				periodicity_simple_harmonic_summing(d_frequency_power_CT, d_power_SNR, d_power_harmonics, d_MSD, (nTimesamples>>1), DMs_per_cycle, per_param.nHarmonics);
				periodicity_simple_harmonic_summing(d_frequency_interbin_CT, d_interbin_SNR, d_interbin_harmonics, d_MSD, nTimesamples, DMs_per_cycle, per_param.nHarmonics);
				timer.Stop();
				printf("     -> harmonic summing took %f ms\n", timer.Elapsed());
				calc_time_per_range = calc_time_per_range + timer.Elapsed();
				//---------<
				
				checkCudaErrors(hipGetLastError());
				
				
				
				//---------> Peak finding
				timer.Start();
				//Peak_find_for_periodicity_search(d_half_C, d_power_harmonics, &d_two_B[0], (nTimesamples>>1), DMs_per_cycle, per_param.sigma_cutoff, local_max_list_size, gmem_power_peak_pos, DM_shift);
				//Peak_find_for_periodicity_search(d_one_A, d_interbin_harmonics, &d_two_B[input_plane_size], nTimesamples, DMs_per_cycle, per_param.sigma_cutoff, local_max_list_size, gmem_interbin_peak_pos, DM_shift);				
				Peak_find_for_periodicity_search(d_power_SNR, d_power_harmonics, d_power_list, (nTimesamples>>1), DMs_per_cycle, per_param.sigma_cutoff, local_max_list_size, gmem_power_peak_pos, DM_shift);
				Peak_find_for_periodicity_search(d_interbin_SNR, d_interbin_harmonics, d_interbin_list, nTimesamples, DMs_per_cycle, per_param.sigma_cutoff, local_max_list_size, gmem_interbin_peak_pos, DM_shift);
				/*
				SNR_limited(d_half_C, &d_two_B[0], d_power_harmonics, d_MSD, 1, DMs_per_cycle, (nTimesamples>>1), 0);
				SNR_limited(d_one_A, &d_two_B[input_plane_size], d_interbin_harmonics, d_MSD, 1, DMs_per_cycle, nTimesamples, 0);
				
				checkCudaErrors(hipGetLastError());
				
				//-----------------------------------------------------------------------------------
				if(i==0 && dm==0 && export_data) Export_data_in_range(&d_two_B[0], nTimesamples/2, nDMs, 483, 493, "power_SNR_data.dat", dm_step[i], dm_low[i]);
				//-----------------------------------------------------------------------------------
				
				//-----------------------------------------------------------------------------------
				if(i==0 && dm==0 && export_data) Export_data_in_range(&d_two_B[input_plane_size], nTimesamples, nDMs, 483, 493, "Interbin_SNR_data.dat", dm_step[i], dm_low[i]);
				//-----------------------------------------------------------------------------------
				
				Peak_find_for_periodicity_search(&d_two_B[0], d_power_harmonics, d_half_C, DMs_per_cycle, (nTimesamples>>1), per_param.sigma_cutoff, local_max_list_size, gmem_power_peak_pos, DM_shift);
				Peak_find_for_periodicity_search(&d_two_B[input_plane_size], d_interbin_harmonics, d_one_A, DMs_per_cycle, nTimesamples, per_param.sigma_cutoff, local_max_list_size, gmem_interbin_peak_pos, DM_shift);
				*/
				timer.Stop();
				printf("     -> Peak finding took %f ms\n", timer.Elapsed());
				calc_time_per_range = calc_time_per_range + timer.Elapsed();
				//---------<
				
				checkCudaErrors(hipGetLastError());
				
				//---------> Transferring peaks to the host
				timer.Start();
				
				checkCudaErrors(hipMemcpy(&temp_host_power_peak_pos, gmem_power_peak_pos, sizeof(int), hipMemcpyDeviceToHost));
				#ifdef GPU_PERIODICITY_SEARCH_DEBUG
				printf("     -> POWER: Total number of peaks found in this range is %d; maximum number of peaks:%d;\n", temp_host_power_peak_pos, local_max_list_size);
				#endif
				
				#ifdef PS_RESCALE_AND_THRESHOLD_LIST
				local_power_list.Allocate(temp_host_power_peak_pos);
				local_power_list.range = i;
				checkCudaErrors(hipMemcpy(local_power_list.data, d_power_list, temp_host_power_peak_pos*4*sizeof(float), hipMemcpyDeviceToHost));
				checkCudaErrors(hipMemcpy(local_power_list.MSD, d_MSD, 3*sizeof(float), hipMemcpyDeviceToHost));
				power_list.push_back(local_power_list);
				#else
				if( (host_power_peak_pos + temp_host_power_peak_pos)<max_host_power_peaks){
					//checkCudaErrors(hipMemcpy(&h_all_power_peaks[host_power_peak_pos*4], d_half_C, temp_host_power_peak_pos*4*sizeof(float), hipMemcpyDeviceToHost));
					//checkCudaErrors(hipMemcpy(&h_all_power_peaks[host_power_peak_pos*4], &d_two_B[0], temp_host_power_peak_pos*4*sizeof(float), hipMemcpyDeviceToHost));
					checkCudaErrors(hipMemcpy(&h_all_power_peaks[host_power_peak_pos*4], d_power_list, temp_host_power_peak_pos*4*sizeof(float), hipMemcpyDeviceToHost));
					host_power_peak_pos = host_power_peak_pos + temp_host_power_peak_pos;
				}
				else printf("     ->      Maximum list size reached! Increase list size or increase sigma cutoff.\n");
				#endif
				
				checkCudaErrors(hipMemcpy(&temp_host_interbin_peak_pos, gmem_interbin_peak_pos, sizeof(int), hipMemcpyDeviceToHost));
				#ifdef GPU_PERIODICITY_SEARCH_DEBUG
				printf("     -> INTERBIN: Total number of peaks found in this range is %d; maximum number of peaks:%d;\n", temp_host_interbin_peak_pos, local_max_list_size);
				#endif
				
				#ifdef PS_RESCALE_AND_THRESHOLD_LIST
				local_interbin_list.Allocate(temp_host_power_peak_pos);
				local_interbin_list.range = i;
				checkCudaErrors(hipMemcpy(local_interbin_list.data, d_power_list, temp_host_power_peak_pos*4*sizeof(float), hipMemcpyDeviceToHost));
				checkCudaErrors(hipMemcpy(local_interbin_list.MSD, d_MSD, 3*sizeof(float), hipMemcpyDeviceToHost));
				interbin_list.push_back(local_interbin_list);
				#else
				if( (host_interbin_peak_pos + temp_host_interbin_peak_pos)<max_host_interbin_peaks){
					//checkCudaErrors(hipMemcpy(&h_all_interbin_peaks[host_interbin_peak_pos*4], d_one_A, temp_host_interbin_peak_pos*4*sizeof(float), hipMemcpyDeviceToHost));
					//checkCudaErrors(hipMemcpy(&h_all_interbin_peaks[host_interbin_peak_pos*4], &d_two_B[input_plane_size], temp_host_interbin_peak_pos*4*sizeof(float), hipMemcpyDeviceToHost));
					checkCudaErrors(hipMemcpy(&h_all_interbin_peaks[host_interbin_peak_pos*4], d_interbin_list, temp_host_interbin_peak_pos*4*sizeof(float), hipMemcpyDeviceToHost));
					host_interbin_peak_pos = host_interbin_peak_pos + temp_host_interbin_peak_pos;
				}
				else printf("     ->      Maximum list size reached! Increase list size or increase sigma cutoff.\n");
				#endif
				
				timer.Stop();
				copy_time_per_range = copy_time_per_range + timer.Elapsed();
				//---------<
				
				DM_shift = DM_shift + DMs_per_cycle;
			} // end of for through batches
			
			//---------> Peak processing on host and export
			char filename[200];

			sprintf(filename, "fourier-dm_%.2f-%.2f.dat", dm_low[i], dm_high[i]);
			Process_and_export_data_to_file(h_all_power_peaks, host_power_peak_pos, filename, nTimesamples, tsamp, dm_step[i], dm_low[i]);
			
			sprintf(filename, "fourier_inter-dm_%.2f-%.2f.dat", dm_low[i], dm_high[i]);
			Process_and_export_data_to_file(h_all_interbin_peaks, host_interbin_peak_pos, filename, nTimesamples, tsamp, dm_step[i], dm_low[i]);
			//---------<
		}
		
		printf("     -----------------------\n");
		printf("     -> This range calculation time: %f ms\n", calc_time_per_range);
		printf("     -> This range copy time:        %f ms\n", copy_time_per_range);
		printf("\n");
		Total_calc_time = Total_calc_time + calc_time_per_range;
		calc_time_per_range = 0;
		Total_copy_time = Total_copy_time + copy_time_per_range;
		copy_time_per_range = 0;
		
		#ifdef PS_RESCALE_AND_THRESHOLD_LIST
		power_list.clear();
		interbin_list.clear();
		#endif
	}

	periodicity_timer.Stop();
	Total_periodicity_time = periodicity_timer.Elapsed();
	
	printf("\nTimer:\n");
	printf("Total calculation time: %f ms\n", Total_calc_time);
	printf("Total copy time:        %f ms\n", Total_copy_time);
	printf("Total periodicity time: %f ms\n", Total_periodicity_time);

	hipDeviceSynchronize();
	
	hipFree(d_MSD);
	#ifdef PS_REUSE_MSD
	hipFree(d_previous_partials);
	#endif
	hipFree(d_one_A);
	hipFree(d_two_B);
	hipFree(d_half_C);
	hipFree(d_power_harmonics);
	hipFree(d_interbin_harmonics);
	hipFree(gmem_power_peak_pos);
	hipFree(gmem_interbin_peak_pos);
	
	#ifdef PS_RESCALE_AND_THRESHOLD_LIST
	power_list.clear();
	interbin_list.clear();
	#else
	free(h_all_power_peaks);
	free(h_all_interbin_peaks);
	#endif
	

	
	
	
	
	
	
	
}


