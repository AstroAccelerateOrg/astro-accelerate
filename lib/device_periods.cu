#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hipfft/hipfft.h>
#include <math.h>
#include "headers/params.h"

#include "headers/device_peak_find.h"
#include "headers/device_MSD_BLN_grid.h"
#include "headers/device_MSD_BLN_pw.h"
#include "headers/device_MSD_limited.h"
#include "headers/device_power.h"
#include "headers/device_harmonic_summing.h"

// define to see debug info
#define GPU_PERIODICITY_SEARCH_DEBUG

void GPU_periodicity(int range, int nsamp, int max_ndms, int processed, float cutoff, float ***output_buffer, int *ndms, int *inBin, float *dm_low, float *dm_high, float *dm_step, float tsamp, int nHarmonics) {
	// processed = maximum number of time-samples through out all ranges
	// nTimesamples = number of time-samples in given range 'i'
	// TODO:
	//     ->Be more clever regarding memory allocations for cuFFT use:
	//			const int NX = 1024;
	//			const int BATCH = 100000;
	//			size_t workSize;
	//			hipfftEstimate1d(NX, HIPFFT_C2C, BATCH, &workSize);
	// 	     or
	//			hipfftHandle plan;
	//			hipfftCreate(&plan);
	//			hipfftGetSize1d(plan, NX, HIPFFT_C2C, BATCH, &workSize);
	//      ->Use callbacks for power calculation
	//      ->Solve peak finding problem from batch to batch (we do not want to find peaks on shared borders)
	//      ->Interbinning which is not performed at the moment
	//      ->max_ndms is possibly the same thing as max_nDMs! Investigate.

//--------------------------------------------------------------------
//------> Starting Periodicity from scratch
	printf("\n");
	printf("------------ STARTING PERIODICITY SEARCH ------------\n\n");
	
	//---------> Initial stuff
	int nTimesamples, nDMs, max_nDMs, itemp;
	size_t max_nDMs_in_memory, max_nDMs_in_range;
	GpuTimer timer, periodicity_timer;
	double Total_periodicity_time = 0, Total_calc_time = 0, calc_time_per_range = 0, Total_copy_time = 0, copy_time_per_range = 0;
	
	periodicity_timer.Start();
	
	//---------> Finding nearest lower power of two (because of FFT algorithm)
	int nearest = (int) floorf(log2f((float) processed));
	nTimesamples = (int) powf(2.0, nearest);
	printf("Decreasing number of processed samples to nearest lower power of two, because of FFT algorithm...\n");
	printf("Number of processed timesamples: %d; nearest power of two: %d\n", processed, nTimesamples);
	processed = nTimesamples;
	
	// determining maximum number of DM trials in through all ranges
	if(range>0){
		max_nDMs = ndms[0];
		for(int f=0; f<range; f++){
			if(ndms[f]>max_nDMs) max_nDMs=ndms[f];
		}
	}
	printf("maximum number of DM trials through all ranges is %d\n", max_nDMs);
	
	//--------> Determining maximum number of DM trials we can fit into memory
	size_t free_mem,total_mem;
	hipMemGetInfo(&free_mem,&total_mem);
	printf("     Memory available:%0.3f MB \n", ((float) free_mem)/(1024.0*1024.0) );
	max_nDMs_in_memory = (free_mem*0.95)/(processed*(5.5*sizeof(float) + 2*sizeof(ushort))); // 1 for real input real, 2 for complex output, 2 for complex cuFFT, 1 for peaks + 1 ushort
	if( (max_nDMs+32)<max_nDMs_in_memory) { //if we can fit all DM trials from largest range into memory then we need to find nearest higher multiple of PHS_NTHREADS
		itemp = (int) (max_nDMs/PHS_NTHREADS);
		if( (max_nDMs%PHS_NTHREADS)>0 ) itemp++;
		max_nDMs_in_memory = itemp*PHS_NTHREADS;
	}
	itemp = (int) (max_nDMs_in_memory/PHS_NTHREADS); // if we cannot fit all DM trials from largest range into memory we find nearest lower multiple of PHS_NTHREADS
	max_nDMs_in_memory = itemp*PHS_NTHREADS;
	printf("     Maximum number of DM trials which fit into memory is %d; Input plane size: %0.2f MB;\n", max_nDMs_in_memory, (((float) max_nDMs_in_memory*processed*sizeof(float))/(1024.0*1024.0)));
	
	
	//--------> Allocation of GPU memory. We allocate such amount of memory as to accommodate maximum number of DM trials from all ranges.
	unsigned int input_plane_size = (processed)*max_nDMs_in_memory;
	float *d_one_A; //for input and interbinned values
	if ( hipSuccess != hipMalloc((void **) &d_one_A,  sizeof(float)*input_plane_size )) printf("Periodicity Allocation error! d_one_A\n");
	
	float *d_two_B; //for cuFFT complex output and peaks
	if ( hipSuccess != hipMalloc((void **) &d_two_B,  sizeof(float)*2*input_plane_size )) printf("Periodicity Allocation error! d_two_B\n");
	
	float *d_half_C; // for power values
	if ( hipSuccess != hipMalloc((void **) &d_half_C,  sizeof(float)*input_plane_size/2 )) printf("Periodicity Allocation error! d_spectra_Real\n");
	
	ushort *d_power_harmonics, *d_interbin_harmonics;
	if ( hipSuccess != hipMalloc((void **) &d_power_harmonics, sizeof(ushort)*input_plane_size )) printf("Periodicity Allocation error! d_harmonics\n");
	if ( hipSuccess != hipMalloc((void **) &d_interbin_harmonics, sizeof(ushort)*input_plane_size )) printf("Periodicity Allocation error! d_harmonics\n");
	
	float *d_dedispersed_data, *d_FFT_complex_output, *d_frequency_power, *d_frequency_interbin, *d_frequency_power_CT, *d_frequency_interbin_CT, *d_power_SNR, *d_interbin_SNR, *d_power_list, *d_interbin_list;
	d_dedispersed_data      = d_one_A;
	d_FFT_complex_output    = d_two_B;
	d_frequency_power       = d_half_C;
	d_frequency_interbin    = d_one_A;
	d_frequency_power_CT    = &d_two_B[0];
	d_frequency_interbin_CT = &d_two_B[input_plane_size];
	d_power_SNR             = d_half_C;
	d_interbin_SNR          = d_one_A;
	d_power_list            = &d_two_B[0];
	d_interbin_list         = &d_two_B[input_plane_size];
	
	int *gmem_power_peak_pos;
	if ( hipSuccess != hipMalloc((void**) &gmem_power_peak_pos, 1*sizeof(int)) )  printf("Periodicity Allocation error! gmem_power_peak_pos\n");
	int *gmem_interbin_peak_pos;
	if ( hipSuccess != hipMalloc((void**) &gmem_interbin_peak_pos, 1*sizeof(int)) )  printf("Periodicity Allocation error! gmem_interbin_peak_pos\n");
	
	float *d_MSD;
	if ( hipSuccess != hipMalloc((void**) &d_MSD, sizeof(float)*4)) {printf("Periodicity Allocation error! d_MSD\n");}
	
	checkCudaErrors(hipGetLastError());
	
	int local_max_list_size = (input_plane_size)/4; //maximum number of peaks per batch
	
	float *h_all_power_peaks, *h_all_interbin_peaks;
	h_all_power_peaks  = (float *)malloc(input_plane_size*sizeof(float));  // this might be too much, but it is very conservative assumption
	h_all_interbin_peaks  = (float *)malloc(input_plane_size*2*sizeof(float));  // this might be too much, but it is very conservative assumption
	size_t max_host_power_peaks = (input_plane_size)/4;
	size_t max_host_interbin_peaks = (input_plane_size*2)/4;
	size_t host_power_peak_pos;
	size_t host_interbin_peak_pos;
	int temp_host_power_peak_pos, temp_host_interbin_peak_pos;

	
	for (int i = 0; i < range; i++) {
		calc_time_per_range = 0; copy_time_per_range = 0;
		host_power_peak_pos = 0; host_interbin_peak_pos = 0;
		
		nTimesamples = processed/inBin[i];
		nDMs = ndms[i];
		max_nDMs_in_range = max_nDMs_in_memory*inBin[i];
		printf("Processing de-dispersion range:%f--%f:%f; inBin:%d; Timesamples:%d; DM trials:%d; max_nDMs:%d;\n", dm_low[i], dm_high[i], dm_step[i], inBin[i], nTimesamples, nDMs, max_nDMs_in_range);
		
		int nRepeats, nRest, DM_shift, DMs_per_cycle;
		std::vector<int> DM_list;
		
		//---------> Setting up batches
		nRepeats = nDMs/max_nDMs_in_range;
		nRest = nDMs - nRepeats*max_nDMs_in_range;
		for(int f=0; f<nRepeats; f++) DM_list.push_back(max_nDMs_in_range);
		if(nRest>0) DM_list.push_back(nRest);
		
		printf("     Periodicity search will run %d batches each containing %d DM trials. Remainder %d DM trials\n", (int) DM_list.size(), max_nDMs_in_range, nRest);
		
		if(DM_list.size()>0){
			DM_shift = 0;
			for(int dm=0; dm<DM_list.size(); dm++) {
				DMs_per_cycle = DM_list[dm];
				printf("\tBatch %d contains %d DM trials.\n",dm,DMs_per_cycle);
				
				hipMemset((void*) gmem_power_peak_pos, 0, sizeof(int));
				hipMemset((void*) gmem_interbin_peak_pos, 0, sizeof(int));
				
				//---------> Copy data from the host
				timer.Start();
				checkCudaErrors( hipMemcpy( d_one_A, output_buffer[i][DM_shift], nTimesamples*DMs_per_cycle*sizeof(float), hipMemcpyHostToDevice));
				timer.Stop();
				copy_time_per_range = copy_time_per_range + timer.Elapsed();
				//---------<

				//---------> cuFFT
				timer.Start();
				hipfftHandle plan_input;
				hipfftResult cufft_error;
				cufft_error = hipfftPlan1d(&plan_input, nTimesamples, HIPFFT_R2C, DMs_per_cycle);
				if ( cufft_error != HIPFFT_SUCCESS) printf("CUFFT error: %d", cufft_error);
				//hipfftExecR2C(plan_input, (hipfftReal *)d_one_A, (hipfftComplex *)d_two_B);
				hipfftExecR2C(plan_input, (hipfftReal *)d_dedispersed_data, (hipfftComplex *)d_FFT_complex_output);
				hipfftDestroy(plan_input);
				timer.Stop();
				printf("     -> cuFFT took %f ms\n", timer.Elapsed());
				calc_time_per_range = calc_time_per_range + timer.Elapsed();
				//---------<
				
				checkCudaErrors(hipGetLastError());
				
				//---------> Calculate powers and interbinning
				timer.Start();
				//simple_power_and_interbin( (float2 *) d_two_B, d_half_C, d_one_A, nTimesamples, DMs_per_cycle);
				simple_power_and_interbin( (float2 *) d_FFT_complex_output, d_frequency_power, d_frequency_interbin, nTimesamples, DMs_per_cycle);
				timer.Stop();
				printf("     -> Calculation of powers and interbining took %f ms\n", timer.Elapsed());
				calc_time_per_range = calc_time_per_range + timer.Elapsed();
				//---------<
				
				checkCudaErrors(hipGetLastError());
				
				//---------> Mean and StDev on powers
				timer.Start();
				MSD_limited(d_frequency_power, d_MSD, DMs_per_cycle, nTimesamples/2, 0);
				//MSD_BLN_pw(d_half_C, d_MSD, DMs_per_cycle, nTimesamples/2, 0, sigma_constant);
				timer.Stop();
				printf("     -> baselevel MSD took %f ms\n", timer.Elapsed());
				calc_time_per_range = calc_time_per_range + timer.Elapsed();
				//---------<
				
				checkCudaErrors(hipGetLastError());
				
				//---------> Corner turn
				timer.Start();
				//corner_turn_SM(d_half_C, &d_two_B[0], nTimesamples, DMs_per_cycle);
				//corner_turn_SM(d_one_A, &d_two_B[input_plane_size], nTimesamples, DMs_per_cycle);
				corner_turn_SM(d_frequency_power, d_frequency_power_CT, nTimesamples, DMs_per_cycle);
				corner_turn_SM(d_frequency_interbin, d_frequency_interbin_CT, nTimesamples, DMs_per_cycle);
				//corner_turn(d_frequency_power, d_frequency_power_CT, nTimesamples, DMs_per_cycle);
				//corner_turn(d_frequency_interbin, d_frequency_interbin_CT, nTimesamples, DMs_per_cycle);
				timer.Stop();
				printf("     -> corner turn took %f ms\n", timer.Elapsed());
				calc_time_per_range = calc_time_per_range + timer.Elapsed();
				//---------<
				
				checkCudaErrors(hipGetLastError());
				
				//---------> Harmonic summing
				timer.Start();
				//periodicity_simple_harmonic_summing(&d_two_B[0], d_half_C, d_power_harmonics, d_MSD, nTimesamples/2, DMs_per_cycle, nHarmonics);
				//periodicity_simple_harmonic_summing(&d_two_B[input_plane_size], d_one_A, d_interbin_harmonics, d_MSD, nTimesamples, DMs_per_cycle, nHarmonics);
				periodicity_simple_harmonic_summing(d_frequency_power_CT, d_power_SNR, d_power_harmonics, d_MSD, nTimesamples/2, DMs_per_cycle, nHarmonics);
				periodicity_simple_harmonic_summing(d_frequency_interbin_CT, d_interbin_SNR, d_interbin_harmonics, d_MSD, nTimesamples, DMs_per_cycle, nHarmonics);
				timer.Stop();
				printf("     -> harmonic summing took %f ms\n", timer.Elapsed());
				calc_time_per_range = calc_time_per_range + timer.Elapsed();
				//---------<
				
				checkCudaErrors(hipGetLastError());
				
				//---------> Peak finding
				timer.Start();
				// BEWARE: Here I have reversed nDMs<->nTimesamples because we have transposed data. Since this peak finding algorithm works in 2D it should not matter. However it means that in output list DM trial in column 0 will be in column for time samples (column 1) and wise versa
				//Peak_find_for_periodicity_search(d_half_C, d_power_harmonics, &d_two_B[0], nTimesamples/2, DMs_per_cycle, cutoff, local_max_list_size, gmem_power_peak_pos, DM_shift);
				//Peak_find_for_periodicity_search(d_one_A, d_interbin_harmonics, &d_two_B[input_plane_size], nTimesamples, DMs_per_cycle, cutoff, local_max_list_size, gmem_interbin_peak_pos, DM_shift);				
				
				Peak_find_for_periodicity_search(d_power_SNR, d_power_harmonics, d_power_list, nTimesamples/2, DMs_per_cycle, cutoff, local_max_list_size, gmem_power_peak_pos, DM_shift);
				Peak_find_for_periodicity_search(d_interbin_SNR, d_interbin_harmonics, d_interbin_list, nTimesamples, DMs_per_cycle, cutoff, local_max_list_size, gmem_interbin_peak_pos, DM_shift);
				timer.Stop();
				printf("     -> Peak finding took %f ms\n", timer.Elapsed());
				calc_time_per_range = calc_time_per_range + timer.Elapsed();
				//---------<
				
				checkCudaErrors(hipGetLastError());
				
				//---------> Transferring peaks to the host
				timer.Start();
				
				checkCudaErrors(hipMemcpy(&temp_host_power_peak_pos, gmem_power_peak_pos, sizeof(int), hipMemcpyDeviceToHost));
				#ifdef GPU_PERIODICITY_SEARCH_DEBUG
				printf("POWER: Total number of peaks found in this range is %d; maximum number of peaks:%d;\n", temp_host_power_peak_pos, local_max_list_size);
				#endif
				if( (host_power_peak_pos + temp_host_power_peak_pos)<max_host_power_peaks){
					checkCudaErrors(hipMemcpy(&h_all_power_peaks[host_power_peak_pos*4], d_power_list, temp_host_power_peak_pos*4*sizeof(float), hipMemcpyDeviceToHost));
					host_power_peak_pos = host_power_peak_pos + temp_host_power_peak_pos;
				}
				else printf("     Maximum list size reached! Increase list size or increase sigma cutoff.\n");
				
				checkCudaErrors(hipMemcpy(&temp_host_interbin_peak_pos, gmem_interbin_peak_pos, sizeof(int), hipMemcpyDeviceToHost));
				#ifdef GPU_PERIODICITY_SEARCH_DEBUG
				printf("INTERBIN: Total number of peaks found in this range is %d; maximum number of peaks:%d;\n", temp_host_interbin_peak_pos, local_max_list_size);
				#endif
				if( (host_interbin_peak_pos + temp_host_interbin_peak_pos)<max_host_interbin_peaks){
					checkCudaErrors(hipMemcpy(&h_all_interbin_peaks[host_interbin_peak_pos*4], d_interbin_list, temp_host_interbin_peak_pos*4*sizeof(float), hipMemcpyDeviceToHost));
					host_interbin_peak_pos = host_interbin_peak_pos + temp_host_interbin_peak_pos;
				}
				else printf("     Maximum list size reached! Increase list size or increase sigma cutoff.\n");
				
				timer.Stop();
				copy_time_per_range = copy_time_per_range + timer.Elapsed();
				//---------<
				
				DM_shift = DM_shift + DMs_per_cycle;
			} // end of for through batches
			
			//---------> Peak processing on host and export
			#pragma omp parallel for
			for (int count = 0; count < host_power_peak_pos; count++){
				// Since columns for DM trial and time samples should be reversed I'm reversing them here to counter that and to keep output list consistent with other outputs 
				h_all_power_peaks[4*count]     = h_all_power_peaks[4*count]*dm_step[i] + dm_low[i];
				h_all_power_peaks[4*count + 1] = h_all_power_peaks[4*count + 1]*(1.0/(tsamp*nTimesamples));
			}
			
			#pragma omp parallel for
			for (int count = 0; count < host_interbin_peak_pos; count++){
				// Since columns for DM trial and time samples should be reversed I'm reversing them here to counter that and to keep output list consistent with other outputs
				h_all_interbin_peaks[4*count]     = h_all_interbin_peaks[4*count]*dm_step[i] + dm_low[i];
				h_all_interbin_peaks[4*count + 1] = h_all_interbin_peaks[4*count + 1]*(1.0/(tsamp*nTimesamples));
			}
			
			FILE *fp_out;
			char filename[200];
			
			if(host_power_peak_pos>0){
				sprintf(filename, "fourier-dm_%.2f-%.2f.dat", dm_low[i], dm_high[i]);
				if (( fp_out = fopen(filename, "wb") ) == NULL)	{
					fprintf(stderr, "Error opening output file!\n");
					exit(0);
				}
				fwrite(h_all_power_peaks, host_power_peak_pos*sizeof(float), 4, fp_out);
				fclose(fp_out);
			}
			
			if(host_interbin_peak_pos>0){
				sprintf(filename, "fourier_inter-dm_%.2f-%.2f.dat", dm_low[i], dm_high[i]);
				if (( fp_out = fopen(filename, "wb") ) == NULL)	{
					fprintf(stderr, "Error opening output file!\n");
					exit(0);
				}
				fwrite(h_all_interbin_peaks, host_interbin_peak_pos*sizeof(float), 4, fp_out);
				fclose(fp_out);
			}
			//---------<
		}
		
		printf("This range calculation time: %f ms\n", calc_time_per_range);
		printf("This range copy time:        %f ms\n", copy_time_per_range);
		Total_calc_time = Total_calc_time + calc_time_per_range;
		calc_time_per_range = 0;
		Total_copy_time = Total_copy_time + copy_time_per_range;
		copy_time_per_range = 0;
	}

	periodicity_timer.Stop();
	Total_periodicity_time = periodicity_timer.Elapsed();
	
	printf("Timer:\n");
	printf("Total calculation time: %f ms\n", Total_calc_time);
	printf("Total copy time:        %f ms\n", Total_copy_time);
	printf("Total periodicity time: %f ms\n", Total_periodicity_time);

	hipDeviceSynchronize();
	
	hipFree(d_MSD);
	hipFree(d_one_A);
	hipFree(d_two_B);
	hipFree(d_half_C);
	hipFree(d_power_harmonics);
	hipFree(d_interbin_harmonics);
}


