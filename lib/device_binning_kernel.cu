#include "hip/hip_runtime.h"
#ifndef BIN_KERNEL_H_
#define BIN_KERNEL_H_

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "headers/params.h"

#define BINARRAYSIZE 2 * BINDIVINT * BINDIVINF

__shared__ float f_line_bin[BINARRAYSIZE];

//{{{ bin
/*
 __global__ void bin(float *bin_buffer, float *input_buffer, int in_nsamp) {

 int	idx = (threadIdx.x + (threadIdx.y * BINDIVINT));
 int	c = ((blockIdx.y * BINDIVINF) + threadIdx.y);

 f_line_bin[idx] = input_buffer[(c*in_nsamp)+(blockIdx.x * BINDIVINT*2)+idx];
 f_line_bin[idx+(BINDIVINT*BINDIVINF)] = input_buffer[(c*in_nsamp)+(blockIdx.x * BINDIVINT*2)+idx+(BINDIVINT*BINDIVINF)];
 __syncthreads();

 int	out_nsamp = in_nsamp / 2;
 int	t_out =  ( (blockIdx.x * BINDIVINT) + threadIdx.x);

 int	shift_one = ((c*out_nsamp) + t_out);
 int	shift_three = (2*threadIdx.x);

 bin_buffer[(shift_one)] = (f_line_bin[(shift_three)] + f_line_bin[shift_three + 1])/2;
 }
 */

__global__ void bin(unsigned short *d_input, float *d_output, int in_nsamp)
{

	int c = ( ( blockIdx.y * BINDIVINF ) + threadIdx.y );
	int out_nsamp = ( in_nsamp ) / 2;
	int t_out = ( ( blockIdx.x * BINDIVINT ) + threadIdx.x );
	int t_in = 2 * t_out;

	int shift_one = ( ( c * out_nsamp ) + t_out );
	int shift_two = ( ( c * in_nsamp ) + t_in );

	d_output[( shift_one )] = (float) ( ( d_input[( shift_two )] + d_input[shift_two + 1] )/2.0f );
//	if ((c+t_out) ==0) {
//		for (int k = 0; k < 10; k++)
//			printf("\n\n\t\tp: %p in_nsamp: %i d_input: %hu d_output: %f", d_input, in_nsamp, d_input[k], d_output[k]);
//	}

}


__global__ void DiT_GPU_v2(float const* __restrict__ d_input, float *d_output, unsigned int nDMs, unsigned int nTimesamples, unsigned int dts) {
	float2 ftemp2;
	unsigned int posx, posy, itemp;
	
	posy = (blockIdx.y*DIT_YSTEP + threadIdx.y);
	posx = (blockIdx.x*DIT_ELEMENTS_PER_THREAD*blockDim.x);
	
	//#pragma unroll
	for(int f=0; f<DIT_ELEMENTS_PER_THREAD; f++){
		itemp = (posx + threadIdx.x + f*blockDim.x);
		if( (2*itemp+1)<nTimesamples ){
			ftemp2.x = d_input[posy*nTimesamples + 2*itemp];
			ftemp2.y = d_input[posy*nTimesamples + 2*itemp+1];
			d_output[posy*dts + itemp] = ftemp2.x + ftemp2.y;
		}
	}
}


#endif

//}}}
