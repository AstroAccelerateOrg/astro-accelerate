#include "hip/hip_runtime.h"
// Added by Karel Adamek 

#ifndef SNR_LIMITED_KERNEL_H_
#define SNR_LIMITED_KERNEL_H_

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "headers/params.h"

__global__ void SNR_GPU_limited(float *d_FIR_input, float *d_SNR_output, float *d_SNR_taps, float *d_MSD, int x_steps, int nTaps, int nColumns, int offset)
{
	int local_id = threadIdx.x & ( WARP - 1 );
	int warp_id = threadIdx.x >> 5;
	int dim_y = blockDim.x >> 5;

	int pos_x, pos_y;
	float old_SNR, new_SNR;

	float signal_mean = d_MSD[0];
	float signal_sd = d_MSD[1];

	pos_y = ( blockIdx.y * dim_y + warp_id ) * nColumns;
	pos_x = blockIdx.x * WARP * x_steps + local_id;

	for (int xf = 0; xf < x_steps; xf++)
	{
		if (pos_x < ( nColumns - offset ))
		{
			old_SNR = d_SNR_output[pos_y + pos_x];
			//new_SNR = (d_FIR_input[pos_y + pos_x]-nTaps*signal_mean)/(sqrt((float) nTaps)*signal_sd);
			new_SNR = ( d_FIR_input[pos_y + pos_x] - signal_mean ) / ( signal_sd );
			if (nTaps == 1)
			{
				//if(new_SNR>3.0){
				//	d_FIR_input[pos_y + pos_x]=d_MSD[0];
				//	d_SNR_output[pos_y + pos_x]=0;
				//	d_SNR_taps[pos_y + pos_x]=nTaps;
				//}
				//else {
				d_SNR_output[pos_y + pos_x] = new_SNR;
				d_SNR_taps[pos_y + pos_x] = nTaps;
				//}
			}
			else if (new_SNR > old_SNR)
			{
				d_SNR_output[pos_y + pos_x] = new_SNR;
				d_SNR_taps[pos_y + pos_x] = nTaps;
			}
		}
		else
		{
			if (pos_x >= ( nColumns - offset ) && pos_x < nColumns)
			{
				d_SNR_output[pos_y + pos_x] = 0;
				d_SNR_taps[pos_y + pos_x] = 0;
			}
		}
		pos_x = pos_x + WARP;
	}

} //-------------------- KERNEL ENDS HERE --------------------------

#endif
