/* This function takes a pointer to the file pointer so that it can update the position of the file pointer
 */
 
#include <hip/hip_runtime_api.h>

#include <hip/hip_vector_types.h>
#include <driver_functions.h>
#include <hip/hip_runtime.h>

// CUDA utilities and system includes
#include <hip/hip_vector_types.h>

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
//#include <omp.h>
#include <hip/hip_runtime.h>
#include "headers/params.h"

#include "headers/device_DDTR_Plan.h"

void allocate_memory_cpu_input(unsigned short **input_buffer, size_t nsamp, size_t nchans) {

	size_t inputsize = nsamp*((size_t) nchans)*sizeof(unsigned short);
	*input_buffer = (unsigned short *) malloc(inputsize);
}

void allocate_memory_cpu_output(float ****output_buffer, DDTR_Plan *DDTR_plan) {
	int nRanges = DDTR_plan->nRanges;
	int num_tchunks = DDTR_plan->num_tchunks;
	size_t host_outputsize = 0;
	
	*output_buffer = (float ***) malloc( nRanges*sizeof(float **) );
	for (int i = 0; i < nRanges; i++) {
		size_t total_samps = 0;
		for (int k = 0; k < num_tchunks; k++)
			total_samps += DDTR_plan->t_processed[i][k];
		//printf("\nTOTSAMPS:\t%d %d", total_samps, i);
		( *output_buffer )[i] = (float **) malloc( DDTR_plan->ndms[i]*sizeof(float *));
		//if((*output_buffer)[i]) printf("\n FAILED! Could not allocate %zu bytes", ndms[i]*sizeof(float *));
		for (int j = 0; j < DDTR_plan->ndms[i]; j++) {
			( *output_buffer )[i][j] = (float *) malloc( total_samps*sizeof(float) );
			if( (*output_buffer)[i][j] == NULL ) printf("\n FAILED! Could not allocate %zu bytes", DDTR_plan->ndms[i]*sizeof(float *));
		}
		host_outputsize += total_samps*DDTR_plan->ndms[i]*sizeof(float);
		printf("\noutput size: %llu", (unsigned long long) sizeof( *output_buffer ) / 1024 / 1024 / 1024);
	}
	
	DDTR_plan->host_outputsize = host_outputsize;
}

void allocate_memory_gpu(unsigned short **d_input, float **d_output, DDTR_Plan *DDTR_plan) {
	size_t time_samps = DDTR_plan->t_processed[0][0] + (size_t) DDTR_plan->maxshift;
	printf("\n\n\n%d\n\n\n", time_samps), fflush(stdout);
	DDTR_plan->gpu_inputsize = time_samps*DDTR_plan->nchans*sizeof(unsigned short);
	checkCudaErrors( hipMalloc((void **) d_input, DDTR_plan->gpu_inputsize) );
	printf("time_samp: %zu; input size: %zu\n", time_samps, DDTR_plan->gpu_inputsize);

	if (DDTR_plan->nchans < DDTR_plan->max_ndms) {
		DDTR_plan->gpu_outputsize = time_samps*DDTR_plan->max_ndms*sizeof(float);
	}
	else {
		DDTR_plan->gpu_outputsize = time_samps*DDTR_plan->nchans*sizeof(float);
	}
	printf("DDTR_plan->gpu_outputsize: %zu;\n", DDTR_plan->gpu_outputsize);
	checkCudaErrors( hipMalloc((void **) d_output, DDTR_plan->gpu_outputsize) );
	checkCudaErrors( hipMemset(*d_output, 0, DDTR_plan->gpu_outputsize) );
}



