//Added by Karel Adamek

#include "headers/params.h"

void PD_FIR_init(void)
{
	//---------> Specific nVidia stuff
	hipDeviceSetCacheConfig (hipFuncCachePreferShared);
	hipDeviceSetSharedMemConfig (hipSharedMemBankSizeEightByte);
}

int PD_FIR(float *d_input, float *d_output, int nTaps, int nDMs, int nTimesamples)
{
	//---------> Task specific
	int ut; //unused timesamples
	int itemp = (int) ( ( nTaps - 1 ) / ( WARP * PD_FIR_ACTIVE_WARPS ) ) + 1;
	int nLoops = PD_FIR_NWINDOWS + itemp;

	//---------> CUDA block and CUDA grid parameters
	int nCUDAblocks_x = (int) ( ( nTimesamples - nTaps + 1 ) / ( PD_FIR_ACTIVE_WARPS * WARP * PD_FIR_NWINDOWS ) );
	int nCUDAblocks_y = nDMs;
	int SM_size = ( PD_FIR_ACTIVE_WARPS * WARP * PD_FIR_NWINDOWS + nTaps - 1 ) * 4;

	dim3 gridSize(nCUDAblocks_x, nCUDAblocks_y, 1);
	dim3 blockSize(PD_FIR_ACTIVE_WARPS * WARP, 1, 1);

	//---------> Pulse detection FIR
	PD_FIR_init();
	PD_FIR_GPU<<<gridSize, blockSize, SM_size>>>(d_input, d_output, nTaps, nLoops, nTimesamples);

	ut = nTimesamples - nCUDAblocks_x * PD_FIR_ACTIVE_WARPS * WARP * PD_FIR_NWINDOWS;
	return ( ut );
}


int GPU_FIRv1_wrapper(float *d_input, float *d_output, int nTaps, unsigned int nDMs, unsigned int nTimesamples){
	//---------> Task specific
	int ut; //unused timesamples
	int itemp=(int) ((nTaps - 1)/(WARP*PD_FIR_ACTIVE_WARPS)) + 1;
	int nLoops=PD_FIR_NWINDOWS + itemp;
	
	//---------> CUDA block and CUDA grid parameters
	int nCUDAblocks_x=(int) ((nTimesamples - nTaps + 1)/(PD_FIR_ACTIVE_WARPS*WARP*PD_FIR_NWINDOWS));
	int nCUDAblocks_y=nDMs; //Head size
	int SM_size=(PD_FIR_ACTIVE_WARPS*WARP*PD_FIR_NWINDOWS + nTaps - 1)*4;
	
	dim3 gridSize(nCUDAblocks_x, nCUDAblocks_y, 1);			//nCUDAblocks_y goes through spectra
	dim3 blockSize(PD_FIR_ACTIVE_WARPS*WARP, 1, 1); 		//nCUDAblocks_x goes through channels
	
	// ----------------------------------------------->
	// --------> Measured part (Pulse detection FIR)	
	
	//---------> Pulse detection FIR
	hipDeviceSetCacheConfig(hipFuncCachePreferShared);
	hipDeviceSetSharedMemConfig(hipSharedMemBankSizeEightByte);
	PD_FIR_GPUv1<<<gridSize,blockSize,SM_size>>>(d_input, d_output, nTaps, nLoops, nTimesamples);

	// --------> Measured part (Pulse detection FIR)
	// ----------------------------------------------->
	
	ut=nTimesamples - nCUDAblocks_x*PD_FIR_ACTIVE_WARPS*WARP*PD_FIR_NWINDOWS;
	return(ut);
}
