#include "hip/hip_runtime.h"
#ifndef CORNERTURN_KERNEL_H_
#define CORNERTURN_KERNEL_H_

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "headers/params.h"

//{{{ corner_turn
__global__ void simple_corner_turn_kernel(unsigned short *d_input, float *d_output, int nchans, int nsamp)
{

	int t = blockIdx.x * blockDim.x + threadIdx.x;
	int c = blockIdx.y * blockDim.y + threadIdx.y;

	d_output[c * nsamp + t] = (float) __ldg(&d_input[t * nchans + c]);

}

__global__ void swap(unsigned short *d_input, float *d_output, int nchans, int nsamp)
{

	int t = blockIdx.x * blockDim.x + threadIdx.x;
	int c = blockIdx.y * blockDim.y + threadIdx.y;

	d_input[c * nsamp + t] = (unsigned short) __ldg(&d_output[c * nsamp + t]);

}

//}}}


__global__ void simple_corner_turn_kernel(float *d_input, float *d_output, int primary_size, int secondary_size){

	int primary = blockIdx.x * blockDim.x + threadIdx.x;
	int secondary = blockIdx.y * blockDim.y + threadIdx.y;

	d_output[primary*secondary_size + secondary] = (float) __ldg(&d_input[secondary*primary_size + primary]);
}


__global__ void corner_turn_SM_kernel(float const* __restrict__ d_input, float *d_output, int primary_size, int secondary_size) {
	__shared__ float s_input[WARP*(WARP+1)*CT_CORNER_BLOCKS];
	
	int i, gpos, spos, itemp, pc, sc;
	
	int warp_id = threadIdx.x>>5;
	int local_id = threadIdx.x & (WARP - 1);
	
	gpos=(blockIdx.y*(blockDim.x>>5) + warp_id)*CT_ROWS_PER_WARP*primary_size + blockIdx.x*CT_CORNER_BLOCKS*WARP + local_id;
	for(int by=0; by<CT_ROWS_PER_WARP; by++){
		spos=local_id*WARP + local_id + warp_id*CT_ROWS_PER_WARP + by;
		for(int bx=0; bx<CT_CORNER_BLOCKS; bx++){ // temporary 
			s_input[spos]=d_input[gpos];
			gpos=gpos + WARP;
			spos=spos + WARP*(WARP+1);
		}
		gpos=gpos + primary_size - CT_CORNER_BLOCKS*WARP;
	}
	
	__syncthreads();
	
	itemp=warp_id*CT_ROWS_PER_WARP*CT_CORNER_BLOCKS;
	for(i=0; i<CT_ROWS_PER_WARP*CT_CORNER_BLOCKS; i++){
		pc = (blockIdx.x*CT_CORNER_BLOCKS*WARP + itemp + i);
		sc = WARP*blockIdx.y + local_id;
		if( pc<primary_size && sc<secondary_size ) {
			gpos=pc*secondary_size + sc;
			spos=(itemp + i)*(WARP+1) + local_id;
			d_output[gpos]=s_input[spos];
		}
	}
}


#endif

