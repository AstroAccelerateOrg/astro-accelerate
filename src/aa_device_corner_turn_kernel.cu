#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "aa_params.hpp"
#include <stdio.h>

namespace astroaccelerate {

  __global__ void simple_corner_turn_kernel(float *d_input, float *d_output, int primary_size, int secondary_size){

    size_t primary = blockIdx.x * blockDim.x + threadIdx.x;
    size_t secondary = blockIdx.y * blockDim.y + threadIdx.y;

    d_output[(size_t)primary*secondary_size + secondary] = (float) __ldg(&d_input[(size_t)secondary*primary_size + primary]);
  }


  __global__ void corner_turn_SM_kernel(float const* __restrict__ d_input, float *d_output, int primary_size, int secondary_size) {
    __shared__ float s_input[WARP*(WARP+1)*CT_CORNER_BLOCKS];
	
    int i, spos, itemp, pc, sc;
    size_t gpos;
	
    int warp_id = threadIdx.x>>5;
    int local_id = threadIdx.x & (WARP - 1);
	
    gpos=(size_t)((size_t)(blockIdx.y*(blockDim.x>>5)) + (size_t)warp_id)*CT_ROWS_PER_WARP*primary_size + (size_t)(blockIdx.x*CT_CORNER_BLOCKS*WARP) + (size_t)local_id;
    for(int by=0; by<CT_ROWS_PER_WARP; by++){
      spos=local_id*WARP + local_id + warp_id*CT_ROWS_PER_WARP + by;
      for(int bx=0; bx<CT_CORNER_BLOCKS; bx++){ // temporary 
	s_input[spos]=d_input[gpos];
	gpos=gpos + (size_t)WARP;
	spos=spos + WARP*(WARP+1);
      }
      gpos=gpos + (size_t)primary_size - (size_t)(CT_CORNER_BLOCKS*WARP);
    }
	
    __syncthreads();
	
    itemp=warp_id*CT_ROWS_PER_WARP*CT_CORNER_BLOCKS;
    for(i=0; i<CT_ROWS_PER_WARP*CT_CORNER_BLOCKS; i++){
      pc = (blockIdx.x*CT_CORNER_BLOCKS*WARP + itemp + i);
      sc = WARP*blockIdx.y + local_id;
      if( pc<primary_size && sc<secondary_size ) {
	gpos=(size_t)(pc*secondary_size) + (size_t)sc;
	spos=(itemp + i)*(WARP+1) + local_id;
	d_output[gpos]=s_input[spos];
      }
    }
  }

  __global__ void simple_corner_turn_kernel(unsigned short *d_input, float *d_output, int nchans, int nsamp) {

    size_t t = blockIdx.x * blockDim.x + threadIdx.x;
    size_t c = blockIdx.y * blockDim.y + threadIdx.y;

    d_output[(size_t)(c * nsamp) + t] = (float) __ldg(&d_input[(size_t)(t * nchans) + c]);

  }

  __global__ void swap(unsigned short *d_input, float *d_output, int nchans, int nsamp) {

    size_t t = blockIdx.x * blockDim.x + threadIdx.x;
    size_t c = blockIdx.y * blockDim.y + threadIdx.y;

    d_input[(size_t)(c * nsamp) + t] = (unsigned short) __ldg(&d_output[(size_t)(c * nsamp) + t]);

  }

  /** \brief Kernel wrapper function for simple_corner_turn_kernel kernel function. */
  void call_kernel_simple_corner_turn_kernel(const dim3 &block_size, const dim3 &grid_size, unsigned short *const d_input, float *const d_output, const int &primary_size, const int &secondary_size) {
    simple_corner_turn_kernel<<<block_size, grid_size>>>(d_input, d_output, primary_size, secondary_size);
  }

  /** \brief Kernel wrapper function for simple_corner_turn_kernel kernel function. */
  void call_kernel_simple_corner_turn_kernel(const dim3 &block_size, const dim3 &grid_size, float *const d_input, float *const d_output, const int &primary_size, const int &secondary_size) {
    simple_corner_turn_kernel<<<block_size, grid_size>>>(d_input, d_output, primary_size, secondary_size);
  }

  /** \brief Kernel wrapper function for corner_turn_SM_kernel kernel function. */
  void call_kernel_corner_turn_SM_kernel(const dim3 &grid_size, const dim3 &block_size, float const *const d_input, float *const d_output, const int &primary_size, const int &secondary_size) {
    corner_turn_SM_kernel<<<grid_size,block_size>>>(d_input, d_output, primary_size, secondary_size);
  }

  /** \brief Kernel wrapper function for swap kernel function. */
  void call_kernel_swap(const dim3 &block_size, const dim3 &grid_size, unsigned short *const d_input, float *const d_output, const int &nchans, const int &nsamp) {
    swap<<<block_size, grid_size>>>(d_input, d_output, nchans, nsamp);
  }

} //namespace astroaccelerate
