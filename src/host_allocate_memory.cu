/* This function takes a pointer to the file pointer so that it can update the
 * position of the file pointer
 */

#include <hip/hip_runtime.h>
#include <driver_functions.h>
#include <hip/hip_vector_types.h>

// CUDA utilities and system includes
#include <hip/hip_vector_types.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <stdio.h>
//#include <omp.h>
#include "host_info.hpp"
#include "params.hpp"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

void allocate_memory_cpu_input(FILE**           fp,
                               size_t           gpu_memory,
                               size_t*          host_memory,
                               int              maxshift,
                               int              num_tchunks,
                               int              max_ndms,
                               int              total_ndms,
                               int              nsamp,
                               int              nchans,
                               int              nbits,
                               int              range,
                               int*             ndms,
                               int**            t_processed,
                               unsigned short** input_buffer,
                               float****        output_buffer,
                               unsigned short** d_input,
                               float**          d_output,
                               size_t*          gpu_inputsize,
                               size_t*          gpu_outputsize,
                               size_t*          inputsize,
                               size_t*          outputsize) {
  //	printf("\nAvailable memory: %zu MiB\n", (size_t)host_memory/1024/1024);
  *inputsize = nsamp * (size_t)nchans * sizeof(unsigned short);
  if(*host_memory < *inputsize) {
    host_mem_error((unsigned int)(*inputsize / 1024.0 / 1024.0),
                   (unsigned int)(*host_memory / 1024.0 / 1024.0),
                   "input");
  }
  *host_memory  = *host_memory - *inputsize;
  *input_buffer = (unsigned short*)malloc(*inputsize);
  //        printf("\nMemory available: %zd", *host_memory/1024/1024);
}

void allocate_memory_cpu_output(FILE**           fp,
                                size_t           gpu_memory,
                                size_t*          host_memory,
                                int              maxshift,
                                int              num_tchunks,
                                int              max_ndms,
                                int              total_ndms,
                                int              nsamp,
                                int              nchans,
                                int              nbits,
                                int              range,
                                int*             ndms,
                                int**            t_processed,
                                unsigned short** input_buffer,
                                float****        output_buffer,
                                unsigned short** d_input,
                                float**          d_output,
                                size_t*          gpu_inputsize,
                                size_t*          gpu_outputsize,
                                size_t*          inputsize,
                                size_t*          outputsize) {
  size_t estimate_outputbuffer_size = 0;
  for(int i = 0; i < range; i++)
    for(int j = 0; j < num_tchunks; j++)
      estimate_outputbuffer_size +=
          (size_t)(t_processed[i][j] * sizeof(float) * ndms[i]);
  //	size_t estimate_outputbuffer_size =
  //(size_t)(total_ndms*sizeof(float)*nsamp); 	printf("\nTotal ndms: %i; nsamp:
  //%i; mem: %zu estimated: %zu \n", total_ndms, nsamp,
  //(size_t)(*host_memory)/1024/1024, estimate_outputbuffer_size/1024/1024);
  if(*host_memory < estimate_outputbuffer_size) {
    host_mem_error((unsigned int)(estimate_outputbuffer_size / 1024.0 / 1024.0),
                   (unsigned int)(*host_memory / 1024.0 / 1024.0),
                   "output");
  }

  *outputsize    = 0;
  *output_buffer = (float***)malloc(range * sizeof(float**));
  for(int i = 0; i < range; i++) {
    int total_samps = 0;
    for(int k = 0; k < num_tchunks; k++)
      total_samps += t_processed[i][k];
    // printf("\nTOTSAMPS:\t%d %d", total_samps, i);
    (*output_buffer)[i] = (float**)malloc(ndms[i] * sizeof(float*));
    // if((*output_buffer)[i]) printf("\n FAILED! Could not allocate %zu bytes",
    // ndms[i]*sizeof(float *));
    for(int j = 0; j < ndms[i]; j++) {
      (*output_buffer)[i][j] = (float*)malloc((total_samps) * sizeof(float));
      // if((*output_buffer)[i][j]) printf("\n FAILED! Could not allocate %zu
      // bytes", ndms[i]*sizeof(float *));
      //			memset((*output_buffer)[i][j],0.0f,(total_samps)*sizeof(float));
    }
    *outputsize += (total_samps)*ndms[i] * sizeof(float);
  }
  *host_memory = *host_memory - *outputsize;
  //      printf("\noutput: Memory available: %zu", (*host_memory)/1024/1024);
  //	printf("\noutput size: %llu",(unsigned long
  //long)(*outputsize/1024/1024));
}

void allocate_memory_gpu(FILE**           fp,
                         size_t           gpu_memory,
                         int              maxshift,
                         int              num_tchunks,
                         int              max_ndms,
                         int              total_ndms,
                         int              nsamp,
                         int              nchans,
                         int              nbits,
                         int              range,
                         int*             ndms,
                         int**            t_processed,
                         unsigned short** input_buffer,
                         float****        output_buffer,
                         unsigned short** d_input,
                         float**          d_output,
                         size_t*          gpu_inputsize,
                         size_t*          gpu_outputsize,
                         size_t*          inputsize,
                         size_t*          outputsize) {

  int time_samps = t_processed[0][0] + maxshift;
  printf("\n\n\n%d\n\n\n", time_samps), fflush(stdout);
  *gpu_inputsize = (size_t)time_samps * (size_t)nchans * sizeof(unsigned short);
  checkCudaErrors(hipMalloc((void**)d_input, *gpu_inputsize));

  if(nchans < max_ndms) {
    *gpu_outputsize = (size_t)time_samps * (size_t)max_ndms * sizeof(float);
  } else {
    *gpu_outputsize = (size_t)time_samps * (size_t)nchans * sizeof(float);
  }
  checkCudaErrors(hipMalloc((void**)d_output, *gpu_outputsize));

  // end_t=omp_get_wtime();
  // time = (float)(end_t-start_t);
  // printf("\nGPU Malloc in: %f ", time);

  (hipMemset(*d_output, 0, *gpu_outputsize));
}

void allocate_memory_MSD(float**           d_MSD_workarea,
                         unsigned short**  d_MSD_output_taps,
                         float**           d_MSD_interpolated,
                         unsigned long int MSD_maxtimesamples,
                         int               MSD_DIT_widths,
                         int               nTimesamples,
                         size_t            MSD_profile_size) {

  checkCudaErrors(hipMalloc((void**)d_MSD_workarea,
                             MSD_maxtimesamples * 5.5 * sizeof(float)));
  checkCudaErrors(hipMalloc((void**)&(*d_MSD_output_taps),
                             sizeof(ushort) * 2 * MSD_maxtimesamples));
  //        checkCudaErrors(hipMallocManaged((void **) gmem_peak_pos,
  //        sizeof(int)));
  //       checkCudaErrors(hipHostMalloc((void **) temp_peak_pos,
  //       sizeof(int)));
  checkCudaErrors(
      hipMalloc((void**)d_MSD_interpolated, sizeof(float) * MSD_profile_size));
}
