#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>

#include "aa_fdas_host.hpp"
#include "aa_log.hpp"
#include "aa_device_spectrum_whitening.hpp"

#define ENABLE_HOST_DERED false
#define ENABLE_HOST_BLOCK_MEDIAN_NORM false

namespace astroaccelerate {

  /** \brief Print fdas parameters. */
  void  fdas_print_params_h()
  {
    printf("\n\nParameters defined in params.h:\n\t-------------------\n");
    //  printf("\nSampling time: TSAMP %g\n", TSAMP);
    printf("\nSpeed of light: SLIGHT %g\n", SLIGHT);
    printf("\nTemplate length for FFT: KERNLEN = RADIX*POTWO %d\n", KERNLEN);
    printf("\nAcceleration step in fourier bins (z): ACCEL_STEP %f\n", ACCEL_STEP);
    printf("\nAcceleration step in fourier bins (z) reciprocal: ACCEL_STEP_R %f\n", ACCEL_STEP_R);
    printf("\nMaximum acceleration in fourier bins (z): ZMAX %d\n", ZMAX);
    printf("\nNumber of templates including zero acceleration: NKERN %d\n", NKERN);
    //  printf("\nLowest acceleration in fourier bins (z) (for harmonic sum): ZLO %d\n", ZLO);
    printf("\nThread block size in x direction for 2-D thread block convolution GPU kernels : TBSIZEX %d\n", TBSIZEX);
    printf("\nThread block size in Y direction for 2-D thread block convolution GPU kernels : TBSIZEY %d\n", TBSIZEY);
    printf("\nThread block size in x direction for 2-D thread block power spectrum GPU kernels : PTBSIZEX %d\n", PTBSIZEX);
    printf("\nThread block size in y direction for 2-D thread block power spectrum GPU kernels : PTBSIZEY %d\n", PTBSIZEY);
    printf("\n\nCustom FFT specific parameters:\n\t------------------\n" );
    printf("\n\n\t--------------\n\n");
  }

  /** \brief Check CUDA devices. */
  void fdas_cuda_check_devices(int devid) {
    int devcount;
    hipError_t e = hipGetDeviceCount(&devcount);
    
    if(e != hipSuccess) {
      LOG(log_level::error, "Could not hipGetDeviceCount in aa_fdas_host.cu (" + std::string(hipGetErrorString(e)) + ")");
    }
    
    printf("\nDetected %d CUDA Capable device(s)\n", devcount);
  }

  /** \brief Allocate GPU arrays for fdas. */
  void fdas_alloc_gpu_arrays(fdas_gpuarrays *arrays,  cmd_args *cmdargs)
  {
    printf("\nAllocating gpu arrays:\n"); 

    if (cmdargs->inbin){
      printf("\nF-fdot array will be interbinned\n");
    }
    double gbyte = 1024.0*1024.0*1024.0;
    //double mbyte = 1024.0*1024.0;

    // Memory allocations for gpu real fft input / output signal
    hipError_t e = hipMalloc((void**)&arrays->d_in_signal, arrays->mem_insig);
    
    if(e != hipSuccess) {
      LOG(log_level::error, "Could not hipMalloc in aa_fdas_host.cu (" + std::string(hipGetErrorString(e)) + ")");
    }

    e = hipMalloc((void**)&arrays->d_fft_signal, arrays->mem_rfft);

    if(e != hipSuccess) {
      LOG(log_level::error, "Could not hipMalloc in aa_fdas_host.cu (" + std::string(hipGetErrorString(e)) + ")");
    }

    //Allocating arrays for fourier domain convolution  
    e = hipMalloc((void**)&arrays->d_ext_data, arrays->mem_extsig);

    if(e != hipSuccess) {
      LOG(log_level::error, "Could not hipMalloc in aa_fdas_host.cu (" + std::string(hipGetErrorString(e)) + ")");
    }

    //templates
    e = hipMalloc((void**)&arrays->d_kernel, KERNLEN*sizeof(float2)*NKERN );

    if(e != hipSuccess) {
      LOG(log_level::error, "Could not hipMalloc in aa_fdas_host.cu (" + std::string(hipGetErrorString(e)) + ")");
    }

    //ffdot planes
    e = hipMalloc((void**)&arrays->d_ffdot_pwr, arrays->mem_ffdot );

    if(e != hipSuccess) {
      LOG(log_level::error, "Could not hipMalloc in aa_fdas_host.cu (" + std::string(hipGetErrorString(e)) + ")");
    }
    
    //initialise array
    e = hipMemset(arrays->d_ffdot_pwr, 0, arrays->mem_ffdot);

    if(e != hipSuccess) {
      LOG(log_level::error, "Could not hipMemset in aa_fdas_host.cu (" + std::string(hipGetErrorString(e)) + ")");
    }

    printf("ffdot x size: %lu",(unsigned long)arrays->mem_ffdot/sizeof(float)/(unsigned long)NKERN);
    if(cmdargs->basic==1){
      e = hipMalloc(&arrays->d_ffdot_cpx, arrays->mem_ffdot_cpx);

      if(e != hipSuccess) {
	LOG(log_level::error, "Could not hipMalloc in aa_fdas_host.cu (" + std::string(hipGetErrorString(e)) + ")");
      }
    }

    if(cmdargs->kfft && cmdargs->inbin){
      //    printf("mem_ipedge = %u ",mem_ipedge/);
      e = hipMalloc(&arrays->ip_edge_points, arrays->mem_ipedge);

      if(e != hipSuccess) {
	LOG(log_level::error, "Could not hipMalloc in aa_fdas_host.cu (" + std::string(hipGetErrorString(e)) + ")");
      }
    }
   
    // Added by KA
    if ( hipSuccess != hipMalloc((void**) &arrays->d_fdas_peak_list, arrays->mem_max_list_size)) printf("Allocation error in FDAS: d_fdas_peak_list\n");
	
    // check allocated/free memory
    size_t mfree,  mtotal;
    e = hipMemGetInfo ( &mfree, &mtotal );

    if(e != hipSuccess) {
      LOG(log_level::error, "Could not hipMemGetInfo in aa_fdas_host.cu (" + std::string(hipGetErrorString(e)) + ")");
    }
    
    printf("\nMemory allocation finished: Total memory for this device: %.2f GB\nAvailable memory left on this device: %.2f GB \n", mtotal/gbyte, mfree/gbyte);
  }

  /** \brief Free GPU arrays for fdas. */
  void fdas_free_gpu_arrays(fdas_gpuarrays *arrays,  cmd_args *cmdargs)
  {

    hipError_t e = hipFree(arrays->d_in_signal);
    
    if(e != hipSuccess) {
      LOG(log_level::error, "Could not hipFree in aa_fdas_host.cu (" + std::string(hipGetErrorString(e)) + ")");
    }
    
    e = hipFree(arrays->d_fft_signal);

    if(e != hipSuccess) {
      LOG(log_level::error, "Could not hipFree in aa_fdas_host.cu (" + std::string(hipGetErrorString(e)) + ")");
    }
    
    e = hipFree(arrays->d_ext_data);

    if(e != hipSuccess) {
      LOG(log_level::error, "Could not hipFree in aa_fdas_host.cu (" + std::string(hipGetErrorString(e)) + ")");
    }
    
    e = hipFree(arrays->d_ffdot_pwr);

    if(e != hipSuccess) {
      LOG(log_level::error, "Could not hipFree in aa_fdas_host.cu (" + std::string(hipGetErrorString(e)) + ")");
    }
    
    e = hipFree(arrays->d_kernel);

    if(e != hipSuccess) {
      LOG(log_level::error, "Could not hipFree in aa_fdas_host.cu (" + std::string(hipGetErrorString(e)) + ")");
    }
    
    if(cmdargs->basic) {
      e = hipFree(arrays->d_ffdot_cpx);
      
      if(e != hipSuccess) {
	LOG(log_level::error, "Could not hipFree in aa_fdas_host.cu (" + std::string(hipGetErrorString(e)) + ")");
      } 
    }

    if(cmdargs->kfft && cmdargs->inbin) {
      e = hipFree(arrays->ip_edge_points);
      
      if(e != hipSuccess) {
	LOG(log_level::error, "Could not hipFree in aa_fdas_host.cu (" + std::string(hipGetErrorString(e)) + ")");
      } 
    }
	
    // Added by KA
    hipFree(arrays->d_fdas_peak_list);
  }

  /**
   * \brief Create kernel templates for the correlation technique (Ransom et. al. 2002), and upload + FFT to GPU memory.
   * \brief Using functions from the original PRESTO accelsearch code, (small adaptations for variables and remove normal interpolation management - input is already interpolated signal).
   * \author Scott Ransom.
   */
  void fdas_create_acc_kernels(hipfftComplex* d_kernel, cmd_args *cmdargs ) {
    int ii;
    int inbin = 1;
    hipfftComplex *h_kernel, *tempkern;
    hipfftHandle templates_plan; // for host kernel fft
    int nrank = 1;
    int n[] = {KERNLEN};
    int idist = n[0], odist =n[0];
    int *inembed = n, *onembed = n;
    int istride =1, ostride = 1;

    //allocate kernel array and prepare fft
    h_kernel = (hipfftComplex*) malloc(NKERN*KERNLEN*sizeof(float2));

    // batched fft plan for the templates array
    hipfftPlanMany( &templates_plan, nrank, n, inembed , istride, 
		   idist, onembed, ostride,
		   odist, HIPFFT_C2C, NKERN); 

    for (ii = 0; ii < NKERN; ii++){
      double z = (-ZMAX+ii*ACCEL_STEP);
      int halfwidth = presto_z_resp_halfwidth(z, LOWACC) ;
      int numkern = 2 * halfwidth * inbin;
      tempkern = presto_gen_z_response(0.0, inbin, z, numkern);
      presto_place_complex_kernel(tempkern, numkern, (h_kernel+ii*KERNLEN), KERNLEN);
      free(tempkern);
    }
  
    //!TEST!: replace templates here. Template width: numkern; padded width: KERNLEN
#ifdef FDAS_CONV_TEST
    for (ii = 0; ii < NKERN; ii++){
      int boxcar_width=ii*FDAS_TEST_FILTER_INCREMENT;
      for(int f=0; f<KERNLEN; f++){
	h_kernel[ii*KERNLEN + f].x = 0;
	h_kernel[ii*KERNLEN + f].y = 0;
	  
	if(f<boxcar_width/2) h_kernel[ii*KERNLEN + f].x = 1.0;
	if(f>=(KERNLEN-boxcar_width/2)) h_kernel[ii*KERNLEN + f].x = 1.0;
      }
    }
#endif
    //!TEST!: replace templates here. Template width: numkern; padded width: KERNLEN
  
    hipError_t e = hipMemcpy( d_kernel, h_kernel, KERNLEN*sizeof(float2)* NKERN, hipMemcpyHostToDevice); // upload kernels to GPU
    
    if(e != hipSuccess) {
      LOG(log_level::error, "Could not hipMemcpy in aa_fdas_host.cu (" + std::string(hipGetErrorString(e)) + ")");
    }
    
#ifndef NOCUST
    //use kerel's non-reordered fft
    if (cmdargs->kfft)
      call_kernel_customfft_fwd_temps_no_reorder(d_kernel);  
#endif
    //use cuFFT to transform the templates
    if (cmdargs->basic)
      hipfftExecC2C(templates_plan, d_kernel, d_kernel, HIPFFT_FORWARD); 

    free(h_kernel);

  }

  /** \brief Create CUDA cufft fftplans for FDAS. */
  void fdas_cuda_create_fftplans(fdas_cufftplan *fftplans, fdas_params *params) {
    /*check plan memory overhead and create plans */
    double mbyte = 1024.0*1024.0;
    //double gbyte = mbyte*1024.0;
 
    //set cufft plan parameters
    size_t sig_worksize, real_worksize;
    int nrank = 1;
    int n[] = {KERNLEN};
    int idist = n[0], odist =n[0];
    int *inembed = n, *onembed = n;
    int istride =1, ostride = 1;

    //estimate plan memory for real fft
    hipfftResult e = hipfftEstimate1d( params->nsamps, HIPFFT_R2C, 1, &real_worksize);
    
    if(e != HIPFFT_SUCCESS) {
      LOG(log_level::error, "Could not hipfftEstimate1d in aa_fdas_host.cu");
    }
    
    printf("\nsignal real fft plan requires extra %f MB of memory\n", real_worksize / mbyte);

    //estimate plan memory for forward fft
    e = hipfftEstimateMany(nrank, n,inembed, istride, idist, onembed, ostride, odist, HIPFFT_C2C, params->nblocks, &sig_worksize);

    if(e != HIPFFT_SUCCESS) {
      LOG(log_level::error, "Could not cudaEstimateMany in aa_fdas_host.cu");
    }
    
    printf("\nsignal forward fft plan requires extra  %f MB of memory\n the same plan is used for the inverse fft", sig_worksize / mbyte);
  
    // real plan
    size_t rworksize;
    int rn[] = {params->nsamps};
    int *rinembed = rn, *ronembed = rn;
    int ridist = rn[0], rodist = params->rfftlen;
 
    hipfftCreate(&fftplans->realplan);
    e = hipfftMakePlanMany( fftplans->realplan, nrank, rn, rinembed, istride, ridist, ronembed, ostride, rodist, HIPFFT_R2C, 1, &rworksize);

    if(e != HIPFFT_SUCCESS) {
      LOG(log_level::error, "Could not hipfftMakePlanMany in aa_fdas_host.cu");
    }
    
    hipDeviceSynchronize();
    //getLastCudaError("\nCuda Error real fft plan\n");

    // forward batched plan - same used for inverse
    e = hipfftCreate(&fftplans->forwardplan);

    if(e != HIPFFT_SUCCESS) {
      LOG(log_level::error, "Could not hipfftCreate in aa_fdas_host.cu");
    }
    
    e = hipfftMakePlanMany( fftplans->forwardplan, nrank, n, inembed, istride, idist, onembed, ostride, odist, HIPFFT_C2C, params->nblocks, &sig_worksize);

    if(e != HIPFFT_SUCCESS) {
      LOG(log_level::error, "Could not hipfftMakePlanMany in aa_fdas_host.cu");
    }
    
    hipDeviceSynchronize();
    //getLastCudaError("\nCuda Error forward fft plan\n");
    printf("\ncuFFT plans done \n");
  }

	void export_fft_data(float2* data, size_t data_size, const char* file){
		std::ofstream FILEOUT;
		FILEOUT.open (file, std::ofstream::out);
		for(size_t f = 0; f < data_size; f++){
			FILEOUT << data[f].x << " " << data[f].y << " " << sqrt(data[f].x*data[f].x + data[f].y*data[f].y) << std::endl;
		}
		FILEOUT.close();
	}

  /** \brief Perform basic fourier domain accelerated search (fdas). */
  void fdas_cuda_basic(fdas_cufftplan *fftplans, fdas_gpuarrays *gpuarrays, cmd_args *cmdargs, fdas_params *params)
  {
    /* Basic GPU fdas algorithm using cuFFT */
    //int inbin;
    int cthreads = TBSIZEX;
    int cblocks = KERNLEN/TBSIZEX;

    dim3 pwthreads(PTBSIZEX, PTBSIZEY);
    dim3 pwblocks((params->sigblock / PTBSIZEX) + 1, NKERN/PTBSIZEY);

    /* if (cmdargs->inbin)
       inbin = 2;
       else
       inbin = 1;
    */
    //real fft
#ifndef FDAS_CONV_TEST
    hipfftExecR2C(fftplans->realplan, gpuarrays->d_in_signal, gpuarrays->d_fft_signal);
#endif
  
#ifdef FDAS_CONV_TEST
    float2 *f2temp;
    float *ftemp;
    ftemp  = (float *)malloc(params->rfftlen*sizeof(float));
    f2temp = (float2 *)malloc(params->rfftlen*sizeof(float2));
    hipError_t e = hipMemcpy(ftemp, gpuarrays->d_in_signal, (params->rfftlen)*sizeof(float), hipMemcpyDeviceToHost);

    if(e != hipSuccess) {
      LOG(log_level::error, "Could not hipMemcpy in aa_fdas_host.cu (" + std::string(hipGetErrorString(e)) + ")");
    }
    
    for(int f=0; f<params->rfftlen; f++){
      f2temp[f].x = ftemp[f];
      f2temp[f].y = 0;
    }
    e = hipMemcpy(gpuarrays->d_fft_signal, f2temp, (params->rfftlen)*sizeof(float2), hipMemcpyHostToDevice);
    
    if(e != hipSuccess) {
      LOG(log_level::error, "Could not hipMemcpy in aa_fdas_host.cu (" + std::string(hipGetErrorString(e)) + ")");
    }
    
    free(ftemp);
    free(f2temp);
#endif

	if (cmdargs->norm){
		if(ENABLE_HOST_DERED){
			// doing deredning on the host
			float2 *fftsig;
			fftsig = (float2*)malloc((params->rfftlen)*sizeof(float2)); 
			
			hipError_t e = hipMemcpy(fftsig, gpuarrays->d_fft_signal, (params->rfftlen)*sizeof(float2), hipMemcpyDeviceToHost);
			
			if(e != hipSuccess) {
				LOG(log_level::error, "Could not hipMemcpy in aa_fdas_host.cu (" + std::string(hipGetErrorString(e)) + ")");
			}
			
			presto_dered_sig(fftsig, params->rfftlen);
			e = hipMemcpy(gpuarrays->d_fft_signal, fftsig, (params->rfftlen)*sizeof(float2), hipMemcpyHostToDevice);
			
			if(e != hipSuccess) {
				LOG(log_level::error, "Could not hipMemcpy in aa_fdas_host.cu (" + std::string(hipGetErrorString(e)) + ")");
			}
			
			free(fftsig);
		}
		else {
			// doing deedning on GPU
			hipStream_t stream; stream = NULL;
			spectrum_whitening_SGP2((float2 *) gpuarrays->d_fft_signal, params->rfftlen, 1, true, stream);
		}
	}

    //overlap-copy
    call_kernel_cuda_overlap_copy(gpuarrays->d_ext_data, gpuarrays->d_fft_signal, params->sigblock, params->rfftlen, params->extlen, params->offset, params->nblocks );

	if (cmdargs->norm && ENABLE_HOST_BLOCK_MEDIAN_NORM){
		//  PRESTO block median normalization
		// TODO: replace with GPU version
		float2 *extsig;
		extsig = (float2*)malloc((params->extlen)*sizeof(float2));
		hipError_t e = hipMemcpy(extsig, gpuarrays->d_ext_data, (params->extlen)*sizeof(float2), hipMemcpyDeviceToHost);
	
		if(e != hipSuccess) {
			LOG(log_level::error, "Could not hipMemcpy in aa_fdas_host.cu (" + std::string(hipGetErrorString(e)) + ")");
		}
		
		for(int b=0; b<params->nblocks; ++b)
			presto_norm(extsig+b*KERNLEN, KERNLEN);
		e = hipMemcpy(gpuarrays->d_ext_data, extsig, (params->extlen)*sizeof(float2), hipMemcpyHostToDevice);
	
		if(e != hipSuccess) {
			LOG(log_level::error, "Could not hipMemcpy in aa_fdas_host.cu (" + std::string(hipGetErrorString(e)) + ")");
		}
		
		free(extsig);
	}

    //complex block fft
    hipfftExecC2C(fftplans->forwardplan, gpuarrays->d_ext_data, gpuarrays->d_ext_data, HIPFFT_FORWARD);

    //complex multiplication kernel
    call_kernel_cuda_convolve_reg_1d_halftemps(cblocks, cthreads, gpuarrays->d_kernel, gpuarrays->d_ext_data, gpuarrays->d_ffdot_cpx, params->extlen, params->scale);

    //inverse fft
    int nTemplates = ((NKERN-1)/2);
    for (int k=0; k < nTemplates; k++){
      hipfftExecC2C(fftplans->forwardplan, gpuarrays->d_ffdot_cpx + k * params->extlen, gpuarrays->d_ffdot_cpx + k *params->extlen, HIPFFT_BACKWARD);
      hipfftExecC2C(fftplans->forwardplan, gpuarrays->d_ffdot_cpx + (NKERN-1-k) * params->extlen, gpuarrays->d_ffdot_cpx + (NKERN-1-k) *params->extlen, HIPFFT_BACKWARD);
    }
    // z=0
    hipfftExecC2C(fftplans->forwardplan, gpuarrays->d_ffdot_cpx + (nTemplates * params->extlen), gpuarrays->d_ffdot_cpx + (nTemplates * params->extlen), HIPFFT_BACKWARD);

    //power spectrum 
    if (cmdargs->inbin){
      call_kernel_cuda_ffdotpow_concat_2d_inbin(pwblocks, pwthreads, gpuarrays->d_ffdot_cpx, gpuarrays->d_ffdot_pwr, params->sigblock, params->offset, params->nblocks, params->extlen, params->siglen);
    }
    else{
      call_kernel_cuda_ffdotpow_concat_2d(pwblocks, pwthreads, gpuarrays->d_ffdot_cpx, gpuarrays->d_ffdot_pwr, params->sigblock, params->offset, params->nblocks, params->extlen, params->siglen);
    }
  }

#ifndef NOCUST
  void fdas_cuda_customfft(fdas_cufftplan *fftplans, fdas_gpuarrays *gpuarrays, cmd_args *cmdargs, fdas_params *params) {
    //int nthreads;
    dim3 cblocks(params->nblocks, NKERN/2); 

    //real fft
#ifndef FDAS_CONV_TEST
    hipfftExecR2C(fftplans->realplan, gpuarrays->d_in_signal, gpuarrays->d_fft_signal);
#endif

#ifdef FDAS_CONV_TEST
    float2 *f2temp;
    float *ftemp;
    ftemp  = (float *)malloc(params->rfftlen*sizeof(float));
    f2temp = (float2 *)malloc(params->rfftlen*sizeof(float2));
    hipError_t e = hipMemcpy(ftemp, gpuarrays->d_in_signal, (params->rfftlen)*sizeof(float), hipMemcpyDeviceToHost);

    if(e != hipSuccess) {
      LOG(log_level::error, "Could not hipMemcpy in aa_fdas_host.cu (" + std::string(hipGetErrorString(e)) + ")");
    }
    
    for(int f=0; f<params->rfftlen; f++){
      f2temp[f].x = ftemp[f];
      f2temp[f].y = 0;
    }
    e = hipMemcpy(gpuarrays->d_fft_signal, f2temp, (params->rfftlen)*sizeof(float2), hipMemcpyHostToDevice);

    if(e != hipSuccess) {
      LOG(log_level::error, "Could not hipMemcpy in aa_fdas_host.cu (" + std::string(hipGetErrorString(e)) + ")");
    }
    
    free(ftemp);
    free(f2temp);
#endif
  

	if (cmdargs->norm){
		if(ENABLE_HOST_DERED){
			// doing deredning on the host
			float2 *fftsig;
			fftsig = (float2*)malloc((params->rfftlen)*sizeof(float2)); 
			
			hipError_t e = hipMemcpy(fftsig, gpuarrays->d_fft_signal, (params->rfftlen)*sizeof(float2), hipMemcpyDeviceToHost);
			if(e != hipSuccess) {
				LOG(log_level::error, "Could not hipMemcpy in aa_fdas_host.cu (" + std::string(hipGetErrorString(e)) + ")");
			}
			
			presto_dered_sig(fftsig, params->rfftlen);
			//export_fft_data(fftsig, params->rfftlen, "presto_dered_sig.dat");
			
			e = hipMemcpy(gpuarrays->d_fft_signal, fftsig, (params->rfftlen)*sizeof(float2), hipMemcpyHostToDevice);
			if(e != hipSuccess) {
				LOG(log_level::error, "Could not hipMemcpy in aa_fdas_host.cu (" + std::string(hipGetErrorString(e)) + ")");
			}
			
			free(fftsig);
		}
		else {
			// doing deredning on GPU
			hipStream_t stream; stream = NULL;
			spectrum_whitening_SGP2((float2 *) gpuarrays->d_fft_signal, params->rfftlen, 1, true, stream);
			
			//float2 *GPU_norm;
			//GPU_norm = (float2*)malloc((params->rfftlen)*sizeof(float2));
			//hipMemcpy(GPU_norm, gpuarrays->d_fft_signal, (params->rfftlen)*sizeof(float2), hipMemcpyDeviceToHost);
			//export_fft_data(GPU_norm, params->rfftlen, "gpu_dered_sig.dat");
			//free(GPU_norm);
		}
	}

    //overlap-copy
    call_kernel_cuda_overlap_copy_smallblk(params->nblocks, gpuarrays->d_ext_data, gpuarrays->d_fft_signal, params->sigblock, params->rfftlen, params->extlen, params->offset, params->nblocks );

	if (cmdargs->norm && ENABLE_HOST_BLOCK_MEDIAN_NORM){
		// PRESTO block median normalization
		// TODO: replace with GPU version
		float2 *extsig;
		extsig = (float2*)malloc((params->extlen)*sizeof(float2));
		hipError_t e = hipMemcpy(extsig, gpuarrays->d_ext_data, (params->extlen)*sizeof(float2), hipMemcpyDeviceToHost);
		
		if(e != hipSuccess) {
			LOG(log_level::error, "Could not hipMemcpy in aa_fdas_host.cu (" + std::string(hipGetErrorString(e)) + ")");
		}
		
		for(int b=0; b<params->nblocks; ++b) {
			presto_norm(extsig+b*KERNLEN, KERNLEN);
		}
		
		e = hipMemcpy(gpuarrays->d_ext_data, extsig, (params->extlen)*sizeof(float2), hipMemcpyHostToDevice);
		if(e != hipSuccess) {
			LOG(log_level::error, "Could not hipMemcpy in aa_fdas_host.cu (" + std::string(hipGetErrorString(e)) + ")");
		}
		
		free(extsig);
	}

    // Custom FFT convolution kernel
    if(cmdargs->inbin){
      call_kernel_cuda_convolve_customfft_wes_no_reorder02_inbin(params->nblocks, gpuarrays->d_kernel, gpuarrays->d_ext_data, gpuarrays->d_ffdot_pwr, params->sigblock, params->extlen, params->siglen, params->offset, params->scale, gpuarrays->ip_edge_points);
    }
    else{
      //-------------------------------------------
      dim3 gridSize(1, 1, 1);
      dim3 blockSize(1, 1, 1);
      
      //-------------------------------------------
      //Four elements per thread
      gridSize.x = params->nblocks;
      gridSize.y = 1;
      gridSize.z = 1;
      blockSize.x = KERNLEN/4;
      call_kernel_GPU_CONV_kFFT_mk11_4elem_2v(gridSize,blockSize, gpuarrays->d_ext_data, gpuarrays->d_ffdot_pwr, gpuarrays->d_kernel, params->sigblock, params->offset, params->nblocks, params->scale);
    }
  }
#endif

  /** \brief Write fdas list to disk. */
  void fdas_write_list(fdas_gpuarrays *gpuarrays, cmd_args *cmdargs, fdas_params *params, float *h_MSD, float dm_low, int dm_count, float dm_step, unsigned int list_size){
    int ibin=1;
    if (cmdargs->inbin) ibin=2;
    double tobs = (double)params->tsamp* (double)params->nsamps*ibin;
	
    if( !isnan(h_MSD[0]) || !isinf(h_MSD[0]) || !isnan(h_MSD[1]) || !isinf(h_MSD[1]) ){
      //printf("Number of peaks:%d; mean:%f; strdev:%f\n", list_size, h_MSD[0], h_MSD[1]);
		
      float *h_fdas_peak_list = (float*)malloc(list_size*4*sizeof(float));
      hipError_t e = hipMemcpy(h_fdas_peak_list, gpuarrays->d_fdas_peak_list, list_size*4*sizeof(float), hipMemcpyDeviceToHost);
      
      if(e != hipSuccess) {
	LOG(log_level::error, "Could not hipMemcpy in aa_fdas_host.cu (" + std::string(hipGetErrorString(e)) + ")");
      }
		
      //prepare file
      const char *dirname= "output_data";
      struct stat st = {0};

      if (stat(dirname, &st) == -1) {
	printf("\nDirectory %s does not exist, creating...\n", dirname);
	mkdir(dirname, 0700);
      }	
		
      FILE *fp_c;
      char pfname[200];
      sprintf(pfname, "acc_list_%f.dat", dm_low + ((float)dm_count)*dm_step);
      if ((fp_c=fopen(pfname, "w")) == NULL) {
	fprintf(stderr, "Error opening %s file for writing: %s\n",pfname, strerror(errno));
      }

      int i_list_size = (int)list_size;
      for(int f=0; f<i_list_size; f++){
	int j;
	double a, acc, acc1, jfreq, pow, SNR;
	a   = h_fdas_peak_list[4*f];
	j   = (int) h_fdas_peak_list[4*f + 1];
	pow = h_fdas_peak_list[4*f + 2];
	SNR = (pow-h_MSD[0])/h_MSD[1];
	jfreq = (double)(j) / tobs;
	acc = (double) (ZMAX - a* ACCEL_STEP);
	acc1 = acc*SLIGHT / jfreq / tobs / tobs;
	fprintf(fp_c, "%.2f\t%.3f\t%u\t%.3f\t%.3f\t%.3f\n", acc, acc1, j , jfreq, pow, SNR);
      }

      fclose(fp_c);
		
      free(h_fdas_peak_list);
    }
    else {
      printf("Error: mean or standard deviation was NaN or Inf!\n");
    }
  }


  /** \brief Write ffdot output data to disk. */
  void fdas_write_ffdot(fdas_gpuarrays *gpuarrays, cmd_args *cmdargs, fdas_params *params, float dm_low, int dm_count, float dm_step ) {
    int ibin=1;
    if (cmdargs->inbin)
      ibin=2;
    // Download, threshold and write ffdot data to file
    //int nsamps = params->nsamps;

    printf("\n\nWrite data for signal with %d samples\nf-fdot size=%u\n",params->nsamps, params->ffdotlen);
    float *h_ffdotpwr = (float*)malloc(params->ffdotlen* sizeof(float));
    //download data
    hipError_t e = hipMemcpy(h_ffdotpwr, gpuarrays->d_ffdot_pwr, params->ffdotlen*sizeof(float), hipMemcpyDeviceToHost);

    if(e != hipSuccess) {
      LOG(log_level::error, "Could not hipMemcpy in aa_fdas_host.cu (" + std::string(hipGetErrorString(e)) + ")");
    }
    
    // calculating statistics
    double total = 0.0;
    double mean;
    double stddev;
    // unsigned int j;
    int i_params_ffdotlen = (int)params->ffdotlen;
    for ( int j = 0; j < i_params_ffdotlen; ++j){
      total += (double)(h_ffdotpwr[j]);
      if(isnan(total)){
	printf("\nnan detected during sum for mean at j=%d\nValue at j:%f\n",j,h_ffdotpwr[j]);
	exit(1);
      }
    }
  
    mean = total / ((double)(params->ffdotlen)); 

    printf("\ntotal ffdot:%e\tmean ffdot: %e", total, mean);
      
    // Calculate standard deviation
    total = 0.0;
    for ( int j = 0; j < i_params_ffdotlen; ++j){
      total += ((double)h_ffdotpwr[j] - mean ) * ((double)h_ffdotpwr[j] - mean);
      if(isnan(total)||isinf(total)){
	printf("\ninf/nan detected during sum for mean at j=%d\nValue at j:%f\n",j,h_ffdotpwr[j]);
	exit(1);
      }
    }
    stddev = sqrt(abs(total) / (double)(params->ffdotlen - 1)); 
    printf("\nmean ffdot: %e\tstd ffdot: %e\n", mean, stddev);

    //prepare file
    const char *dirname= "output_data";
    struct stat st = {0};

    if (stat(dirname, &st) == -1) {
      printf("\nDirectory %s does not exist, creating...\n", dirname);
      mkdir(dirname, 0700);
    }

    FILE *fp_c;
    char pfname[200];
    //  char *infilename;
    //  infilename = basename(cmdargs->afname);
    // filename needs to be acc_dm_%f, dm_low[i] + ((float)dm_count)*dm_step[i]
    //sprintf(pfname, "%s/out_inbin%d_%s",dirname,ibin,infilename);
    sprintf(pfname, "acc_%f.dat", dm_low + ((float)dm_count)*dm_step);
    printf("\nwriting results to file %s\n",pfname);
    if ((fp_c=fopen(pfname, "w")) == NULL) {
      fprintf(stderr, "Error opening %s file for writing: %s\n",pfname, strerror(errno));
      exit(1);
    }
    float pow, sigma;
    double tobs = (double)params->tsamp * (double)(params->nsamps)*ibin;
    unsigned int numindep = params->siglen*(NKERN+1)*ACCEL_STEP/6.95; // taken from PRESTO

    //write to file
    printf("\nWriting ffdot data to file...\n");

    for(int a = 0; a < NKERN; a++) {
      double acc = (double) (ZMAX - a* ACCEL_STEP);
      for( int j = 0; j < ibin*params->siglen; j++){
	pow =  h_ffdotpwr[a * ibin*params->siglen + j]; //(h_ffdotpwr[a * params->siglen + j]-mean)/stddev;
		
		if( pow > cmdargs->thresh) {
			sigma = candidate_sigma(pow, cmdargs->nharms, numindep);//power, number of harmonics, number of independed searches=1...2^harms
			//  sigma=1.0;
			double jfreq = (double)(j) / tobs;
			double acc1 = acc*SLIGHT / jfreq / tobs / tobs;
			fprintf(fp_c, "%.2f\t%.3f\t%u\t%.3f\t%.3f\t%.3f\n", acc, acc1, j , jfreq, pow, sigma);
		}    
      }
    }

    fclose(fp_c);
    printf("\nFinished writing file %s\n",pfname);
    
    free(h_ffdotpwr);

  }


  /** \brief Write test ffdot to disk. */
  void fdas_write_test_ffdot(fdas_gpuarrays *gpuarrays, cmd_args *cmdargs, fdas_params *params, float dm_low, int dm_count, float dm_step ) {
    int ibin=1;
    if (cmdargs->inbin)
      ibin=2;
    /* Download, threshold and write ffdot data to file */
    //int nsamps = params->nsamps;

    printf("\n\nWrite data for signal with %d samples\nf-fdot size=%u\n",params->nsamps, params->ffdotlen);
    float *h_ffdotpwr = (float*)malloc(params->ffdotlen* sizeof(float));
    //download data
    hipError_t e = hipMemcpy(h_ffdotpwr, gpuarrays->d_ffdot_pwr, params->ffdotlen*sizeof(float), hipMemcpyDeviceToHost);

    if(e != hipSuccess) {
      LOG(log_level::error, "Could not hipMemcpy in aa_fdas_host.cu (" + std::string(hipGetErrorString(e)) + ")");
    }

    // calculating statistics
    double total = 0.0;
    double mean;
    double stddev;
    // unsigned int j;
    int i_params_ffdotlen = (int)params->ffdotlen;
    for ( int j = 0; j < i_params_ffdotlen; ++j){
      total += (double)(h_ffdotpwr[j]);
      if(isnan(total)){
	printf("\nnan detected during sum for mean at j=%d\nValue at j:%f\n",j,h_ffdotpwr[j]);
	exit(1);
      }
    }
  
    mean = total / ((double)(i_params_ffdotlen)); 

    printf("\ntotal ffdot:%lf\tmean ffdot: %lf", total, mean);
      
    // Calculate standard deviation
    total = 0.0;
    for ( int j = 0; j < i_params_ffdotlen; ++j){
      total += ((double)h_ffdotpwr[j] - mean ) * ((double)h_ffdotpwr[j] - mean);
      if(isnan(total)||isinf(total)){
	printf("\ninf/nan detected during sum for mean at j=%d\nValue at j:%f\n",j,h_ffdotpwr[j]);
	exit(1);
      }
    }
    stddev = sqrt(abs(total) / (double)(i_params_ffdotlen - 1)); 
    printf("\nmean ffdot: %f\tstd ffdot: %lf\n", mean, stddev);

    //prepare file
    const char *dirname= "output_data";
    struct stat st = {0};

    if (stat(dirname, &st) == -1) {
      printf("\nDirectory %s does not exist, creating...\n", dirname);
      mkdir(dirname, 0700);
    }

    FILE *fp_c;
    char pfname[200];
    sprintf(pfname, "acc_fdas_conv_test.dat");
    printf("\nwriting results to file %s\n",pfname);
    if ((fp_c=fopen(pfname, "w")) == NULL) {
      fprintf(stderr, "Error opening %s file for writing: %s\n",pfname, strerror(errno));
      exit(1);
    }
    float pow;

    //write to file
    printf("\nWriting ffdot data to file...\n");

    for(int a = 0; a < NKERN; a++) {
      for( int j = 0; j < ibin*params->siglen; j++){
	pow =  h_ffdotpwr[a * ibin*params->siglen + j]; //(h_ffdotpwr[a * params->siglen + j]-mean)/stddev;
	fprintf(fp_c, "%u\t%u\t%f\n", a, j, pow); 
      }
    }

    fclose(fp_c);
    printf("\nFinished writing file %s\n",pfname);
    
    free(h_ffdotpwr);

  }
} //namespace astroaccelerate
