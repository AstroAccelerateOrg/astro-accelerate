#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hipfft/hipfft.h>
#include <time.h>
#include <string>

#include "aa_log.hpp"
#include "aa_params.hpp"
#include "aa_device_stats.hpp"
#include "aa_device_stretch.hpp"
#include "aa_device_set_stretch.hpp"
#include "aa_device_power.hpp"

namespace astroaccelerate {

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
  inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true) {
    if (code != hipSuccess) {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
    }
  }

  /**
   * \brief Example FFT.
   * \todo Clarify the difference between this function and other fdas functions.
   */
  void acceleration(int range, int nsamp, int max_ndms, int processed, int nboots, int num_trial_bins, int navdms, float narrow, float wide, int nsearch, float aggression, float cutoff, float ***output_buffer, int *ndms, int *inBin, float *dm_low, float *dm_high, float *dm_step, float tsamp) {
    printf("\n");

    printf("[1DCUFFT] is starting...\n");

    size_t size;
    int a; //j;
    float mean, stddev;

    for (int i = 0; i < range; i++) {

      //double total = 0.0;

      hipStream_t stream_e;
      //hipError_t result_e;
      gpuErrchk(hipStreamCreate(&stream_e));

      hipEvent_t event_e;
      hipEventCreate(&event_e);

      hipStream_t stream_o;
      //hipError_t result_o;
      gpuErrchk(hipStreamCreate(&stream_o));

      hipEvent_t event_o;
      hipEventCreate(&event_o);

      int samps = processed / inBin[i];

      printf("\nsamps:\t%d", samps);
      int nearest = (int) floorf(log2f((float) samps));
      printf("\nnearest:\t%d", nearest);
      samps = (int) powf(2.0, nearest);
      printf("\nsamps:\t%d", samps);

      // Allocate memory for signal even
      float* d_signal_in_e;
      size = samps * sizeof(float);
      printf("\nSize of GPU input signal:\t%zu MB", size / 1024 / 1024);
      hipError_t e = hipMalloc((void** )&d_signal_in_e, size);

      if(e != hipSuccess) {
	LOG(log_level::error, "Could not hipMalloc in aa_host_acceleration.cu (" + std::string(hipGetErrorString(e)) + ")");
      }

      float* d_signal_transformed_e;
      size = samps * sizeof(float);
      printf("\nSize of GPU stretched signal:\t%zu MB", size / 1024 / 1024);
      e = hipMalloc((void** )&d_signal_transformed_e, size);

      if(e != hipSuccess) {
	LOG(log_level::error, "Could not hipMalloc in aa_host_acceleration.cu (" + std::string(hipGetErrorString(e)) + ")");
      }
      
      hipfftComplex* d_signal_fft_e;
      size = ( samps / 2 + 1 ) * sizeof(hipfftComplex);
      printf("\nSize of GPU output signal:\t%zu MB", size / 1024 / 1024);
      e = hipMalloc((void** )&d_signal_fft_e, size);

      if(e != hipSuccess) {
	LOG(log_level::error, "Could not hipMalloc in aa_host_acceleration.cu (" + std::string(hipGetErrorString(e)) + ")");
      }

      float* d_signal_power_e;
      size = sizeof(float) * ( samps / 2 ) * ( 2 * ACCMAX + ACCSTEP ) / ACCSTEP;
      printf("\nSize of GPU power signal:\t%zu MB", size / 1024 / 1024);
      e = hipMalloc((void** )&d_signal_power_e, size);

      if(e != hipSuccess) {
	LOG(log_level::error, "Could not hipMalloc in aa_host_acceleration.cu (" + std::string(hipGetErrorString(e)) + ")");
      }

      float2* h_signal_e;
      size = ( samps ) * sizeof(float2);
      printf("\nSize of host output signal:\t%zu MB", size / 1024 / 1024);
      e = hipHostMalloc((void** )&h_signal_e, size);

      if(e != hipSuccess) {
	LOG(log_level::error, "Could not hipMalloc in aa_host_acceleration.cu (" + std::string(hipGetErrorString(e)) + ")");
      }

      float* h_signal_transformed_e;
      size = samps * sizeof(float);
      printf("\nSize of GPU stretched signal:\t%zu MB", size / 1024 / 1024);
      e = hipHostMalloc((void** )&h_signal_transformed_e, size);

      if(e != hipSuccess) {
	LOG(log_level::error, "Could not hipMalloc in aa_host_acceleration.cu (" + std::string(hipGetErrorString(e)) + ")");
      }

      float* h_signal_power_e;
      size = sizeof(float) * ( samps / 2 ) * ( 2 * ACCMAX + ACCSTEP ) / ACCSTEP;
      printf("\nSize of total host power signal:\t%zu MB", size / 1024 / 1024), fflush(stdout);
      e = hipHostMalloc((void** )&h_signal_power_e, size);

      if(e != hipSuccess) {
	LOG(log_level::error, "Could not hipMalloc in aa_host_acceleration.cu (" + std::string(hipGetErrorString(e)) + ")");
      }

      // Allocate memory for signal odd
      float* d_signal_in_o;
      size = samps * sizeof(float);
      printf("\nSize of GPU input signal:\t%zu MB", size / 1024 / 1024);
      e = hipMalloc((void** )&d_signal_in_o, size);

      if(e != hipSuccess) {
	LOG(log_level::error, "Could not hipMalloc in aa_host_acceleration.cu (" + std::string(hipGetErrorString(e)) + ")");
      }

      float* d_signal_transformed_o;
      size = samps * sizeof(float);
      printf("\nSize of GPU stretched signal:\t%zu MB", size / 1024 / 1024);
      e = hipMalloc((void** )&d_signal_transformed_o, size);

      if(e != hipSuccess) {
	LOG(log_level::error, "Could not hipMalloc in aa_host_acceleration.cu (" + std::string(hipGetErrorString(e)) + ")");
      }

      hipfftComplex* d_signal_fft_o;
      size = ( samps / 2 + 1 ) * sizeof(hipfftComplex);
      printf("\nSize of GPU output signal:\t%zu MB", size / 1024 / 1024);
      e = hipMalloc((void** )&d_signal_fft_o, size);

      if(e != hipSuccess) {
	LOG(log_level::error, "Could not hipMalloc in aa_host_acceleration.cu (" + std::string(hipGetErrorString(e)) + ")");
      }

      float* d_signal_power_o;
      size = sizeof(float) * ( samps / 2 ) * ( 2 * ACCMAX + ACCSTEP ) / ACCSTEP;
      printf("\nSize of GPU power signal:\t%zu MB", size / 1024 / 1024);
      e = hipMalloc((void** )&d_signal_power_o, size);

      if(e != hipSuccess) {
	LOG(log_level::error, "Could not hipMalloc in aa_host_acceleration.cu (" + std::string(hipGetErrorString(e)) + ")");
      }

      float2* h_signal_o;
      size = ( samps ) * sizeof(float2);
      printf("\nSize of host output signal:\t%zu MB", size / 1024 / 1024);
      e = hipHostMalloc((void** )&h_signal_o, size);

      if(e != hipSuccess) {
	LOG(log_level::error, "Could not hipMalloc in aa_host_acceleration.cu (" + std::string(hipGetErrorString(e)) + ")");
      }

      float* h_signal_transformed_o;
      size = samps * sizeof(float);
      printf("\nSize of GPU stretched signal:\t%zu MB", size / 1024 / 1024);
      e = hipHostMalloc((void** )&h_signal_transformed_o, size);

      if(e != hipSuccess) {
	LOG(log_level::error, "Could not hipMalloc in aa_host_acceleration.cu (" + std::string(hipGetErrorString(e)) + ")");
      }

      float* h_signal_power_o;
      size = sizeof(float) * ( samps / 2 ) * ( 2 * ACCMAX + ACCSTEP ) / ACCSTEP;
      printf("\nSize of total host power signal:\t%zu MB", size / 1024 / 1024), fflush(stdout);
      e = hipHostMalloc((void** )&h_signal_power_o, size);

      if(e != hipSuccess) {
	LOG(log_level::error, "Could not hipMalloc in aa_host_acceleration.cu (" + std::string(hipGetErrorString(e)) + ")");
      }

      // CUFFT plan even
      hipfftHandle plan_e;
      hipfftPlan1d(&plan_e, samps, HIPFFT_R2C, 1);
      hipfftSetStream(plan_e, stream_e);

      // CUFFT plan odd
      hipfftHandle plan_o;
      hipfftPlan1d(&plan_o, samps, HIPFFT_R2C, 1);
      hipfftSetStream(plan_o, stream_o);

      int trials = ( 2 * ACCMAX + ACCSTEP ) / ACCSTEP;

      // Transfer even memory asynchronously
      //TEST:checkCudaErrors(hipMemcpyAsync(d_signal_in_e, output_buffer[i][230],   samps*sizeof(float), hipMemcpyHostToDevice, stream_e));
      e = hipMemcpyAsync(d_signal_in_e, output_buffer[i][0], samps * sizeof(float), hipMemcpyHostToDevice, stream_e);

      if(e != hipSuccess) {
	LOG(log_level::error, "Could not hipMalloc in aa_host_acceleration.cu (" + std::string(hipGetErrorString(e)) + ")");
      }
      
      hipEventRecord(event_e, stream_e);

      // Cacluclate even dm
      for (a = 0; a < trials; a++)
	{
	  int acc = -ACCMAX + a * ACCSTEP;
	  float mean = 127.959f;
	  set_stretch_gpu(event_e, stream_e, samps, mean, d_signal_transformed_e);
	  stretch_gpu(event_e, stream_e, acc, samps, tsamp, d_signal_in_e, d_signal_transformed_e);
	  hipStreamWaitEvent(stream_e, event_e, 0);
	  hipfftResult e = hipfftExecR2C(plan_e, (float * )d_signal_transformed_e, (hipfftComplex * )d_signal_fft_e);

	  if(e != HIPFFT_SUCCESS) {
	    LOG(log_level::error, "Could not hipfftExecR2C in aa_host_acceleration.cu");
	  }
	  
	  power_gpu(event_e, stream_e, samps, a, d_signal_fft_e, d_signal_power_e);
	}

      for (int dm_count = 1; dm_count < ndms[i] - 1; dm_count += 2)
	{
	  hipStreamWaitEvent(stream_o, event_o, 0);
	  hipError_t e = hipMemcpyAsync(d_signal_in_o, output_buffer[i][dm_count], samps * sizeof(float), hipMemcpyHostToDevice, stream_o);
	  
	  if(e != hipSuccess) {
	    LOG(log_level::error, "Could not hipMemcpyAsync in aa_host_acceleration.cu (" + std::string(hipGetErrorString(e)) + ")");
	  }
	  
	  hipEventRecord(event_o, stream_o);

	  // Cacluclate odd dm
	  for (a = 0; a < trials; a++)
	    {
	      int acc = -ACCMAX + a * ACCSTEP;
	      float mean = 127.959f;
	      set_stretch_gpu(event_o, stream_o, samps, mean, d_signal_transformed_o);
	      stretch_gpu(event_o, stream_o, acc, samps, tsamp, d_signal_in_o, d_signal_transformed_o);
	      hipfftResult cufft_e = hipfftExecR2C(plan_o, (float * )d_signal_transformed_o, (hipfftComplex * )d_signal_fft_o);
	      
	      if(cufft_e != HIPFFT_SUCCESS) {
		LOG(log_level::error, "Could not hipfftExecR2C in aa_host_acceleration.cu");
	      }
	      
	      hipStreamWaitEvent(stream_o, event_o, 0);
	      power_gpu(event_o, stream_o, samps, a, d_signal_fft_o, d_signal_power_o);
	    }

	  // Threshold even f-fdot plane
	  hipStreamSynchronize(stream_e);
	  stats_gpu(event_e, stream_e, samps, &mean, &stddev, h_signal_power_e, d_signal_power_e);
	    
	  e = hipMemcpyAsync(d_signal_in_e, output_buffer[i][dm_count + 1], samps * sizeof(float), hipMemcpyHostToDevice, stream_e);
	  
	  if(e != hipSuccess) {
	    LOG(log_level::error, "Could not hipMemcpyAsync in aa_host_acceleration.cu (" + std::string(hipGetErrorString(e)) + ")");
	  }
	  
	  hipEventRecord(event_e, stream_e);

	  // Cacluclate even dm
	  for (a = 0; a < trials; a++)
	    {
	      int acc = -ACCMAX + a * ACCSTEP;
	      float mean = 127.959f;
	      set_stretch_gpu(event_e, stream_e, samps, mean, d_signal_transformed_e);
	      stretch_gpu(event_e, stream_e, acc, samps, tsamp, d_signal_in_e, d_signal_transformed_e);
	      hipStreamWaitEvent(stream_e, event_e, 0);
	      hipfftResult e = hipfftExecR2C(plan_e, (float * )d_signal_transformed_e, (hipfftComplex * )d_signal_fft_e);
	      
	      if(e != HIPFFT_SUCCESS) {
		LOG(log_level::error, "Could not hipfftExecR2C in aa_host_acceleration.cu");
	      }
	      
	      power_gpu(event_e, stream_e, samps, a, d_signal_fft_e, d_signal_power_e);
	    }

	  // Threshold odd f-fdot plane
	  hipStreamSynchronize(stream_o);
	  stats_gpu(event_o, stream_o, samps, &mean, &stddev, h_signal_power_o, d_signal_power_o);
	}

      //Destroy CUFFT context
      hipfftDestroy(plan_e);
      hipfftDestroy(plan_o);

      //Destroy streams
      gpuErrchk(hipStreamDestroy(stream_e));
      gpuErrchk(hipStreamDestroy(stream_o));

      // cleanup even memory
      hipHostFree(h_signal_e);
      hipHostFree(h_signal_power_e);
      hipFree(d_signal_in_e);
      hipFree(d_signal_fft_e);
      hipFree(d_signal_power_e);
      hipFree(d_signal_transformed_e);

      // cleanup odd memory
      hipHostFree(h_signal_o);
      hipHostFree(h_signal_power_o);
      hipFree(d_signal_in_o);
      hipFree(d_signal_fft_o);
      hipFree(d_signal_power_o);
      hipFree(d_signal_transformed_o);
    }
  }
} //namespace astroaccelerate
