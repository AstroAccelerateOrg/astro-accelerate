#include "hip/hip_runtime.h"
// Added by Karel Adamek

#include "aa_device_threshold_kernel.hpp"
#include "aa_device_peak_find_shared_kernel_functions.cuh"
#include "aa_device_threshold_shared_kernel_functions.cuh"
#include "aa_device_cuda_deprecated_wrappers.cuh"

namespace astroaccelerate {

  __global__ void THR_GPU_WARP(float const* __restrict__ d_input, ushort *d_input_taps, unsigned int *d_output_list_DM, unsigned int *d_output_list_TS, float *d_output_list_SNR, unsigned int *d_output_list_BW, int *gmem_pos, float threshold, int nTimesamples, int offset, int shift, int max_list_size, int DIT_value) {
    int local_id;
    local_id = threadIdx.x & (WARP - 1);
    int warp_id;
    warp_id = threadIdx.x>>5;
	
    int pos_x, pos_y, list_pos, mask, leader;
    float R;
	
    pos_y = (blockIdx.y*THR_WARPS_PER_BLOCK + warp_id)*nTimesamples;
    pos_x = blockIdx.x*WARP*THR_ELEM_PER_THREAD + local_id;
	
    for(int i=0; i<THR_ELEM_PER_THREAD; i++){
      if(pos_x<offset){
	R=__ldg(&d_input[pos_x + pos_y]);
	if(R > threshold) {
	  mask=aa_ballot(AA_ASSUME_MASK,1);
	  leader=__ffs(mask)-1;
	  if(local_id==leader) list_pos=atomicAdd(gmem_pos,__popc(mask));
	  list_pos=aa_shfl(AA_ASSUME_MASK,list_pos,leader);
	  list_pos=list_pos+__popc(mask&((1<<local_id)-1));
	  if(list_pos<max_list_size){
	    d_output_list_DM[list_pos]  = blockIdx.y*THR_WARPS_PER_BLOCK + warp_id + shift;
	    d_output_list_TS[list_pos]  = DIT_value*pos_x + d_input_taps[(pos_x + pos_y)]/2;
	    d_output_list_SNR[list_pos] = R;
	    d_output_list_BW[list_pos]  = (unsigned int) d_input_taps[(pos_x + pos_y)];
	  }
	}
      }
      pos_x = pos_x + WARP;
    }
  }

  __global__ void GPU_Threshold_for_periodicity_kernel_old(float const* __restrict__ d_input, ushort *d_input_harms, float *d_output_list, int *gmem_pos, float *d_MSD, float threshold, int primary_size, int secondary_size, int DM_shift, int max_list_size, int DIT_value) {
    int pos_p, pos_s, pos, list_pos, mask, leader;
    float R;
    float hrms;
    float mean = d_MSD[0];
    float sd   = d_MSD[1];
	
    pos_p = blockIdx.x*blockDim.x*THR_ELEM_PER_THREAD + threadIdx.x;
    pos_s = blockIdx.y*blockDim.y + threadIdx.y;
	
	
    for(int i=0; i<THR_ELEM_PER_THREAD; i++){
      if( (pos_p<primary_size) && (pos_s<secondary_size)){
	pos = pos_s*primary_size + pos_p;
			
	//--------> Thresholding
	R = __ldg(&d_input[pos]);
	if(R > threshold) {
	  mask=aa_ballot(AA_ASSUME_MASK,1);
	  leader=__ffs(mask)-1;
	  if(threadIdx.x==leader) list_pos=atomicAdd(gmem_pos,__popc(mask));
	  list_pos=aa_shfl(AA_ASSUME_MASK,list_pos,leader);
	  list_pos=list_pos+__popc(mask&((1<<threadIdx.x)-1));
	  if(list_pos<max_list_size){
	    d_output_list[4*list_pos]   = pos_p + DM_shift;
	    d_output_list[4*list_pos+1] = pos_s/DIT_value;
	    hrms = (float) d_input_harms[pos];
	    d_output_list[4*list_pos+3] = hrms;
	    d_output_list[4*list_pos+2] = inverse_white_noise(&R,&hrms,&mean,&sd);
	  }
	}
	//-------------------------<
			
      }
      pos_p = pos_p + blockDim.x;
    }
  }

  __global__ void GPU_Threshold_for_periodicity_kernel(float const* __restrict__ d_input, ushort *d_input_harms, float *d_output_list, int *gmem_pos, float const* __restrict__ d_MSD, float threshold, int primary_size, int secondary_size, int DM_shift, int max_list_size, int DIT_value) {
    int pos_p, pos_s, pos, list_pos, mask, leader;
    float R;
    int hrms;
	
    pos_p = blockIdx.x*blockDim.x*THR_ELEM_PER_THREAD + threadIdx.x;
    pos_s = blockIdx.y*blockDim.y + threadIdx.y;
	
	
    for(int i=0; i<THR_ELEM_PER_THREAD; i++){
      if( (pos_p<primary_size) && (pos_s<secondary_size)){
	pos = pos_s*primary_size + pos_p;
			
	//--------> Thresholding
	R = __ldg(&d_input[pos]);
	if(R > threshold) {
	  mask=aa_ballot(AA_ASSUME_MASK,1);
	  leader=__ffs(mask)-1;
	  if(threadIdx.x==leader) list_pos=atomicAdd(gmem_pos,__popc(mask));
	  list_pos=aa_shfl(AA_ASSUME_MASK,list_pos,leader);
	  list_pos=list_pos+__popc(mask&((1<<threadIdx.x)-1));
	  if(list_pos<max_list_size){
	    d_output_list[4*list_pos]   = pos_p + DM_shift;
	    d_output_list[4*list_pos+1] = pos_s/DIT_value;
	    hrms = (int) d_input_harms[pos];
	    d_output_list[4*list_pos+3] = hrms;
	    d_output_list[4*list_pos+2] = R*__ldg(&d_MSD[2*hrms+1]) + __ldg(&d_MSD[2*hrms]);
	  }
	}
	//-------------------------<
			
      }
      pos_p = pos_p + blockDim.x;
    }
  }

  /** \brief Kernel wrapper function for THR_GPU_WARP kernel function. */
  void call_kernel_THR_GPU_WARP(const dim3 &grid_size, const dim3 &block_size,
				float const *const d_input, ushort *const d_input_taps,
				unsigned int *const d_output_list_DM, unsigned int *const d_output_list_TS,
				float *const d_output_list_SNR, unsigned int *const d_output_list_BW,
				int *const gmem_pos, const float &threshold, const int &nTimesamples, const int &offset,
				const int &shift, const int &max_list_size, const int &DIT_value) {
    THR_GPU_WARP<<<grid_size, block_size>>>(d_input, d_input_taps, d_output_list_DM, d_output_list_TS,
					    d_output_list_SNR, d_output_list_BW,
					    gmem_pos, threshold, nTimesamples, offset,
					    shift, max_list_size, DIT_value);
  }

  /** \brief Kernel wrapper function for GPU_Threshold_for_periodicity_kernel_old kernel function. */
  void call_kernel_GPU_Threshold_for_periodicity_kernel_old(const dim3 &grid_size, const dim3 &block_size,
							    float const *const d_input, ushort *const d_input_harms,
							    float *const d_output_list, int *const gmem_pos, float *const d_MSD,
							    const float &threshold, const int &primary_size, const int &secondary_size,
							    const int &DM_shift, const int &max_list_size, const int &DIT_value) {
    GPU_Threshold_for_periodicity_kernel_old<<<grid_size, block_size>>>(d_input, d_input_harms, d_output_list,
									gmem_pos, d_MSD, threshold, primary_size,
									secondary_size, DM_shift, max_list_size, DIT_value);

  }

  /** \brief Kernel wrapper function for GPU_Threshold_for_periodicity_kernel kernel function. */
  void call_kernel_GPU_Threshold_for_periodicity_kernel(const dim3 &grid_size, const dim3 &block_size,
							float const *const d_input, ushort *const d_input_harms,
							float *const d_output_list, int *const gmem_pos, float const *const d_MSD,
							const float &threshold, const int &primary_size,
							const int &secondary_size, const int &DM_shift, const int &max_list_size, const int &DIT_value) {
    GPU_Threshold_for_periodicity_kernel<<<grid_size, block_size>>>(d_input, d_input_harms,
								    d_output_list, gmem_pos, d_MSD,
								    threshold, primary_size,
								    secondary_size, DM_shift, max_list_size, DIT_value);
  }
} //namespace astroaccelerate
