#include <stdio.h>
#include "aa_params.hpp"
#include "aa_device_stretch_kernel.hpp"

namespace astroaccelerate {
  /** \brief Doppler stretch. Simple corner turn on the GPU. */
  void stretch_gpu(hipEvent_t event, hipStream_t stream, int acc, int samps, float tsamp, float *d_input, float *d_output) {
    int divisions_in_t = 32;
    int num_blocks_t = samps / divisions_in_t;

    float t_zero = ( (double) tsamp ) / ( 1.0 + ( ( acc * samps * (double) tsamp ) / 599584916.0 ) );
    float multiplier = ( t_zero * acc * (double) tsamp ) / 599584916.0;
    float tsamp_inverse = 1.0 / tsamp;

    dim3 threads_per_block(divisions_in_t);
    dim3 num_blocks(num_blocks_t);

    hipStreamWaitEvent(stream, event, 0);
    call_kernel_stretch_kernel(num_blocks, threads_per_block, 0, stream, acc, samps, tsamp, d_input, d_output, t_zero, multiplier, tsamp_inverse);
    //getLastCudaError("stretch_kernel failed");
    hipEventRecord(event, stream);
  }
} //namespace astroaccelerate
