#include "hip/hip_runtime.h"
#ifndef ASTRO_ACCELERATE_DEVICE_CORNER_TURN_KERNEL_CU
#define ASTRO_ACCELERATE_DEVICE_CORNER_TURN_KERNEL_CU

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "params.hpp"

__global__ void simple_corner_turn_kernel(float *d_input, float *d_output, int primary_size, int secondary_size){

  int primary = blockIdx.x * blockDim.x + threadIdx.x;
  int secondary = blockIdx.y * blockDim.y + threadIdx.y;

  d_output[primary*secondary_size + secondary] = (float) __ldg(&d_input[secondary*primary_size + primary]);
}


__global__ void corner_turn_SM_kernel(float const* __restrict__ d_input, float *d_output, int primary_size, int secondary_size) {
  __shared__ float s_input[WARP*(WARP+1)*CT_CORNER_BLOCKS];
	
  int i, gpos, spos, itemp, pc, sc;
	
  int warp_id = threadIdx.x>>5;
  int local_id = threadIdx.x & (WARP - 1);
	
  gpos=(blockIdx.y*(blockDim.x>>5) + warp_id)*CT_ROWS_PER_WARP*primary_size + blockIdx.x*CT_CORNER_BLOCKS*WARP + local_id;
  for(int by=0; by<CT_ROWS_PER_WARP; by++){
    spos=local_id*WARP + local_id + warp_id*CT_ROWS_PER_WARP + by;
    for(int bx=0; bx<CT_CORNER_BLOCKS; bx++){ // temporary 
      s_input[spos]=d_input[gpos];
      gpos=gpos + WARP;
      spos=spos + WARP*(WARP+1);
    }
    gpos=gpos + primary_size - CT_CORNER_BLOCKS*WARP;
  }
	
  __syncthreads();
	
  itemp=warp_id*CT_ROWS_PER_WARP*CT_CORNER_BLOCKS;
  for(i=0; i<CT_ROWS_PER_WARP*CT_CORNER_BLOCKS; i++){
    pc = (blockIdx.x*CT_CORNER_BLOCKS*WARP + itemp + i);
    sc = WARP*blockIdx.y + local_id;
    if( pc<primary_size && sc<secondary_size ) {
      gpos=pc*secondary_size + sc;
      spos=(itemp + i)*(WARP+1) + local_id;
      d_output[gpos]=s_input[spos];
    }
  }
}

__global__ void simple_corner_turn_kernel(unsigned short *d_input, float *d_output, int nchans, int nsamp) {

  int t = blockIdx.x * blockDim.x + threadIdx.x;
  int c = blockIdx.y * blockDim.y + threadIdx.y;

  d_output[c * nsamp + t] = (float) __ldg(&d_input[t * nchans + c]);

}

__global__ void swap(unsigned short *d_input, float *d_output, int nchans, int nsamp) {

  int t = blockIdx.x * blockDim.x + threadIdx.x;
  int c = blockIdx.y * blockDim.y + threadIdx.y;

  d_input[c * nsamp + t] = (unsigned short) __ldg(&d_output[c * nsamp + t]);

}

void call_kernel_simple_corner_turn_kernel(dim3 block_size, dim3 grid_size, unsigned short *d_input, float *d_output, int primary_size, int secondary_size) {
  simple_corner_turn_kernel<<<block_size, grid_size>>>(d_input, d_output, primary_size, secondary_size);
}

void call_kernel_simple_corner_turn_kernel(dim3 block_size, dim3 grid_size, float *d_input, float *d_output, int primary_size, int secondary_size) {
  simple_corner_turn_kernel<<<block_size, grid_size>>>(d_input, d_output, primary_size, secondary_size);
}

void call_kernel_corner_turn_SM_kernel(dim3 grid_size, dim3 block_size, float const* __restrict__ d_input, float *d_output, int primary_size, int secondary_size) {
  corner_turn_SM_kernel<<<grid_size,block_size>>>(d_input, d_output, primary_size, secondary_size);
}

void call_kernel_swap(dim3 block_size, dim3 grid_size, unsigned short *d_input, float *d_output, int nchans, int nsamp) {
  swap<<<block_size, grid_size>>>(d_input, d_output, nchans, nsamp);
}

#endif

