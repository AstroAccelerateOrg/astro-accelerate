#include "hip/hip_runtime.h"
//#define GPU_ANALYSIS_DEBUG
//#define MSD_BOXCAR_TEST
//#define GPU_PARTIAL_TIMER
#define GPU_TIMER

#include <iostream>
#include <tuple>
#include <vector>

#include <stdio.h>
#include <stdlib.h>

#include "params.hpp"

#include "device_BC_plan.hpp"
#include "device_peak_find.hpp"
#include "device_MSD_plane_profile.hpp"
#include "device_SPS_long.hpp"
#include "device_SPS_plan"
#include "device_threshold.hpp"

#include "gpu_timer.hpp"


//TODO:
// Make BC_plan for arbitrary long pulses, by reusing last element in the plane



void Create_list_of_boxcar_widths(std::vector<int> *boxcar_widths, std::vector<int> *BC_widths, int max_boxcar_width){
	int DIT_value, DIT_factor, width;
	DIT_value = 1;
	DIT_factor = 2;
	width = 0;
	for(int f=0; f<(int) BC_widths->size(); f++){
		for(int b=0; b<BC_widths->operator[](f); b++){
			width = width + DIT_value;
			if(width<=max_boxcar_width){
				boxcar_widths->push_back(width);
			}
		}
		DIT_value = DIT_value*DIT_factor;
	}
}


void analysis_GPU(bool verbose, float* d_SPS_input, float *h_candidate_list, size_t &number_candidates, size_t max_candidates, SPS_Plan &spsplan){
	// Definition of some local variables
	float local_tsamp  = spsplan.GetCurrentSamplingTime(); // SPS_data.sampling_time*SPS_data.inBin; // corrected sampling time
	size_t nTimesamples = spsplan.GetCurrentTimeSamples();
	size_t nDMs         = spsplan.GetNumberDMs();
	if(verbose) {
		std::cout << "----------> Single Pulse GPU analysis" << std::endl;
		printf("  Dimensions: nTimesamples:%zu; nDMs:%zu; inBin:%d; sampling time: %f; corrected s. time: %f;\n", nTimesamples, nDMs, spsplan.GetCurrentBinningFactor(), spsplan.GetOriginalSamplingTime(), local_tsamp);
	}
	
	//--------> Definition of SPDT boxcar plan
	int max_desired_boxcar_width = spsplan.GetCurrentMaxBoxcarWidth();
	int max_width_performed = 0, max_iteration = 0;
	
	std::tuple<float, float, float> dm_limits = spsplan.GetDMLimits();
	// Old version
	//int t_BC_widths[10]={PD_MAXTAPS,16,16,16,8,8,8,8,8,8};
	//std::vector<int> BC_widths(t_BC_widths,t_BC_widths+sizeof(t_BC_widths)/sizeof(int));
	//std::vector<PulseDetection_plan> PD_plan;
	//Create_PD_plan(&PD_plan, &BC_widths, nTimesamples); //PD_plan is independent on maximum boxcar width. which is wrong?
	//max_iteration = Get_max_iteration(max_desired_boxcar_width, &BC_widths, &max_width_performed);
	//std::vector<int> h_boxcar_widths;
	//Create_list_of_boxcar_widths(&h_boxcar_widths, &BC_widths, max_width_performed);
	
	
	//New version
	std::vector<PulseDetection_plan> PD_plan;
	max_iteration = SPS_params->get_max_iteration(&max_width_performed, max_desired_boxcar_width);
	SPS_params->Create_PD_plan(&PD_plan, &max_width_performed, max_desired_boxcar_width, nTimesamples);
	if(verbose) 
		printf("  Selected iteration:%d; maximum boxcar width requested:%d; maximum boxcar width performed:%d;\n", max_iteration, max_desired_boxcar_width, max_width_performed);
	std::vector<int> h_boxcar_widths;
	SPS_params->Create_list_of_boxcar_widths(&h_boxcar_widths, max_width_performed);
	
	//printf("old size: %d; new size: %d;\n", (int) h_boxcar_widths.size(), (int) new_h_boxcar_widths.size());
	//if(h_boxcar_widths.size() == new_h_boxcar_widths.size()){
	//	int error;
	//	for(int f=0; f<(int) h_boxcar_widths.size(); f++){
	//		error = h_boxcar_widths[f] - new_h_boxcar_widths[f];
	//		if(error!=0) printf("%d-%d=%d at f=%f\n", h_boxcar_widths[f], new_h_boxcar_widths[f], error, f);
	//	}
	//}
	
	
	/*
	printf("Old calculation:\n");
	printf("max_iteration: %d; max_desired_boxcar_width: %d; max_width_performed: %d;\n", max_iteration, max_desired_boxcar_width, max_width_performed);
	printf("dec_nTs: "); for(int f=0; f<(int)PD_plan.size(); f++) printf("%d\t", PD_plan[f].decimated_timesamples); printf("\n");
	printf("dtm:     "); for(int f=0; f<(int)PD_plan.size(); f++) printf("%d\t", PD_plan[f].dtm); printf("\n");
	printf("iter:    "); for(int f=0; f<(int)PD_plan.size(); f++) printf("%d\t", PD_plan[f].iteration); printf("\n");
	printf("nBoxc:   "); for(int f=0; f<(int)PD_plan.size(); f++) printf("%d\t", PD_plan[f].nBoxcars); printf("\n");
	printf("nBlocks: "); for(int f=0; f<(int)PD_plan.size(); f++) printf("%d\t", PD_plan[f].nBlocks); printf("\n");
	printf("out_shf: "); for(int f=0; f<(int)PD_plan.size(); f++) printf("%d\t", PD_plan[f].output_shift); printf("\n");
	printf("shift:   "); for(int f=0; f<(int)PD_plan.size(); f++) printf("%d\t", PD_plan[f].shift); printf("\n");
	printf("s_taps:  "); for(int f=0; f<(int)PD_plan.size(); f++) printf("%d\t", PD_plan[f].startTaps); printf("\n");
	printf("un_samp: "); for(int f=0; f<(int)PD_plan.size(); f++) printf("%d\t", PD_plan[f].unprocessed_samples); printf("\n");
	printf("tot_ut:  "); for(int f=0; f<(int)PD_plan.size(); f++) printf("%d\t", PD_plan[f].total_ut); printf("\n");
	printf("------------------------------------------\n");
	printf("\n");
	
	int new_max_iteration, new_max_width_performed;
	std::vector<PulseDetection_plan> new_PD_plan;
	new_max_iteration = SPS_params->get_max_iteration(&new_max_width_performed, max_desired_boxcar_width);
	SPS_params->Create_PD_plan(&new_PD_plan, &max_width_performed, max_desired_boxcar_width, nTimesamples);
	printf("New calculation:\n");
	printf("max_iteration: %d; max_desired_boxcar_width: %d; max_width_performed: %d;\n", max_iteration, max_desired_boxcar_width, max_width_performed);
	printf("dec_nTs: "); for(int f=0; f<(int)new_PD_plan.size(); f++) printf("%d\t", new_PD_plan[f].decimated_timesamples); printf("\n");
	printf("dtm:     "); for(int f=0; f<(int)new_PD_plan.size(); f++) printf("%d\t", new_PD_plan[f].dtm); printf("\n");
	printf("iter:    "); for(int f=0; f<(int)new_PD_plan.size(); f++) printf("%d\t", new_PD_plan[f].iteration); printf("\n");
	printf("nBoxc:   "); for(int f=0; f<(int)new_PD_plan.size(); f++) printf("%d\t", new_PD_plan[f].nBoxcars); printf("\n");
	printf("nBlocks: "); for(int f=0; f<(int)new_PD_plan.size(); f++) printf("%d\t", new_PD_plan[f].nBlocks); printf("\n");
	printf("out_shf: "); for(int f=0; f<(int)new_PD_plan.size(); f++) printf("%d\t", new_PD_plan[f].output_shift); printf("\n");
	printf("shift:   "); for(int f=0; f<(int)new_PD_plan.size(); f++) printf("%d\t", new_PD_plan[f].shift); printf("\n");
	printf("s_taps:  "); for(int f=0; f<(int)new_PD_plan.size(); f++) printf("%d\t", new_PD_plan[f].startTaps); printf("\n");
	printf("un_samp: "); for(int f=0; f<(int)new_PD_plan.size(); f++) printf("%d\t", new_PD_plan[f].unprocessed_samples); printf("\n");
	printf("tot_ut:  "); for(int f=0; f<(int)new_PD_plan.size(); f++) printf("%d\t", new_PD_plan[f].total_ut); printf("\n");
	printf("------------------------------------------\n");
	printf("\n");	
	
	if(PD_plan.size()>=new_PD_plan.size()){
		printf("Difference:\n");
		printf("max_iteration: %d; max_desired_boxcar_width: %d; max_width_performed: %d;\n", max_iteration-new_max_iteration, max_desired_boxcar_width, max_width_performed-new_max_width_performed);
		printf("dec_nTs: "); for(int f=0; f<(int)new_PD_plan.size(); f++) printf("%d\t", new_PD_plan[f].decimated_timesamples-PD_plan[f].decimated_timesamples); printf("\n");
		printf("dtm:     "); for(int f=0; f<(int)new_PD_plan.size(); f++) printf("%d\t", new_PD_plan[f].dtm-PD_plan[f].dtm); printf("\n");
		printf("iter:    "); for(int f=0; f<(int)new_PD_plan.size(); f++) printf("%d\t", new_PD_plan[f].iteration-PD_plan[f].iteration); printf("\n");
		printf("nBoxc:   "); for(int f=0; f<(int)new_PD_plan.size(); f++) printf("%d\t", new_PD_plan[f].nBoxcars-PD_plan[f].nBoxcars); printf("\n");
		printf("nBlocks: "); for(int f=0; f<(int)new_PD_plan.size(); f++) printf("%d\t", new_PD_plan[f].nBlocks-PD_plan[f].nBlocks); printf("\n");
		printf("out_shf: "); for(int f=0; f<(int)new_PD_plan.size(); f++) printf("%d\t", new_PD_plan[f].output_shift-PD_plan[f].output_shift); printf("\n");
		printf("shift:   "); for(int f=0; f<(int)new_PD_plan.size(); f++) printf("%d\t", new_PD_plan[f].shift-PD_plan[f].shift); printf("\n");
		printf("s_taps:  "); for(int f=0; f<(int)new_PD_plan.size(); f++) printf("%d\t", new_PD_plan[f].startTaps-PD_plan[f].startTaps); printf("\n");
		printf("un_samp: "); for(int f=0; f<(int)new_PD_plan.size(); f++) printf("%d\t", new_PD_plan[f].unprocessed_samples-PD_plan[f].unprocessed_samples); printf("\n");
		printf("tot_ut:  "); for(int f=0; f<(int)new_PD_plan.size(); f++) printf("%d\t", new_PD_plan[f].total_ut-PD_plan[f].total_ut); printf("\n");
		printf("------------------------------------------\n");
		printf("\n");
	}
	*/
	// It should be like this:
	//   SPS_params should contain BC_widths
	//   SPS_params should also contain function get_maximum_iteration which would give number of iterations required to achieve user defined value in form of max_desired_boxcar_width
	//   Based on maximum_iteration SPS should build PD_plan
	//   Proper error check must be placed so SPS would not die if user chooses wrong maximum search width
	
	//--------> Benchmarking
	double total_time=0, MSD_time=0, SPDT_time=0, PF_time=0;
	

	//---------------------------------------------------------------------------
	//----------> GPU part
	GpuTimer total_timer, timer;
	total_timer.Start();
	
	
	size_t free_mem,total_mem;
	hipMemGetInfo(&free_mem,&total_mem);
	if(verbose) printf("  Memory required by boxcar filters:%0.3f MB\n",(4.5*nTimesamples*nDMs*sizeof(float) + 2*nTimesamples*nDMs*sizeof(ushort))/(1024.0*1024) );
	if(verbose) printf("  Memory available:%0.3f MB \n", ((float) free_mem)/(1024.0*1024.0) );
	
	
	//-------------------------------------------------------------------------
	//---------> Comparison between interpolated values and computed values
	#ifdef MSD_BOXCAR_TEST
		MSD_plane_profile_boxcars(d_SPS_input, nTimesamples, nDMs, &h_boxcar_widths, MSD_params->OR_sigma_multiplier, std::get<0>(dm_limits), std::get<1>(dm_limits), spsplan.GetCurrentStartTime());
	#endif
	//---------> Comparison between interpolated values and computed values
	//-------------------------------------------------------------------------
	
	
	
	//-------------------------------------------------------------------------
	//------------ Using MSD_plane_profile
	size_t MSD_profile_size_in_bytes, MSD_DIT_profile_size_in_bytes, workarea_size_in_bytes;
	hipMemGetInfo(&free_mem,&total_mem);
	Get_MSD_plane_profile_memory_requirements(&MSD_profile_size_in_bytes, &MSD_DIT_profile_size_in_bytes, &workarea_size_in_bytes, nTimesamples, nDMs, &h_boxcar_widths);
	double dit_time, MSD_only_time;
	float *d_MSD_interpolated;
	float *d_MSD_DIT = NULL;
	float *temporary_workarea;
	hipMalloc((void **) &d_MSD_interpolated, MSD_profile_size_in_bytes);
	hipMalloc((void **) &temporary_workarea, workarea_size_in_bytes);
	
	MSD_plane_profile(d_MSD_interpolated, d_SPS_input, d_MSD_DIT, temporary_workarea, false, nTimesamples, nDMs, &h_boxcar_widths, spsplan.GetCurrentStartTime(), std::get<0>(dm_limits), std::get<1>(dm_limits), MSD_params->OR_sigma_multiplier, MSD_params->enable_outlier_rejection, false, &MSD_time, &dit_time, &MSD_only_time);
	
	#ifdef GPU_PARTIAL_TIMER
		printf("    MSD time: Total: %f ms; DIT: %f ms; MSD: %f ms;\n", MSD_time, dit_time, MSD_only_time);
	#endif
	
	hipFree(temporary_workarea);
	//------------ Using MSD_plane_profile
	//-------------------------------------------------------------------------	
	
	
	//-------------------------------------------------------------------------
	//------------ Splitting input data into chunks
	std::vector<int> DM_list;
	unsigned long int max_timesamples=(free_mem*0.95)/(5.5*sizeof(float) + 2*sizeof(ushort));
	int DMs_per_cycle = max_timesamples/nTimesamples;
	int nRepeats, nRest, DM_shift, itemp, local_max_list_size;//BC_shift,
	
	itemp = (int) (DMs_per_cycle/THR_WARPS_PER_BLOCK);
	DMs_per_cycle = itemp*THR_WARPS_PER_BLOCK;
	
	nRepeats = nDMs/DMs_per_cycle;
	nRest = nDMs - nRepeats*DMs_per_cycle;
	local_max_list_size = (DMs_per_cycle*nTimesamples)/4;
	
	for(int f=0; f<nRepeats; f++) DM_list.push_back(DMs_per_cycle);
	if(nRest>0) DM_list.push_back(nRest);
	
	if( (int) DM_list.size() > 1 ) 
		printf("  SPS will run %d batches each containing %d DM trials. Remainder %d DM trials\n", (int) DM_list.size(), DMs_per_cycle, nRest);
	else 
		printf("  SPS will run %d batch containing %d DM trials.\n", (int) DM_list.size(), nRest);
	//------------ Splitting input data into chunks
	//-------------------------------------------------------------------------	
	
	
	
	if(DM_list.size()>0){
		DMs_per_cycle = DM_list[0];
		
		float *d_peak_list;
		if ( hipSuccess != hipMalloc((void**) &d_peak_list, sizeof(float)*DMs_per_cycle*nTimesamples)) printf("Allocation error! peaks\n");
		
		float *d_decimated;
		if ( hipSuccess != hipMalloc((void **) &d_decimated,  sizeof(float)*(((DMs_per_cycle*nTimesamples)/2)+PD_MAXTAPS) )) printf("Allocation error! dedispered\n");
		
		float *d_boxcar_values;
		if ( hipSuccess != hipMalloc((void **) &d_boxcar_values,  sizeof(float)*DMs_per_cycle*nTimesamples)) printf("Allocation error! boxcars\n");
		
		float *d_output_SNR;
		if ( hipSuccess != hipMalloc((void **) &d_output_SNR, sizeof(float)*2*DMs_per_cycle*nTimesamples)) printf("Allocation error! SNR\n");
		
		ushort *d_output_taps;
		if ( hipSuccess != hipMalloc((void **) &d_output_taps, sizeof(ushort)*2*DMs_per_cycle*nTimesamples)) printf("Allocation error! taps\n");
		
		int *gmem_peak_pos;
		hipMalloc((void**) &gmem_peak_pos, 1*sizeof(int));
		hipMemset((void*) gmem_peak_pos, 0, sizeof(int));
		
		DM_shift = 0;
		for(int f=0; f<DM_list.size(); f++) {
			//-------------- SPDT
			timer.Start();
			SPDT_search_long_MSD_plane(&d_SPS_input[DM_shift*nTimesamples], d_boxcar_values, d_decimated, d_output_SNR, d_output_taps, d_MSD_interpolated, &PD_plan, max_iteration, nTimesamples, DM_list[f]);
			timer.Stop();
			SPDT_time += timer.Elapsed();
			#ifdef GPU_PARTIAL_TIMER
			printf("    SPDT took:%f ms\n", timer.Elapsed());
			#endif
			//-------------- SPDT
			
			checkCudaErrors(hipGetLastError());
			
			#ifdef GPU_ANALYSIS_DEBUG
			printf("    BC_shift:%d; DMs_per_cycle:%d; f*DMs_per_cycle:%d; max_iteration:%d;\n", DM_shift*nTimesamples, DM_list[f], DM_shift, max_iteration);
			#endif
			
			if (spsplan.GetSPSAlgorithm() == 1) {
				//-------------- Thresholding
				timer.Start();
				THRESHOLD(d_output_SNR, d_output_taps, d_peak_list, gmem_peak_pos, spsplan.GetSigmaCutoff(), DM_list[f], nTimesamples, DM_shift, &PD_plan, max_iteration, local_max_list_size, std::get<2>(dm_limits), std::get<0>(dm_limits), local_tsamp, spsplan.GetCurrentBinningFactor(), spsplan.GetCurrentStartTime());
				timer.Stop();
				PF_time += timer.Elapsed();
				#ifdef GPU_PARTIAL_TIMER
				printf("    Thresholding took:%f ms\n", timer.Elapsed());
				#endif
				//-------------- Thresholding
			} else if (spsplan.GetSPSAlgorithm() == 0) {
				//-------------- Peak finding
				timer.Start();
				PEAK_FIND(d_output_SNR, d_output_taps, d_peak_list, DM_list[f], nTimesamples, spsplan.GetSigmaCutoff(), local_max_list_size, gmem_peak_pos, DM_shift, &PD_plan, max_iteration, std::get<2>(dm_limits), std::get<0>(dm_limits), local_tsamp, spsplan.GetCurrentBinningFactor(), spsplan.GetCurrentStartTime());
				timer.Stop();
				PF_time = timer.Elapsed();
				#ifdef GPU_PARTIAL_TIMER
				printf("    Peak finding took:%f ms\n", timer.Elapsed());
				#endif
				//-------------- Peak finding
			}
			
			checkCudaErrors(hipGetLastError());
			
			int temp_peak_pos = 0;
			checkCudaErrors(hipMemcpy(&temp_peak_pos, gmem_peak_pos, sizeof(int), hipMemcpyDeviceToHost));
			#ifdef GPU_ANALYSIS_DEBUG
			printf("    Candidates found:%d; Total #candidates for this chunk:%zu; Maximum #candidates:%zu; Local max. #candidates:%d;\n", temp_peak_pos, (number_candidates), max_candidates, local_max_list_size);
			#endif
			if( temp_peak_pos>=local_max_list_size ) {
				printf("    WARNING: Maximum list size reached! Not all candidates will be saved. You can increase sigma cutoff.\n");
				temp_peak_pos = local_max_list_size;
			}
			if( ((number_candidates) + temp_peak_pos)<max_candidates){
				checkCudaErrors(hipMemcpy(&h_candidate_list[(number_candidates)*4], d_peak_list, temp_peak_pos*4*sizeof(float), hipMemcpyDeviceToHost));
				number_candidates = (number_candidates) + temp_peak_pos;
			}
			else printf("    ERROR: Not enough memory to store all candidates on the host!\n");

			DM_shift = DM_shift + DM_list[f];
			hipMemset((void*) gmem_peak_pos, 0, sizeof(int));
		}
		
		hipFree(d_peak_list);
		hipFree(d_boxcar_values);
		hipFree(d_decimated);
		hipFree(d_output_SNR);
		hipFree(d_output_taps);
		hipFree(gmem_peak_pos);
		hipFree(d_MSD_DIT);
		hipFree(d_MSD_interpolated);

	}
	else printf("Error not enough memory to search for pulses\n");

	total_timer.Stop();
	total_time = total_timer.Elapsed();
	#ifdef GPU_TIMER
	printf("\n  TOTAL TIME OF SPS:%f ms\n", total_time);
	printf("  MSD_time: %f ms; SPDT time: %f ms; Candidate selection time: %f ms;\n", MSD_time, SPDT_time, PF_time);
	printf("----------<\n\n");
	#endif
	//----------> GPU part
	//---------------------------------------------------------------------------
	
}