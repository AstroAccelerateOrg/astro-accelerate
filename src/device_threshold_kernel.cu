#include "hip/hip_runtime.h"
// Added by Karel Adamek

#include "device_threshold_kernel.hpp"
#include "device_peak_find_shared_kernel_functions.cuh"
#include "device_threshold_shared_kernel_functions.cuh"

__global__ void THR_GPU_WARP(float const* __restrict__ d_input, ushort *d_input_taps, float *d_output_list, int *gmem_pos, float threshold, int nTimesamples, int offset, int shift, int max_list_size, int DIT_value, float dm_step, float dm_low, float sampling_time, float inBin, float start_time) {
	int local_id;
	local_id = threadIdx.x & (WARP - 1);
	int warp_id;
	warp_id = threadIdx.x>>5;
	
	int pos_x, pos_y, list_pos, mask, leader;
	float R;

	pos_y = (blockIdx.y*THR_WARPS_PER_BLOCK + warp_id)*nTimesamples;
	pos_x = blockIdx.x*WARP*THR_ELEM_PER_THREAD + local_id;
	
	for(int i=0; i<THR_ELEM_PER_THREAD; i++){
		if(pos_x<offset){
			R=__ldg(&d_input[pos_x + pos_y]);
			if(R > threshold) {
				mask=__ballot(1);
				leader=__ffs(mask)-1;
				if(local_id==leader) list_pos=atomicAdd(gmem_pos,__popc(mask));
				list_pos=__shfl(list_pos,leader);
				list_pos=list_pos+__popc(mask&((1<<local_id)-1));
				if(list_pos<max_list_size){
					d_output_list[4*list_pos]   = (blockIdx.y*THR_WARPS_PER_BLOCK + warp_id + shift)*dm_step + dm_low;
					d_output_list[4*list_pos+1] = (DIT_value*pos_x + (float) d_input_taps[(pos_x + pos_y)]/2.0)*sampling_time + start_time;
					d_output_list[4*list_pos+2] = R;
					d_output_list[4*list_pos+3] = ((float) d_input_taps[(pos_x + pos_y)])*inBin;
				}
			}
		}
		pos_x = pos_x + WARP;
	}
}

__global__ void GPU_Threshold_for_periodicity_kernel_old(float const* __restrict__ d_input, ushort *d_input_harms, float *d_output_list, int *gmem_pos, float *d_MSD, float threshold, int primary_size, int secondary_size, int DM_shift, int max_list_size, int DIT_value) {
	int pos_p, pos_s, pos, list_pos, mask, leader;
	float R;
	float hrms;
	float mean = d_MSD[0];
	float sd   = d_MSD[1];
	
	pos_p = blockIdx.x*blockDim.x*THR_ELEM_PER_THREAD + threadIdx.x;
	pos_s = blockIdx.y*blockDim.y + threadIdx.y;
	
	
	for(int i=0; i<THR_ELEM_PER_THREAD; i++){
		if( (pos_p<primary_size) && (pos_s<secondary_size)){
			pos = pos_s*primary_size + pos_p;
			
			//--------> Thresholding
			R = __ldg(&d_input[pos]);
			if(R > threshold) {
				mask=__ballot(1);
				leader=__ffs(mask)-1;
				if(threadIdx.x==leader) list_pos=atomicAdd(gmem_pos,__popc(mask));
				list_pos=__shfl(list_pos,leader);
				list_pos=list_pos+__popc(mask&((1<<threadIdx.x)-1));
				if(list_pos<max_list_size){
					d_output_list[4*list_pos]   = pos_p + DM_shift;
					d_output_list[4*list_pos+1] = pos_s/DIT_value;
					hrms = (float) d_input_harms[pos];
					d_output_list[4*list_pos+3] = hrms;
					d_output_list[4*list_pos+2] = inverse_white_noise(&R,&hrms,&mean,&sd);
				}
			}
			//-------------------------<
			
		}
		pos_p = pos_p + blockDim.x;
	}
}

__global__ void GPU_Threshold_for_periodicity_kernel(float const* __restrict__ d_input, ushort *d_input_harms, float *d_output_list, int *gmem_pos, float const* __restrict__ d_MSD, float threshold, int primary_size, int secondary_size, int DM_shift, int max_list_size, int DIT_value) {
	int pos_p, pos_s, pos, list_pos, mask, leader;
	float R;
	int hrms;
	
	pos_p = blockIdx.x*blockDim.x*THR_ELEM_PER_THREAD + threadIdx.x;
	pos_s = blockIdx.y*blockDim.y + threadIdx.y;
	
	
	for(int i=0; i<THR_ELEM_PER_THREAD; i++){
		if( (pos_p<primary_size) && (pos_s<secondary_size)){
			pos = pos_s*primary_size + pos_p;
			
			//--------> Thresholding
			R = __ldg(&d_input[pos]);
			if(R > threshold) {
				mask=__ballot(1);
				leader=__ffs(mask)-1;
				if(threadIdx.x==leader) list_pos=atomicAdd(gmem_pos,__popc(mask));
				list_pos=__shfl(list_pos,leader);
				list_pos=list_pos+__popc(mask&((1<<threadIdx.x)-1));
				if(list_pos<max_list_size){
					d_output_list[4*list_pos]   = pos_p + DM_shift;
					d_output_list[4*list_pos+1] = pos_s/DIT_value;
					hrms = (int) d_input_harms[pos];
					d_output_list[4*list_pos+3] = hrms;
					d_output_list[4*list_pos+2] = R*__ldg(&d_MSD[2*hrms+1]) + __ldg(&d_MSD[2*hrms]);
				}
			}
			//-------------------------<
			
		}
		pos_p = pos_p + blockDim.x;
	}
}

void call_kernel_THR_GPU_WARP(dim3 grid_size, dim3 block_size,
			      float const* d_input, ushort *d_input_taps,
			      unsigned int *d_output_list_DM, unsigned int *d_output_list_TS,
			      float *d_output_list_SNR, unsigned int *d_output_list_BW,
			      int *gmem_pos, float threshold, int nTimesamples, int offset,
			      int shift, int max_list_size, int DIT_value) {
  THR_GPU_WARP<<<grid_size, block_size>>>(d_input, d_input_taps, d_output_list_DM, d_output_list_TS,
					  d_output_list_SNR, d_output_list_BW,
					  gmem_pos, threshold, nTimesamples, offset,
					  shift, max_list_size, DIT_value);
}

void call_kernel_GPU_Threshold_for_periodicity_kernel_old(dim3 grid_size, dim3 block_size,
							  float const* d_input, ushort *d_input_harms,
							  float *d_output_list, int *gmem_pos, float *d_MSD,
							  float threshold, int primary_size, int secondary_size,
							  int DM_shift, int max_list_size, int DIT_value) {
  GPU_Threshold_for_periodicity_kernel_old<<<grid_size, block_size>>>(d_input, d_input_harms, d_output_list,
								      gmem_pos, d_MSD, threshold, primary_size,
								      secondary_size, DM_shift, max_list_size, DIT_value);

}

void call_kernel_GPU_Threshold_for_periodicity_kernel(dim3 grid_size, dim3 block_size,
						      float const* d_input, ushort *d_input_harms,
						      float *d_output_list, int *gmem_pos, float const* d_MSD,
						      float threshold, int primary_size,
						      int secondary_size, int DM_shift, int max_list_size, int DIT_value) {
  GPU_Threshold_for_periodicity_kernel<<<grid_size, block_size>>>(d_input, d_input_harms,
								 d_output_list, gmem_pos, d_MSD,
								 threshold, primary_size,
								 secondary_size, DM_shift, max_list_size, DIT_value);
}
