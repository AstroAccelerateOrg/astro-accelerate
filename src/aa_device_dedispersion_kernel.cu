#include "hip/hip_runtime.h"
#include "aa_device_dedispersion_kernel.hpp"
//union magic_int {
//	unsigned int i;
//	unsigned short int j[2];
//	unsigned short short int k[4];
//}

#include "float.h"
#include <stdio.h>
#include "aa_timelog.hpp"

namespace astroaccelerate {

	union magic_int {
		unsigned int i;
		unsigned short j[2];
		unsigned char k[4];
	};

  //{{{ shared_dedisperse_loop

  __device__ __shared__ ushort2 f_line[UNROLLS][ARRAYSIZE + 2];
  __device__ __shared__ uchar4 test[UNROLLS][ARRAYSIZE+4];
  __device__ __constant__ int i_nsamp, i_nchans, i_t_processed_s;
  __device__ __constant__ float dm_shifts[8192];

 __global__ void test_kernel(int bin, unsigned short *d_input, float *d_output, float mstartdm, float mdmstep){

   ushort temp_f;

   	if ((threadIdx.x == 0) & (threadIdx.y == 0) & (blockIdx.x == 0) & (blockIdx.y == 0)) printf("*****************************\n");
    int i, c, unroll, stage;
    magic_int local;

    int shift;
    int local_kernel_one[SNUMREG];
    int local_kernel_two[SNUMREG];
    int local_kernel_three[SNUMREG];
    int local_kernel_four[SNUMREG];

    float findex = (threadIdx.x*4 + 3); //max = 13*4+3 = 55;  orig:27;

    for (i = 0; i < SNUMREG; i++)
      {
        local_kernel_one[i] = 0;
        local_kernel_two[i] = 0;
	local_kernel_three[i] = 0;
	local_kernel_four[i] = 0;
      }

    int idx       = ( threadIdx.x + ( threadIdx.y * SDIVINT ) );  // 559
    int nsamp_counter = ( idx + ( blockIdx.x * ( 4 * SNUMREG * SDIVINT ) ) ); //max: 559 + 168*4*8*14 = 75823  -- 75712 

    float shift_two = ( mstartdm + ( __int2float_rz(blockIdx.y) * SFDIVINDM * mdmstep ) ); //max 0 + 37*40*1562.5 = 2312500
    float shift_one = ( __int2float_rz(threadIdx.y) * mdmstep ); // max : 39*1562.5 = 60937.5

    for (c = 0; c < i_nchans; c ++){
	     __syncthreads();
	     temp_f = ( __ldg(( d_input + ( __float2int_rz(dm_shifts[c]*shift_two) ) )  +  nsamp_counter) );
		test[0][idx + 3].x = temp_f;
		test[0][idx + 2].y = temp_f;
		test[0][idx + 1].z = temp_f;
		test[0][idx].w = temp_f;

		shift = __float2int_rz(shift_one*dm_shifts[c] + findex);
		//max 60937.5*0.000923 + 55
		//orig 60937.5*0.000923 + 27

		 nsamp_counter = ( nsamp_counter + ( UNROLLS * i_nsamp ) );

	       __syncthreads();

	       for (i = 0; i < SNUMREG; i++){
	            local.i = 0;
	            unroll = ( i * 4 * SDIVINT );
		    //max: 7*4*14 = 392
		    //orif 7*2*14 = 196
	            
		    stage = *(int*) &test[0][( shift + unroll )]; //196+27=223; now:447
        	    local.i += stage;
 		    
		    local_kernel_one[i] += local.k[0];
	            local_kernel_two[i] += local.k[1];
		    local_kernel_three[i] += local.k[2];
		    local_kernel_four[i] += local.k[3];
               }
    }
    if ((threadIdx.x == 0) & (threadIdx.y == 0) & (blockIdx.x == 0) & (blockIdx.y == 0)){
	    printf("terno jak to jede: %d %d %d %d %d\n", local_kernel_one[0], local_kernel_two[0], local_kernel_three[0], local_kernel_four[0], i_t_processed_s);
    }
	local.i = ( ( ( ( blockIdx.y * SDIVINDM ) + threadIdx.y ) * ( i_t_processed_s ) ) + ( blockIdx.x * 4 * SNUMREG * SDIVINT ) ) + 4 * threadIdx.x;
	//max: (37*40 + 39)*75712 + 168*4*8*14 + 4*13 // 75264 + 52
	//orig: (37*40 + 39)*75712 + 337*2*8*14 + 2*13 // 75488 + 26

#pragma unroll
    for (i = 0; i < SNUMREG; i++)
      {
        *( (float4*) ( d_output + local.i + ( i * 4 * SDIVINT ) ) ) = make_float4((float)local_kernel_one[i] / i_nchans/bin,
                                                                                  (float)local_kernel_two[i] / i_nchans/bin,
										  (float)local_kernel_three[i] / i_nchans/bin,
										  (float)local_kernel_four[i] / i_nchans/bin 
										  );
      }


 }

  __global__ void shared_dedisperse_kernel(int bin, unsigned short *d_input, float *d_output, float mstartdm, float mdmstep) {
    ushort temp_f;

    int i, j, c, local, unroll, stage;

    int shift[UNROLLS];
    int local_kernel_one[SNUMREG];
    int local_kernel_two[SNUMREG];

    float findex = (( threadIdx.x * 2 ) + 1 );

    for (i = 0; i < SNUMREG; i++)
      {
	local_kernel_one[i] = 0;
	local_kernel_two[i] = 0;
      }

    int idx 	  = ( threadIdx.x + ( threadIdx.y * SDIVINT ) );  // 559
    int nsamp_counter = ( idx + ( blockIdx.x * ( 2 * SNUMREG * SDIVINT ) ) ); //max: 559 + 337*2*8*14 = 76047; 559 + 168*4*8*14 = 75823  -- 75712 

    float shift_two = ( mstartdm + ( __int2float_rz(blockIdx.y) * SFDIVINDM * mdmstep ) ); //max 0 + 37*40*1562.5 = 2312500
    float shift_one = ( __int2float_rz(threadIdx.y) * mdmstep ); // max : 39*1562.5 = 60937.5

    for (c = 0; c < i_nchans; c += UNROLLS)
      {

	__syncthreads();

	for (j = 0; j < UNROLLS; j++)
	  {
	    temp_f = ( __ldg(( d_input + ( __float2int_rz(dm_shifts[c + j] * shift_two) ) )  + ( nsamp_counter + ( j * i_nsamp ) )) );
	    //0.000923 * 2312500 + 76047+insamp*c + 0*78130 = 20001332
	    if ((idx == 559) & (c == 255) & (blockIdx.y == 37) & (blockIdx.x == 337)) printf("SSS %d %d\n", idx, nsamp_counter );
	    if ( (idx < 2) & (c == 0) & (blockIdx.x == 0) & (blockIdx.y == 0) ){
		    printf("%d z\n", i_nsamp);
		    printf("\nidx: %d Zyyyyyyyyyyyyyyyyyyyyyy %hu %d %d %lf\n", idx, temp_f, d_input[0], d_input[1], shift_two);
	    }


	    f_line[j][idx + 1].x = temp_f;
	    f_line[j][idx    ].y = temp_f;

	    shift[j] = __float2int_rz(shift_one * dm_shifts[c + j] + findex);
//	    if (c==255) printf("c: %lf shif: %d %lf\n", shift_one, shift[j], dm_shifts[c+j]);
	  }

	nsamp_counter = ( nsamp_counter + ( UNROLLS * i_nsamp ) );

	__syncthreads();

	for (i = 0; i < SNUMREG; i++)
	  {
	    local = 0;
	    unroll = ( i * 2 * SDIVINT );
	    for (j = 0; j < UNROLLS; j++)
	      {
		stage = *(int*) &f_line[j][( shift[j] + unroll )];
		local += stage;
	      }
	    local_kernel_one[i] += (local & 0x0000FFFF);
	    local_kernel_two[i] += (local & 0xFFFF0000) >> 16;
	    if ( (idx == 0) & (c == 0) & (blockIdx.x == 0) & (blockIdx.y == 0) ){
		    printf("TTTTTT local_one: %d two:%d local:%d %d\n", local_kernel_one[i], local_kernel_two[i], local, shift[0]);
	    }
	  }
      }

    // Write the accumulators to the output array. 
    local = ( ( ( ( blockIdx.y * SDIVINDM ) + threadIdx.y ) * ( i_t_processed_s ) ) + ( blockIdx.x * 2 * SNUMREG * SDIVINT ) ) + 2 * threadIdx.x;

    if ( (idx < 2) & (blockIdx.x == 0) & (blockIdx.y == 0)) printf("sum results: %d %d %d %d\n", local_kernel_one[0], local_kernel_two[0], local_kernel_one[1], local_kernel_two[1]);
#pragma unroll
    for (i = 0; i < SNUMREG; i++)
      {
	*( (float2*) ( d_output + local + ( i * 2 * SDIVINT ) ) ) = make_float2((float)local_kernel_one[i] / i_nchans/bin,
										(float)local_kernel_two[i] / i_nchans/bin);
      }
  }



	__global__ void shared_dedisperse_kernel_nchan8192p(int bin, unsigned short *d_input, float *d_output, float *d_dm_shifts, float mstartdm, float mdmstep) {
		ushort temp_f;
		
		int i, j, c, local, unroll, stage;
		
		int shift[UNROLLS];
		int local_kernel_one[SNUMREG];
		int local_kernel_two[SNUMREG];
		
		float findex = (( threadIdx.x * 2 ) + 1 );
		
		for (i = 0; i < SNUMREG; i++) {
			local_kernel_one[i] = 0;
			local_kernel_two[i] = 0;
		}
		
		int idx 	  = ( threadIdx.x + ( threadIdx.y * SDIVINT ) );
		int nsamp_counter = ( idx + ( blockIdx.x * ( 2 * SNUMREG * SDIVINT ) ) );
		
		float shift_two = ( mstartdm + ( __int2float_rz(blockIdx.y) * SFDIVINDM * mdmstep ) );
		float shift_one = ( __int2float_rz(threadIdx.y) * mdmstep );
		
		for (c = 0; c < i_nchans; c += UNROLLS) {
			
			__syncthreads();
			for (j = 0; j < UNROLLS; j++) {
				temp_f = ( __ldg(( d_input + ( __float2int_rz(d_dm_shifts[c + j] * shift_two) ) )  + ( nsamp_counter + ( j * i_nsamp ) )) );
				
				f_line[j][idx + 1].x = temp_f;
				f_line[j][idx    ].y = temp_f;
				
				shift[j] = __float2int_rz(shift_one * d_dm_shifts[c + j] + findex);
			}
			
			nsamp_counter = ( nsamp_counter + ( UNROLLS * i_nsamp ) );
			
			__syncthreads();
		
			for (i = 0; i < SNUMREG; i++) {
				local = 0;
				unroll = ( i * 2 * SDIVINT );
				for (j = 0; j < UNROLLS; j++) {
					stage = *(int*) &f_line[j][( shift[j] + unroll )];
					local += stage;
				}
				local_kernel_one[i] += (local & 0x0000FFFF);
				local_kernel_two[i] += (local & 0xFFFF0000) >> 16;
			}
		}
		
		// Write the accumulators to the output array. 
		local = ( ( ( ( blockIdx.y * SDIVINDM ) + threadIdx.y ) * ( i_t_processed_s ) ) + ( blockIdx.x * 2 * SNUMREG * SDIVINT ) ) + 2 * threadIdx.x;
		
		#pragma unroll
		for (i = 0; i < SNUMREG; i++) {
			*( (float2*) ( d_output + local + ( i * 2 * SDIVINT ) ) ) = make_float2((float)local_kernel_one[i]/i_nchans/bin, (float)local_kernel_two[i]/i_nchans/bin);
		}
	}



  __global__ void shared_dedisperse_kernel_16(int bin, unsigned short *d_input, float *d_output, float mstartdm, float mdmstep) {
    int i, c;
    int shift;

    ushort temp_f;
    int local, unroll;

    float findex = ( threadIdx.x * 2 );
    float local_kernel_one[SNUMREG];
    float local_kernel_two[SNUMREG];

    for (i = 0; i < SNUMREG; i++)
      {
	local_kernel_one[i] = 0.0f;
	local_kernel_two[i] = 0.0f;
      }

    int idx = ( threadIdx.x + ( threadIdx.y * SDIVINT ) );
    int nsamp_counter = ( idx + ( blockIdx.x * ( 2 * SNUMREG * SDIVINT ) ) );

    float shift_two = ( mstartdm + ( __int2float_rz(blockIdx.y) * SFDIVINDM * mdmstep ) );
    float shift_one = ( __int2float_rz(threadIdx.y) * mdmstep );

    for (c = 0; c < i_nchans; c ++)
      {

	__syncthreads();

	temp_f = ( __ldg(( d_input + ( __float2int_rz(dm_shifts[c] * shift_two) ) ) + ( nsamp_counter )) );
	if ( (idx < 2) & (c == 0) & (blockIdx.x == 0) & (blockIdx.y == 0) ){
		printf("ladlgkgeapog %hu\n", temp_f);
	}

	f_line[0][idx].x = temp_f;
	if (idx > 0)
	  {
	    f_line[0][idx - 1].y = temp_f;
	  }
	shift = __float2int_rz(shift_one * dm_shifts[c] + findex);

	nsamp_counter = ( nsamp_counter + i_nsamp );

	__syncthreads();

	for (i = 0; i < SNUMREG; i++)
	  {
	    unroll = ( i * 2 * SDIVINT );
	    local = *(int*) &f_line[0][( shift + unroll )];
	    local_kernel_one[i] += ( (ushort2*) ( &local ) )->x;
	    local_kernel_two[i] += ( (ushort2*) ( &local ) )->y;
            if ( (idx == 0) & (c == 0) & (blockIdx.x == 0) & (blockIdx.y == 0) ){
                    printf("16TTTTTT local_one: %lf %lf %d\n", local_kernel_one[i], local_kernel_two[i], local);
            }

	  }
      }

    // Write the accumulators to the output array. 
    local = ( ( ( ( blockIdx.y * SDIVINDM ) + threadIdx.y ) * ( i_t_processed_s ) ) + ( blockIdx.x * 2 * SNUMREG * SDIVINT ) ) + 2 * threadIdx.x;

#pragma unroll
    for (i = 0; i < SNUMREG; i++)
      {
	*( (float2*) ( d_output + local + ( i * 2 * SDIVINT ) ) ) = make_float2(local_kernel_one[i] / i_nchans / bin, local_kernel_two[i] / i_nchans / bin);
      }
  }

  
	__global__ void shared_dedisperse_kernel_16_nchan8192p(int bin, unsigned short *d_input, float *d_output, float *d_dm_shifts, float mstartdm, float mdmstep) {
		int i, c;
		int shift;
		
		ushort temp_f;
		int local, unroll;
		
		float findex = ( threadIdx.x * 2 );
		float local_kernel_one[SNUMREG];
		float local_kernel_two[SNUMREG];
		
		for (i = 0; i < SNUMREG; i++) {
			local_kernel_one[i] = 0.0f;
			local_kernel_two[i] = 0.0f;
		}
		
		int idx = ( threadIdx.x + ( threadIdx.y * SDIVINT ) );
		int nsamp_counter = ( idx + ( blockIdx.x * ( 2 * SNUMREG * SDIVINT ) ) );
		
		float shift_two = ( mstartdm + ( __int2float_rz(blockIdx.y) * SFDIVINDM * mdmstep ) );
		float shift_one = ( __int2float_rz(threadIdx.y) * mdmstep );
		
		for (c = 0; c < i_nchans; c ++) {
			
			__syncthreads();
			
			temp_f = ( __ldg(( d_input + ( __float2int_rz(d_dm_shifts[c] * shift_two) ) ) + ( nsamp_counter )) );
			
			f_line[0][idx].x = temp_f;
			if (idx > 0) {
				f_line[0][idx - 1].y = temp_f;
			}
			shift = __float2int_rz(shift_one * d_dm_shifts[c] + findex);
			
			nsamp_counter = ( nsamp_counter + i_nsamp );
			
			__syncthreads();
			
			for (i = 0; i < SNUMREG; i++) {
				unroll = ( i * 2 * SDIVINT );
				local = *(int*) &f_line[0][( shift + unroll )];
				local_kernel_one[i] += ( (ushort2*) ( &local ) )->x;
				local_kernel_two[i] += ( (ushort2*) ( &local ) )->y;
			}
		}
		
		// Write the accumulators to the output array. 
		local = ( ( ( ( blockIdx.y * SDIVINDM ) + threadIdx.y ) * ( i_t_processed_s ) ) + ( blockIdx.x * 2 * SNUMREG * SDIVINT ) ) + 2 * threadIdx.x;
		
		#pragma unroll
		for (i = 0; i < SNUMREG; i++) {
			*( (float2*) ( d_output + local + ( i * 2 * SDIVINT ) ) ) = make_float2(local_kernel_one[i]/i_nchans/bin, local_kernel_two[i]/i_nchans/bin);
		}
	}

  
  __global__ void cache_dedisperse_kernel(int bin, unsigned short *d_input, float *d_output, float mstartdm, float mdmstep) {
    int   shift;	
    float local_kernel;

    int t  = blockIdx.x * SDIVINT  + threadIdx.x;
	
    // Initialise the time accumulators
    local_kernel = 0.0f;

    float shift_temp = mstartdm + ((blockIdx.y * SDIVINDM + threadIdx.y) * mdmstep);
	
    // Loop over the frequency channels.
    for(int c = 0; c < i_nchans; c++) {


      // Calculate the initial shift for this given frequency
      // channel (c) at the current despersion measure (dm) 
      // ** dm is constant for this thread!!**
      shift = (c * (i_nsamp) + t) + __float2int_rz (dm_shifts[c] * shift_temp);
		
      local_kernel += (float)__ldg(&d_input[shift]);
    }

    // Write the accumulators to the output array. 
    shift = ( ( ( blockIdx.y * SDIVINDM ) + threadIdx.y ) * ( i_t_processed_s ) ) + t;

    d_output[shift] = (local_kernel / i_nchans / bin);

  }

  
	__global__ void cache_dedisperse_kernel_nchan8192p(int bin, unsigned short *d_input, float *d_output, float *d_dm_shifts, float mstartdm, float mdmstep) {
		int   shift;	
		float local_kernel;

		int t  = blockIdx.x * SDIVINT  + threadIdx.x;
	
		// Initialise the time accumulators
		local_kernel = 0.0f;

		float shift_temp = mstartdm + ((blockIdx.y * SDIVINDM + threadIdx.y) * mdmstep);
	
		// Loop over the frequency channels.
		for(int c = 0; c < i_nchans; c++) {
			// Calculate the initial shift for this given frequency
			// channel (c) at the current despersion measure (dm) 
			// ** dm is constant for this thread!!**
			shift = (c * (i_nsamp) + t) + __float2int_rz (d_dm_shifts[c]*shift_temp);
			
			local_kernel += (float)__ldg(&d_input[shift]);
		}

		// Write the accumulators to the output array. 
		shift = ( ( ( blockIdx.y * SDIVINDM ) + threadIdx.y ) * ( i_t_processed_s ) ) + t;

		d_output[shift] = (local_kernel / i_nchans / bin);
  }
 

	//-------------------------------- wrapper functions
  
  /** \brief Kernel wrapper function to set device constants for dedispersion_kernel kernel function. */
  void set_device_constants_dedispersion_kernel(const int &nchans, const int &length, const int &t_processed, const float *const dmshifts) {
    hipMemcpyToSymbol(HIP_SYMBOL(dm_shifts), dmshifts, nchans * sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(i_nchans), &nchans, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(i_nsamp), &length, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(i_t_processed_s), &t_processed, sizeof(int));
    //checkCudaErrors(hipGetLastError());
  }

  /** \brief Kernel wrapper function to set device constants for kernel dedispersion_kernel function. */
  void set_device_constants_dedispersion_kernel(const long int &length, const int &t_processed) {
    hipMemcpyToSymbol(HIP_SYMBOL(i_nsamp), &length, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(i_t_processed_s), &t_processed, sizeof(int));
  }

  /** \brief Kernel wrapper function for dedisperse_kernel  kernel function. */
  void call_kernel_shared_dedisperse_kernel(const dim3 &block_size, const dim3 &grid_size,
					    const int &bin, unsigned short *const d_input, float *const d_output, const float &mstartdm, const float &mdmstep) {
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(shared_dedisperse_kernel), hipFuncCachePreferShared);
    shared_dedisperse_kernel<<<block_size, grid_size>>>(bin, d_input, d_output, mstartdm, mdmstep);
	dim3 blok(14,40,1);
	dim3 grid(169,38,1);
	printf("zzzzzzzzzzzzzzzzzzzz\n");
    test_kernel<<<grid,blok>>>(bin, d_input, d_output, mstartdm, mdmstep);
  }
  
	/** \brief Kernel wrapper function for dedispersion GPU kernel which works with number of channels greater than 8192. */
	void call_kernel_shared_dedisperse_kernel_nchan8192p(const dim3 &block_size, const dim3 &grid_size, const int &bin, unsigned short *const d_input, float *const d_output, float *const d_dm_shifts, const float &mstartdm, const float &mdmstep) {
		hipFuncSetCacheConfig(reinterpret_cast<const void*>(shared_dedisperse_kernel_nchan8192p), hipFuncCachePreferShared);
		shared_dedisperse_kernel_nchan8192p<<<block_size, grid_size>>>(bin, d_input, d_output, d_dm_shifts, mstartdm, mdmstep);
	}
  

  /** \brief Kernel wrapper function for dedisperse_kernel_16 kernel function. */
  void call_kernel_shared_dedisperse_kernel_16(const dim3 &block_size, const dim3 &grid_size,
					       const int &bin, unsigned short *const d_input, float *const d_output, const float &mstartdm, const float &mdmstep) {
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(shared_dedisperse_kernel_16), hipFuncCachePreferShared);
    shared_dedisperse_kernel_16<<<block_size, grid_size>>>(bin, d_input, d_output, mstartdm, mdmstep);
  }
  
	/** \brief Kernel wrapper function for dedispersion kernel which works with 16bit data and when number of channels is greater than 8192 kernel function. */
	void call_kernel_shared_dedisperse_kernel_16_nchan8192p(const dim3 &block_size, const dim3 &grid_size, const int &bin, unsigned short *const d_input, float *const d_output, float *const d_dm_shifts, const float &mstartdm, const float &mdmstep) {
		hipFuncSetCacheConfig(reinterpret_cast<const void*>(shared_dedisperse_kernel_16_nchan8192p), hipFuncCachePreferShared);
		shared_dedisperse_kernel_16<<<block_size, grid_size>>>(bin, d_input, d_output, mstartdm, mdmstep);
	}

  /** \brief Kernel wrapper function for cache_dedisperse_kernel kernel function. */
  void call_kernel_cache_dedisperse_kernel(const dim3 &block_size, const dim3 &grid_size,
					   const int &bin, unsigned short *const d_input, float *const d_output, const float &mstartdm, const float &mdmstep) {
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(cache_dedisperse_kernel), hipFuncCachePreferL1);
    cache_dedisperse_kernel<<<block_size, grid_size>>>(bin, d_input, d_output, mstartdm, mdmstep);
  }
  
	/** \brief Kernel wrapper function for cache_dedisperse_kernel kernel function. */
	void call_kernel_cache_dedisperse_kernel_nchan8192p(const dim3 &block_size, const dim3 &grid_size, const int &bin, unsigned short *const d_input, float *const d_output, float *const d_dm_shifts, const float &mstartdm, const float &mdmstep) {
		hipFuncSetCacheConfig(reinterpret_cast<const void*>(cache_dedisperse_kernel_nchan8192p), hipFuncCachePreferL1);
		cache_dedisperse_kernel_nchan8192p<<<block_size, grid_size>>>(bin, d_input, d_output, d_dm_shifts, mstartdm, mdmstep);
	}

} //namespace astroaccelerate
