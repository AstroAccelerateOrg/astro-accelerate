#include "hip/hip_runtime.h"
#include "device_set_stretch_kernel.hpp"
#include "params.hpp"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

//{{{ Set stretch
__global__ void set_stretch_kernel(int samps, float mean, float* d_input) {

  int t = blockIdx.x * blockDim.x + threadIdx.x;

  if(t >= 0 && t < samps)
    d_input[t] = mean;
}

void call_kernel_set_stretch_kernel(const dim3&         block_size,
                                    const dim3&         grid_size,
                                    const int&          smem_bytes,
                                    const hipStream_t& stream,
                                    const int&          samps,
                                    const float&        mean,
                                    float* const        d_input) {
  set_stretch_kernel<<<block_size, grid_size, smem_bytes, stream>>>(
      samps, mean, d_input);
}

//}}}
