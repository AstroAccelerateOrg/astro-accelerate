#include "hip/hip_runtime.h"
#include <vector>
#include "aa_device_convolution.hpp"

namespace astroaccelerate {

	void CONV_init(){
		//---------> Specific nVidia stuff
		hipDeviceSetCacheConfig(hipFuncCachePreferShared);
		hipDeviceSetSharedMemConfig(hipSharedMemBankSizeEightByte);
	}


	void forwardCustomFFT(float2 *d_filters, int FFT_size, int nFilters){
		dim3 gridSize(nFilters, 1, 1);
		dim3 blockSize(FFT_size/4, 1, 1);
		
		call_kernel_k_customFFT_GPU_forward(gridSize, blockSize, d_filters, d_filters, FFT_size);
	}


	void conv_OLS_customFFT(
			float2 *d_input_signal, 
			float *d_output_plane, 
			float2 *d_filters, 
			int64_t signal_length, 
			int64_t convolution_length, 
			int64_t useful_part_size, 
			int64_t offset, 
			int64_t nConvolutions, 
			int64_t nFilters, 
			float scale
	){
		dim3 gridSize(nConvolutions, 1, 1);
		dim3 blockSize(convolution_length/4, 1, 1);
		
		call_kernel_k_GPU_conv_OLS_via_customFFT(gridSize, blockSize, d_input_signal, d_output_plane, d_filters, signal_length, useful_part_size, offset, nConvolutions, nFilters, scale, convolution_length);
	}


	void convolve_signal_C2C(
			float2 *d_input_signal, 
			float *d_output_plane, 
			float2 *d_filters, 
			int64_t signal_length, 
			int64_t nFilters, 
			int64_t filter_halfwidth, 
			int64_t convolution_length, 
			float scale
	){
		int64_t useful_part_size = convolution_length - 2*filter_halfwidth + 1;
		int64_t nSegments        = (signal_length + useful_part_size - 1)/useful_part_size;
		conv_OLS_customFFT(d_input_signal, d_output_plane, d_filters, signal_length, convolution_length, useful_part_size, filter_halfwidth, nSegments, nFilters, scale);
	}

}
