#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "aa_params.hpp"
#include "aa_device_single_pulse_search_kernel.hpp"
#include "aa_device_SPS_inplace_kernel.hpp"

namespace astroaccelerate {

  __global__ void PD_SEARCH_GPU(float const* __restrict__ d_input, float *d_output, float *d_output_taps, float *d_MSD, int maxTaps, int nTimesamples)
  {
    extern __shared__ float s_input[]; //dynamically allocated memory for now

    int f, spos, itemp;
    size_t gpos;
    float res_SNR[PD_NWINDOWS], SNR, FIR_value, ftemp, signal_mean, signal_sd;
    int res_Taps[PD_NWINDOWS];
    signal_mean = d_MSD[0];
    signal_sd = d_MSD[1];

    //----------------------------------------------
    //----> Reading data
    gpos = (size_t)(blockIdx.y*nTimesamples) + (size_t)(blockIdx.x*PD_NTHREADS*PD_NWINDOWS);
    spos = threadIdx.x;
    for (f = 0; f < PD_NWINDOWS; f++)
      {
	s_input[spos] = __ldg(&d_input[gpos + (size_t)spos]);
	spos = spos + blockDim.x;
      }

    itemp = PD_NTHREADS*PD_NWINDOWS + maxTaps - 1;
    while (spos < itemp)
      {
	s_input[spos] = __ldg(&d_input[gpos + (size_t)spos]);
	spos = spos + blockDim.x;
      }

    //----> SNR for nTaps=1
    spos = threadIdx.x;
    for (f = 0; f < PD_NWINDOWS; f++)
      {
	res_SNR[f] = ( s_input[spos] - signal_mean )/signal_sd;
	res_Taps[f] = 1;
	spos = spos + blockDim.x;
      }

    __syncthreads();

    //----------------------------------------------
    //----> FIR calculation loop
    for (f = 2; f <= maxTaps; f++)
      {
	ftemp = c_sqrt_taps[f]*signal_sd;
	for (int i = 0; i < PD_NWINDOWS; i++)
	  {
	    spos = threadIdx.x + i*blockDim.x;
	    FIR_value = 0;
	    for (int t = 0; t < f; t++)
	      {
		FIR_value += s_input[spos + t];
	      }
	    SNR = ( FIR_value - f*signal_mean )/( ftemp );
	    if (SNR > res_SNR[i])
	      {
		res_SNR[i] = SNR;
		res_Taps[i] = f;
	      }
	  }
      }

    //----------------------------------------------
    //---- Writing data
    gpos = (size_t)(blockIdx.y*nTimesamples) + (size_t)(blockIdx.x*PD_NTHREADS*PD_NWINDOWS);
    spos = threadIdx.x;
    for (int i = 0; i < PD_NWINDOWS; i++)
      {
	d_output[gpos + (size_t)spos] = res_SNR[i];
	d_output_taps[gpos + (size_t)spos] = res_Taps[i];
	spos = spos + blockDim.x;
      }
  }

  /** \brief Kernel wrapper function for PD_SEARCH_GPU kernel function. */
  void call_kernel_PD_SEARCH_GPU(const dim3 &grid_size, const dim3 &block_size, const int &sm_size,
				 float const *const d_input, float *const d_output, float *const d_output_taps, float *const d_MSD, const int &maxTaps, const int &nTimesamples) {
    PD_SEARCH_GPU<<<grid_size, block_size, sm_size>>>(d_input, d_output, d_output_taps, d_MSD, maxTaps, nTimesamples);
  }

} //namespace astroaccelerate
