#include "hip/hip_runtime.h"
#include <time.h>
#include <stdio.h>
#include <hipfft/hipfft.h>
#include "aa_params.hpp"
#include "aa_device_stats_kernel.hpp"
#include "aa_log.hpp"


namespace astroaccelerate {
  /** \brief Returns stats. */
  void stats_gpu(hipEvent_t event, hipStream_t stream, int samps, float *mean, float *stddev, float *h_signal_power, float *d_signal_power) {
    int a;
    int trials = ( 2 * ACCMAX + ACCSTEP ) / ACCSTEP;
    
    int half_samps = samps / 2;
    int acc_size = half_samps * trials;

    int divisions = STATST;
    int blocks = (int) floor((float) acc_size / divisions / STATSLOOP);

    dim3 threads_per_block(divisions);
    dim3 num_blocks(blocks);

    int size = (int) floor((float) acc_size / STATSLOOP);

    float* d_sum;
    hipError_t e = hipMalloc((void** )&d_sum, size * sizeof(float));

    if(e != hipSuccess) {
      LOG(log_level::error, "Could not hipMalloc in aa_device_stats.cu (" + std::string(hipGetErrorString(e)) + ")");
    }

    float* d_sum_square;
    e = hipMalloc((void** )&d_sum_square, size * sizeof(float));

    if(e != hipSuccess) {
      LOG(log_level::error, "Could not hipMalloc in aa_device_stats.cu (" + std::string(hipGetErrorString(e)) + ")");
    }

    float* h_sum;
    e = hipHostMalloc((void** )&h_sum, size * sizeof(float));

    if(e != hipSuccess) {
      LOG(log_level::error, "Could not hipHostMalloc in aa_device_stats.cu (" + std::string(hipGetErrorString(e)) + ")");
    }

    float* h_sum_square;
    e = hipHostMalloc((void** )&h_sum_square, size * sizeof(float));

    if(e != hipSuccess) {
      LOG(log_level::error, "Could not hipHostMalloc in aa_device_stats.cu (" + std::string(hipGetErrorString(e)) + ")");
    }

    hipStreamWaitEvent(stream, event, 0);
    call_kernel_stats_kernel(num_blocks, threads_per_block, 0, stream, half_samps, d_sum, d_sum_square, d_signal_power);
    //getLastCudaError("power_kernel failed");
    hipEventRecord(event, stream);

    hipStreamWaitEvent(stream, event, 0);
    e = hipMemcpyAsync(h_sum, d_sum, size * sizeof(float), hipMemcpyDeviceToHost, stream);

    if(e != hipSuccess) {
      LOG(log_level::error, "Could not hipMemcpyAsync in aa_device_stats.cu (" + std::string(hipGetErrorString(e)) + ")");
    }

    
    e = hipMemcpyAsync(h_sum_square, d_sum_square, size * sizeof(float), hipMemcpyDeviceToHost, stream);

    if(e != hipSuccess) {
      LOG(log_level::error, "Could not hipMemcpyAsync in aa_device_stats.cu (" + std::string(hipGetErrorString(e)) + ")");
    }
    
    hipEventRecord(event, stream);
    hipStreamSynchronize(stream);

    float total_sum = 0.0;
    float total_sum_square = 0.0;

    for (a = 0; a < size; a++)
      {
	total_sum += ( h_sum[a] );
	total_sum_square += ( h_sum_square[a] );
      }
    *mean = (float) ( total_sum / ( acc_size ) );  // Mean for data sample
    *stddev = (float) sqrt(( total_sum_square - acc_size * ( *mean ) * ( *mean ) ) / ( acc_size - 1 ));

    hipFree(d_sum);
    hipFree(d_sum_square);
    hipHostFree(h_sum);
    hipHostFree(h_sum_square);

  }
} //namespace astroaccelerate
